#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// -----------------------------
// CUDA helpers
#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) { if (err==hipSuccess) return;
    fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); hipDeviceReset(); exit(EXIT_FAILURE);
}
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
// -----------------------------

__device__ static char* g_seq = NULL;
__device__ static int g_len = 0;

#define my_len g_len
#define my_seq g_seq
#define my_P g_P
#define my_dev __device__
#include "librna_impl.h"

// -----------------------------
// transfer helpers
static paramT *cg_P=NULL;
static char* cg_seq=NULL; // CPU pointers
__global__ static void _initP(paramT* params) { g_P=params; }
void initP() { paramT* P = get_scaled_parameters(); cuMalloc(cg_P,sizeof(paramT)); cuPut(P,cg_P,sizeof(paramT),NULL); _initP<<<1,1>>>(cg_P); free(P); }
void freeP() { cuFree(cg_P); }

__global__ static void _initSeq(char* seq, int len) { g_seq=seq; g_len=len; }
void initSeq(const char* str) {
  size_t i,len=strlen(str); char* seq=(char*)malloc((len+1)*sizeof(char));
  for (i=0;i<len;++i) switch(str[i]) {
    case 'a': seq[i]=1; break; case 'c': seq[i]=2; break; case 'g': seq[i]=3; break; case 'u': seq[i]=4; break;
    default: fprintf(stderr,"Bad character '%c' (%d) in the provided sequence.\n",str[i],str[i]); exit(1);
  }
  seq[len]=0; cuMalloc(cg_seq,len); cuPut(seq,cg_seq,len,NULL); _initSeq<<<1,1>>>(cg_seq,len); free(seq);
}
void freeSeq() { cuFree(cg_seq); }
// -----------------------------


// -----------------------------------------------------------------------------
// EXAMPLE APPLICATION
// -----------------------------------------------------------------------------

#include "vienna/vienna.c"
#include "vienna/energy_par.c"

__global__ void testKern(int* out) {
	*out =
    ext_mismatch_energy(0, 9) + termau_energy(0,9) + // dlr(0,9)
    sr_energy(0,9) + // stack(0,9)
    sr_energy(1,8) + hl_energy(2,7); // hairpin (1,8)

}

int main() {
    read_parameter_file("vienna/rna_turner2004.par");
    initP();
    initSeq("guaugagaua");

    // execution here
    int c;
    int *g;
    cuMalloc(g,sizeof(int));
    testKern<<<1,1>>>(g);
    cuGet(&c,g,sizeof(int),NULL);
    printf("Result = %d\n",c);

    cuFree(g);


    freeSeq();
    freeP();
    return 0;
}
