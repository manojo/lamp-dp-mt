#include "hip/hip_runtime.h"
/* RNA energy library, CUDA wrapper for Vienna-Tables */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "vienna/vienna.h" // paramT

enum base_t { N_BASE, A_BASE, C_BASE, G_BASE, U_BASE, GAP_BASE };
enum iupac_t { N_IUPAC = 0, B_IUPAC = 6, D_IUPAC = 7, H_IUPAC = 8, R_IUPAC = 9, V_IUPAC = 10, Y_IUPAC = 11 };
enum bp_t { N_BP, CG_BP, GC_BP, GU_BP, UG_BP, AU_BP, UA_BP, NO_BP };
typedef unsigned int rsize;

// -----------------------------------------------------------------------------
// Header

__device__ int termau_energy(rsize i, rsize j);
__device__ int hl_energy(rsize i, rsize j);
__device__ int hl_energy_stem(rsize i, rsize j);
__device__ int il_energy(rsize i, rsize j, rsize k, rsize l);
__device__ int bl_energy(rsize bl, rsize i, rsize j, rsize br, rsize Xright);
__device__ int br_energy(rsize bl, rsize i, rsize j, rsize br, rsize Xleft);
__device__ int sr_energy(rsize i, rsize j);
__device__ int sr_pk_energy(char a, char b, char c, char d);
__device__ int dl_energy(rsize i, rsize j);
__device__ int dr_energy(rsize i, rsize j, rsize n);
__device__ int dli_energy(rsize i, rsize j);
__device__ int dri_energy(rsize i, rsize j);
__device__ int ext_mismatch_energy(rsize i, rsize j);
__device__ int ml_mismatch_energy(rsize i, rsize j);
__device__ int ml_energy();
__device__ int ul_energy();
__device__ int sbase_energy();
__device__ int ss_energy(rsize i, rsize j);

__device__ int dl_dangle_dg(enum base_t dangle, enum base_t i, enum base_t j);
__device__ int dr_dangle_dg(enum base_t i, enum base_t j, enum base_t dangle);

__device__ double mk_pf(double x);
__device__ double scale(int x);

__device__ bool iupac_match(enum base_t base, unsigned char iupac_base);

// -----------------------------------------------------------------------------
// Implementation

__device__ paramT *g_P = NULL;
__device__ const char* g_seq = NULL;
__device__ int g_len = 0;

// -----------------------------------------------------------------------------

__device__ static int bp_index(char x, char y) {
  switch (x) {
    case A_BASE: if (y==U_BASE) return AU_BP; break;
    case C_BASE: if (y==G_BASE) return CG_BP; break;
    case G_BASE : switch (y) {
        case C_BASE : return GC_BP;
        case U_BASE : return GU_BP;
      }
      break;
    case U_BASE : switch (y) {
        case G_BASE : return UG_BP;
        case A_BASE : return UA_BP;
      }
      break;
  }
  return NO_BP;
}
#define _bp(i,j) bp_index(g_seq[i],g_seq[j])

__device__ static rsize noGaps(rsize i, rsize j) {
  rsize noGaps=0; for (rsize k=i; k<=j; ++k) if (g_seq[k] == GAP_BASE) ++noGaps;
  return noGaps;
}

__device__ static size_t ungapRegion(rsize i, rsize j, char *ungapped) {
  rsize pos=0; for (rsize y=i; y<=j; ++y) if (g_seq[y] != GAP_BASE) ungapped[pos++] = g_seq[y];
  return pos;
}

__device__ static rsize getNext(rsize pos, rsize steps, rsize rightBorder) {
  //assert(steps>0);
  rsize nongaps=0, x=pos+1; if (x>rightBorder) return rightBorder;
  do { if (g_seq[x] != GAP_BASE) ++nongaps; } while (nongaps < steps && ++x < rightBorder);
  return x;
}

__device__ static rsize getPrev(rsize pos, rsize steps, rsize leftBorder) {
  // assert(pos>0); assert(steps>0);
  rsize nongaps=0, x=pos-1; if (x<=leftBorder) return leftBorder;
  do { if (g_seq[x] != GAP_BASE) ++nongaps; } while (nongaps < steps && --x > leftBorder);
  return x;
}

#define _next(pos,steps,left) g_seq[getNext(pos,steps,left)]
#define _prev(pos,steps,right) g_seq[getPrev(pos,steps,right)]

__device__ static void decode(char *s, const char *x, const int len) {
	unsigned int i;
	for (i = 0; i < len; ++i) switch (x[i]) {
      case 0 : s[i] = 'N'; break;
      case 1 : s[i] = 'A'; break;
      case 2 : s[i] = 'C'; break;
      case 3 : s[i] = 'G'; break;
      case 4 : s[i] = 'U'; break;
      case 5 : s[i] = '_'; break;
      default: s[i] = '?'; //abort();
	}
}

__device__ static int jacobson_stockmayer(rsize l) { return (int)(g_P->lxc*log((l)/(1.0 * MAXLOOP))); }
__device__ static int hl_ent(rsize l) { return (l>MAXLOOP) ? g_P->hairpin[MAXLOOP]+jacobson_stockmayer(l) : g_P->hairpin[l]; }
__device__ static int hl_stack(rsize i, rsize j) { return g_P->mismatchH[_bp(i,j)][_next(i,1,j-1)][_prev(j,1,i+1)]; }

__device__ int termau_energy(rsize i, rsize j) {
  if ((g_seq[i]==G_BASE && g_seq[j]==C_BASE) || (g_seq[i]==C_BASE && g_seq[j]==G_BASE)) return 0;
  else return g_P->TerminalAU;
}

__device__ static char *strstr2(const char *s1, const char *s2) {
  const char* p=s1;
  do {
    int n=0; while(s2[n]==p[n] && p[n]) ++n;
    if (s2[n]==0) return (char*)p;
  } while (*(++p));
  return NULL;
}

__device__ int hl_energy(rsize i, rsize j) {
  // assert(j-i>1);
  rsize size = j-i-1 - noGaps(i+1,j-1);
  int entropy = hl_ent(size);
  int stack_mismatch = hl_stack(i,j);

  if (size < 3) return 600;
  if (size == 3 || size == 4 || size == 6) {
	char ungapped[20]; // XXX: j-i+1
	int sizeUngapped = ungapRegion(i,j,ungapped);
    char loop[20]; // XXX: sizeUngapped+1
    loop[sizeUngapped] = 0;
	decode(loop, ungapped, sizeUngapped);
	if (sizeUngapped == 3+2) {
	  char *ts; loop[5]=0;
	  if ((ts=strstr2(g_P->Triloops, loop))) return (g_P->Triloop_E[(ts - g_P->Triloops)/6]);
	} else if (sizeUngapped == 4+2) { //special tetraloop cases
	  char *ts; loop[6]=0;
	  if ((ts=strstr2(g_P->Tetraloops, loop))) return (g_P->Tetraloop_E[(ts - g_P->Tetraloops)/7]);
	} else if (sizeUngapped == 6+2) { //special hexaloop cases
	  char *ts; loop[8]=0;
	  if ((ts=strstr2(g_P->Hexaloops, loop))) return (g_P->Hexaloop_E[(ts - g_P->Hexaloops)/9]);
	}
  }
  if (size == 3) return entropy + termau_energy(i, j); //normal hairpins of loop size 3
  else return entropy + stack_mismatch; //normal hairpins of loop sizes larger than three
}

__device__ int hl_energy_stem(rsize i, rsize j) {
  int r = hl_energy(i,j);
  rsize size = j-i-1 - noGaps(i+1,j-1);
  if (size >= 4) return r - hl_stack(i,j);
  return r;
}

__device__ static int il11_energy(rsize i, rsize k, rsize l, rsize j) {
  int enclosedBP = _bp(getPrev(j,2,l),getNext(i,2,k)); //we know that the enclosed base pair is at exactly this position, since both unpaired regions have size 1.  Note, basepair is reversed to preserver 5'-3' order.
  return g_P->int11[_bp(i,j)][enclosedBP][_next(i,1,k)][_prev(j,1,l)];
}

__device__ static int il12_energy(rsize i, rsize k, rsize l, rsize j) {
  int enclosedBP = bp_index(_prev(j,3,l), _next(i,2,k));
  return g_P->int21[_bp(i,j)][enclosedBP][_next(i,1,k)][_prev(j,2,l)][_prev(j,1,l)];
}

__device__ static int il21_energy(rsize i, rsize k, rsize l, rsize j) {
  int closingBP = bp_index(_prev(j,2,l), _next(i,3,k));
  return g_P->int21[closingBP][_bp(i,j)][_prev(j,1,l)][_next(i,1,k)][_next(i,2,k)];
}

__device__ static int il22_energy(rsize i, rsize k, rsize l, rsize j) {
  int enclosedBP = bp_index(_prev(j,3,l), _next(i,3,k));
  return g_P->int22[_bp(i,j)][enclosedBP][_next(i,1,k)][_next(i,2,k)][_prev(j,2,l)][_prev(j,1,l)];
}

__device__ static int il_ent(rsize l) { // assert(l>1);
  return g_P->internal_loop[MAXLOOP] + (l > MAXLOOP ? jacobson_stockmayer(l) : 0);
}

__device__ static int il_stack(rsize i, rsize k, rsize l, rsize j) { // Note, basepair and stacking bases are reversed to preserver 5'-3' order
  return g_P->mismatchI[_bp(i,j)][_next(i,1,j-1)][_prev(j,1,i+1)] + g_P->mismatchI[_bp(l,k)][_next(l,1,j-1)][_prev(k,1,i+1)];
}

__device__ static int il_asym(rsize sl, rsize sr) {
  int r = abs((int)sl-(int)sr) * g_P->ninio[0];
  return (r < g_P->ninio[1]) ? r : g_P->ninio[1];
}

__device__ int il_energy(rsize i, rsize k, rsize l, rsize j) {
  rsize sl = k-i-1 - noGaps(i+1, k-1);
  rsize sr = j-l-1 - noGaps(l+1, j-1);

  int out_closingBP = _bp(i,j);
  int out_lbase = _next(i,1,j-1);
  int out_rbase = _prev(j,1,i+1);
  int in_closingBP = _bp(l,k); // Note, basepair and stacking bases are reversed to preserver 5'-3' order
  int in_lbase = _next(l,1,j-1);
  int in_rbase = _prev(k,1,i+1);

  if (sl == 0) return br_energy(i, l+1, j-1, j, k); //internal loop really is an right bulge, because left unpaired region is just a gap
  if (sr == 0) return bl_energy(i, i+1, k-1, j, l); //internal loop really is an left bulge, because right unpaired region is just a gap

  if (sl == 1) {
	if (sr == 1) return il11_energy(i, k, l, j);
	else if (sr == 2) return il12_energy(i, k, l, j);
	else return il_ent(sl+sr) + il_asym(sl,sr) + g_P->mismatch1nI[out_closingBP][out_lbase][out_rbase] + g_P->mismatch1nI[in_closingBP][in_lbase][in_rbase];
  } else if (sl == 2) {
	if (sr == 1) return il21_energy(i, k, l, j);
	else if (sr == 2) return il22_energy(i, k, l, j);
	else if (sr == 3) return g_P->internal_loop[5]+g_P->ninio[0] + g_P->mismatch23I[out_closingBP][out_lbase][out_rbase] + g_P->mismatch23I[in_closingBP][in_lbase][in_rbase];
  } else if ((sl == 3) && (sr == 2)) {
	return g_P->internal_loop[5]+g_P->ninio[0] + g_P->mismatch23I[out_closingBP][out_lbase][out_rbase] + g_P->mismatch23I[in_closingBP][in_lbase][in_rbase];
  } else if (sr == 1) {
    return il_ent(sl+sr) + il_asym(sl,sr) + g_P->mismatch1nI[out_closingBP][out_lbase][out_rbase] + g_P->mismatch1nI[in_closingBP][in_lbase][in_rbase];
  }
  return il_ent(sl+sr) + il_stack(i, k, l, j) + il_asym(sl, sr);
}

__device__ static int bl_ent(rsize l) {
  // assert(l>0);
  if (l>MAXLOOP) return g_P->bulge[MAXLOOP] + jacobson_stockmayer(l);
  else return g_P->bulge[l];
}

__device__ int bl_energy(rsize i, rsize k, rsize l, rsize j, rsize Xright) {
  // assert(j >= 2); // this is of no biological relevance, just to avoid an underflow
  rsize size = l-k+1 - noGaps(k, l);
  if (size==0) return g_P->stack[_bp(i,j)][_bp(getPrev(j,1,Xright),l+1)];
  if (size==1) return bl_ent(size) + g_P->stack[_bp(i,j)][_bp(getPrev(j,1,Xright),l+1)];
  if (size>1) return bl_ent(size) + termau_energy(i,j) + termau_energy(getPrev(j,1,Xright), l+1);
  return -1000000; // error
}

__device__ int br_energy(rsize i, rsize k, rsize l, rsize j, rsize Xleft) {
  // assert(j >= 1); // this is of no biological relevance, just to avoid an underflow
  rsize size = l-k+1 - noGaps(k, l);
  if (size == 0) return g_P->stack[_bp(i,j)][_bp(k-1,getNext(i,1,Xleft))];
  if (size == 1) return bl_ent(size) + g_P->stack[_bp(i,j)][_bp(k-1,getNext(i,1,Xleft))];
  if (size > 1) return bl_ent(size) + termau_energy(i, j) + termau_energy(k-1, getNext(i,1,Xleft));
  return -1000000; // error
}

__device__ int sr_energy(rsize i, rsize j) { return g_P->stack[_bp(i,j)][_bp(j-1,i+1)]; }
__device__ int sr_pk_energy(char a, char b, char c, char d) { return g_P->stack[_bp(a,b)][_bp(d,c)]; }

__device__ int dl_energy(rsize i, rsize j) {
  if (i == 0) return 0;
  int dd = g_P->dangle5[_bp(i,j)][_prev(i,1,0)];
  return (dd>0) ? 0 : dd; // must be <= 0
}

__device__ int dr_energy(rsize i, rsize j, rsize n) {
  if ((j+1) >= n) return 0;
  int dd = g_P->dangle3[_bp(i,j)][_next(j,1,n)];
  return (dd>0) ? 0 : dd; // must be <= 0
}

__device__ int dli_energy(rsize i, rsize j) {
  int dd = g_P->dangle3[_bp(j,i)][_next(i,1,j-1)];
  return (dd>0) ? 0 : dd; // must be <= 0
}

__device__ int dri_energy(rsize i, rsize j) {
  int dd = g_P->dangle5[_bp(j,i)][_prev(j,1,i+1)];
  return (dd>0) ? 0 : dd; // must be <= 0
}

__device__ int ext_mismatch_energy(rsize i, rsize j) {
  if ((i > 0) && ((j+1) < g_len)) return g_P->mismatchExt[_bp(i,j)][_prev(i,1,0)][_next(j,1,g_len)];
  else if (i > 0) return dl_energy(i,j);
  else if ((j+1) < g_len) return dr_energy(i,j,g_len);
  else return 0;
}

__device__ int ml_mismatch_energy(rsize i, rsize j) {
  return g_P->mismatchM[_bp(j,i)][_prev(j,1,i+1)][_next(i,1,j-1)]; // Note, basepairs and stacking bases are reversed to preserver 5'-3' order
}

__device__ int ml_energy() { return g_P->MLclosing; }
__device__ int ul_energy() { return g_P->MLintern[0]; }
__device__ int sbase_energy() { return 0; }
__device__ int ss_energy(rsize i, rsize j) { return 0; }

__device__ double mk_pf(double x) { return exp((-1.0 * x/100.0) / (GASCONST/1000 * (g_P->temperature + K0))); }

__device__ double scale(int x) {
  double mean_nrg= -0.1843;  // mean energy for random sequences: 184.3*length cal
  double mean_scale = exp (-1.0 * mean_nrg / (GASCONST/1000 * (g_P->temperature + K0)));
  return (1.0 / pow(mean_scale, x));
}

__device__ int dl_dangle_dg(enum base_t dangle, enum base_t i, enum base_t j) {
  int dd = g_P->dangle5[_bp(i,j)][dangle]; return (dd>0) ? 0 : dd;  // must be <= 0
}

__device__ int dr_dangle_dg(enum base_t i, enum base_t j, enum base_t dangle) {
  int dd = g_P->dangle3[_bp(i,j)][dangle]; return (dd>0) ? 0 : dd;  // must be <= 0
}

// added by gsauthof, 2012
__device__ static const bool map_base_iupac[5][12] = {
    /*      { N    , A     , C     , G     , U     , _     , B     , D     , H     , R     , V     , Y     } */
    /* N */ { true , true  , true  , true  , true  , true  , true  , true  , true  , true  , true  , true  }  ,
    /* A */ { true , true  , false , false , false , false , false , true  , true  , true  , true  , false }  ,
    /* C */ { true , false , true  , false , false , false , true  , false , true  , false , true  , true  }  ,
    /* G */ { true , false , false , true  , false , false , true  , true  , false , true  , true  , false }  ,
    /* U */ { true , false , false , false , true  , false , true  , true  , true  , false , false , true  }  ,
};

__device__ bool iupac_match(enum base_t base, unsigned char iupac_base) {
  // assert(iupac_base<12);
  return map_base_iupac[base][iupac_base];
}

// -----------------------------------------------------------------------------
// EXAMPLE APPLICATION
// -----------------------------------------------------------------------------

#include "vienna/vienna.c"
#include "vienna/energy_par.c"

// -----------------------------
#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) { if (err==hipSuccess) return;
	fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); hipDeviceReset(); exit(EXIT_FAILURE);
}
// Device memory
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
// -----------------------------

static paramT *cg_P=NULL;
static char* cg_seq=NULL; // CPU pointers
__global__ void _initP(paramT* params) { g_P=params; }
void initP() { paramT* P = get_scaled_parameters(); cuMalloc(cg_P,sizeof(paramT)); cuPut(P,cg_P,sizeof(paramT),NULL); _initP<<<1,1>>>(cg_P); free(P); }
void freeP() { cuFree(cg_P); }

__global__ void _initSeq(const char* seq, int len) { g_seq=seq; g_len=len; }
void initSeq(const char* str) {
  size_t i,len=strlen(str); char* seq=(char*)malloc((len+1)*sizeof(char));
  for (i=0;i<len;++i) switch(str[i]) {
    case 'a': seq[i]=1; break; case 'c': seq[i]=2; break; case 'g': seq[i]=3; break; case 'u': seq[i]=4; break;
    default: fprintf(stderr,"Bad character '%c' (%d) in the provided sequence.\n",str[i],str[i]); exit(1);
  }
  seq[len]=0; cuMalloc(cg_seq,len); cuPut(seq,cg_seq,len,NULL); _initSeq<<<1,1>>>(cg_seq,len); free(seq);
}
void freeSeq() { cuFree(cg_seq); }

int main() {
	read_parameter_file("vienna/rna_turner2004.par");
	initP();
	initSeq("guaugagaua");
	// execution here
	freeSeq();
	freeP();
	return 0;
}
