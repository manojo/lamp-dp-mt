#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// -----------------------------
// CUDA helpers
#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) { if (err==hipSuccess) return;
    fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); hipDeviceReset(); exit(EXIT_FAILURE);
}
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
// -----------------------------

#include "vienna/vienna.h" // paramT

#define my_len g_len
#define my_seq g_seq
#define my_P g_P
#define my_dev __device__
#include "librna_impl.h"

// -----------------------------
// transfer helpers
static paramT *cg_P=NULL;
static char* cg_seq=NULL; // CPU pointers
__global__ static void _initP(paramT* params) { g_P=params; }
void initP() { paramT* P = get_scaled_parameters(); cuMalloc(cg_P,sizeof(paramT)); cuPut(P,cg_P,sizeof(paramT),NULL); _initP<<<1,1>>>(cg_P); free(P); }
void freeP() { cuFree(cg_P); }

__global__ static void _initSeq(char* seq, int len) { g_seq=seq; g_len=len; }
void initSeq(const char* str) {
  size_t i,len=strlen(str); char* seq=(char*)malloc((len+1)*sizeof(char));
  for (i=0;i<len;++i) switch(str[i]) {
    case 'a': seq[i]=1; break; case 'c': seq[i]=2; break; case 'g': seq[i]=3; break; case 'u': seq[i]=4; break;
    default: fprintf(stderr,"Bad character '%c' (%d) in the provided sequence.\n",str[i],str[i]); exit(1);
  }
  seq[len]=0; cuMalloc(cg_seq,len); cuPut(seq,cg_seq,len,NULL); _initSeq<<<1,1>>>(cg_seq,len); free(seq);
}
void freeSeq() { cuFree(cg_seq); }
// -----------------------------


// -----------------------------------------------------------------------------
// EXAMPLE APPLICATION
// -----------------------------------------------------------------------------

#include "vienna/vienna.c"
#include "vienna/energy_par.c"

int main() {
    read_parameter_file("vienna/rna_turner2004.par");
    initP();
    initSeq("guaugagaua");
    // execution here
    freeSeq();
    freeP();
    return 0;
}
