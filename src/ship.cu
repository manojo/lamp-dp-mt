#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdint.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>


// This is a performance tes for shipping data to GPU

#include "mrand.h"

#define _inline __attribute__((unused)) static inline
#define ASSERT(cond) extern int __assert__[1-2*(!(cond))];

// -----------------------------------------------------------------------------

#define cuErr(err) (hipError_t(err,__FILE__,__LINE__))
#define cuSync(stream) cuErr(hipStreamSynchronize(stream))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
_inline void hipError_t(hipError_t err, const char *file,  int line) { if (err!=hipSuccess) { fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); exit(EXIT_FAILURE); } }
_inline void cuInfo(bool full=true) {
	int deviceCount=0; hipError_t err=hipGetDeviceCount(&deviceCount);
	if (err==38 || deviceCount==0) { fprintf(stderr,"No CUDA device\n"); exit(EXIT_FAILURE); }
	else if (err!=hipSuccess) { fprintf(stderr,"CUDA error %d: %s.\n", err, hipGetErrorString(err)); exit(EXIT_FAILURE); }
	int driverVersion=0, runtimeVersion=0; hipDriverGetVersion(&driverVersion); hipRuntimeGetVersion(&runtimeVersion);
	printf("Found %d CUDA device(s), driver %d.%d, runtime %d.%d.\n", deviceCount, driverVersion/1000, driverVersion%100, runtimeVersion/1000, runtimeVersion%100);
	for (int dev=0; dev<deviceCount; ++dev) {
		hipDeviceProp_t prop; hipGetDeviceProperties(&prop, dev);
		hipDeviceReset();
		printf("- Device %d: '%s' (capability %d.%d, watchdog %s)\n", dev, prop.name, prop.major, prop.minor, prop.kernelExecTimeoutEnabled?"on":"off");
		if (full) {
			printf("    - Memory    : %dMb @ %dMhz mapHost=%d, unifiedAddr=%d, asyncCopy=%d\n",(int)round(prop.totalGlobalMem/1048576.0),
				prop.memoryClockRate>>10,prop.canMapHostMemory,prop.unifiedAddressing,prop.asyncEngineCount);
			int m=prop.computeMode; const char* mode=(m==0?"default":(m==1?"exclusive":(m==2?"prohibited":"exclusiveProcess")));
			printf("    - Processors: %d @ %dMHz, maxThreads=%d, warp=%d, concurrency=%d, mode=%s\n",prop.multiProcessorCount,
				prop.clockRate>>10,prop.maxThreadsPerMultiProcessor,prop.warpSize,prop.concurrentKernels,mode);
			printf("    - Limits    : %d regs/block, %ldK sharedMem/proc, %d thr/block %d thr/proc, %d blocks\n",prop.regsPerBlock,
				prop.sharedMemPerBlock>>10,prop.maxThreadsPerBlock,prop.maxThreadsPerMultiProcessor,prop.maxGridSize[0]);
		}
	}
}


#define OP(a,b,idx) { int d=a[idx]-b[idx]; a[idx]+=b[idx]; b[idx]=d<0?-d:d; }

__global__ void fun(int* a, int* b, unsigned size) {
	unsigned idx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	if (idx<size) { OP(a,b,idx) }
}

int main(int argc,char** argv) {
	const unsigned size=3<<24; // max for 512MB GPU
	const unsigned int ms=size*sizeof(int);

	// 3<<24 = 48M pairs of (4+4) chars => 384Mb data to and back from GPU
	// Note that it goes faster with -d64 on my CPU (!)

	int* ca = (int*)malloc(ms);
	int* cb = (int*)malloc(ms);
	int* ga; cuErr(hipMalloc(&ga,ms));
	int* gb; cuErr(hipMalloc(&gb,ms));

/*
With -m64
CPU time: 330.254 ms <- cold caches
GPU time: 511.160 ms
CPU time: 110.565 ms
GPU time: 510.759 ms
CPU time: 106.342 ms
GPU time: 374.137 ms
CPU time: 110.514 ms
GPU time: 285.962 ms
CPU time: 105.509 ms
GPU time: 286.003 ms
CPU time: 105.665 ms
GPU time: 286.128 ms
CPU time: 106.243 ms
GPU time: 285.868 ms
CPU time: 108.147 ms
GPU time: 285.921 ms
CPU time: 105.941 ms
GPU time: 285.827 ms
CPU time: 107.504 ms
GPU time: 285.893 ms <- hot caches 2.6x slowdown vs CPU

Without -m64
CPU time: 417.344 ms
GPU time: 511.177 ms
CPU time: 149.060 ms
GPU time: 510.771 ms
CPU time: 149.459 ms
GPU time: 360.264 ms
CPU time: 147.718 ms
GPU time: 286.457 ms
CPU time: 147.381 ms
GPU time: 285.966 ms
CPU time: 150.956 ms
GPU time: 283.353 ms
CPU time: 146.627 ms
GPU time: 283.340 ms
CPU time: 149.717 ms
GPU time: 283.385 ms
CPU time: 146.716 ms
GPU time: 283.412 ms
CPU time: 146.732 ms
GPU time: 286.180 ms <- 1.95x slowdown vs CPU

fun() only, -m64
CPU time: 332.379 ms
GPU time: 0.062 ms
CPU time: 104.286 ms
GPU time: 0.027 ms
CPU time: 107.368 ms
GPU time: 0.025 ms
CPU time: 104.345 ms
GPU time: 0.028 ms
CPU time: 104.347 ms
GPU time: 0.027 ms
CPU time: 103.969 ms
GPU time: 0.026 ms
CPU time: 115.863 ms
GPU time: 0.025 ms
CPU time: 132.301 ms
GPU time: 0.026 ms
CPU time: 126.471 ms
GPU time: 0.027 ms
CPU time: 107.523 ms
GPU time: 0.024 ms
*/

for (int i=0;i<10;++i) {
	struct timeval ts,te;
	gettimeofday(&ts, NULL);
	for (unsigned i=0;i<size;++i) { OP(ca,cb,i) }

	gettimeofday(&te, NULL);
	printf("CPU time: %.3f ms\n", (float)( 1000.0*(te.tv_sec-ts.tv_sec) + 0.001*(te.tv_usec-ts.tv_usec) ) );

	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));

	gettimeofday(&ts, NULL);

	cuPut(ca,ga,ms,stream);
	cuPut(cb,gb,ms,stream);
	fun<<<size/32, 32, 0, stream>>>(ga,gb,size);

	cuGet(ca,ga,ms,stream);
	cuGet(cb,gb,ms,stream);
	cuSync(stream);

	gettimeofday(&te, NULL);
	printf("GPU time: %.3f ms\n", (float)( 1000.0*(te.tv_sec-ts.tv_sec) + 0.001*(te.tv_usec-ts.tv_usec) ) );
}

	free(ca);
	free(cb);
	hipFree(ga);
	hipFree(gb);

	hipDeviceReset();
	return 0;
}