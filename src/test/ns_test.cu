#include "hip/hip_runtime.h"
#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 1024LU  // matrix dimension
#define M_H 1024LU  // matrix dimension

// -----------------------------------------------------------------------------
#include "include/ns_prob.h" // problem definitions
#include "include/ns.h"      // common functions
#include "include/ns_cpu.h"  // cpu implementation
#include "include/ns_gpu.h"  // gpu implementation
// -----------------------------------------------------------------------------

__global__ void gpu_solve(TI* in0, TI* in1, TC* cost, TB* back, volatile unsigned* lock) {
	const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x; // * (  + blockIdx.y*gridDim.x );
	const unsigned tN = blockDim.x * gridDim.x;
	const unsigned tB = blockIdx.x;
	unsigned tP=0; // block progress

#ifdef SH_RECT
	// 1 block = 88ms, 128x128, correct, down to 30ms with multi-blocks
	// 1043ms for 1024x1024 => we are above "Optimizing DP on GPU via adaptive thread parallelism"
	// => we need to use the same technique as they do (but first compare on the same problem)
	const unsigned jjN = tN + M_W;
	for (unsigned i=tI; i<M_H; i+=tN) for (unsigned jj=0; jj<jjN; ++jj) {
		unsigned j = jj-tI;
		if (j<M_W) {


	//for (unsigned i=0; i<M_H; ++i) { // 560ms, 128x128
	//	for (unsigned j=0; j<M_W; ++j) {
#endif
#ifdef SH_TRI
	// not sure we want to proceed by //gram-blocks or by diagonal
	for (unsigned ii=0; ii<M_H; ++ii) {
		unsigned i=M_H-1-ii;
		for (unsigned j=i; j<M_W; ++j) {
#endif
#ifdef SH_PARA
	for (unsigned jj=0; jj<M_W; ++jj) {
		for (unsigned i=0; i<M_H; ++i) {
			unsigned j=jj+i;
#endif

			TB b=BT_STOP; TC c=0,c2; // stop
			if (!INIT(i,j)) {
				p_kernel
			}
			cost[idx(i,j)] = c;
			back[idx(i,j)] = b;
		}
		// Sync between blocks
		__threadfence();
		if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
		__syncthreads();
	}
}

void g_solve() {
	unsigned blk_size = 32;
	unsigned blk_num = (M_H+blk_size-1)/blk_size;

	unsigned* lock;
	cuMalloc(lock,sizeof(unsigned)*blk_num);
	cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
	gpu_solve<<<blk_num, blk_size, 0, NULL>>>(g_in[0], g_in[1], g_cost, g_back, lock);
	cuFree(lock);
}

// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	// CPU solving
	for (int i=0;i<2;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");

	// GPU solving
	for (int i=0;i<2;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");

	dbg_compare();
	// XXX: also compare backtrack

	//dbg_print(false,stdout);
	//dbg_track(false,stdout);

	dbg_cleanup();
	return 0;
}
