#include "hip/hip_runtime.h"
#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 1024LU  // matrix dimension
#define M_H 1024LU  // matrix dimension

// -----------------------------------------------------------------------------
#include "include/ns_prob.h" // problem definitions
#include "include/ns.h"      // common functions
#include "include/ns_cpu.h"  // cpu implementation
#include "include/ns_gpu.h"  // gpu implementation
// -----------------------------------------------------------------------------

/*
PRELIMINARY RESULTS (i7-3720QM-1core vs GeForce GT 650M-attached)
-----------------------------------------------------------------


*/

__global__ void gpu_solve(const TI* in0, const TI* in1, TC* cost, TB* back, volatile unsigned* lock) {
	const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x; // * (  + blockIdx.y*gridDim.x );
	const unsigned tN = blockDim.x * gridDim.x;
	const unsigned tB = blockIdx.x;
	unsigned tP=0; // block progress
	/*
	 * Optimizations to implement when we are confident about the problem structure:
	 * SH_RECT: Since we sweep the rectangle with a diagonal, at iteration M_W+(tN+1)/2
	 *          half of them will we unused. At this point, break the loop and construct
	 *          a new loop where 2 threads are assigned the same cell, that is thread tI
	 *          is assigned to cell (i/2, j). We would require tI/2 shared cost cells to
	 *          exchange maximal cost, the cell with cost=maximum writes (cost,backtrack)
	 *          to the original matrices.
	 * SH_TRI : Similarly, when half of the threads go out of the triangle, we can reassign
	 *          two threads per cell, then repeat the operation at 4 and 8 (possibly 16 and 32).
	 * SH_PARA: No optimization possible since every pair of thread has very different dependences
     */

#ifdef SH_RECT
	// 1 block = 88ms, 128x128, correct, down to 30ms with multi-blocks
	// 1043ms for 1024x1024 => we are above "Optimizing DP on GPU via adaptive thread parallelism"
	// => we need to use the same technique as they do (but first compare on the same problem)
	const unsigned jjN = tN + M_W;
	for (unsigned i=tI; i<M_H; i+=tN) {
		for (unsigned jj=0; jj<jjN; ++jj) {
			unsigned j = jj-tI;
			if (j<M_W) {
	//for (unsigned i=0; i<M_H; ++i) {
	//	for (unsigned j=0; j<M_W; ++j) { {
#endif
#ifdef SH_TRI
	for (unsigned ii=tI; ii<M_H; ii+=tN) {
		unsigned i=M_H-1-ii;
		for (unsigned j=i; j<M_W+i; ++j) {
			if (j<M_W) {
	//for (unsigned ii=0; ii<M_H; ++ii) { unsigned i=M_H-1-ii;
	//	for (unsigned j=i; j<M_W; ++j) { {
#endif
#ifdef SH_PARA
	for (unsigned jj=0; jj<M_W; ++jj) {
		for (unsigned i=tI; i<M_H; i+=tN) {
			unsigned j=jj+i;
			{
	//for (unsigned jj=0; jj<M_W; ++jj) {
	//	for (unsigned i=0; i<M_H; ++i) { unsigned j=jj+i; {
#endif
				TB b=BT_STOP; TC c=0,c2; // stop
				if (!INIT(i,j)) { p_kernel }
				cost[idx(i,j)] = c;
				back[idx(i,j)] = b;
			}
			// Sync between blocks, removing __threadfence() is incorrect but works
			// __threadfence(); 
			#ifdef SH_PARA // wait for all blocks
			__syncthreads();
			++tP; if (threadIdx.x==0) lock[tB]=tP;
			for (unsigned b=threadIdx.x;b<gridDim.x;b+=blockDim.x) { // sync with all blocks
				while(lock[(tB+b)%gridDim.x]<tP) {}
			}
			#else // wait for previous block only
			if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
			#endif
			__syncthreads();
		}
	}
}

void g_solve() {
	unsigned blk_size = 32; // = warp size
	unsigned blk_num = (M_H+blk_size-1)/blk_size;
	unsigned* lock;
	cuMalloc(lock,sizeof(unsigned)*blk_num);
	cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
	gpu_solve<<<blk_num, blk_size, 0, NULL>>>(g_in[0], g_in[1], g_cost, g_back, lock);
	cuFree(lock);
}

// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);
	// GPU solving
	for (int i=0;i<10;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");

	dbg_compare();
	// XXX: also compare backtrack

	//dbg_print(false,stdout);
	//dbg_track(false,stdout);

	dbg_cleanup();
	return 0;
}
