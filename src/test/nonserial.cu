#include "hip/hip_runtime.h"
#include "include/common.h"

// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
// System parameters
// - Data types
#define TI char    // input data type
#define TC int     // cost type
#define TB char    // backtrack type
#define TW int     // wavefront type (if not defined, no wavefront)
// - Problem dimensions
#define B_W 8LU    // block width
#define B_H 4LU    // block height
#define M_W 79LU   // matrix dimension
#define M_H 37LU   // matrix dimension
// - Problem shape (RECT, TRIANG, PARALL)
#define SH_RECT
//#define SH_TRI
//#define SH_PARA
// Initialization
#define INIT(i,j)  ((i)==0 || (j)==0) // matrix initialization at [stop]

// Problem definition SWat(S,T) with arbitrary gap cost.
// By convention, S is the longest string and put vertically
// M[i,j] = max { 0                                 stop } = B[i,j]
//              { M[i-1,j-1]+cost(S[i],T[j])        NW   }
//              { max{1<=k<=j-1} M[i,j-k] - gap(k)  N(k) }
//              { max{1<=k<=i-1} M[i-k,j] - gap(k)  W(k) }
//

// Input string that can be free()d
TI* p_input(bool horz=false) {
	static unsigned sh=time(NULL), sv=time(NULL)+573; // keep consistent
	const char alph[4]={'A','C','G','T'};
	unsigned n = horz?M_W:M_H; mseed(horz?sh:sv);
	TI* in = (char*)malloc(n*sizeof(TI));
	for (unsigned i=0;i<n;++i) in[i]=alph[mrand()%4];
	return in;
}
// problem specific helpers
inline TC p_gap(int k) { return 20-k; }
inline TC p_cost(char s, char t) { return s==t?1:0; }
// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------

#include "include/nonserial.h" // must be included after problem definition

// XXX: shall we assume that we could go sub-byte granularity for types ?
// XXX: need a full rework here, depending the configuration

// XXX: the memory is corrupted at some point, the full matrix is not initialized properly
inline unsigned idx(unsigned i, unsigned j) { return B_H*((j%M_W)+(i%B_H)) + (i%B_H) + (i/B_H)*M_W*B_H; }

void c_solve() {
	for (size_t i=0; i<M_H; ++i) {
		for (size_t j=0; j<M_W; ++j) {
			if (INIT(i,j)) { c_back[idx(i,j)]=BSTOP; c_cost[idx(i,j)]=0; }
			else {
				TB b=BSTOP; TC c=0,c2;  // stop

				b=2;

				/*
				for (size_t k=1; k<j; ++k) { c2=c_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=c_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = c_cost[idx(i-1,j-1)]+p_cost(c_in[0][i],c_in[1][j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,0); }
				*/
				c_cost[idx(i,j)] = c;
				c_back[idx(i,j)] = b;
			}
		}
	}
}

// simply return the pair of indices (i,j) that are in the backtrack
void c_backtrack() {
	// XXX: separate find max and backtrack itself ?
}

void c_print(FILE* f) {
	fprintf(f,"Matrix(%ldx%ld), blocks(%ldx%ld)\n",M_H,M_W,B_H,B_W);
	fprintf(f,"  |");
	// header
	for (size_t j=0;j<M_W;++j) { fprintf(f," %c",c_in[1][j]); if (j%B_W==B_W-1) fprintf(f," |"); }
	fprintf(f,"\n");
	for (size_t i=0;i<M_H;++i) {
		// spacer
		if (i%B_H==0) {
			fprintf(f,"--+");
			for (size_t j=0;j<M_W;++j) { fprintf(f,"--"); if (j%B_W==B_W-1) fprintf(f,"-+"); }
			fprintf(f,"\n");
		}
		// content
		fprintf(f,"%c |",c_in[0][i]);
		for (size_t j=0;j<M_W;++j) {
			char d[]={'#','|','-','\\'};
			fprintf(f," %c",d[BD(c_back[idx(i,j)])]);
			if (j%B_W==B_W-1) fprintf(f," |");
		}
		fprintf(f,"\n");
	}
}

// -----------------------------------------------------------------------------

// GPU structures

// - allocate memory
// - access memory[cost,backtrack,wavefront,input] at index (i,j)

// -----------------------------------------------------------------------------

/*
*/


int main(int argc, char** argv) {
	c_init();
	c_solve();
	c_print(stdout);
	c_free();


//	g_init();
//	g_free();



/*
	#ifdef SH_RECT
	printf("rectangle\n");
	#endif
	#ifdef SH_TRI
	printf("triangle\n");
	#endif
	#ifdef SH_PARA
	printf("parallelogram\n");
	#endif
*/

	return 0;
}


#if 0

// macro helpers
#define B_LEN(n) (B_H*B_W*(n) + B_H*(B_H-1)-1) // memory length for N contiguous(line) blocks
#define B_IN(i,j) ( ((i)/B_H)*M_W*B_H + ((j)/B_W)*B_W*B_H ) // returns offset of the block containing (i,j)
#define B_AT(bi,bj) ( (bi)*M_W*B_H + (bj)*B_W*B_H ) // return the block (bi,bj)

inline off_t idx(size_t i, size_t j) { return B_H*(j+(i%B_H)) + (i%B_H) + (i/B_H)*M_W*B_H; }

typedef struct {
	unsigned bi, bj;     // block offset
	unsigned mi,mj;     // valid content maximal position
	const TI* in[2];  // related input 0=vert(S),1=horz(T)
	TC* cost;         // cost matrix
	TB* back;         // backtrack matrix
	//bool device;    // whether valid memory is CUDA or CPU memory
	bool wr_back;     // whether memory needs to be written back
} blk_t;

// -----------------------------------------------------------------------------
// Memory manager and problem data structures
// > all-in-main-memory implementation

const TI* g_in[2]={NULL,NULL}; // 0=vert(S), 1=horizontal(T) with |S| >= |T|
TC* g_cost=NULL;
TB* g_back=NULL;
#ifdef TW
TW* g_wave[3]={NULL,NULL,NULL}; // 0=vert,1=horizontal,2=diagonal
#endif

// initialize structures
void init() {
	size_t mem = M_W*M_H+B_H*B_H;
	g_cost=(TC*)malloc(sizeof(TC)*mem);
	g_back=(TB*)malloc(sizeof(TB)*mem);
	#ifdef TW
		g_wave[0]=(TW*)malloc(sizeof(TW)*C_H);
		g_wave[1]=(TW*)malloc(sizeof(TW)*C_W);
		g_wave[2]=(TW*)malloc(sizeof(TW)*MAX(M_H/B_H,M_W/B_W));
	#endif
	g_in[0]=p_S; // also duplicate into CUDA
	g_in[1]=p_T;
}

// cleanup structures
void cleanup() {
	if (g_cost) free(g_cost); g_cost=NULL;
	if (g_back) free(g_back); g_back=NULL;
	#ifdef TW
	for (int i=0;i<3;++i) if (g_wave[i]) { free(g_wave[i]); g_wave[i]=NULL; }
	#endif
}

// memory manager
// XXX: keep track of allocated zones, both function must be mutex-protected
// XXX: we need an atomic list of cost blocks, of input blocks and of backtrack blocks
// pthread_mutex_t* mutex;
// typedef struct { off_t bi,bj; bool cost; bool dev; void* ptr; unsigned retained; } mem_t;
	// XXX: test whether these zone were already allocated in CUDA memory to
	// avoid using duplicate memory
 // all these loads need to be atomically done, also released on demand
 // ==> allocate an array of pointers (both main and device mem) and do atomic CAS on them to get pointer + counter
 // See http://www.boost.org/doc/libs/1_39_0/boost/interprocess/detail/atomic.hpp
 // instead take into account blocks that have been written to disk so that reloading into memory is easier
 // also note that writing on disk must reorder differently to avoid writing dirty surrounding data
void* mm_alloc(off_t bi, off_t bj, bool cost=true, bool device=false) {
	// attempt to alloc, if cannot, then try to free all the blocks with 0 retain,
	// possibly write-back to disk, then try again
	if (cost) return &g_cost[B_AT(bi,bj)];
	else return &g_back[B_AT(bi,bj)];
}
void mm_free(void* ptr) {}

blk_t blk_get(off_t bi, off_t bj, bool device=false) {
	blk_t b; b.bi=bi; b.bj=bj;
	b.mi=C_H-bi*B_H; if (b.mi>B_H) b.mi=B_H;
	b.mj=C_W-bj*B_W; if (b.mj>B_W) b.mj=B_W;
	b.in[0]=&g_in[0][bi*B_H]; // XXX: depends whether we're on device
	b.in[1]=&g_in[1][bj*B_W];
	b.wr_back=false;
	b.cost=(TC*)mm_alloc(bi,bj,true);
	b.back=(TB*)mm_alloc(bi,bj,false);
	return b;
}

void blk_free(blk_t* blk) {
	if (blk->wr_back) {
		// XXX: write-back to main memory or to storage
	}
	mm_free(blk->cost);
	mm_free(blk->back);
}

void print(FILE* f) {
	fprintf(f,"Matrix(%ldx%ld), data(%ldx%ld), blocks(%ldx%ld)\n",M_H,M_W,C_H,C_W,B_H,B_W);
	fprintf(f,"  ");
	// header
	for (size_t j=0;j<M_W;++j) { fprintf(f," %c",j<C_W?(char)g_in[1][j]:'#'); if (j%B_W==B_W-1) fprintf(f," |"); }
	fprintf(f,"\n");
	for (size_t i=0;i<M_H;++i) {
		// content
		fprintf(f," %c",i<C_H?(char)g_in[0][i]:'#');
		for (size_t j=0;j<M_W;++j) {
			if (i>=C_H||j>=C_W) fprintf(f," .");
			else {
				// TABLE CONTENT
				char c = g_back[idx(i,j)]; fprintf(f," %c",c?c:' ');
				//fprintf(f,"  ");
			}
			if (j%B_W==B_W-1) fprintf(f," |");
		}
		fprintf(f,"\n");
		// spacer
		if (i%B_H==B_H-1) {
			fprintf(f,"--");
			for (size_t j=0;j<M_W;++j) { fprintf(f,"--"); if (j%B_W==B_W-1) fprintf(f,"-+"); }
			fprintf(f,"\n");
		}
	}
}

// -----------------------------------------------------------------------------
// reference implementation

void solve1() {
	// Recurrence (embeds initialization)
	for (size_t i=0; i<C_H; ++i) {
		for (size_t j=0; j<C_W; ++j) {
			if (i==0 || j==0) {
				g_back[idx(i,j)]='/'; g_cost[idx(i,j)]=0;
			} else {
				TB b='/'; TC c=0,c2;  // stop
				for (size_t k=1; k<j; ++k) { c2=g_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } } // XXX: missing the k information
				for (size_t k=1; k<i; ++k) { c2=g_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				c2 = g_cost[idx(i-1,j-1)]+p_cost(g_in[0][i],g_in[1][j]); if (c2>=c) { c=c2; b='\\'; }
				g_cost[idx(i,j)] = c;
				g_back[idx(i,j)] = b;
			}
		}
	}
}

// -----------------------------------------------------------------------------
// block-split

#define COST_BLEFT(i,k) (cl_left[(k-1)/B_W][ i*(B_H+1)+ (B_W-k%B_W)%B_W *B_H ])
#define COST_BTOP(k,j) (cl_top[k/B_H][idx((B_H-(k%B_H))%B_H,j)])

// Handle non-serial dependencies out of the block
void blk_precompute2(blk_t* blk, TC** cl_top, TC** cl_left) {
	const unsigned long oi=blk->bi*B_H, oj=blk->bj*B_W; // block offset in global memory
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // default(0,stop)
			if (!INIT(oi+i,oj+j)) {
				// Non-serial partial dependencies
				for (unsigned k=1; k<oj; ++k) { c2=COST_BLEFT(i,k) - p_gap(j+k); if (c2>c) { c=c2; b=p_left[j+k]; } }
				for (unsigned k=1; k<oi; ++k) { c2=COST_BTOP(k,j) - p_gap(k+i); if (c2>=c) { c=c2; b=p_up[k+i]; } }

			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

// Get the cost value with backward search of at most 1 block
#define COST_B1(i,j) ({ int _i=(i),_j=(j); TC* _c=_i<0?(_j<0?c_diag:c_top):(_j<0?c_left:blk->cost); _i=(_i+B_H)%B_H; _j=(_j+B_W)%B_W; _c[B_H*(_j+_i)+_i]; })

void blk_solve2(blk_t* blk, TC* c_top, TC* c_left, TC* c_diag) {
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // stop
			if (!INIT(blk->bi*B_H+i,blk->bj*B_W+j)) {
				c2=blk->cost[idx(i,j)];
				// Non-serial partial result
				if (c2>c) { c=c2; b=blk->back[idx(i,j)]; }
				// Finish non-serial
				for (size_t k=1; k<=j; ++k) { c2=blk->cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } }
				for (size_t k=1; k<=i; ++k) { c2=blk->cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				// Monadic diagonal
				c2=COST_B1(i-1,j-1) + p_cost(blk->in[0][i],blk->in[1][j]); if (c2>=c) { c=c2; b='\\'; }
			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

void solve2() {
	// we need to manage concurrency at CPU level here to call multiple blocks
	for (unsigned bi=0;bi<M_H/B_H;++bi) {
		for (unsigned bj=0;bj<M_W/B_W;++bj) {
			blk_t blk = blk_get(bi,bj);
			// --------- Solving non-serial dependencies
			#if NONSERIAL>0
			TC** c_list[2]={NULL,NULL}; // previous blocks list (0=vert,1=horz,2=diag)
			#if (NONSERIAL)&DIR_VERT
			c_list[0]=(TC**)malloc(bi*sizeof(TC**)); for (unsigned k=0;k<bi;++k) c_list[0][k]=(TC*)mm_alloc(bi-k-1,bj);
			#endif
			#if (NONSERIAL)&DIR_HORZ
			c_list[1]=(TC**)malloc(bj*sizeof(TC**)); for (unsigned k=0;k<bj;++k) c_list[1][k]=(TC*)mm_alloc(bi,bj-k-1);
			#endif
			blk_precompute2(&blk,c_list[0],c_list[1]);
			#if (NONSERIAL)&DIR_VERT
			for (unsigned k=0;k<bi;++k) mm_free(c_list[0][k]); free(c_list[0]);
			#endif
			#if (NONSERIAL)&DIR_HORZ
			for (unsigned k=0;k<bj;++k) mm_free(c_list[1][k]); free(c_list[1]);
			#endif
			#endif
			// --------- Block processing
			TC* c_prev[3]={NULL,NULL,NULL};
			#if (POLYADIC)&(DIR_VERT|DIR_DIAG)
			if (bi>0) c_prev[0]=(TC*)mm_alloc(bi-1,bj);
			#endif
			#if (POLYADIC)&(DIR_HORZ|DIR_DIAG)
			if (bj>0) c_prev[1]=(TC*)mm_alloc(bi,bj-1);
			#endif
			#if (POLYADIC)&DIR_DIAG
			if (bi>0 && bj>0) c_prev[2]=(TC*)mm_alloc(bi-1,bj-1);
			#endif
			blk_solve2(&blk,c_prev[0],c_prev[1],c_prev[2]);
			#if (POLYADIC)&(DIR_VERT|DIR_DIAG)
			if (bi>0) mm_free(c_prev[0]);
			#endif
			#if (POLYADIC)&(DIR_HORZ|DIR_DIAG)
			if (bj>0) mm_free(c_prev[1]);
			#endif
			#if (POLYADIC)&DIR_DIAG
			if (bi>0 && bj>0) mm_free(c_prev[2]);
			#endif
		}
	}
}

#include <unistd.h>
#include <stdarg.h>
pid_t sys_exec(const char* path, ...) {
	pid_t f; int r; char** argv=NULL; char* a; va_list ap; int n=2;
	va_start(ap,path); while ((a=va_arg(ap,char*))) ++n; va_end(ap);
	argv=(char**)malloc(n*sizeof(char*)); if (!argv) return -1; argv[0]=(char*)path; n=1;
	va_start(ap,path); while ((a=va_arg(ap,char*))) argv[n++]=a; va_end(ap); argv[n]=NULL;
	switch(f=fork()) { case -1: return -1; case 0: execvp(path,(char** const)argv); _exit(1);
		default: free(argv); if (f!=-1 && waitpid(f,&r,0)!=-1) return WEXITSTATUS(r); else return -1;
	}
}

int main(int argc, char** argv) {

	init();

	FILE* f;
	f=fopen("ref.txt","w"); solve1(); print(f); fclose(f);
	f=fopen("t.txt","w"); solve2(); print(f); fclose(f);
	int r=sys_exec("diff","-dur","ref.txt","t.txt",NULL);
	printf("\n\n");
	sys_exec("cat","t.txt",NULL);
	printf("\nCorrectness: %s\n",!r?"no differences":"FAILURE !!!");
	unlink("ref.txt"); unlink("t.txt");

	cleanup();
	return 0;
}

#endif