#include "hip/hip_runtime.h"
#include "include/common.h"

// Problem shape (SH_RECT, SH_TRI, SH_PARA)
//#define SH_RECT
#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 32LU  // matrix dimension
#define M_H 32LU  // matrix dimension

// -----------------------------------------------------------------------------
#ifdef SH_RECT
// Smith-Waterman with arbitrary gap cost function (rectangular matrix).
// SWat(S,T); by convention, S is the longest string and put vertically
// M[i,j] = max { 0                                 stop } = B[i,j]
//              { M[i-1,j-1]+cost(S[i],T[j])        NW   }
//              { max{1<=k<=j-1} M[i,j-k] - gap(k)  N(k) }
//              { max{1<=k<=i-1} M[i-k,j] - gap(k)  W(k) }
// Data types
#define TI char     // input data type
#define TI_CHR(X) X // conversion to char
#define TC int      // cost type
#define TB short    // backtrack type (2 bits for direction + 14 for value)
//#define TW int    // wavefront type (if not defined, no wavefront)

// Initialization
#define INIT(i,j)  ((i)==0 || (j)==0) // matrix initialization at [stop]
// Input
TI* p_input(bool horz=false) {
	static unsigned sh=time(NULL), sv=time(NULL)+573; // keep consistent
	const char alph[4]={'A','C','G','T'};
	unsigned n = horz?M_W:M_H; mseed(horz?sh:sv);
	TI* in = (TI*)malloc(n*sizeof(TI));
	for (unsigned i=0;i<n;++i) in[i]=alph[mrand()%4];
	return in;
}
// Helpers
_hostdev _inline TC p_gap(int k) { return 20-k; }
_hostdev _inline TC p_cost(char s, char t) { return s==t?1:0; }
#endif
// -----------------------------------------------------------------------------
#ifdef SH_TRI
// Matrix multiplication parenthesizing (triangular matrix)
// M[i,j]= min {i<=k<j} M[i,k] + M [k+1,j] + r_i * c_k * c_j
// Now we need to flip the i axis to normalize progression
//
//   M[i,j]= min {i<=k<j} M[i,k] + M [j+i-(1+k),j] + r_i * c_k * c_j
//
typedef struct { unsigned rows,cols; char print() { return 'X'; } } mat_t;
// Data types
#define TI mat_t         // input data type
#define TI_CHR(X) ('0'+(X).rows) // conversion to char (debug)
#define TC unsigned long // cost type
#define TB short         // backtrack type (2 bits for direction + 14 for value)
//#define TW int         // wavefront type (if not defined, no wavefront)
// Initialization
#define INIT(i,j) (i+j<=M_H-1) // matrix initialization at [stop]
#define COST_MAX  (0x78888888UL)

// Input
TI* p_input() {
	static unsigned s=time(NULL); mseed(s); // keep consistent
	TI* in = (TI*)malloc(M_H*sizeof(TI));
	#define RNZ ({ unsigned x; do { x=mrand()&0x7; } while (!x); x; })
	in[0].rows=RNZ;
	for (int i=1;i<M_H;++i) { in[i-1].cols=in[i].rows=RNZ; }
	in[M_H-1].cols=RNZ;
	return in;
}

#endif
// -----------------------------------------------------------------------------
#ifdef SH_PARA
// Polygon triangulation (parallelogram matrix)
#endif
// -----------------------------------------------------------------------------

#include "include/nonserial.h" // must be included after problem definition

void c_solve() {
	mat_t* in0=c_in[0];
	for (unsigned i=0; i<M_H; ++i) {
		#ifdef SH_RECT
		for (unsigned j=0; j<M_W; ++j)
		#endif
		#ifdef SH_TRI
		for (unsigned j=M_H-1-i; j<M_W; ++j)
		#endif
		#ifdef SH_PARA
		for (unsigned j=i; j<M_W+i; ++j)
		#endif
		{
			TB b=BSTOP; TC c=0,c2;  // stop
			if (!INIT(i,j)) {
				#ifdef SH_RECT // SWat with arbitrary cost
				for (unsigned k=1; k<j; ++k) { c2=c_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (unsigned k=1; k<i; ++k) { c2=c_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = c_cost[idx(i-1,j-1)]+p_cost(c_in[0][i],c_in[1][j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
				#endif
				#ifdef SH_TRI // Matrix multiplication
				c = COST_MAX;
				for (unsigned k=M_H-1-i; k<j; ++k) {
					c2=c_cost[idx(i,k)] + c_cost[idx(M_H-k-2,j)] + in0[i].rows * in0[k].cols * in0[j].cols;
					if (c2<c) { c=c2; b=B(DIR_DIAG,k); }
				}
				#endif
			}
			c_cost[idx(i,j)] = c;
			c_back[idx(i,j)] = b;
		}
	}
}

// -----------------------------------------------------------------------------

#if 0

#define M_STRIPES ((M_H+B_H-1)/B_H) // number of CUDA block stripes (of height B_H) in the matrix
__global__ void gpu_solve(TI* in0, TI* in1, TC* cost, TB* back, volatile unsigned* sync) { // 437ms -> 555ms for volatile (but ensures correctness)
/*
	const int tx=threadIdx.x;
	if (threadIdx.x==0)
	for (unsigned i=0; i<M_H; ++i) {
		for (unsigned j=0; j<M_W; ++j) {
			TB b=BSTOP; TC c=0,c2;  // stop
			if (!INIT(i,j)) {
				for (size_t k=1; k<j; ++k) { c2=cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = cost[idx(i-1,j-1)]+p_cost(in0[i],in1[j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
			}
			cost[idx(i,j)] = c;
			back[idx(i,j)] = b;
		}
	}
*/

	const int tx=threadIdx.x;
	__shared__ unsigned max[32]; // one min per block
	for (unsigned i=0; i<M_H; ++i) {
		for (unsigned j=0; j<M_W; ++j) {
			TB b=BSTOP; TC c=0,c2;  // stop
			max[0]=0;

			if (!INIT(i,j)) {
				for (size_t k=1; k<j; ++k) { c2=cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }

				if (threadIdx.x==0) {
					c2 = cost[idx(i-1,j-1)]+p_cost(in0[i],in1[j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
				}

				if (tx>=16) max[tx-16]=c;
				if (tx<16 && c>max[tx]) max[tx]=c;
				if (tx<8 && (c2=max[tx+8])>max[tx]) max[tx]=c2;
				if (tx<4 && (c2=max[tx+4])>max[tx]) max[tx]=c2;
				if (tx<2 && (c2=max[tx+2])>max[tx]) max[tx]=c2;
				if (tx==0 && (c2=max[1])>max[0]) max[0]=c2;

			}
			if (c==max[0]) {
				cost[idx(i,j)] = c;
				back[idx(i,j)] = b;
			}
			__threadfence();
		}
	}
	// assign 4 threads to search minimum
	//	const unsigned tx = threadIdx.x+B_H*blockIdx.x;
	// XXX: dynamic number of sub-threads to compute minimum

/* 128x128 = 176.43ms
	const unsigned tx = threadIdx.x+B_H*blockIdx.x;
	const int i=tx;
	for (int jj=0; jj<M_W+M_H; ++jj) {
		int j=jj-tx;

		// Do computations in the valid part of the matrix
		if (j>=0 && j<M_W && i<M_H) {
			TB b=BSTOP; TC c=0,c2; // stop
			if (!INIT(i,j)) {
				for (size_t k=1; k<j; ++k) { c2=cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = cost[idx(i-1,j-1)]+p_cost(in0[i],in1[j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
			}
			cost[idx(i,j)] = c;
			back[idx(i,j)] = b;
		}

		// Synchronize between blocks
		__threadfence();
		if (threadIdx.x==0) {
			sync[blockIdx.x]=jj;
			if (blockIdx.x>0) {
				while (sync[blockIdx.x-1]<jj) {}
			}
		}
		__syncthreads();
	}
*/
}
#endif

void g_solve() {
/*
	//const unsigned tx = threadIdx.x + blockDim.x * ( blockIdx.x + blockIdx.y*gridDim.x );
	unsigned* sync;
	cuMalloc(sync,sizeof(int)*M_STRIPES); // 64 blocks
	hipMemset(sync,0,sizeof(unsigned)*M_STRIPES);
	//gpu_solve<<<M_STRIPES,B_H,0,NULL>>>(g_in[0],g_in[1],g_cost,g_back,sync);

	gpu_solve<<<1,32,0,NULL>>>(g_in[0],g_in[1],g_cost,g_back,sync);

	cuFree(sync);
*/
}

TC g_backtrack(unsigned** bt, unsigned* size) { return 0; }



// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	// CPU solving
	for (int i=0;i<2;++i) { t.start(); c_solve(); t.stop(); }
	printf("CPU solve: "); t.print(); printf("\n");

	// GPU solving
	// for (int i=0;i<2;++i) { t.start(); g_solve(); t.stop(); }
	// printf("GPU solve: "); t.print(); printf("\n");

	dbg_print(false,stdout);
	dbg_track(false,stdout);
	// dbg_compare();

	dbg_cleanup();
	return 0;
}




























#if 0
// macro helpers
#define B_LEN(n) (B_H*B_W*(n) + B_H*(B_H-1)-1) // memory length for N contiguous(line) blocks
#define B_IN(i,j) ( ((i)/B_H)*M_W*B_H + ((j)/B_W)*B_W*B_H ) // returns offset of the block containing (i,j)
#define B_AT(bi,bj) ( (bi)*M_W*B_H + (bj)*B_W*B_H ) // return the block (bi,bj)

inline off_t idx(size_t i, size_t j) { return B_H*(j+(i%B_H)) + (i%B_H) + (i/B_H)*M_W*B_H; }

typedef struct {
	unsigned bi, bj;     // block offset
	unsigned mi,mj;     // valid content maximal position
	const TI* in[2];  // related input 0=vert(S),1=horz(T)
	TC* cost;         // cost matrix
	TB* back;         // backtrack matrix
	//bool device;    // whether valid memory is CUDA or CPU memory
	bool wr_back;     // whether memory needs to be written back
} blk_t;

// -----------------------------------------------------------------------------
// Memory manager and problem data structures
// > all-in-main-memory implementation

// memory manager
// XXX: keep track of allocated zones, both function must be mutex-protected
// XXX: we need an atomic list of cost blocks, of input blocks and of backtrack blocks
// pthread_mutex_t* mutex;
// typedef struct { off_t bi,bj; bool cost; bool dev; void* ptr; unsigned retained; } mem_t;
	// XXX: test whether these zone were already allocated in CUDA memory to
	// avoid using duplicate memory
 // all these loads need to be atomically done, also released on demand
 // ==> allocate an array of pointers (both main and device mem) and do atomic CAS on them to get pointer + counter
 // See http://www.boost.org/doc/libs/1_39_0/boost/interprocess/detail/atomic.hpp
 // instead take into account blocks that have been written to disk so that reloading into memory is easier
 // also note that writing on disk must reorder differently to avoid writing dirty surrounding data

#define COST_BLEFT(i,k) (cl_left[(k-1)/B_W][ i*(B_H+1)+ (B_W-k%B_W)%B_W *B_H ])
#define COST_BTOP(k,j) (cl_top[k/B_H][idx((B_H-(k%B_H))%B_H,j)])

// Handle non-serial dependencies out of the block
void blk_precompute2(blk_t* blk, TC** cl_top, TC** cl_left) {
	const unsigned long oi=blk->bi*B_H, oj=blk->bj*B_W; // block offset in global memory
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // default(0,stop)
			if (!INIT(oi+i,oj+j)) {
				// Non-serial partial dependencies
				for (unsigned k=1; k<oj; ++k) { c2=COST_BLEFT(i,k) - p_gap(j+k); if (c2>c) { c=c2; b=p_left[j+k]; } }
				for (unsigned k=1; k<oi; ++k) { c2=COST_BTOP(k,j) - p_gap(k+i); if (c2>=c) { c=c2; b=p_up[k+i]; } }

			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

// Get the cost value with backward search of at most 1 block
#define COST_B1(i,j) ({ int _i=(i),_j=(j); TC* _c=_i<0?(_j<0?c_diag:c_top):(_j<0?c_left:blk->cost); _i=(_i+B_H)%B_H; _j=(_j+B_W)%B_W; _c[B_H*(_j+_i)+_i]; })

void blk_solve2(blk_t* blk, TC* c_top, TC* c_left, TC* c_diag) {
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // stop
			if (!INIT(blk->bi*B_H+i,blk->bj*B_W+j)) {
				c2=blk->cost[idx(i,j)];
				// Non-serial partial result
				if (c2>c) { c=c2; b=blk->back[idx(i,j)]; }
				// Finish non-serial
				for (size_t k=1; k<=j; ++k) { c2=blk->cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } }
				for (size_t k=1; k<=i; ++k) { c2=blk->cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				// Monadic diagonal
				c2=COST_B1(i-1,j-1) + p_cost(blk->in[0][i],blk->in[1][j]); if (c2>=c) { c=c2; b='\\'; }
			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

int main(int argc, char** argv) {
	init();
	FILE* f;
	f=fopen("ref.txt","w"); solve1(); print(f); fclose(f);
	f=fopen("t.txt","w"); solve2(); print(f); fclose(f);
	int r=sys_exec("diff","-dur","ref.txt","t.txt",NULL);
	printf("\n\n");
	sys_exec("cat","t.txt",NULL);
	printf("\nCorrectness: %s\n",!r?"no differences":"FAILURE !!!");
	unlink("ref.txt"); unlink("t.txt");

	return 0;
}


/*
//GPU lock-free synchronization function
__device__ void __gpu_sync(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	// thread ID in a block
	int tid_in_blk = threadIdx.x * blockDim.y + threadIdx.y;
	int nBlockNum = gridDim.x * gridDim.y;
	int bid = blockIdx.x * gridDim.y + blockIdx.y;
	// only thread 0 is used for synchronization
	if (tid_in_blk == 0) Arrayin[bid] = goalVal;
	if (bid == 1) {
		if (tid_in_blk < nBlockNum)	{
			while (Arrayin[tid_in_blk] != goalVal) {}
			__syncthreads();
		}
		if (tid_in_blk<nBlockNum) Arrayout[tid_in_blk] = goalVal;
	}
	if (tid_in_blk == 0) {
		while (Arrayout[bid] != goalVal) {}
	}
	__syncthreads();
}
*/

#endif
