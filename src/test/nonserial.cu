#include "hip/hip_runtime.h"
#include "include/common.h"

// We want to support 3 types of problems:
// - SH_RECT SWat arbitrary gap
// - SH_TRI  Matrix multiplication
// - SH_PARA Triangulation

// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
// System parameters
// - Data types
#define TI char    // input data type
#define TC int     // cost type
#define TB short   // backtrack type (2 bits for direction + 14 for value)
#define TW int     // wavefront type (if not defined, no wavefront)
// - Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 249LU   // matrix dimension
#define M_H 267LU   // matrix dimension
// - Problem shape (RECT, TRIANG, PARALL)
#define SH_RECT
//#define SH_TRI
//#define SH_PARA
// Initialization
#define INIT(i,j)  ((i)==0 || (j)==0) // matrix initialization at [stop]

// Problem definition SWat(S,T) with arbitrary gap cost.
// By convention, S is the longest string and put vertically
// M[i,j] = max { 0                                 stop } = B[i,j]
//              { M[i-1,j-1]+cost(S[i],T[j])        NW   }
//              { max{1<=k<=j-1} M[i,j-k] - gap(k)  N(k) }
//              { max{1<=k<=i-1} M[i-k,j] - gap(k)  W(k) }
//

// Input string that can be free()d
TI* p_input(bool horz=false) {
	static unsigned sh=time(NULL), sv=time(NULL)+573; // keep consistent
	const char alph[4]={'A','C','G','T'};
	unsigned n = horz?M_W:M_H; mseed(horz?sh:sv);
	TI* in = (char*)malloc(n*sizeof(TI));
	for (unsigned i=0;i<n;++i) in[i]=alph[mrand()%4];
	return in;
}
// problem specific helpers
_hostdev _inline TC p_gap(int k) { return 20-k; }
_hostdev _inline TC p_cost(char s, char t) { return s==t?1:0; }
// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------

#include "include/nonserial.h" // must be included after problem definition

// simply return the pair of indices (i,j) that are in the backtrack
// by default we use the direction-length backtrack
TC c_backtrack(unsigned** bt, unsigned* size) {
	unsigned i,j;
	// Find the position with maximal cost
	unsigned mi=0; TC ci=0;
	unsigned mj=0; TC cj=0;
	for (unsigned i=0; i<M_H; ++i) { TC c=c_cost[idx(i,M_W-1)]; if (c>ci) { mi=i; ci=c; } }
	for (unsigned j=0; j<M_W; ++j) { TC c=c_cost[idx(M_H-1,j)]; if (c>cj) { mj=j; cj=c; } }
	if (ci>cj) { i=mi; j=M_W-1; } else { i=M_H-1; j=mj; }

	TC cost = c_cost[idx(i,j)];
	// Backtrack, returns a pair of coordinates in reverse order
	if (bt && size) {
		TB b;
		*bt=(unsigned*)malloc((M_W+M_H)*2*sizeof(unsigned));
		unsigned sz=0;
		unsigned* track=*bt;
		do {
			track[0]=i; track[1]=j; track+=2; ++sz;
			b = c_back[idx(i,j)];
			switch(BD(b)) {
				case DIR_LEFT: j-=BV(b); break;
				case DIR_UP: i-=BV(b); break;
				case DIR_DIAG: i-=BV(b); j-=BV(b); break;
			}
		} while (b!=BSTOP);
		*size=sz;
	}
	return cost;
}

void c_solve() {
	for (unsigned i=0; i<M_H; ++i) {
		for (unsigned j=0; j<M_W; ++j) {
			if (INIT(i,j)) { c_back[idx(i,j)]=BSTOP; c_cost[idx(i,j)]=0; }
			else {
				TB b=BSTOP; TC c=0,c2;  // stop
				for (size_t k=1; k<j; ++k) { c2=c_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=c_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = c_cost[idx(i-1,j-1)]+p_cost(c_in[0][i],c_in[1][j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
				c_cost[idx(i,j)] = c;
				c_back[idx(i,j)] = b;
			}
		}
	}
}

/*
//GPU lock-free synchronization function
__device__ void __gpu_sync(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	// thread ID in a block
	int tid_in_blk = threadIdx.x * blockDim.y + threadIdx.y;
	int nBlockNum = gridDim.x * gridDim.y;
	int bid = blockIdx.x * gridDim.y + blockIdx.y;

	// only thread 0 is used for synchronization
	if (tid_in_blk == 0) Arrayin[bid] = goalVal;

	if (bid == 1) {
		if (tid_in_blk < nBlockNum)	{
			while (Arrayin[tid_in_blk] != goalVal) {}
			__syncthreads();
		}
		if (tid_in_blk<nBlockNum) Arrayout[tid_in_blk] = goalVal;
	}

	if (tid_in_blk == 0) {
		while (Arrayout[bid] != goalVal) {}
	}
	__syncthreads();

}
*/

__global__ void gpu_solve(TI* in0, TI* in1, TC* cost, TB* back, unsigned* sync) {
	const unsigned tx = threadIdx.x+blockDim.x * blockIdx.x;
	const unsigned bx = blockDim.x * gridDim.x;


// XXX: BUGGY BUT TIMING IS BEARABLE

	int i,j;
	// initialization
	for (i=tx;i<M_H;i+=bx) { cost[idx(i,0)]=0; back[idx(i,0)]=BSTOP; }
	for (j=tx;j<M_W;j+=bx) { cost[idx(0,j)]=0; back[idx(0,j)]=BSTOP; }

	if (threadIdx.x==0) sync[blockIdx.x]=0;


	for (int i=1+tx; i<M_H; i+=bx) {
		unsigned p=idx(i,1);
		unsigned p1=idx(i-1,0);
		for (int j=1-tx; j<M_W; ++j) {
			if (j>0) {
				TC c=cost[p1]+p_cost(in0[i],in1[j]),c2;
				TB b=B(DIR_DIAG,1);
				for (size_t k=1; k<j; ++k) { c2=cost[p-k*B_W]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				cost[p] = c;
				back[p] = b;
				p+=B_W;
				p1+=B_W;
			}
			// XXX: missing synchro between blocks

			// XXX: sync between blocks: wait for previous block to have released its lock
			__syncthreads();
			// XXX: flush to memory
			if (threadIdx.x==0) {
				const unsigned b = blockIdx.x;
				sync[b]=i;
				if (b>0) { while (sync[b-1]<i) {} }
			}
			__syncthreads();
		}
	}


}

void g_solve() {
	//const unsigned tx = threadIdx.x + blockDim.x * ( blockIdx.x + blockIdx.y*gridDim.x );
	unsigned* sync;

	unsigned num_blocks = 64; // XXX: make sure that block*threads CAN be < M_H

	cuMalloc(sync,sizeof(unsigned)*num_blocks); // 64 blocks
	gpu_solve<<<64,32,0,NULL>>>(g_in[0],g_in[1],g_cost,g_back,sync);
	cuFree(sync);
}

TC g_backtrack(unsigned** bt, unsigned* size) {
	return 0;
}




// -----------------------------------------------------------------------------
// GPU structures
// - allocate memory
// - access memory[cost,backtrack,wavefront,input] at index (i,j)

void dbg_track(bool gpu, FILE* f) {
	// XXX: add for GPU

	unsigned* bt;
	unsigned sz;
	unsigned score = c_backtrack(&bt,&sz);
	fprintf(f,"Backtrack with best score : %d\n",score);
	for (unsigned i=sz-1;;--i) {
		printf("(%d,%d) ",bt[i*2],bt[i*2+1]);
		if (!i) break;
	}
	printf("\n");
	free(bt);
}

// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuInfo();

	c_init();
	cuTimer t;
	for (int i=0;i<4;++i) { t.start(); c_solve(); t.stop(); }
	printf("CPU solve: "); t.print(); printf("\n");
	// dbg_print(false,stdout);
	// dbg_track(false,stdout);

	g_init();
	for (int i=0;i<4;++i) { t.start(); g_solve(); t.stop(); }
	printf("GPU solve: "); t.print(); printf("\n");

	/*
	TC* tc=(TC*)malloc(sizeof(TC)*MEM_MATRIX); cuGet(tc,g_cost,sizeof(TC)*MEM_MATRIX,NULL);
	TB* tb=(TB*)malloc(sizeof(TB)*MEM_MATRIX); cuGet(tb,g_back,sizeof(TB)*MEM_MATRIX,NULL);
	for (int i=0;i<M_H;++i) {
		for (int j=0;j<M_W;++j) {
			if (tc[idx(i,j)]!=c_cost[idx(i,j)]) printf("C");
			if (tb[idx(i,j)]!=c_back[idx(i,j)]) printf("B");
		}
	}
	printf("\n");
	free(tc);
	free(tb);
	*/

	c_free();
	g_free();

/*
	#ifdef SH_RECT
	printf("rectangle\n");
	#endif
	#ifdef SH_TRI
	printf("triangle\n");
	#endif
	#ifdef SH_PARA
	printf("parallelogram\n");
	#endif
*/
	hipDeviceReset();
	return 0;
}



















#if 0
// macro helpers
#define B_LEN(n) (B_H*B_W*(n) + B_H*(B_H-1)-1) // memory length for N contiguous(line) blocks
#define B_IN(i,j) ( ((i)/B_H)*M_W*B_H + ((j)/B_W)*B_W*B_H ) // returns offset of the block containing (i,j)
#define B_AT(bi,bj) ( (bi)*M_W*B_H + (bj)*B_W*B_H ) // return the block (bi,bj)

inline off_t idx(size_t i, size_t j) { return B_H*(j+(i%B_H)) + (i%B_H) + (i/B_H)*M_W*B_H; }

typedef struct {
	unsigned bi, bj;     // block offset
	unsigned mi,mj;     // valid content maximal position
	const TI* in[2];  // related input 0=vert(S),1=horz(T)
	TC* cost;         // cost matrix
	TB* back;         // backtrack matrix
	//bool device;    // whether valid memory is CUDA or CPU memory
	bool wr_back;     // whether memory needs to be written back
} blk_t;

// -----------------------------------------------------------------------------
// Memory manager and problem data structures
// > all-in-main-memory implementation

const TI* g_in[2]={NULL,NULL}; // 0=vert(S), 1=horizontal(T) with |S| >= |T|
TC* g_cost=NULL;
TB* g_back=NULL;
#ifdef TW
TW* g_wave[3]={NULL,NULL,NULL}; // 0=vert,1=horizontal,2=diagonal
#endif

// initialize structures
void init() {
	size_t mem = M_W*M_H+B_H*B_H;
	g_cost=(TC*)malloc(sizeof(TC)*mem);
	g_back=(TB*)malloc(sizeof(TB)*mem);
	#ifdef TW
		g_wave[0]=(TW*)malloc(sizeof(TW)*M_H);
		g_wave[1]=(TW*)malloc(sizeof(TW)*M_W);
		g_wave[2]=(TW*)malloc(sizeof(TW)*MAX(M_H/B_H,M_W/B_W));
	#endif
	g_in[0]=p_S; // also duplicate into CUDA
	g_in[1]=p_T;
}

// cleanup structures
void cleanup() {
	if (g_cost) free(g_cost); g_cost=NULL;
	if (g_back) free(g_back); g_back=NULL;
	#ifdef TW
	for (int i=0;i<3;++i) if (g_wave[i]) { free(g_wave[i]); g_wave[i]=NULL; }
	#endif
}

// memory manager
// XXX: keep track of allocated zones, both function must be mutex-protected
// XXX: we need an atomic list of cost blocks, of input blocks and of backtrack blocks
// pthread_mutex_t* mutex;
// typedef struct { off_t bi,bj; bool cost; bool dev; void* ptr; unsigned retained; } mem_t;
	// XXX: test whether these zone were already allocated in CUDA memory to
	// avoid using duplicate memory
 // all these loads need to be atomically done, also released on demand
 // ==> allocate an array of pointers (both main and device mem) and do atomic CAS on them to get pointer + counter
 // See http://www.boost.org/doc/libs/1_39_0/boost/interprocess/detail/atomic.hpp
 // instead take into account blocks that have been written to disk so that reloading into memory is easier
 // also note that writing on disk must reorder differently to avoid writing dirty surrounding data
void* mm_alloc(off_t bi, off_t bj, bool cost=true, bool device=false) {
	// attempt to alloc, if cannot, then try to free all the blocks with 0 retain,
	// possibly write-back to disk, then try again
	if (cost) return &g_cost[B_AT(bi,bj)];
	else return &g_back[B_AT(bi,bj)];
}
void mm_free(void* ptr) {}

blk_t blk_get(off_t bi, off_t bj, bool device=false) {
	blk_t b; b.bi=bi; b.bj=bj;
	b.mi=M_H-bi*B_H; if (b.mi>B_H) b.mi=B_H;
	b.mj=M_W-bj*B_W; if (b.mj>B_W) b.mj=B_W;
	b.in[0]=&g_in[0][bi*B_H]; // XXX: depends whether we're on device
	b.in[1]=&g_in[1][bj*B_W];
	b.wr_back=false;
	b.cost=(TC*)mm_alloc(bi,bj,true);
	b.back=(TB*)mm_alloc(bi,bj,false);
	return b;
}

void blk_free(blk_t* blk) {
	if (blk->wr_back) {
		// XXX: write-back to main memory or to storage
	}
	mm_free(blk->cost);
	mm_free(blk->back);
}

// -----------------------------------------------------------------------------
// reference implementation

void solve1() {
	// Recurrence (embeds initialization)
	for (size_t i=0; i<C_H; ++i) {
		for (size_t j=0; j<C_W; ++j) {
			if (i==0 || j==0) {
				g_back[idx(i,j)]='/'; g_cost[idx(i,j)]=0;
			} else {
				TB b='/'; TC c=0,c2;  // stop
				for (size_t k=1; k<j; ++k) { c2=g_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } } // XXX: missing the k information
				for (size_t k=1; k<i; ++k) { c2=g_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				c2 = g_cost[idx(i-1,j-1)]+p_cost(g_in[0][i],g_in[1][j]); if (c2>=c) { c=c2; b='\\'; }
				g_cost[idx(i,j)] = c;
				g_back[idx(i,j)] = b;
			}
		}
	}
}

// -----------------------------------------------------------------------------
// block-split

#define COST_BLEFT(i,k) (cl_left[(k-1)/B_W][ i*(B_H+1)+ (B_W-k%B_W)%B_W *B_H ])
#define COST_BTOP(k,j) (cl_top[k/B_H][idx((B_H-(k%B_H))%B_H,j)])

// Handle non-serial dependencies out of the block
void blk_precompute2(blk_t* blk, TC** cl_top, TC** cl_left) {
	const unsigned long oi=blk->bi*B_H, oj=blk->bj*B_W; // block offset in global memory
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // default(0,stop)
			if (!INIT(oi+i,oj+j)) {
				// Non-serial partial dependencies
				for (unsigned k=1; k<oj; ++k) { c2=COST_BLEFT(i,k) - p_gap(j+k); if (c2>c) { c=c2; b=p_left[j+k]; } }
				for (unsigned k=1; k<oi; ++k) { c2=COST_BTOP(k,j) - p_gap(k+i); if (c2>=c) { c=c2; b=p_up[k+i]; } }

			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

// Get the cost value with backward search of at most 1 block
#define COST_B1(i,j) ({ int _i=(i),_j=(j); TC* _c=_i<0?(_j<0?c_diag:c_top):(_j<0?c_left:blk->cost); _i=(_i+B_H)%B_H; _j=(_j+B_W)%B_W; _c[B_H*(_j+_i)+_i]; })

void blk_solve2(blk_t* blk, TC* c_top, TC* c_left, TC* c_diag) {
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // stop
			if (!INIT(blk->bi*B_H+i,blk->bj*B_W+j)) {
				c2=blk->cost[idx(i,j)];
				// Non-serial partial result
				if (c2>c) { c=c2; b=blk->back[idx(i,j)]; }
				// Finish non-serial
				for (size_t k=1; k<=j; ++k) { c2=blk->cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } }
				for (size_t k=1; k<=i; ++k) { c2=blk->cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				// Monadic diagonal
				c2=COST_B1(i-1,j-1) + p_cost(blk->in[0][i],blk->in[1][j]); if (c2>=c) { c=c2; b='\\'; }
			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

void solve2() {
	// we need to manage concurrency at CPU level here to call multiple blocks
	for (unsigned bi=0;bi<M_H/B_H;++bi) {
		for (unsigned bj=0;bj<M_W/B_W;++bj) {
			blk_t blk = blk_get(bi,bj);
			// --------- Solving non-serial dependencies
			#if NONSERIAL>0
			TC** c_list[2]={NULL,NULL}; // previous blocks list (0=vert,1=horz,2=diag)
			#if (NONSERIAL)&DIR_VERT
			c_list[0]=(TC**)malloc(bi*sizeof(TC**)); for (unsigned k=0;k<bi;++k) c_list[0][k]=(TC*)mm_alloc(bi-k-1,bj);
			#endif
			#if (NONSERIAL)&DIR_HORZ
			c_list[1]=(TC**)malloc(bj*sizeof(TC**)); for (unsigned k=0;k<bj;++k) c_list[1][k]=(TC*)mm_alloc(bi,bj-k-1);
			#endif
			blk_precompute2(&blk,c_list[0],c_list[1]);
			#if (NONSERIAL)&DIR_VERT
			for (unsigned k=0;k<bi;++k) mm_free(c_list[0][k]); free(c_list[0]);
			#endif
			#if (NONSERIAL)&DIR_HORZ
			for (unsigned k=0;k<bj;++k) mm_free(c_list[1][k]); free(c_list[1]);
			#endif
			#endif
			// --------- Block processing
			TC* c_prev[3]={NULL,NULL,NULL};
			#if (POLYADIC)&(DIR_VERT|DIR_DIAG)
			if (bi>0) c_prev[0]=(TC*)mm_alloc(bi-1,bj);
			#endif
			#if (POLYADIC)&(DIR_HORZ|DIR_DIAG)
			if (bj>0) c_prev[1]=(TC*)mm_alloc(bi,bj-1);
			#endif
			#if (POLYADIC)&DIR_DIAG
			if (bi>0 && bj>0) c_prev[2]=(TC*)mm_alloc(bi-1,bj-1);
			#endif
			blk_solve2(&blk,c_prev[0],c_prev[1],c_prev[2]);
			#if (POLYADIC)&(DIR_VERT|DIR_DIAG)
			if (bi>0) mm_free(c_prev[0]);
			#endif
			#if (POLYADIC)&(DIR_HORZ|DIR_DIAG)
			if (bj>0) mm_free(c_prev[1]);
			#endif
			#if (POLYADIC)&DIR_DIAG
			if (bi>0 && bj>0) mm_free(c_prev[2]);
			#endif
		}
	}
}

#include <unistd.h>
#include <stdarg.h>
pid_t sys_exec(const char* path, ...) {
	pid_t f; int r; char** argv=NULL; char* a; va_list ap; int n=2;
	va_start(ap,path); while ((a=va_arg(ap,char*))) ++n; va_end(ap);
	argv=(char**)malloc(n*sizeof(char*)); if (!argv) return -1; argv[0]=(char*)path; n=1;
	va_start(ap,path); while ((a=va_arg(ap,char*))) argv[n++]=a; va_end(ap); argv[n]=NULL;
	switch(f=fork()) { case -1: return -1; case 0: execvp(path,(char** const)argv); _exit(1);
		default: free(argv); if (f!=-1 && waitpid(f,&r,0)!=-1) return WEXITSTATUS(r); else return -1;
	}
}

int main(int argc, char** argv) {
	init();
	FILE* f;
	f=fopen("ref.txt","w"); solve1(); print(f); fclose(f);
	f=fopen("t.txt","w"); solve2(); print(f); fclose(f);
	int r=sys_exec("diff","-dur","ref.txt","t.txt",NULL);
	printf("\n\n");
	sys_exec("cat","t.txt",NULL);
	printf("\nCorrectness: %s\n",!r?"no differences":"FAILURE !!!");
	unlink("ref.txt"); unlink("t.txt");

	cleanup();
	return 0;
}

#endif