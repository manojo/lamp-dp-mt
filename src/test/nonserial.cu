#include "hip/hip_runtime.h"
#include "include/common.h"

// We want to support 3 types of problems:
// - SH_RECT SWat arbitrary gap
// - SH_TRI  Matrix multiplication
// - SH_PARA Triangulation

// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------
// System parameters
// - Data types
#define TI char    // input data type
#define TC int     // cost type
#define TB short   // backtrack type (2 bits for direction + 14 for value)
#define TW int     // wavefront type (if not defined, no wavefront)
// - Problem dimensions
#define B_W 64LU    // block width
#define B_H 64LU    // block height
#define M_W 512LU  // matrix dimension
#define M_H 512LU  // matrix dimension
// - Problem shape (RECT, TRIANG, PARALL)
#define SH_RECT
//#define SH_TRI
//#define SH_PARA
// Initialization
#define INIT(i,j)  ((i)==0 || (j)==0) // matrix initialization at [stop]

// Problem definition SWat(S,T) with arbitrary gap cost.
// By convention, S is the longest string and put vertically
// M[i,j] = max { 0                                 stop } = B[i,j]
//              { M[i-1,j-1]+cost(S[i],T[j])        NW   }
//              { max{1<=k<=j-1} M[i,j-k] - gap(k)  N(k) }
//              { max{1<=k<=i-1} M[i-k,j] - gap(k)  W(k) }
//

// Input string that can be free()d
TI* p_input(bool horz=false) {
	static unsigned sh=time(NULL), sv=time(NULL)+573; // keep consistent
	const char alph[4]={'A','C','G','T'};
	unsigned n = horz?M_W:M_H; mseed(horz?sh:sv);
	TI* in = (char*)malloc(n*sizeof(TI));
	for (unsigned i=0;i<n;++i) in[i]=alph[mrand()%4];
	return in;
}
// problem specific helpers
_hostdev _inline TC p_gap(int k) { return 20-k; }
_hostdev _inline TC p_cost(char s, char t) { return s==t?1:0; }
// -----------------------------------------------------------------------------
// -----------------------------------------------------------------------------

#include "include/nonserial.h" // must be included after problem definition

void c_solve() {
	for (unsigned i=0; i<M_H; ++i) {
		for (unsigned j=0; j<M_W; ++j) {
			if (INIT(i,j)) { c_back[idx(i,j)]=BSTOP; c_cost[idx(i,j)]=0; }
			else {
				TB b=BSTOP; TC c=0,c2;  // stop
				for (size_t k=1; k<j; ++k) { c2=c_cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=c_cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = c_cost[idx(i-1,j-1)]+p_cost(c_in[0][i],c_in[1][j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
				c_cost[idx(i,j)] = c;
				c_back[idx(i,j)] = b;
			}
		}
	}
}

// simply return the pair of indices (i,j) that are in the backtrack
// by default we use the direction-length backtrack
TC c_backtrack(unsigned** bt, unsigned* size) {
	unsigned i,j;
	// Find the position with maximal cost
	unsigned mi=0; TC ci=0;
	unsigned mj=0; TC cj=0;
	for (unsigned i=0; i<M_H; ++i) { TC c=c_cost[idx(i,M_W-1)]; if (c>ci) { mi=i; ci=c; } }
	for (unsigned j=0; j<M_W; ++j) { TC c=c_cost[idx(M_H-1,j)]; if (c>cj) { mj=j; cj=c; } }
	if (ci>cj) { i=mi; j=M_W-1; } else { i=M_H-1; j=mj; }

	TC cost = c_cost[idx(i,j)];
	// Backtrack, returns a pair of coordinates in reverse order
	if (bt && size) {
		TB b;
		*bt=(unsigned*)malloc((M_W+M_H)*2*sizeof(unsigned));
		unsigned sz=0;
		unsigned* track=*bt;
		do {
			track[0]=i; track[1]=j; track+=2; ++sz;
			b = c_back[idx(i,j)];
			switch(BD(b)) {
				case DIR_LEFT: j-=BV(b); break;
				case DIR_UP: i-=BV(b); break;
				case DIR_DIAG: i-=BV(b); j-=BV(b); break;
			}
		} while (b!=BSTOP);
		*size=sz;
	}
	return cost;
}

/*
//GPU lock-free synchronization function
__device__ void __gpu_sync(int goalVal, volatile int *Arrayin, volatile int *Arrayout) {
	// thread ID in a block
	int tid_in_blk = threadIdx.x * blockDim.y + threadIdx.y;
	int nBlockNum = gridDim.x * gridDim.y;
	int bid = blockIdx.x * gridDim.y + blockIdx.y;

	// only thread 0 is used for synchronization
	if (tid_in_blk == 0) Arrayin[bid] = goalVal;

	if (bid == 1) {
		if (tid_in_blk < nBlockNum)	{
			while (Arrayin[tid_in_blk] != goalVal) {}
			__syncthreads();
		}
		if (tid_in_blk<nBlockNum) Arrayout[tid_in_blk] = goalVal;
	}

	if (tid_in_blk == 0) {
		while (Arrayout[bid] != goalVal) {}
	}
	__syncthreads();

}
*/

// -----------------------------------------------------------------------------

#define M_STRIPES ((M_H+B_H-1)/B_H) // number of CUDA block stripes (of height B_H) in the matrix

__global__ void gpu_solve(TI* in0, TI* in1, TC* cost, TB* back, volatile unsigned* sync) { // 437ms -> 555ms for volatile (but ensures correctness)
	const unsigned tx = threadIdx.x+B_H*blockIdx.x;
	const int i=tx;
	for (int jj=0; jj<M_W+M_H; ++jj) {
		int j=jj-tx;

		// Do computations in the valid part of the matrix
		if (j>=0 && j<M_W && i<M_H) {
			TB b=BSTOP; TC c=0,c2; // stop
			if (!INIT(i,j)) {
				for (size_t k=1; k<j; ++k) { c2=cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=B(DIR_LEFT,k); } }
				for (size_t k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=B(DIR_UP,k); } }
				c2 = cost[idx(i-1,j-1)]+p_cost(in0[i],in1[j]); if (c2>=c) { c=c2; b=B(DIR_DIAG,1); }
			}
			cost[idx(i,j)] = c;
			back[idx(i,j)] = b;
		}

		// Synchronize between blocks
		__threadfence();
		if (threadIdx.x==0) {
			sync[blockIdx.x]=jj;
			if (blockIdx.x>0) {
				while (sync[blockIdx.x-1]<jj) {}
			}
		}
		__syncthreads();
	}
}

void g_solve() {
	//const unsigned tx = threadIdx.x + blockDim.x * ( blockIdx.x + blockIdx.y*gridDim.x );
	unsigned* sync;
	cuMalloc(sync,sizeof(int)*M_STRIPES); // 64 blocks
	hipMemset(sync,0,sizeof(unsigned)*M_STRIPES);
	gpu_solve<<<M_STRIPES,B_H,0,NULL>>>(g_in[0],g_in[1],g_cost,g_back,sync);
	cuFree(sync);
}

TC g_backtrack(unsigned** bt, unsigned* size) { return 0; }

// -----------------------------------------------------------------------------
// GPU structures
// - allocate memory
// - access memory[cost,backtrack,wavefront,input] at index (i,j)

void dbg_track(bool gpu, FILE* f) {
	// XXX: add for GPU

	unsigned* bt;
	unsigned sz;
	unsigned score = c_backtrack(&bt,&sz);
	fprintf(f,"Backtrack with best score : %d\n",score);
	for (unsigned i=sz-1;;--i) {
		printf("(%d,%d) ",bt[i*2],bt[i*2+1]);
		if (!i) break;
	}
	printf("\n");
	free(bt);
}

void dbg_compare(bool full=false) {
	hipMemset(g_cost,0xff,sizeof(TC)*MEM_MATRIX);
	hipMemset(g_back,0xff,sizeof(TB)*MEM_MATRIX);
	g_solve();
	TC* tc=(TC*)malloc(sizeof(TC)*MEM_MATRIX); cuGet(tc,g_cost,sizeof(TC)*MEM_MATRIX,NULL);
	TB* tb=(TB*)malloc(sizeof(TB)*MEM_MATRIX); cuGet(tb,g_back,sizeof(TB)*MEM_MATRIX,NULL);
	int err=0;
	for (int i=0;i<M_H;++i) {
		for (int j=0;j<M_W;++j) {
			if (tc[idx(i,j)]!=c_cost[idx(i,j)]) { ++err; if (full) printf(" (%d,%d)",i,j); }
		}
	}
	printf("Compare CPU/GPU: %d errors.\n",err);
	free(tc);
	free(tb);
}

// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuInfo();
	printf("Matrix: %ldx%ld, blocks: %ldx%ld.\n",M_H,M_W,B_H,B_W);

	c_init();
	g_init();

	cuTimer t;
	for (int i=0;i<6;++i) { t.start(); c_solve(); t.stop(); }
	printf("CPU solve: "); t.print(); printf("\n");

	// dbg_print(false,stdout);
	// dbg_track(false,stdout);
	dbg_compare();

	for (int i=0;i<6;++i) { t.start(); g_solve(); t.stop(); }
	printf("GPU solve: "); t.print(); printf("\n");

	c_free();
	g_free();

	// #ifdef SH_RECT
	// #ifdef SH_TRI
	// #ifdef SH_PARA
	hipDeviceReset();
	return 0;
}




























#if 0
// macro helpers
#define B_LEN(n) (B_H*B_W*(n) + B_H*(B_H-1)-1) // memory length for N contiguous(line) blocks
#define B_IN(i,j) ( ((i)/B_H)*M_W*B_H + ((j)/B_W)*B_W*B_H ) // returns offset of the block containing (i,j)
#define B_AT(bi,bj) ( (bi)*M_W*B_H + (bj)*B_W*B_H ) // return the block (bi,bj)

inline off_t idx(size_t i, size_t j) { return B_H*(j+(i%B_H)) + (i%B_H) + (i/B_H)*M_W*B_H; }

typedef struct {
	unsigned bi, bj;     // block offset
	unsigned mi,mj;     // valid content maximal position
	const TI* in[2];  // related input 0=vert(S),1=horz(T)
	TC* cost;         // cost matrix
	TB* back;         // backtrack matrix
	//bool device;    // whether valid memory is CUDA or CPU memory
	bool wr_back;     // whether memory needs to be written back
} blk_t;

// -----------------------------------------------------------------------------
// Memory manager and problem data structures
// > all-in-main-memory implementation

// memory manager
// XXX: keep track of allocated zones, both function must be mutex-protected
// XXX: we need an atomic list of cost blocks, of input blocks and of backtrack blocks
// pthread_mutex_t* mutex;
// typedef struct { off_t bi,bj; bool cost; bool dev; void* ptr; unsigned retained; } mem_t;
	// XXX: test whether these zone were already allocated in CUDA memory to
	// avoid using duplicate memory
 // all these loads need to be atomically done, also released on demand
 // ==> allocate an array of pointers (both main and device mem) and do atomic CAS on them to get pointer + counter
 // See http://www.boost.org/doc/libs/1_39_0/boost/interprocess/detail/atomic.hpp
 // instead take into account blocks that have been written to disk so that reloading into memory is easier
 // also note that writing on disk must reorder differently to avoid writing dirty surrounding data

#define COST_BLEFT(i,k) (cl_left[(k-1)/B_W][ i*(B_H+1)+ (B_W-k%B_W)%B_W *B_H ])
#define COST_BTOP(k,j) (cl_top[k/B_H][idx((B_H-(k%B_H))%B_H,j)])

// Handle non-serial dependencies out of the block
void blk_precompute2(blk_t* blk, TC** cl_top, TC** cl_left) {
	const unsigned long oi=blk->bi*B_H, oj=blk->bj*B_W; // block offset in global memory
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // default(0,stop)
			if (!INIT(oi+i,oj+j)) {
				// Non-serial partial dependencies
				for (unsigned k=1; k<oj; ++k) { c2=COST_BLEFT(i,k) - p_gap(j+k); if (c2>c) { c=c2; b=p_left[j+k]; } }
				for (unsigned k=1; k<oi; ++k) { c2=COST_BTOP(k,j) - p_gap(k+i); if (c2>=c) { c=c2; b=p_up[k+i]; } }

			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

// Get the cost value with backward search of at most 1 block
#define COST_B1(i,j) ({ int _i=(i),_j=(j); TC* _c=_i<0?(_j<0?c_diag:c_top):(_j<0?c_left:blk->cost); _i=(_i+B_H)%B_H; _j=(_j+B_W)%B_W; _c[B_H*(_j+_i)+_i]; })

void blk_solve2(blk_t* blk, TC* c_top, TC* c_left, TC* c_diag) {
	for (unsigned i=0;i<blk->mi;++i) {
		for (unsigned j=0;j<blk->mj;++j) {
			TB b='/'; TC c=0,c2;  // stop
			if (!INIT(blk->bi*B_H+i,blk->bj*B_W+j)) {
				c2=blk->cost[idx(i,j)];
				// Non-serial partial result
				if (c2>c) { c=c2; b=blk->back[idx(i,j)]; }
				// Finish non-serial
				for (size_t k=1; k<=j; ++k) { c2=blk->cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=p_left[k]; } }
				for (size_t k=1; k<=i; ++k) { c2=blk->cost[idx(i-k,j)]-p_gap(k); if (c2>=c) { c=c2; b=p_up[k]; } }
				// Monadic diagonal
				c2=COST_B1(i-1,j-1) + p_cost(blk->in[0][i],blk->in[1][j]); if (c2>=c) { c=c2; b='\\'; }
			}
			blk->cost[idx(i,j)] = c;
			blk->back[idx(i,j)] = b;
		}
	}
}

int main(int argc, char** argv) {
	init();
	FILE* f;
	f=fopen("ref.txt","w"); solve1(); print(f); fclose(f);
	f=fopen("t.txt","w"); solve2(); print(f); fclose(f);
	int r=sys_exec("diff","-dur","ref.txt","t.txt",NULL);
	printf("\n\n");
	sys_exec("cat","t.txt",NULL);
	printf("\nCorrectness: %s\n",!r?"no differences":"FAILURE !!!");
	unlink("ref.txt"); unlink("t.txt");

	cleanup();
	return 0;
}

#endif