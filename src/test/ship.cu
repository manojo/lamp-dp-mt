#include "hip/hip_runtime.h"
#include "include/common.h"

#define OP(a,b,idx) { int d=a[idx]-b[idx]; a[idx]+=b[idx]; b[idx]=d<0?-d:d; }

__global__ void fun(int* a, int* b, unsigned size) {
	unsigned idx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	if (idx<size) { OP(a,b,idx) }
}

__global__ void fun1(int* a, int* b, unsigned size) {
	unsigned idx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	if (idx==0) for (unsigned i=0;i<size;++i) { OP(a,b,i) }

}

int main(int argc,char** argv) {
	hipDeviceReset();
	const unsigned size=1<<22; // 3<<24 = max can be reserved for 512MB GPU
	const unsigned int ms=size*sizeof(int);

	// 3<<24 = 48M pairs of (4+4) chars => 384Mb data to and back from GPU
	// Note that it goes faster with -d64 on my CPU (!)

	int* ca = (int*)malloc(ms);
	int* cb = (int*)malloc(ms);
	int* ga; cuErr(hipMalloc(&ga,ms));
	int* gb; cuErr(hipMalloc(&gb,ms));

for (int i=0;i<10;++i) {
	struct timeval ts,te;
	gettimeofday(&ts, NULL);
	for (unsigned i=0;i<size;++i) { OP(ca,cb,i) }

	gettimeofday(&te, NULL);
	printf("CPU time: %.3f ms\n", (float)( 1000.0*(te.tv_sec-ts.tv_sec) + 0.001*(te.tv_usec-ts.tv_usec) ) );

	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));

	// Test 1: forth and back memory transfer
	gettimeofday(&ts, NULL);
	cuPut(ca,ga,ms,stream);
	cuPut(cb,gb,ms,stream);
	fun<<<size/32, 32, 0, stream>>>(ga,gb,size);
	cuGet(ca,ga,ms,stream);
	cuGet(cb,gb,ms,stream);
	cuSync(stream);
	gettimeofday(&te, NULL);
	printf("GPU time1: %.3f ms\n", (float)( 1000.0*(te.tv_sec-ts.tv_sec) + 0.001*(te.tv_usec-ts.tv_usec) ) );

	// Test 2: backtrack on GPU
	gettimeofday(&ts, NULL);
	fun1<<<1, 1, 0, stream>>>(ga,gb,size);
	cuSync(stream);
	gettimeofday(&te, NULL);
	printf("GPU time2: %.3f ms\n", (float)( 1000.0*(te.tv_sec-ts.tv_sec) + 0.001*(te.tv_usec-ts.tv_usec) ) );
}

	free(ca);
	free(cb);
	hipFree(ga);
	hipFree(gb);

	hipDeviceReset();
	return 0;
}


/*
With -m64
CPU time: 330.254 ms <- cold caches
GPU time: 511.160 ms
CPU time: 110.565 ms
GPU time: 510.759 ms
CPU time: 106.342 ms
GPU time: 374.137 ms
CPU time: 110.514 ms
GPU time: 285.962 ms
CPU time: 105.509 ms
GPU time: 286.003 ms
CPU time: 105.665 ms
GPU time: 286.128 ms
CPU time: 106.243 ms
GPU time: 285.868 ms
CPU time: 108.147 ms
GPU time: 285.921 ms
CPU time: 105.941 ms
GPU time: 285.827 ms
CPU time: 107.504 ms
GPU time: 285.893 ms <- hot caches 2.6x slowdown vs CPU

Without -m64
CPU time: 417.344 ms
GPU time: 511.177 ms
CPU time: 149.060 ms
GPU time: 510.771 ms
CPU time: 149.459 ms
GPU time: 360.264 ms
CPU time: 147.718 ms
GPU time: 286.457 ms
CPU time: 147.381 ms
GPU time: 285.966 ms
CPU time: 150.956 ms
GPU time: 283.353 ms
CPU time: 146.627 ms
GPU time: 283.340 ms
CPU time: 149.717 ms
GPU time: 283.385 ms
CPU time: 146.716 ms
GPU time: 283.412 ms
CPU time: 146.732 ms
GPU time: 286.180 ms <- 1.95x slowdown vs CPU

fun() only, -m64
CPU time: 332.379 ms
GPU time: 0.062 ms
CPU time: 104.286 ms
GPU time: 0.027 ms
CPU time: 107.368 ms
GPU time: 0.025 ms
CPU time: 104.345 ms
GPU time: 0.028 ms
CPU time: 104.347 ms
GPU time: 0.027 ms
CPU time: 103.969 ms
GPU time: 0.026 ms
CPU time: 115.863 ms
GPU time: 0.025 ms
CPU time: 132.301 ms
GPU time: 0.026 ms
CPU time: 126.471 ms
GPU time: 0.027 ms
CPU time: 107.523 ms
GPU time: 0.024 ms
*/