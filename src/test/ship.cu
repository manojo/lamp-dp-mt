#include "hip/hip_runtime.h"
#include "include/common.h"

#define OP(a,b,idx) { int d=a[idx]-b[idx]; a[idx]+=b[idx]; b[idx]=d<0?-d:d; }

__global__ void fun(int* a, int* b, unsigned size) {
	unsigned idx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	if (idx<size) { OP(a,b,idx) }
}

__global__ void fun1(int* a, int* b, unsigned size) {
	unsigned idx = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*blockDim.x*gridDim.x;
	if (idx==0) for (unsigned i=0;i<size;++i) { OP(a,b,i) }

}

/*
 * Processing 992 Mb of data
 * CPU: 162.10 ms
 * GPU: 483.20 ms (including transfers)
 * GPU: 122.69 ms (without transfers)
 * => 5.3743 GB/s for transfer
 */

int main(int argc,char** argv) {
	hipDeviceReset();
	// We transfer 2 pairs of int, hence 8 bytes, my card mem is 2^30 bytes
	const unsigned size=31<<22; // we need some memory for the program itself
	fprintf(stderr,"Processing %ld Mb of data:\n",size*sizeof(int)*2 /1024/1024);

	const unsigned int ms=size*sizeof(int);
	int* ca = (int*)malloc(ms);
	int* cb = (int*)malloc(ms);
	int* ga; cuMalloc(ga,ms);
	int* gb; cuMalloc(gb,ms);

	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));
	cuTimer tc,tg,tp;

	for (int i=0;i<20;++i) {
		// CPU benchmark
		tc.start();
		for (unsigned i=0;i<size;++i) { OP(ca,cb,i) }
		tc.stop();

		// GPU + transfer benchmark
		cuSync(stream);
		tg.start();
		cuPut(ca,ga,ms,stream);
		cuPut(cb,gb,ms,stream);
		fun<<<size/32, 32, 0, stream>>>(ga,gb,size);
		cuGet(ca,ga,ms,stream);
		cuGet(cb,gb,ms,stream);
		cuSync(stream);
		tg.stop();

		// GPU: processing only
		tp.start();
		fun<<<size/32, 32, 0, NULL>>>(ga,gb,size);
		tp.stop();
	}

	fprintf(stderr,"CPU: "); tc.print(); fprintf(stderr,"\n");
	fprintf(stderr,"GPU: "); tg.print(); fprintf(stderr,"\n");
	fprintf(stderr,"Prc: "); tp.print(); fprintf(stderr,"\n");

	cuErr(hipStreamDestroy(stream));
	cuFree(ga); cuFree(gb);
	free(ca); free(cb);
	hipDeviceReset();
	return 0;
}
