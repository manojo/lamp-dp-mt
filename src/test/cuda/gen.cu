#include "hip/hip_runtime.h"
#include "include/common.h"

#define _unroll _Pragma("unroll 5") // Optimized by hand for my GPU
#define M_W 9UL // == size(input1) + 1
#define M_H 9UL // == M_W || size(input2) + 1 if (twotracks)

// GENERATED
// -----------------------------------------------------------------------------

// Type: sequence parser
// Rule #0  'm1'     : id=1  alt=2  cat=1  min=1  max=-1
// Rule #1  'aggr'   : id=0  alt=1  cat=2  min=3  max=-1
// Rule #2  'm2'     : id=3  alt=1  cat=0  min=1  max=-1
typedef struct __T2ii T2ii;
typedef struct __T3iii T3iii;
typedef struct __bt0 bt0;
typedef struct __bt1 bt1;
typedef struct __bt2 bt2;
struct __T2ii { int _1; int _2; };
struct __T3iii { int _1; int _2; int _3; };
struct __bt0 { short rule; };
struct __bt1 { short rule; short pos[1]; };
struct __bt2 { short rule; short pos[2]; };
#define input_t T2ii
typedef struct { T3iii m1; T3iii aggr; T3iii m2; } cost_t;
typedef struct { bt1 m1; bt2 aggr; bt0 m2; } back_t;
typedef struct { short i,j,rule; short pos[2]; } trace_t;
#define input_t T2ii
const unsigned trace_len[4] = {2,1,1,0};
void g_init(input_t* in1, input_t* in2);
void g_free();
void g_solve();
T3iii g_backtrack(trace_t** trace, unsigned* size);

#define MEM_MATRIX ((M_H*(M_H+1))/2)
#define idx(i,j) ({ unsigned _i=(i),_d=M_H+1+_i-(j); MEM_MATRIX - (_d*(_d-1))/2 +_i; })
static input_t *g_in1 = NULL, *g_in2 = NULL;
static cost_t *g_cost = NULL;
static back_t *g_back = NULL;

__device__ inline T3iii fun0(T2ii i) { return (T3iii){i._1,0,i._2}; }
__device__ inline int fun1(T3iii a) { return a._2; }
__device__ inline T3iii fun2(T3iii l, T3iii r) { return (T3iii){l._1, l._2 + r._2 + l._1 * l._3 * r._3, r._3}; }
__device__ inline bool fun3(int i, int j) { return i%2==j%2; }

__global__ void gpu_solve(const input_t* in1, const input_t* in2, cost_t* cost, back_t* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
  const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned tN = blockDim.x * gridDim.x;
  const unsigned tB = blockIdx.x;
  unsigned tP=s_start; // block progress
  for (unsigned jj=s_start; jj<s_stop; ++jj) {
    for (unsigned ii=tI; ii<M_H; ii+=tN) {
      unsigned i = M_H-1-ii, j = i+jj;
      if (j<M_W) {
        back_t _back = {{-1,{0}},{-1,{0,0}},{-1}};
        cost_t _cost = {}; // init to 0
        #define VALID(I,J,RULE) (back[idx(I,J)].RULE.rule!=-1)
        /* --- m1[i,j] --- */
        if (i+1==j) {
          T3iii _c=fun0(in1[i]); if (fun1(_c)<fun1(_cost.m1) || _back.m1.rule==-1) { _cost.m1=_c; _back.m1=(bt1){1}; }
        }
        _unroll for(int k=i+1; k<j; ++k) {
          T3iii _c=fun2(cost[idx(i,k)].m1,cost[idx(k,j)].m1); if (fun1(_c)<fun1(_cost.m1) || _back.m1.rule==-1) { _cost.m1=_c; _back.m1=(bt1){2,{k}}; }
        }
        /* --- aggr[i,j] --- */
        for(int k=i+1,ku=j-2; k<=ku; ++k) {
          T3iii _c1; bt0 _b1={-1};
          if (i+1<=k && 1==0) {
            T3iii _c=fun2((T3iii){},cost[idx(i,k)].m1); if (fun1(_c)<fun1(_c1) || _b1.rule==-1) { _c1=_c; _b1=(bt0){0}; }
          }
          T3iii _c2; bt1 _b2={-1,{}};
          _unroll for(int l=k+1; l<j; ++l) {
            if (fun3(k,l)) {
              T3iii _c=fun2(cost[idx(k,l)].m1,cost[idx(l,j)].m1); if (fun1(_c)<fun1(_c2) || _b2.rule==-1) { _c2=_c; _b2=(bt1){0,{l}}; }
            }
          }
          if (_b1.rule!=-1 && _b2.rule!=-1) {
            T3iii _c=fun2(_c1,_c2); if (fun1(_c)<fun1(_cost.aggr) || _back.aggr.rule==-1) { _cost.aggr=_c; _back.aggr=(bt2){0,{k,_b2.pos[0]}}; }
          }
        }
        /* --- m2[i,j] --- */
        if (i+1<=j) {
          T3iii _c=cost[idx(i,j)].m1; if (fun1(_c)<fun1(_cost.m2) || _back.m2.rule==-1) { _cost.m2=_c; _back.m2=(bt0){3}; }
        }
        cost[idx(i,j)] = _cost;
        back[idx(i,j)] = _back;
      }
    }
    // Sync between blocks, removing __threadfence() here is incorrect but works
    // __threadfence();
    if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
    __syncthreads();
  }
}

__global__ void gpu_backtrack(trace_t* trace, unsigned* size, back_t* back, int i0, int j0) {
  const unsigned trace_len[4] = {2,1,1,0};  trace_t *rd=trace, *wr=trace; *size=0;
  #define PUSH_BACK(I,J,RULE) { wr->i=I; wr->j=J; wr->rule=RULE; ++wr; ++(*size); }
  PUSH_BACK(i0,j0,1);
  for(;rd<wr;++rd) {
    bt2* bt;
    switch (rd->rule) {
      case 0: bt=(bt2*)&back[idx(rd->i,rd->j)].aggr; break;
      case 1: bt=(bt2*)&back[idx(rd->i,rd->j)].m1; break;
      case 2: bt=(bt2*)&back[idx(rd->i,rd->j)].m1; break;
      case 3: bt=(bt2*)&back[idx(rd->i,rd->j)].m2; break;
    }
    rd->rule=bt->rule;
    for (int i=0,l=trace_len[rd->rule]; i<l; ++i) rd->pos[i]=bt->pos[i];
    switch (rd->rule) {
      case 0: PUSH_BACK(rd->i+0,rd->pos[0],1); PUSH_BACK(rd->pos[0],rd->pos[1],1); PUSH_BACK(rd->pos[1],rd->j,1); break;
      case 1: break;
      case 2: PUSH_BACK(rd->i,rd->pos[0],1); PUSH_BACK(rd->pos[0],rd->j,1); break;
      case 3: PUSH_BACK(rd->i,rd->j,1); break;
    }
  }
}

void g_init(input_t* in1, input_t* in2) {
  cuMalloc(g_in1,sizeof(input_t)*(M_H-1));
  cuPut(in1,g_in1,sizeof(input_t)*(M_H-1),NULL);
  g_in2=NULL;
  cuMalloc(g_cost,sizeof(cost_t)*MEM_MATRIX);
  cuMalloc(g_back,sizeof(back_t)*MEM_MATRIX);
}

void g_free() { cuFree(g_in1); cuFree(g_cost); cuFree(g_back); hipDeviceReset(); }

void g_solve() {
  #define WARP_SIZE 32 // constant over CUDA devices
  unsigned blk_size = WARP_SIZE;
  unsigned blk_num = (M_H+blk_size-1)/blk_size;
  unsigned* lock; cuMalloc(lock,sizeof(unsigned)*blk_num);
  cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
  gpu_solve<<<blk_num, blk_size, 0, NULL>>>(g_in1, g_in2, g_cost, g_back, lock, 0, (M_W));
  cuFree(lock);
}

T3iii g_backtrack(trace_t** trace, unsigned* size) {
  unsigned i0=0, j0=M_W-1;
  if (trace && size) {
    unsigned mem=(M_W+M_H)*sizeof(trace_t);
    trace_t *g_trace=NULL; cuMalloc(g_trace,mem);
    unsigned *g_size=NULL; cuMalloc(g_size,sizeof(unsigned));
    gpu_backtrack<<<1,1,0,NULL>>>(g_trace, g_size, g_back, i0, j0);
    cuGet(size,g_size,sizeof(unsigned),NULL); cuFree(g_size); mem=(*size)*sizeof(trace_t);
    *trace=(trace_t*)malloc(mem); cuGet(*trace,g_trace,mem,NULL); cuFree(g_trace);
  }
  T3iii res; cuGet(&res,&g_cost[idx(i0,j0)].m1,sizeof(T3iii),NULL);
  return res;
}

// -----------------------------------------------------------------------------
// GENERATED END

int main() {
	hipDeviceReset();
	input_t a[8] = {{1,2},{2,20},{20,2},{2,4}, {4,2},{2,1},{1,7},{7,3}};
	// gapc.eu: 1,2,2,20,20,2,2,4,4,2,2,1,1,7,7,3 -> 122, 1, 3
	g_init(a,NULL);
	g_solve();
	trace_t* t; unsigned sz;
	T3iii res = g_backtrack(&t,&sz);
	for (unsigned i=0;i<sz;++i) {
		printf("Trace: %2d,%2d : %2d [%d,%d]\n",t[i].i,t[i].j,t[i].rule,t[i].pos[0],t[i].pos[1]);
	}
	printf("Result = (%d, %d, %d)\n",res._1,res._2,res._3);
	g_free();
	hipDeviceReset();
	return 0;
}
