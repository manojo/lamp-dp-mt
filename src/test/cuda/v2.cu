#include "hip/hip_runtime.h"
#include "include/common.h"

#define M_W 128LU   // in1.size
#define M_H 128LU   // in0.size
//#define SPLITS 8  // number of kernels to be successively launched
#define WARP_SIZE 32

// XXX: fix addressing from [0,1] .. [n-1,n] and n+1 for the rectangular case (0,0) to (n,n)

// -----------------------------------------------------------------------------
// Memory addressing strategies
#if defined(SH_TRI)
	#undef SH_RECT
	#undef M_W
	#define M_W M_H
	// compact triangle address: full main diagonal, then the second, ...
	// idx(i,j) = |M| - |/\| +i  (with |M|=MEM_MATRIX)
	//     |/\| = d*(d+1)/2      (smallest triangle including position element)
	//        d = M_H+1+_i-_j
	#define MEM_MATRIX ((M_H*(M_H+1))/2) // lower right triangle, including diagonal
	#define idx(i,j) ({ unsigned _i=(i),_d=M_H+1+_i-(j); MEM_MATRIX - (_d*(_d-1))/2 +_i; })
#elif defined(SH_RECT)
	// block-lines address: smaller parallelograms in lines of height B_H
	#define B_H WARP_SIZE
	#define MEM_MATRIX (M_W* ((M_H+B_H-1)/B_H)*B_H  +B_H*B_H)
	#define idx(i,j) ({ unsigned _i=(i); (B_H*((j)+(_i%B_H)) + (_i%B_H) + (_i/B_H)*M_W*B_H); })
#else
	#error "Matrix must have a shape"
#endif
#if !defined(M_H) || !defined(M_W)
#error "Matrix dimensions undefined"
#endif

// -----------------------------------------------------------------------------
// Maximal cost
#define COST_MAX ((1ULL<<(sizeof(TC)*8-1))-1) // max signed value

// Backtracking
#define BT_STOP ((TB)(~0ULL)) // = 0xff...ff        // base case, stop backtracking
// - (direction,value), sizeof(TB)*8-2 bits available for the value
#define DIR_UP   0x1 // up direction
#define DIR_LEFT 0x2 // left direction
#define DIR_DIAG 0x3 // up-left diagonal direction
#define BT(dir,value) ((dir) | ((value)<<2)) // new (direction,value) backtrack
#define BT_V(b) ((b)>>2)                      // extract the value
#define BT_D(b) ((b)&3)                       // extract the direction

// -----------------------------------------------------------------------------

#define _unroll _Pragma ("unroll 5") // Optimized by hand for my GPU
#ifdef SH_RECT      // Smith-Waterman with arbitrary gap cost function (rectangular matrix).
#define TI char     // input data type
#define TI_CHR(X) X // conversion to char
#define TC int      // cost type
#define TB short    // backtrack type (2 bits for direction + 14 for value)
// Initialization
#define INIT(i,j)  ((i)==0 || (j)==0) // matrix initialization at [stop]
// Input (returns padded strings)
TI* p_input(bool horz=false) {
	static unsigned sh=time(NULL), sv=time(NULL)+573; // keep consistent
	const char alph[4]={'A','C','G','T'}; unsigned n = horz?M_W:M_H; mseed(horz?sh:sv);
	TI* in = (TI*)malloc(n*sizeof(TI)); in[0]='#'; // padding
	for (unsigned i=1;i<n;++i) in[i]=alph[mrand()%4]; return in;
}
// Helpers functions
_hostdev _inline TC p_gap(int k) { return 20-k; }
_hostdev _inline TC p_cost(char s, char t) { return s==t?1:0; }
// Computation kernel
#define p_kernel \
	_unroll for (unsigned k=1; k<i; ++k) { c2=cost[idx(i-k,j)]-p_gap(k); if (c2>c) { c=c2; b=BT(DIR_UP,k); } }
	_unroll for (unsigned k=1; k<j; ++k) { c2=cost[idx(i,j-k)]-p_gap(k); if (c2>c) { c=c2; b=BT(DIR_LEFT,k); } }
	c2 = cost[idx(i-1,j-1)]+p_cost(in0[i],in1[j]); if (c2>=c) { c=c2; b=BT(DIR_DIAG,1); }
#endif

#ifdef SH_TRI
// Matrix multiplication parenthesizing (triangular matrix)
typedef struct { unsigned rows,cols; } mat_t;
#define TI mat_t         // input data type
#define TI_CHR(X) ('0'+(X).rows) // conversion to char (debug)
#define TC unsigned long // cost type
#define TB short         // backtrack type (2 bits for direction + 14 for value)
// Initialization
#define INIT(i,j) (j<=i) // matrix initialization at [stop]
// Input
TI* p_input() {
	static unsigned s=time(NULL); mseed(s); // keep consistent
	TI* in = (TI*)malloc(M_H*sizeof(TI));
	#define RNZ ({ unsigned x; do { x=mrand()%10; } while (!x); x; })
	in[0].rows=RNZ; for (unsigned i=1;i<M_H;++i) { in[i-1].cols=in[i].rows=RNZ; } in[M_H-1].cols=RNZ;
	return in;
}
// Computation kernel
#define p_kernel c=COST_MAX; \
	_unroll for (unsigned k=i; k<j; ++k) { c2=cost[idx(i,k)] + cost[idx(k+1,j)] + in0[i].rows * in0[k].cols * in0[j].cols; if (c2<c) { c=c2; b=(TB)k; } }
#endif

// -----------------------------------------------------------------------------

__global__ void gpu_solve(const TI* in0, const TI* in1, TC* cost, TB* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
	const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x;
	const unsigned tN = blockDim.x * gridDim.x;
	const unsigned tB = blockIdx.x;
	unsigned tP=s_start; // block progress
#ifdef SH_RECT
	#ifdef SPLITS
	s_start += (tN*s_start)/M_W;
	s_stop += (tN*s_stop)/M_W;
	tP=s_start;
	#else
	s_stop+=tN;
	#endif
	for (unsigned jj=s_start; jj<s_stop; ++jj) {
		for (unsigned i=tI; i<M_H; i+=tN) {
			unsigned j = jj-tI;
			if (j<M_W) {
#endif
#ifdef SH_TRI
	for (unsigned jj=s_start; jj<s_stop; ++jj) {
		for (unsigned ii=tI; ii<M_H; ii+=tN) {
			unsigned i=M_H-1-ii;
			unsigned j=i+jj;
			if (j<M_W) {
#endif
				TB b=BT_STOP; TC c=0,c2; // stop
				if (!INIT(i,j)) { p_kernel }
				cost[idx(i,j)] = c;
				back[idx(i,j)] = b;
			}
		}
		// Sync between blocks, removing __threadfence() is incorrect but works
		if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
		__syncthreads();
	}
}

// -----------------------------------------------------------------------------
// GPU helpers
#ifdef __HIPCC__
static TI* g_in[2]={NULL,NULL};
static TC* g_cost = NULL;
static TB* g_back = NULL;
#endif

int main(int argc, char** argv) {
	// Init
	cuMalloc(g_in[0],sizeof(TI)*M_H);
	//cuPut(in0,g_in[0],sizeof(TI)*M_H,NULL);
	#ifdef SH_RECT
		cuMalloc(g_in[1],sizeof(TI)*M_W);
		//cuPut(in1,g_in[1],sizeof(TI)*M_W,NULL);
	#else
		g_in[1]=g_in[0];
	#endif
	cuMalloc(g_cost,sizeof(TC)*MEM_MATRIX);
	cuMalloc(g_back,sizeof(TB)*MEM_MATRIX);

	// Compute
	unsigned blk_size = WARP_SIZE;
	unsigned blk_num = (M_H+blk_size-1)/blk_size;
	unsigned* lock;
	cuMalloc(lock,sizeof(unsigned)*blk_num);
	cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
#ifdef SPLITS
	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));
	for (int i=0;i<SPLITS;++i) {
		unsigned s0=(M_W*i)/SPLITS;
		unsigned s1=(M_W*(i+1))/SPLITS;
		gpu_solve<<<blk_num, blk_size, 0, stream>>>(g_in[0], g_in[1], g_cost, g_back, lock, s0, s1);
	}
	cuSync(stream);
	cuErr(hipStreamDestroy(stream));
#else
	gpu_solve<<<blk_num, blk_size, 0, NULL>>>(g_in[0], g_in[1], g_cost, g_back, lock, 0, M_W);
#endif
	cuFree(lock);

	// Collect score
	// XXX

	// Collect backtrack
	// XXX

	// Free
	cuFree(g_in[0]);
	#ifdef SH_RECT
	cuFree(g_in[1]);
	#endif
	cuFree(g_cost); cuFree(g_back);

	return 0;
}
