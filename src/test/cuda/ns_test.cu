#include "hip/hip_runtime.h"
#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 4096LU  // matrix dimension (at most 14336LU for in-memory, 12288LU OK)
#define M_H 4096LU  // matrix dimension
#define SPLITS 8    // number of kernels to be successively launched


// -----------------------------------------------------------------------------
#include "include/ns_prob.h" // problem definitions
#include "include/ns.h"      // common functions
#include "include/ns_cpu.h"  // cpu implementation
#include "include/ns_gpu.h"  // gpu implementation
// -----------------------------------------------------------------------------

	/*
	 * Optimizations to implement when we are confident about the problem structure:
	 * SH_RECT: Since we sweep the rectangle with a diagonal, at iteration M_W+(tN+1)/2
	 *          half of them will we unused. At this point, break the loop and construct
	 *          a new loop where 2 threads are assigned the same cell, that is thread tI
	 *          is assigned to cell (i/2, j). We would require tI/2 shared cost cells to
	 *          exchange maximal cost, the cell with cost=maximum writes (cost,backtrack)
	 *          to the original matrices.
	 * SH_TRI : Similarly, when half of the threads go out of the triangle, we can reassign
	 *          two threads per cell, then repeat the operation at 4 and 8 (possibly 16 and 32).
	 * SH_PARA: No optimization possible since every pair of thread has very different dependences
     */
	// 1 block = 88ms, 128x128, correct, down to 30ms with multi-blocks
	// 1043ms for 1024x1024 => we are above "Optimizing DP on GPU via adaptive thread parallelism"
	// => we need to use the same technique as they do (but first compare on the same problem)

__global__ void gpu_solve(const TI* in0, const TI* in1, TC* cost, TB* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
	const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x; // * (  + blockIdx.y*gridDim.x );
	const unsigned tN = blockDim.x * gridDim.x;
	const unsigned tB = blockIdx.x;
	unsigned tP=s_start; // block progress

#ifdef SH_RECT
	#ifdef SPLITS
	if (s_start) { tP+=tN*s_start/B_W; s_start+=tN*s_start/B_W; }
	s_stop+=tN*s_stop/B_W;
	#else
	s_stop+=tN;
	#endif


	for (unsigned jj=s_start; jj<s_stop; ++jj) {
		for (unsigned i=tI; i<M_H; i+=tN) {
			unsigned j = jj-tI;
			if (j<M_W) {
#endif
#ifdef SH_TRI
	for (unsigned jj=s_start; jj<s_stop; ++jj) {
		for (unsigned ii=tI; ii<M_H; ii+=tN) {
			unsigned i=M_H-1-ii;
			unsigned j=i+jj;
			if (j<M_W) {
#endif
#ifdef SH_PARA
	for (unsigned jj=s_start; jj<s_stop; ++jj) {
		for (unsigned i=tI; i<M_H; i+=tN) {
			unsigned j=jj+i;
			{
#endif
				TB b=BT_STOP; TC c=0,c2; // stop
				if (!INIT(i,j)) { p_kernel }
				cost[idx(i,j)] = c;
				back[idx(i,j)] = b;
			}
		}

		// Sync between blocks, removing __threadfence() is incorrect but works
		// __threadfence();
		#ifdef SH_PARA // wait for all blocks
		__syncthreads();
		++tP; if (threadIdx.x==0) lock[tB]=tP;
		for (unsigned b=threadIdx.x;b<gridDim.x;b+=blockDim.x) { // sync with all blocks
			while(lock[(tB+b)%gridDim.x]<tP) {}
		}
		#else // wait for previous block only
		if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
		#endif
		__syncthreads();
	}
}

void g_solve() {
	unsigned blk_size = 32; // = warp size
	unsigned blk_num = (M_H+blk_size-1)/blk_size;
#ifdef SH_PARA // 384 cores (GF650M) XXX: find out why deadlock at >32 blocks
	if (blk_num>32) blk_num=32;
#endif
	unsigned* lock;
	cuMalloc(lock,sizeof(unsigned)*blk_num);
	cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
#ifdef SPLITS
	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));
	for (int i=0;i<SPLITS;++i) {
		unsigned s0=(M_W*i)/SPLITS;
		unsigned s1=(M_W*(i+1))/SPLITS;
		gpu_solve<<<blk_num, blk_size, 0, stream>>>(g_in[0], g_in[1], g_cost, g_back, lock, s0, s1);
	}
	cuSync(stream);
	cuErr(hipStreamDestroy(stream));
#else
	gpu_solve<<<blk_num, blk_size, 0, NULL>>>(g_in[0], g_in[1], g_cost, g_back, lock, 0, M_W);
#endif
	cuFree(lock);
}




TC g_backtrack(unsigned** bt, unsigned* size) {
	TC cost;
	unsigned i,j;

	// Find the position with maximal cost along bottom+right borders
	unsigned mi=0; TC ci=0;
	unsigned mj=0; TC cj=0;
	for (unsigned i=0; i<M_H; ++i) { TC c=c_cost[idx(i,M_W-1)]; if (c>ci) { mi=i; ci=c; } }
	for (unsigned j=0; j<M_W; ++j) { TC c=c_cost[idx(M_H-1,j)]; if (c>cj) { mj=j; cj=c; } }
	if (ci>cj) { i=mi; j=M_W-1; } else { i=M_H-1; j=mj; }

	cost = c_cost[idx(i,j)];
	// Backtrack, returns a pair of coordinates in reverse order
	if (bt && size) {
		TB b;
		*bt=(unsigned*)malloc((M_W+M_H)*2*sizeof(unsigned));
		unsigned sz=0;
		unsigned* track=*bt;
		do {
			track[0]=i; track[1]=j; track+=2; ++sz;
			b = c_back[idx(i,j)];
			switch(BT_D(b)) {
				case DIR_LEFT: j-=BT_V(b); break;
				case DIR_UP: i-=BT_V(b); break;
				case DIR_DIAG: i-=BT_V(b); j-=BT_V(b); break;
			}
		} while (b!=BT_STOP);
		*size=sz;
	}


	return cost;
}




// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	// CPU solving
	//for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	//fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	//fflush(stderr);
	// GPU solving
	for (int i=0;i<1;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");

	// dbg_compare();
	// XXX: also compare backtrack

	//dbg_print(false,stdout);
	//dbg_track(false,stdout);

	dbg_cleanup();
	return 0;
}
