#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <stdint.h>
#include <string.h>
#include <unistd.h>
#include <sys/time.h>

#define _inline __attribute__((unused)) static inline
#define ASSERT(cond) extern int __assert__[1-2*(!(cond))];


// -----------------------------------------------------------------------------

#define cuErr(err) (hipError_t(err,__FILE__,__LINE__))
#define cuSync(stream) cuErr(hipStreamSynchronize(stream))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
_inline void hipError_t(hipError_t err, const char *file,  int line) { if (err!=hipSuccess) { fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); exit(EXIT_FAILURE); } }
_inline void cuInfo(bool full=true) {
	int deviceCount=0; hipError_t err=hipGetDeviceCount(&deviceCount);
	if (err==38 || deviceCount==0) { fprintf(stderr,"No CUDA device\n"); exit(EXIT_FAILURE); }
	else if (err!=hipSuccess) { fprintf(stderr,"CUDA error %d: %s.\n", err, hipGetErrorString(err)); exit(EXIT_FAILURE); }
	int driverVersion=0, runtimeVersion=0; hipDriverGetVersion(&driverVersion); hipRuntimeGetVersion(&runtimeVersion);
	printf("Found %d CUDA device(s), driver %d.%d, runtime %d.%d.\n", deviceCount, driverVersion/1000, driverVersion%100, runtimeVersion/1000, runtimeVersion%100);
	for (int dev=0; dev<deviceCount; ++dev) {
		hipDeviceProp_t prop; hipGetDeviceProperties(&prop, dev);
		hipDeviceReset();
		printf("- Device %d: '%s' (capability %d.%d, watchdog %s)\n", dev, prop.name, prop.major, prop.minor, prop.kernelExecTimeoutEnabled?"on":"off");
		if (full) {
			printf("    - Memory    : %dMb @ %dMhz mapHost=%d, unifiedAddr=%d, asyncCopy=%d\n",(int)round(prop.totalGlobalMem/1048576.0),
				prop.memoryClockRate>>10,prop.canMapHostMemory,prop.unifiedAddressing,prop.asyncEngineCount);
			int m=prop.computeMode; const char* mode=(m==0?"default":(m==1?"exclusive":(m==2?"prohibited":"exclusiveProcess")));
			printf("    - Processors: %d @ %dMHz, maxThreads=%d, warp=%d, concurrency=%d, mode=%s\n",prop.multiProcessorCount,
				prop.clockRate>>10,prop.maxThreadsPerMultiProcessor,prop.warpSize,prop.concurrentKernels,mode);
			printf("    - Limits    : %d regs/block, %ldK sharedMem/proc, %d thr/block %d thr/proc, %d blocks\n",prop.regsPerBlock,
				prop.sharedMemPerBlock>>10,prop.maxThreadsPerBlock,prop.maxThreadsPerMultiProcessor,prop.maxGridSize[0]);
		}
	}
}


#define BLOCK_SIZE  32

// vertically t, horizontally s

// size_t>= size_s
// chunk=block_size

#define C0 1

 #define max(a,b) ({ __typeof__(a)_a = (a); __typeof__ (b) _b= (b); _a > _b ? _a : _b; })

__device__ unsigned int cost_f(unsigned left, unsigned top, unsigned diag, char s, char t, back* back) {
	unsigned cl=left+C0;
	unsigned ct=top+C0;
	unsigned cd=diag+(s==t?0:C0);
	if (cl>=max(ct,cd)) { *back='l'; }
	if (ct>=max(cl,cd)) { *back='t'; }
	if (cd>=max(cl,cr)) { *back='D'; }
	return max(max(cl,cr),ct);
}

__global__ void dp1(unsigned s_size, char* s, unsigned t_size, char* t, char* back) {
	__shared__ unsigned cost[BLOCK_SIZE*3];
	unsigned idx = threadIdx.x;

	// init, go vertically down along a part of t
	unsigned d0=0;
	unsigned d1=BLOCK_SIZE;   // diag with delay 1
	unsigned d2=BLOCK_SIZE*2; // diag with delay 2

	for (unsigned i=0;i<blockDim.x;++i) {
		if (idx<=i) {
			if (idx==0 || i-idx==0) {
				back[idx][i-idx]='.'; // at start
				cost[d0+idx]=0;
			} else {
				cost[d0+idx]=cost_f(cost[d1+idx-1],cost[d1+idx],cost[d2+idx-1],s[i-idx],t[idx],back[i-idx][idx]);
			}
		}
		unsigned dt=d2; d2=d1; d1=d0; d0=dt; // shifting diagonal
		__syncthreads();
	}

	// continu

	// termine

	//unsigned index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

}

int main() {
	cuInfo();

	cuErr(hipDeviceReset());
	hipStream_t stream;
	cuErr(hipStreamCreate(&stream));

	dp1<<<1,1,0,stream>>>(NULL,0,NULL,0,0);


	hipStreamDestroy(stream);

	hipDeviceReset();
	return 0;
}