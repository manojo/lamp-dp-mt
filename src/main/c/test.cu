#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 128LU   // matrix dimension (at most 14336LU for in-memory, 12288LU OK)
#define M_H 128LU   // matrix dimension
//#define SPLITS 8  // number of kernels to be successively launched

// -----------------------------------------------------------------------------
#include "include/small_prob.h" // problem definitions
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation
// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();
	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);

	#ifdef __HIPCC__
	// GPU solving
	for (int i=0;i<4;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");
	#endif

	dbg_compare(); // XXX: also compare backtrack

	//dbg_print(false,stdout);
	dbg_track(false,stdout);
	dbg_track(true,stdout);

	dbg_cleanup();
	return 0;
}
