#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
//#define M_W 4096LU   // matrix dimension (<=14336LU theo., <=12288LU actually)
//#define M_H 4096LU   // matrix dimension
#define SPLITS splits  // number of kernels to be successively launched
int splits = 1;

// -----------------------------------------------------------------------------
#include "include/small_prob.h" // problem definitions
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation
#include "include/small_dbg.h"  // debug helpers
// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	splits = (M_W+M_H)/2048;
	if (splits<=0) splits=1;
	splits = splits*splits*splits;
	printf("Splits = %d\n",splits);

#if 0
	// GPU solving
	printf("- "); fflush(stdout); for (int i=0;i<1;++i) g_solve(); printf("GPU: "); fflush(stdout);
	for (int i=0;i<7;++i) { t.start(); g_solve(); double dt=t.stop(); printf("  %8.3f",dt/1000.0); fflush(stdout); }
	printf("\n"); fflush(stdout);

	// CPU solving
	printf("- "); fflush(stdout); c_solve(); printf("CPU: "); fflush(stdout);
	for (int i=0;i<3;++i) { t.start(); c_solve(); double dt=t.stop(); printf("  %8.3f",dt/1000.0); fflush(stdout); }
	printf("\n"); fflush(stdout);
#else
	// GPU solving
	printf("- "); fflush(stdout); for (int i=0;i<10;++i) g_solve(); printf("GPU: "); fflush(stdout);
	for (int i=0;i<20;++i) { t.start(); g_solve(); double dt=t.stop(); printf("  %8.3f",dt/1000.0); fflush(stdout); }
	printf("\n"); fflush(stdout);

	// CPU solving
	printf("- "); fflush(stdout); c_solve(); printf("CPU: "); fflush(stdout);
	for (int i=0;i<20;++i) { t.start(); c_solve(); double dt=t.stop(); printf("  %8.3f",dt/1000.0); fflush(stdout); }
	printf("\n"); fflush(stdout);
#endif
	/*
	#define FAST
	#ifndef FAST
	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);
	#endif

	#ifdef __HIPCC__
	// GPU solving
		#ifdef FAST
		const unsigned loops=1;
		#else
		const unsigned loops=4;
		#endif
	for (int i=0;i<loops;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");
	#endif

	#ifndef FAST
	dbg_compare();
	//dbg_print(false,stdout);
	dbg_track(false,stdout);
	dbg_track(true,stdout);
	#endif
	*/

	dbg_cleanup();
	return 0;
}
