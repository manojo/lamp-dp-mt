#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
//#define M_W 4096LU   // matrix dimension (<=14336LU theo., <=12288LU actually)
//#define M_H 4096LU   // matrix dimension
#define SPLITS splits  // number of kernels to be successively launched
int splits = 1;

// -----------------------------------------------------------------------------
#include "include/small_prob.h" // problem definitions
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation
#include "include/small_dbg.h"  // debug helpers
// -----------------------------------------------------------------------------

int main(int argc, char** argv) {
	cuTimer t;

	splits = (M_W+M_H)/2048; if (splits<=0) splits=1;
	splits = splits*splits*splits;
	dbg_init();

#if 1
	#define WARM_GPU 10
	#define RUNS_GPU 20
	#define RUNS_CPU 20
#else
	#define WARM_GPU 1
	#define RUNS_GPU 7
	#define RUNS_CPU 3
#endif
	// GPU solving
	printf("sprintf('%c.3f',",'%'); fflush(stdout); for (int i=0;i<10;++i) g_solve(); printf("median(["); fflush(stdout);
	for (int i=0;i<20;++i) { t.start(); g_solve(); double dt=t.stop(); printf(" %.3f",dt/1000.0); fflush(stdout); }
	printf(" ])) %c CUDA (%ld)\n",'%',M_W); fflush(stdout);
	// CPU solving
	printf("sprintf('%c.3f',",'%'); fflush(stdout); c_solve(); printf("median(["); fflush(stdout);
	for (int i=0;i<20;++i) { t.start(); c_solve(); double dt=t.stop(); printf(" %.3f",dt/1000.0); fflush(stdout); }
	printf(" ])) %c CPU (%ld)\n",'%',M_W); fflush(stdout);

	/*
	#define FAST
	#ifndef FAST
	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);
	#endif

	#ifdef __HIPCC__
	// GPU solving
		#ifdef FAST
		const unsigned loops=1;
		#else
		const unsigned loops=4;
		#endif
	for (int i=0;i<loops;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");
	#endif

	#ifndef FAST
	dbg_compare();
	//dbg_print(false,stdout);
	dbg_track(false,stdout);
	dbg_track(true,stdout);
	#endif
	*/

	dbg_cleanup();
	return 0;
}
