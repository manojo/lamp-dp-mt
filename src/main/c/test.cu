#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
#define M_W 128LU   // matrix dimension (<=14336LU theo., <=12288LU actually)
#define M_H 128LU   // matrix dimension
//#define SPLITS 8  // number of kernels to be successively launched

// -----------------------------------------------------------------------------
#include "include/small_prob.h" // problem definitions
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation
// -----------------------------------------------------------------------------

//#define FAST

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();
	#ifndef FAST
	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);
	#endif

	#ifdef __HIPCC__
	// GPU solving
		#ifdef FAST
		const unsigned loops=1;
		#else
		const unsigned loops=4;
		#endif
	for (int i=0;i<loops;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");
	#endif

	#ifndef FAST
	dbg_compare();
	//dbg_print(false,stdout);
	dbg_track(false,stdout);
	dbg_track(true,stdout);
	#endif

	dbg_cleanup();
	return 0;
}
