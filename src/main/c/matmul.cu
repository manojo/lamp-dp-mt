#include "hip/hip_runtime.h"
#include "include/common.h" // XXX: fix path
#define SH_TRI
#define B_W 32LU
#define B_H 32LU
#define M_W 128LU
#define M_H 128LU

////////////////////////////////////////////////////////////////////////////////
// Data types
typedef struct { unsigned rows,cols; char print() { return 'X'; } } mat_t;
#define TI mat_t         // input data type
#define TI_CHR(X) ('0'+(X).rows) // conversion to char (debug)
#define TC unsigned long // cost type
#define TB short         // backtrack type (2 bits for direction + 14 for value)

// Initialization
#define INIT(i,j) (j<=i) // matrix initialization at [stop]

// Matrix multiplication parenthesizing (triangular matrix)
//
//   M[i,j]= min {i<=k<j} M[i,k] + M [k+1,j] + r_i * c_k * c_j
//
#define p_kernel \
	_infinity \
	_min_loop(k,i,j,  _cost(i,k) + _cost(k+1,j) + _in(i).rows * _in(k).cols * _in(j).cols, k )

// Once generated: execute
// nvcc -arch=sm_21 -O2 <FILE>.cu -DSH_RECT -o <OUT>
// optirun <OUT>
////////////////////////////////////////////////////////////////////////////////

// Input
TI* p_input() {
	static unsigned s=time(NULL); mseed(s); // keep consistent
	TI* in = (TI*)malloc(M_H*sizeof(TI));
	#define RNZ ({ unsigned x; do { x=mrand()%10; } while (!x); x; })
	in[0].rows=RNZ;
	for (unsigned i=1;i<M_H;++i) { in[i-1].cols=in[i].rows=RNZ; }
	in[M_H-1].cols=RNZ;
	return in;
}

// XXX: fix path
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation

int main(int argc, char** argv) {
	g_init();
	g_solve();

	unsigned *bt,size;
	TC cost=g_backtrack(&bt,&size);
	printf("Cost = %lu\n",cost);
	printf("Backtrack = \n");
	if (size) {
		unsigned i=size;
		do { --i; printf("(%d,%d) ",bt[i*2],bt[i*2+1]); } while (i);
	}
	free(bt);
	printf("\n");

	g_free();
	return 0;
}
