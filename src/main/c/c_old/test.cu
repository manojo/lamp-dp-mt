#include "include/common.h"
// Problem style (one among the 3 below)
//#define SH_RECT
//#define SH_TRI
//#define SH_PARA

// Problem dimensions
#define B_W 32LU    // block width
#define B_H 32LU    // block height
//#define M_W 128LU   // matrix dimension (<=14336LU theo., <=12288LU actually)
//#define M_H 128LU   // matrix dimension
//#define SPLITS 8  // number of kernels to be successively launched

#define SPLITS splits
int splits = 1;

// -----------------------------------------------------------------------------
#include "include/small_prob.h" // problem definitions
#include "include/small.h"      // common functions
#include "include/small_cpu.h"  // cpu implementation
#include "include/small_gpu.h"  // gpu implementation
#include "include/small_dbg.h"  // debug helpers
// -----------------------------------------------------------------------------

//#define FAST

int main(int argc, char** argv) {
	cuTimer t;
	dbg_init();

	splits = (M_W+M_H)/4096; if (splits==0) splits=1; splits=splits*splits*splits;
	printf("Splits = %d\n",splits);

/*
	c_solve(); // warmup
	printf("- CPU:");
	for (int i=0;i<10;++i) {
		t.start(); c_solve(); double dt = t.stop(); printf(" %8.3f",dt/1000); fflush(stdout);
	}
	printf("\n");
*/

	//for (int i=0;i<10;++i) g_solve(); // warmup
	printf("- GPU:");
	for (int i=0;i<3;++i) {
		t.start(); g_solve(); double dt = t.stop(); printf(" %8.3f",dt/1000); fflush(stdout);
	}
	printf("\n");


	/*
	#ifndef FAST
	// CPU solving
	for (int i=0;i<1;++i) { t.start(); c_solve(); t.stop(); }
	fprintf(stderr,"- CPU: "); t.print(); fprintf(stderr,"\n");
	fflush(stderr);
	#endif

	#ifdef __HIPCC__
	// GPU solving
		#ifdef FAST
		const unsigned loops=1;
		#else
		const unsigned loops=4;
		#endif
	for (unsigned i=0;i<loops;++i) { t.start(); g_solve(); t.stop(); }
	fprintf(stderr,"- GPU: "); t.print(); fprintf(stderr,"\n");
	#endif

	#ifndef FAST
	dbg_compare();
	//dbg_print(false,stdout);
	dbg_track(false,stdout);
	dbg_track(true,stdout);
	#endif
	*/

	dbg_cleanup();
	return 0;
}
