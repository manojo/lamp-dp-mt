#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#include <sys/stat.h>
#include <sys/types.h>

#include "../common/cudalign.hpp"
#include "../common/buffer/Buffer.hpp"
#include "../common/buffer/FileBuffer.hpp"
#include "../common/buffer/SocketBuffer.hpp"

// includes, project
#include <cutil_inline.h>

#include <map>
using namespace std;

#define SW_BUS_FLUSH

typedef int2 bus_t;

typedef int cell_t;
typedef int2 cell2_t;
typedef int3 cell3_t;
typedef int4 cell4_t;


__shared__   int  s_colx [2][THREADS_COUNT]; // Could be THREADS_COUNT-1
__shared__   int  s_coly [2][THREADS_COUNT];

texture<unsigned int, 1, hipReadModeElementType> t_seq0;
texture<unsigned char, 1, hipReadModeElementType> t_seq1;
texture<        bus_t, 1, hipReadModeElementType> t_busH;

__constant__ int d_split[BLOCKS_COUNT+1];

/**
 * Returns true only to the thread with the maximum value.
 *
 * @param idx the thread Id.
 * @param val the value that will be compared
 * @return true to the thread with the maximum value. False otherwise. If
 *              more than one thread has the maximum value, only one of them
 *              will return true.
 */
__device__ int findMax(int idx, int val) {
    __shared__ int s_max[THREADS_COUNT];
    __shared__ int s_idx;
    int count = THREADS_COUNT>>1;
    s_max[idx] = val;
    __syncthreads();

    while (count > 0) {
        if (idx < count) {
            if (s_max[idx] < s_max[idx+count]) {
                s_max[idx] = s_max[idx+count];
            }
        }
        count = count >> 1;
        __syncthreads();
    }
    if (s_max[0] == val) {
        s_idx = idx;
    }
    __syncthreads();
    return (s_idx == idx);
}

/**
 * Returns the maximum of two numbers.
 * @param a first value
 * @param b second value
 */
static __device__ int my_max(int a, int b) {
    return (a>b)?a:b;
}

static __device__ int my_max4(int a, int b, int c, int d) {
    return my_max(my_max(a,b), my_max(c,d));
}


/**
 * This procedure makes the smith waterman computation for the cell (i,j).
 *
 * @param idx the Thread Id.
 * @param i line of the cell (0..|seq0|-1)
 * @param j column of the cell (0..|seq1|-1)
 * @param busH the last special row saved
 * @param ss the variable containing seq0[i] information. If DNA/RNA, ss is the nucleotide.
 *           otherwise ss is the substitution vector for aminoaciad seq0[i].
 * @param max the maximum score found until now.
 * @param pos the best score found for each block.
 * @param e00 Input: value of E(i,j-1). Output: value of E(i,j)
 * @param h01 Input: value of H(i,j-1). Output: value of H(i,j)
 * @param h11 Input: value of H(i-1,j-1)
 * @param h10 Output: value of H(i-1,j)
 * @param load index of s_col for read (s_col[load] is used for reading; s_col[1-load] is used for writing).
 */
__device__ void kernel_sw(unsigned char s1, const int idx, const int i, const int j, bus_t* busH, const unsigned char ss, int *max, int2 *pos, cell_t *e00, cell_t *f00, const cell_t h01, const cell_t h11, const cell_t h10, cell_t *h00, const int inc, const bool store) {

    *e00 = my_max(h01-DNA_GAP_OPEN, *e00)-DNA_GAP_EXT;
    *f00 = my_max(h10-DNA_GAP_OPEN, *f00)-DNA_GAP_EXT;
    int v1 = h11+((ss!=s1)?DNA_MISMATCH:DNA_MATCH);
    *h00 = my_max4(0, v1, *e00, *f00);

    if (*max < *h00) {
        *max = *h00;
        pos->x = j;
        if (store) {
            pos->y = i+inc;
        } else {
            pos->y = inc;
        }
    }
}

__device__ void kernel_load(const int idx, const int bank, const int j, bus_t* busH, int *h, int *f, unsigned char *s) {
    *s = tex1Dfetch(t_seq1, j);
    if (idx) {
        *h = s_colx[bank][idx];
        *f = s_coly[bank][idx];
    } else {
        int2 temp = tex1Dfetch(t_busH,j);
        *h = temp.x;
        *f = temp.y;
    }
}

__device__ void kernel_flush(const int idx, const int bank, const int j, bus_t* busH, const int h, const int f) {
    if (idx == THREADS_COUNT-1) {
        int2 temp = make_int2(h, f);
        busH[j] = temp;
    } else {
        s_colx[bank][idx+1] = h;
        s_coly[bank][idx+1] = f;
    }
}

/**
 * After incrementing the column j, this procedure must be called to check if j overflows the limit of the sequence seq_1.
 * If this happens, we must set j to zero and continue the computation in the proper line and all registers must be
 * reinitialized in order to represent the first column of the matrix (only in this situation the output params will
 * be updated, otherwise the values will be kept).
 *
 * @param seq0_len size of the sequence seq0.
 * @param seq1_len size of the sequence seq1.
 * @param i [in/out] the row of the cell to be processed.
 * @param j [in/out] the column of the cell to be processed.
 * @param ss [in/out] the variable containing seq0[i] information. If DNA/RNA, ss is the nucleotide.
 *           otherwise ss is the substitution vector for aminoaciad seq0[i].
 * @param e00 [in/out] value of E(i,j-1)
 * @param h01 [in/out] value of H(i,j-1)
 * @param h11 [in/out] value of H(i-1,j-1)
 * @param jump [in/out] true if the line i, after update, is out of bound of seq_0.
 * @param H how many lines must be jumped if j overflows.
 */
template <bool LOAD_VERTICAL_BUS, bool STORE_VERTICAL_BUS>
__device__ void kernel_check_bound(int seq0_len, int seq1_len, int *i, int *j,
        uchar4* ss, cell4_t *ee, cell_t *h10,  cell4_t *h00, int *jump,
		const int4* busInH, const int4* busInE, int4* busOutH, int4* busOutE, const int idx,
        const int H) {
    if (*j>=seq1_len) {
		if (STORE_VERTICAL_BUS) {
            busOutH[idx] = *h00;
            busOutE[idx] = *ee;
		}

		*j=0;
        *i+=H;
        *jump = (*i>=seq0_len);

		if (LOAD_VERTICAL_BUS) {
			*h10 = busInH[idx].w;
			*h00 = busInH[idx+1];
			*ee = busInE[idx];
		} else {
			*ee=make_int4(0,0,0,0);
			*h00=make_int4(0,0,0,0);
			*h10=0;
		}


        /*
		*ee=make_int4(0,0,0,0);
        *h00=make_int4(0,0,0,0);
        *h10=0;*/


        unsigned int k = tex1Dfetch(t_seq0,(*i)>>2);
        ss->x = k&0xFF; k >>= 8;
        ss->y = k&0xFF; k >>= 8;
        ss->z = k&0xFF; k >>= 8;
        ss->w = k&0xFF;
    }
}



template <bool LOAD_VERTICAL_BUS, bool STORE_VERTICAL_BUS>
__launch_bounds__(THREADS_COUNT,MIN_BLOCKS_PER_SM)
__global__ void kernel_sw_short_phase(const int seq0_len, const int seq1_len, const int step, int3 *blockResult,
						bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3, const int4* busInH, const int4* busInE, int4* busOutH, int4* busOutE)
{

    int bx = blockIdx.x;
    int by = step-bx;
    if (by < 0) return;

    int idx = threadIdx.x;


    const volatile int x0 = d_split[bx];
    const int xLen = d_split[bx+1] - x0;

    int i=(by*THREADS_COUNT)+idx;
    int j=x0-idx;
    int tidx = (i % (blockDim.x*gridDim.x));
    i <<= 2;
    if (j<=0) {
        j += seq1_len;
        i -= (blockDim.x*gridDim.x)*ALPHA;
    }
    s_colx[0][idx] = s_colx[1][idx] = busV3[tidx].x; // TODO poderia ser pego de busV1[tidx].w. Certo?
    s_coly[0][idx] = s_coly[1][idx] = busV3[tidx].y;

    __syncthreads();

    int2 max_pos;
    max_pos.x = blockResult[blockIdx.x].x;
    max_pos.y = blockResult[blockIdx.x].y;
    int max = blockResult[blockIdx.x].z;

    if (i < seq0_len) {
        cell_t h11;
        cell4_t h01;
        cell4_t ee;
		h01=busV1[tidx];
		ee =busV2[tidx];
		h11=busV3[tidx].z;

        uchar4 ss;
        unsigned int k = tex1Dfetch(t_seq0,(i>>2));
        ss.x = k&0xFF; k >>= 8;
        ss.y = k&0xFF; k >>= 8;
        ss.z = k&0xFF; k >>= 8;
        ss.w = k&0xFF;

        __syncthreads();

        int jump = (i<0);

		// We need THREADS_COUNT-1 Steps to complete the pending cells
        int _k = THREADS_COUNT>>1;
        for (; _k>1; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;

			kernel_check_bound<LOAD_VERTICAL_BUS, STORE_VERTICAL_BUS>(seq0_len, seq1_len, &i, &j, &ss, &ee, &h11, &h01, &jump, busInH, busInE, busOutH, busOutE, idx, blockDim.x*gridDim.x*4);
			if (!jump) {
                unsigned char s1;
                kernel_load(idx, 1, j, busH, &h10, &f00, &s1);
                kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, 0, true);
                kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, 1, true);
                kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, 2, true);
                kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, 3, true);
                kernel_flush(idx, 0, j, busH, h00.w, f00);
            }
            j++;
            __syncthreads();

			kernel_check_bound<LOAD_VERTICAL_BUS, STORE_VERTICAL_BUS>(seq0_len, seq1_len, &i, &j, &ss, &ee, &h10, &h00, &jump, busInH, busInE, busOutH, busOutE, idx, blockDim.x*gridDim.x*4);
			if (!jump) {
                unsigned char s1;
                kernel_load(idx, 0, j, busH, &h11, &f00, &s1);
                kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h00.x, h10  , h11  , &h01.x, 0, true);
                kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h00.y, h00.x, h01.x, &h01.y, 1, true);
                kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h00.z, h00.y, h01.y, &h01.z, 2, true);
                kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h00.w, h00.z, h01.z, &h01.w, 3, true);
                kernel_flush(idx, 1, j, busH, h01.w, f00);
            }
            j++;
            __syncthreads();
        }

        {
            cell4_t h00;
            cell_t h10;
            cell_t f00;

			kernel_check_bound<LOAD_VERTICAL_BUS, STORE_VERTICAL_BUS>(seq0_len, seq1_len, &i, &j, &ss, &ee, &h11, &h01, &jump, busInH, busInE, busOutH, busOutE, idx, blockDim.x*gridDim.x*4);
			if (!jump) {
                unsigned char s1;
                kernel_load(idx, 1, j, busH, &h10, &f00, &s1);
                kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, 0, true);
                kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, 1, true);
                kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, 2, true);
                kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, 3, true);
                kernel_flush(idx, 0, j, busH, h00.w, f00);
            }
            j++;
            __syncthreads();

			kernel_check_bound<LOAD_VERTICAL_BUS, STORE_VERTICAL_BUS>(seq0_len, seq1_len, &i, &j, &ss, &ee, &h10, &h00, &jump, busInH, busInE, busOutH, busOutE, idx, blockDim.x*gridDim.x*4);

			busV1[tidx]=h00;
			busV2[tidx]=ee;
			busV3[tidx].x=s_colx[0][idx];
			busV3[tidx].y=s_coly[0][idx];
			busV3[tidx].z=h10;
        }
    }

    if (findMax(idx, max)) {
        blockResult[blockIdx.x].x = max_pos.x;
        blockResult[blockIdx.x].y = max_pos.y;
        blockResult[blockIdx.x].z = max;
    }
}

__launch_bounds__(THREADS_COUNT,MIN_BLOCKS_PER_SM)
__global__ void kernel_sw_long_phase(const int seq0_len, const int seq1_len, const int step, int3 *blockResult,
        bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3)
{
    int bx = blockIdx.x;
    int by = step-bx;
    if (by < 0) return;

    int idx = threadIdx.x;


    const int x0 = d_split[bx]+(THREADS_COUNT-1);
    const int xLen = d_split[bx+1] - x0;

    int i=(by*THREADS_COUNT)+idx;
    int j=x0-idx;
    int tidx = (i % (blockDim.x*gridDim.x));
    i <<= 2;

    s_colx[0][idx] = s_colx[1][idx] = busV3[tidx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV3[tidx].y;

    __syncthreads();

    int2 max_pos;
    max_pos.x = -1;//blockResult[blockIdx.x].x;
    max_pos.y = -1;//blockResult[blockIdx.x].y;
    int max = blockResult[blockIdx.x].z;
    if (i < seq0_len) {
        /*cell4_t h01=tex1Dfetch(t_busV1,tidx);
        cell4_t ee =tex1Dfetch(t_busV2,tidx);*/
        cell4_t h01=busV1[tidx];
        cell4_t ee =busV2[tidx];
        cell_t  h11=busV3[tidx].z;

        uchar4 ss;
        unsigned int k = tex1Dfetch(t_seq0,(i>>2));
        ss.x = k&0xFF; k >>= 8;
        ss.y = k&0xFF; k >>= 8;
        ss.z = k&0xFF; k >>= 8;
        ss.w = k&0xFF;

        int _k = xLen;
        if (_k&1) { // if odd
            cell4_t h00;
            cell_t h10;
            cell_t f00;

            unsigned char s1;
            kernel_load(idx, 1, j, busH, &h10, &f00, &s1);
            kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, 0, false);
            kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, 1, false);
            kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, 2, false);
            kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, 3, false);
            kernel_flush(idx, 0, j, busH, h00.w, f00);
            j++;
            __syncthreads();
            s_colx[1][idx] = s_colx[0][idx];
            s_coly[1][idx] = s_coly[0][idx];
            h11 = h10;
            h01 = h00;
            __syncthreads();
            _k--;
        }
        _k >>= 1;
        for (; _k; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;

            unsigned char s1;
            kernel_load(idx, 1, j, busH, &h10, &f00, &s1);
            kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, 0, false);
            kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, 1, false);
            kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, 2, false);
            kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, 3, false);
            kernel_flush(idx, 0, j, busH, h00.w, f00);
            j++;
            __syncthreads();

            //s1 = tex1Dfetch(t_seq1, j); // TODO QUE EH ISSO???? REPETIU
            kernel_load(idx, 0, j, busH, &h11, &f00, &s1);
            kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h00.x, h10  , h11  , &h01.x, 0, false);
            kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h00.y, h00.x, h01.x, &h01.y, 1, false);
            kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h00.z, h00.y, h01.y, &h01.z, 2, false);
            kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h00.w, h00.z, h01.z, &h01.w, 3, false);
            kernel_flush(idx, 1, j, busH, h01.w, f00);
            j++;
            __syncthreads();

        }
        busV1[tidx]=h01;
        busV2[tidx]=ee;
        busV3[tidx].x=s_colx[1][idx];
        busV3[tidx].y=s_coly[1][idx];
        busV3[tidx].z=h11;
    }

    if (findMax(idx, max)) {
        if (max_pos.x != -1) {
            blockResult[blockIdx.x].x = max_pos.x;
            blockResult[blockIdx.x].y = max_pos.y+i;
            blockResult[blockIdx.x].z = max;
        }
    }
}


template <bool LOAD_VERTICAL_BUS, bool STORE_VERTICAL_BUS>
__launch_bounds__(THREADS_COUNT,1)
__global__ void kernel_sw_single_phase(const int seq0_len, const int seq1_len, const int step, int3 *blockResult,
							 bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3, const int4* busInH, const int4* busInE, int4* busOutH, int4* busOutE)
{

    int idx = threadIdx.x;

    s_colx[0][idx] = s_colx[1][idx] = busV3[idx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV3[idx].y;

    __syncthreads();

    const int xLen = seq1_len;
    int by = step;
    int j=-idx;
    int i=(by*THREADS_COUNT)+idx;
    i <<= 2;
    if (j<=0) {
        j+=seq1_len;
        i-=THREADS_COUNT*4;
    }

    int2 max_pos;
    max_pos.x = blockResult[0].x;
    max_pos.y = blockResult[0].y;
    int max = blockResult[0].z;
    if (i < seq0_len) {
        cell_t h11;
        cell4_t h01;
        cell4_t ee;

		h01=busV1[idx];
		ee =busV2[idx];
		h11=busV3[idx].z;

        uchar4 ss;
        unsigned int k = tex1Dfetch(t_seq0,(i>>2));
        ss.x = k&0xFF; k >>= 8;
        ss.y = k&0xFF; k >>= 8;
        ss.z = k&0xFF; k >>= 8;
        ss.w = k&0xFF;

        __syncthreads();

        int jump = (i<0);
        int _k = xLen;
        int index = 1;
        for (; _k; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;
			kernel_check_bound<LOAD_VERTICAL_BUS, STORE_VERTICAL_BUS>(seq0_len, seq1_len, &i, &j, &ss, &ee, &h11, &h01, &jump, busInH, busInE, busOutH, busOutE, idx, THREADS_COUNT*4);
            if (!jump) {
                unsigned char s1;
                kernel_load(idx, index, j, busH, &h10, &f00, &s1);
                kernel_sw(s1, idx, i, j, busH, ss.x, &max, &max_pos, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, 0, true);
                kernel_sw(s1, idx, i, j, busH, ss.y, &max, &max_pos, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, 1, true);
                kernel_sw(s1, idx, i, j, busH, ss.z, &max, &max_pos, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, 2, true);
                kernel_sw(s1, idx, i, j, busH, ss.w, &max, &max_pos, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, 3, true);
                kernel_flush(idx, 1-index, j, busH, h00.w, f00);
            }
            index = 1-index;
            j++;
            h11 = h10;
            h01 = h00;
            __syncthreads();
        }
        busV1[idx]=h01;
        busV2[idx]=ee;
        busV3[idx].x=s_colx[index][idx];
        busV3[idx].y=s_coly[index][idx];
        busV3[idx].z=h11;
    }

    if (findMax(idx, max)) {
		blockResult[0].x = max_pos.x;
		blockResult[0].y = max_pos.y;
		blockResult[0].z = max;
    }
}

int3 find_best(int3 *d_blockResult, int size) {
    int3 result[BLOCKS_COUNT];
    cutilSafeCall(hipMemcpy(&result, d_blockResult, sizeof(uint3)*size, hipMemcpyDeviceToHost));
    int best = 0;
    if (size > 1) {
        for (int i=1; i< size; i++) {
            //printf("%2d ", result[i].z);
            if (result[best].z < result[i].z) {
                best = i;
            }
        }
        //printf("\n");
    }
    return result[best];
}

void init_score(const char* order_blosum, const char score_blosum[24][24], char out_score[32][32]) {
    memset(out_score, 0, sizeof(out_score));
    for (int i=0; order_blosum[i]; i++) {
        for (int j=0; order_blosum[j]; j++) {
            char c0 = order_blosum[i];
            char c1 = order_blosum[j];
            out_score[c0-'A'][c1-'A'] = score_blosum[i][j];
        }
    }
}

static void flush_bus(Job* job, int step, int baseY, int *h_split, bus_t* h_busH, bus_t* d_busH) {
    static map<int, SpecialRowWriter*> specialRows;

    if (job->flush_interval > 0) {
        for (int k=0; k<job->blocks && k<=step; k++) {
            int bx = k;
            const int x0 = h_split[bx];
            const int xLen = h_split[bx+1] - x0;
            int by = step-bx;
            int i=(by*THREADS_COUNT)*ALPHA;
            if (by % job->flush_interval == 0 && by>0 && i<job->seq[0].getLen()) {
                hipStreamSynchronize(0);
                cutilSafeCall(hipMemcpy(h_busH+x0, d_busH+x0, xLen*sizeof(cell2_t), hipMemcpyDeviceToHost));
                hipStreamSynchronize(0);

                if (bx==0) {
                    SpecialRowWriter* row = job->fopenNewSpecialRow(STAGE_1, i+baseY, 0);
                    specialRows[i+baseY] = row;
                    row->open();
                    //printf("Flush: %d\n", i+baseY);
                }
                specialRows[i+baseY]->write(&h_busH[x0], xLen);
                if (bx==job->blocks-1) {
                    SpecialRowWriter* row = specialRows[i+baseY];
                    specialRows.erase(i+baseY);
                    row->close();
                    delete row;
                }
            }
        }
    }
}




typedef struct {
	//cell2_t* h_busOut;
	bus_t* h_busH;
	//bus_t* h_busBase;
	int4* h_busOutH;
	int4* h_busOutE;
	int* h_busOut;
	int4* h_busInH;
	int4* h_busInE;
	int* h_busIn;
} host_structures_t;

static void allocHostStructures(Sequence* seq_vertical, Sequence* seq_horizontal, host_structures_t* host_structures) {
	const int seq0_len = seq_vertical->getLen();
	const int seq1_len = seq_horizontal->getLen();
	const int bus_size = seq1_len*sizeof(bus_t);

	host_structures->h_busH = (bus_t*)malloc(bus_size + sizeof(bus_t)); // extra padding for first read at openning
	host_structures->h_busOutH = (int4*)malloc(THREADS_COUNT*sizeof(int4));
	host_structures->h_busOutE = (int4*)malloc(THREADS_COUNT*sizeof(int4));
	host_structures->h_busOut = (int*)malloc(2*THREADS_COUNT*sizeof(int4));
	host_structures->h_busInH = (int4*)malloc((THREADS_COUNT + 1)*sizeof(int4));
	host_structures->h_busInE = (int4*)malloc((THREADS_COUNT + 1)*sizeof(int4));
	host_structures->h_busIn = (int*)malloc(2*(THREADS_COUNT+1)*sizeof(int4));
}

static void freeHostStructures(host_structures_t* host_structures) {
	free(host_structures->h_busH);
    free(host_structures->h_busIn);
    free(host_structures->h_busInH);
    free(host_structures->h_busInE);
    free(host_structures->h_busOut);
    free(host_structures->h_busOutH);
    free(host_structures->h_busOutE);
}


typedef struct {
	unsigned char* d_seq0;
	unsigned char* d_seq1;
	bus_t* d_busH;
	cell4_t* d_busV1;
	cell4_t* d_busV2;
	cell3_t* d_busV3;
	int3* d_blockResult;
	int4* d_busOutH;
	int4* d_busOutE;
	int4* d_busInH;
	int4* d_busInE;
} cuda_structures_t;

static void allocCudaStructures(Sequence* seq_vertical, Sequence* seq_horizontal,
						 cuda_structures_t* cuda_structures) {
	const int seq0_len = seq_vertical->getLen();
	const int seq1_len = seq_horizontal->getLen();
	const int bus_size = seq1_len*sizeof(bus_t);

	cuda_structures->d_seq0 = allocCudaSeq(seq_vertical);
	cuda_structures->d_seq1 = allocCudaSeq(seq_horizontal);

	cutilSafeCall(hipBindTexture(0, t_seq0, cuda_structures->d_seq0, seq0_len+GRID_HEIGHT));
	cutilSafeCall(hipBindTexture(0, t_seq1, cuda_structures->d_seq1, seq1_len+GRID_HEIGHT));

	cuda_structures->d_busV1    = (cell4_t*)allocCuda0(GRID_HEIGHT*sizeof(cell4_t));
	cuda_structures->d_busV2    = (cell4_t*)allocCuda0(GRID_HEIGHT*sizeof(cell4_t));
	cuda_structures->d_busV3    = (cell3_t*)allocCuda0(GRID_HEIGHT*sizeof(cell3_t));
	cuda_structures->d_blockResult = (int3*)allocCuda0(BLOCKS_COUNT*sizeof(int3));
	cuda_structures->d_busH     =   (bus_t*)allocCuda0(bus_size);
	cuda_structures->d_busInH    = (int4*)allocCuda0((THREADS_COUNT+1)*sizeof(int4));
	cuda_structures->d_busInE    = (int4*)allocCuda0((THREADS_COUNT+1)*sizeof(int4));
	cuda_structures->d_busOutH    = (int4*)allocCuda0(THREADS_COUNT*sizeof(int4));
	cuda_structures->d_busOutE    = (int4*)allocCuda0(THREADS_COUNT*sizeof(int4));
}

static void freeCudaStructures(cuda_structures_t* cuda) {
	cutilSafeCall(hipFree(cuda->d_seq0));
	cutilSafeCall(hipFree(cuda->d_seq1));
	cutilSafeCall(hipUnbindTexture(t_seq1));
	cutilSafeCall(hipUnbindTexture(t_seq0));
	cutilSafeCall(hipFree(cuda->d_busV1));
	cutilSafeCall(hipFree(cuda->d_busV2));
	cutilSafeCall(hipFree(cuda->d_busV3));
	cutilSafeCall(hipFree(cuda->d_blockResult));
	cutilSafeCall(hipFree(cuda->d_busH));
	cutilSafeCall(hipFree(cuda->d_busInH));
	cutilSafeCall(hipFree(cuda->d_busInE));
	cutilSafeCall(hipFree(cuda->d_busOutH));
	cutilSafeCall(hipFree(cuda->d_busOutE));
}



static void flush_column(Job* job, Buffer* buffer, int step, host_structures_t &host, cuda_structures_t &cuda) {
	if (buffer == NULL) return;
	int i = step*THREADS_COUNT;
	if (i >= 0) {
		int len = THREADS_COUNT;
		/*if ((i+len)*ALPHA > job->seq[0].getLen()) {
			len = (job->seq[0].getLen()+(ALPHA-1))/ALPHA - i;
		}*/
		if (len > 0) {
			cutilSafeCall(hipMemcpy(host.h_busOutH, cuda.d_busOutH, len*sizeof(int4), hipMemcpyDeviceToHost));
			cutilSafeCall(hipMemcpy(host.h_busOutE, cuda.d_busOutE, len*sizeof(int4), hipMemcpyDeviceToHost));
			//buffer->writeBuffer((char*)host.h_busOutH, sizeof(int4), len);
			for (int i=0; i<len; i++) {
				host.h_busOut[i*8] = host.h_busOutH[i].x;
				host.h_busOut[i*8+1] = host.h_busOutE[i].x;
				host.h_busOut[i*8+2] = host.h_busOutH[i].y;
				host.h_busOut[i*8+3] = host.h_busOutE[i].y;
				host.h_busOut[i*8+4] = host.h_busOutH[i].z;
				host.h_busOut[i*8+5] = host.h_busOutE[i].z;
				host.h_busOut[i*8+6] = host.h_busOutH[i].w;
				host.h_busOut[i*8+7] = host.h_busOutE[i].w;
			}
			buffer->writeBuffer((char*)host.h_busOut, sizeof(int), 8*len);
		}
	}
}

static void load_column(Job* job, Buffer* buffer, int step, host_structures_t &host, cuda_structures_t &cuda) {
    if (buffer == NULL) return;
    int i = step*THREADS_COUNT;
    int len = THREADS_COUNT;
    if ((i+len)*ALPHA > job->seq[0].getLen()) {
        len = 0;
        //len = (job->seq[0].getLen()+(ALPHA-1))/ALPHA - i;
    }
    if (len > 0) {
        if (step == 0) {
            host.h_busInH[0] = make_int4(0,0,0,0);
            host.h_busInE[0] = make_int4(0,0,0,0);
        } else {
            host.h_busInH[0] = host.h_busInH[THREADS_COUNT];
            host.h_busInE[0] = host.h_busInE[THREADS_COUNT];
        }
        int rc = buffer->readBuffer((char*)(host.h_busIn+8), sizeof(int), 8*len);
        for (int i=1; i<=len; i++) {
            host.h_busInH[i].x = host.h_busIn[i*8];
            host.h_busInE[i].x = host.h_busIn[i*8+1];
            host.h_busInH[i].y = host.h_busIn[i*8+2];
            host.h_busInE[i].y = host.h_busIn[i*8+3];
            host.h_busInH[i].z = host.h_busIn[i*8+4];
            host.h_busInE[i].z = host.h_busIn[i*8+5];
            host.h_busInH[i].w = host.h_busIn[i*8+6];
            host.h_busInE[i].w = host.h_busIn[i*8+7];
        }

        cutilSafeCall(hipMemcpy(cuda.d_busInH, host.h_busInH, (len+1)*sizeof(int4), hipMemcpyHostToDevice));
        cutilSafeCall(hipMemcpy(cuda.d_busInE, host.h_busInE, (len+1)*sizeof(int4), hipMemcpyHostToDevice));
    }
}


void processExternalDiagonal(Job* job, Sequence* seq_vertical, Sequence* seq_horizontal, int d, int baseY, bool smallAlgorithm, cuda_structures_t &cuda, host_structures_t &host, int* h_split, Buffer* readBuffer, Buffer* writeBuffer) {
	const int seq0_len = seq_vertical->getLen();
	const int seq1_len = seq_horizontal->getLen();
	static dim3 threads( THREADS_COUNT, 1, 1);
	static dim3 one(1, 1, 1);

	//bool output = 1;
	if (smallAlgorithm) {
		load_column(job, readBuffer, d, host, cuda);
		kernel_sw_single_phase<true, true><<< one, threads, 0>>>(seq0_len, seq1_len, d, cuda.d_blockResult, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_busInH, cuda.d_busInE, cuda.d_busOutH, cuda.d_busOutE);
		flush_column(job, writeBuffer, d-1, host, cuda);
	} else {
		load_column(job, readBuffer, d, host, cuda);

		dim3  grid( job->blocks, 1, 1);
		kernel_sw_short_phase<true, true><<< grid, threads, 0>>>(seq0_len, seq1_len, d, cuda.d_blockResult, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_busInH, cuda.d_busInE, cuda.d_busOutH, cuda.d_busOutE);

		#ifdef SW_BUS_FLUSH
		flush_bus(job, d, baseY, h_split, host.h_busH, cuda.d_busH);
		#endif

		flush_column(job, writeBuffer, d-job->blocks, host, cuda);

		kernel_sw_long_phase<<< grid, threads, 0>>>(seq0_len, seq1_len, d, cuda.d_blockResult, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3);
	}
	cutilCheckMsg("Kernel execution failed");
}

static void createSplitPositions(int j0, int xLen, int* h_split, int blocks) {
	for (int i = 0; i < blocks; i++) {
		int xMod = xLen % blocks;
		h_split[i] = j0 + ((xLen / blocks) * i + (xMod > i ? i : xMod));
	}
	h_split[blocks] = (j0+xLen) - 1;
}


void stage1(Job* job) {
	FILE* stats = job->fopenStatistics(STAGE_1);
	job->loadSpecialRows(STAGE_1);

	Sequence* seq_vertical = new Sequence(job->seq[0], false);
	Sequence* seq_horizontal = new Sequence(job->seq[1], false);

	int seq0_len = seq_vertical->getLen();
	int seq1_len = seq_horizontal->getLen();

	if (job->flush_limit > 0) {
		job->flush_interval = (seq0_len*8/(job->flush_limit/seq1_len))/(THREADS_COUNT*ALPHA)+1; // TODO constante no lugar de 8
		long long special_lines_count = (seq0_len/(THREADS_COUNT*ALPHA*job->flush_interval));
		fprintf(stats, "special lines: %lld\n", special_lines_count);
		fprintf(stats, "total size: %lld\n", special_lines_count*seq1_len*8LL);  // TODO 8
	} else {
		job->flush_interval = 0;
		job->flush_limit = 0;
	}
	fprintf(stats, "Flush Interval: %d\n", job->flush_interval);
	fprintf(stats, "Flush limit: %lld\n", job->flush_limit);

	fprintf(stats, "SW PARAM: %d/%d/%d/%d\n", DNA_MATCH, DNA_MISMATCH, DNA_GAP_FIRST, DNA_GAP_EXT);

	fprintf(stats, "--Alignment sequences:\n", job);
	fprintf(stats, ">%s (%d)\n", seq_vertical->name.c_str(), seq0_len);
	fprintf(stats, ">%s (%d)\n", seq_horizontal->name.c_str(), seq1_len);
	fflush(stats);

    hipDeviceProp_t prop = selectGPU(job->gpu, stats);
    if (job->blocks <= 0) {
        job->blocks = prop.multiProcessorCount*8;
    }

	Timer timer2;

	hipEvent_t ev_step = timer2.createEvent("STEP");
	hipEvent_t ev_start = timer2.createEvent("START");
	hipEvent_t ev_end = timer2.createEvent("END");
	hipEvent_t ev_copy = timer2.createEvent("COPY");
	hipEvent_t ev_alloc = timer2.createEvent("ALLOC");
	hipEvent_t ev_kernel = timer2.createEvent("KERNEL");
	hipEvent_t ev_writeback = timer2.createEvent("WRITEBACK");

	timer2.eventRecord(ev_start);

	printDevMem(stats);

	SpecialRowReader* lastSpecialRow = job->special_lines1->front();
	int baseY = lastSpecialRow->getRow();

	int baseX = 0;
	/*static SpecialRowWriter* specialColumn = job->fopenNewSpecialColumn(STAGE_1, 0, seq1_len);
	specialColumn->open();*/

	Buffer* writeColumn;
	writeColumn = job->loadBufferFromURL(job->flush_column_url);
	if (writeColumn != NULL) {
		writeColumn->autoFlush();
	}
	//writeColumn = new SocketBuffer("", 32111);
	//writeColumn.writeToFile("column.bin");
	//((SocketBuffer*)writeColumn)->writeToSocket();


	seq_vertical->trim(baseY+1, -1);


	host_structures_t host;
	cuda_structures_t cuda;
	allocHostStructures(seq_vertical, seq_horizontal, &host);
	allocCudaStructures(seq_vertical, seq_horizontal, &cuda); // todo retirar baseY

	const int bus_size = seq1_len*sizeof(bus_t);
	//bus_t* h_busH = (bus_t*)malloc(bus_size+sizeof(bus_t)); // extra padding for first read at openning
	if (baseY > 0)  {
		lastSpecialRow->open(host.h_busH, seq1_len); // TODO fazer leitura direta também, em vez de invertida
		int r = lastSpecialRow->read(host.h_busH, 0);
		printf("%d\n", r);
		lastSpecialRow->close();

		cutilSafeCall( hipMemcpy(cuda.d_busH, host.h_busH, bus_size, hipMemcpyHostToDevice));

		printf("Last checkpoint restored (%X)\n", baseY);
		// TODO TODO TODO FIXME restaurar valores de "bestValue" de todos os blocos, senão pode
		// ocorrer de perdermos o melhor score e o best_position.
	}

	timer2.eventRecord(ev_copy);
    cutilSafeCall(hipBindTexture(0, t_busH, cuda.d_busH, bus_size));

	int h_split[BLOCKS_COUNT + 1];
	createSplitPositions ( 0, seq1_len+1, h_split, job->blocks );
	cutilSafeCall ( hipMemcpyToSymbol(HIP_SYMBOL( d_split), h_split, sizeof ( h_split ) ) );

	timer2.eventRecord(ev_alloc);

    // execute the kernel
    //int3* d_blockResult = (int3*)allocCuda0(sizeof(int3)*BLOCKS_COUNT);
    int h=((seq0_len-baseY)/THREADS_COUNT/4+1);
    float total, left;
	job->cells_updates = ((long long)seq0_len)*seq1_len;
	float mcells = job->cells_updates/1000000.0f;
    int sum = 0;
    int steps;
    int smallAlgorithm;
    if (seq1_len < 2*job->blocks*THREADS_COUNT) {
        smallAlgorithm = 1;
        steps = h+1; // Precisa do +1?
        printf("SMALL\n");
    } else {
        smallAlgorithm = 0;
        steps = job->blocks+h+1; // Precisa do +1?
    }

    int3 best = make_int3(0,0,0);
    printf("START: %d steps\n", steps);
	FILE* statusFile = fopen("status.out", "wt");
	Buffer* readColumn = NULL;
	readColumn = job->loadBufferFromURL(job->load_column_url);
	if (readColumn != NULL) {
		readColumn->autoLoad();
	}
	/*readColumn = new SocketBuffer("localhost", 32111);
	((SocketBuffer*)readColumn)->readFromSocket();*/

	//readColumn.readFromFile("column.bin");
	for (int d=0; d<steps; d++) {
		//printf("PROGRESS: %d/%d\n", d, steps);
		bool output = (d==steps-1);

		processExternalDiagonal(job, seq_vertical, seq_horizontal, d, baseY, smallAlgorithm, cuda, host, h_split, readColumn, writeColumn);
		if (output) best = find_best(cuda.d_blockResult, smallAlgorithm ? 1:job->blocks);

		timer2.eventRecord(ev_step);
		hipStreamSynchronize(0);

		if (timer2.intervalElapsed(2.0) || (d==steps-1)) {
			best = find_best(cuda.d_blockResult, smallAlgorithm ? 1:job->blocks);
			fprintf(statusFile, "%d,%d,%d,%d\n", 0, best.x, best.y, best.z);
			fflush(statusFile);

			float t = timer2.totalTime();
			int hour = (int)(t/3600);
			int min = (int)(t - hour*3600)/60;
			int sec = (int)(t - hour*3600 - min*60);
			fprintf(stderr, "(%dh%02dm%02ds) PROGRESS: %4d/%4d  best:(%d,%d,%d)\n",
					hour, min, sec, d+1, steps, best.x, best.y, best.z);

			output = true;
		}

		/*if (output) {
			total = timer2.totalTime();
			left = total*(((float)steps)/(d+1))-total;
			sum += d<BLOCKS_COUNT?(d+1):BLOCKS_COUNT;
			printf("STEP %05d/%05d: MCUPS: %5.0f (Elapsed: %2d:%02d  Left: %2d:%02d) ",
			d+1, steps, total>0?sum*mcells/(BLOCKS_COUNT*h)/total:0.0f,
			(int)(total/60), (int)(fmod(total,60)), (int)(left/60), (int)(fmod(left,60)));

			printf("MAX: %d (%d,%d)\n", best.z, best.y+1, best.x+1+baseY);
		} */

    }
	fclose(statusFile);
	if (readColumn != NULL) {
		delete readColumn;
		readColumn = NULL;
	}
	if (writeColumn != NULL) {
		delete writeColumn;
		writeColumn = NULL;
	}
	timer2.eventRecord(ev_kernel);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

	timer2.eventRecord(ev_end);

    hipStreamSynchronize(0);

	fprintf(stats, "Best Score: %d\n", best.z);
	fprintf(stats, "Best Position: (%d,%d)\n", best.y+1+baseY, best.x+1);
	fprintf(stats, "CUDA times:\n");
	float diff = timer2.printStatistics(stats);

	fprintf(stats, "        total: %.4f\n", diff);
	fprintf(stats, "     Mi.Cells: %.4e\n", (float)job->cells_updates);
	fprintf(stats, "        MCUPS: %.4f\n", job->cells_updates/1000000.0f/(diff/1000.0f));

	printDevMem(stats);
	fprintf(stats, " FreeCudaStructures\n");
	freeCudaStructures(&cuda);
	freeHostStructures(&host);
	printDevMem(stats);

	MidpointsFile* midpointsFile = job->fopenMidpointsFile(0);
	midpointsFile->write(best.y+1+baseY, best.x+1, best.z, 0);
	midpointsFile->close();
    /*job->phase2_i0 = best.y+1;
    job->phase2_j0 = best.x+1+baseY;
    job->phase2_max = best.z;*/

	fclose(stats);
	hipDeviceReset();
	cutilCheckMsg("hipDeviceReset failed");
}
