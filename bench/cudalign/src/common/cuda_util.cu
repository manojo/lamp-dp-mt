#include "hip/hip_runtime.h"
#include <cutil_inline.h>

#include "cudalign.hpp"


#include <sys/time.h>
#include <sys/resource.h>

unsigned char* allocCudaSeq(Sequence* seq, int offset) {
	printf("%X\n", seq);
	int len = seq->getLen()-offset;
	printf("allocCudaSeq(...): %d\n", len+GRID_HEIGHT);
	unsigned char* out = (unsigned char*)allocCuda0(len+GRID_HEIGHT);
	printf("allocCudaSeq(...): %d: OK (%X)\n", len+GRID_HEIGHT, out);
	//cutilSafeCall( hipMalloc((void**) &out, seq->getLen()+GRID_HEIGHT-offset));
	cutilSafeCall( hipMemcpy(out, seq->forward_data+offset, len, hipMemcpyHostToDevice));
    return out;
}

unsigned char* allocCudaSeq(Sequence* seq) {
    return allocCudaSeq(seq, 0);
}

/*unsigned char* allocCudaSeqRev(Sequence* seq, int offset) {
    unsigned char* out;
    printf("allocCudaSeqRev(...): %d\n", seq->getLen()+GRID_HEIGHT-offset);
    cutilSafeCall( hipMalloc((void**) &out, seq->getLen()+GRID_HEIGHT-offset));
    cutilSafeCall( hipMemcpy(out, seq->reverse_data+offset, seq->getLen()+GRID_HEIGHT-offset, hipMemcpyHostToDevice));
    return out;
}

unsigned char* allocCudaSeqRev(Sequence* seq) {
    return allocCudaSeqRev(seq, 0);
}*/

void* allocCuda(void* in, int size) {
    unsigned char* out;
    printf("allocCuda(...,%d)\n", size);
    cutilSafeCall( hipMalloc((void**) &out, size));
    cutilSafeCall( hipMemcpy(out, in, size, hipMemcpyHostToDevice));
    return out;
}

void* allocCuda0(int size) {
    void* out;
    printf("allocCuda0(...,%d)\n", size);
    cutilSafeCall( hipMalloc((void**) &out, size));
    cutilSafeCall( hipMemset(out, 0, size));
    return out;
}

void deallocCuda(void* var, int size, void* out) {
    cutilSafeCall(hipMemcpy( out, var, size, hipMemcpyDeviceToHost));
    cutilSafeCall(hipFree(var));
}

int tempIntCuda(int* var) {
    int out;
    cutilSafeCall(hipMemcpy(&out, var, sizeof(int), hipMemcpyDeviceToHost));
    return out;
}

void printDevProp(hipDeviceProp_t devProp, FILE* file) {
	fprintf(file, "Major revision number:         %d\n",  devProp.major);
	fprintf(file, "Minor revision number:         %d\n",  devProp.minor);
	fprintf(file, "Name:                          %s\n",  devProp.name);
	fprintf(file, "Total global memory:           %u\n",  devProp.totalGlobalMem);
	fprintf(file, "Total shared memory per block: %u\n",  devProp.sharedMemPerBlock);
	fprintf(file, "Total registers per block:     %d\n",  devProp.regsPerBlock);
	fprintf(file, "Warp size:                     %d\n",  devProp.warpSize);
	fprintf(file, "Maximum memory pitch:          %u\n",  devProp.memPitch);
	fprintf(file, "Maximum threads per block:     %d\n",  devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		fprintf(file, "Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		fprintf(file, "Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	fprintf(file, "Clock rate:                    %d\n",  devProp.clockRate);
	fprintf(file, "Total constant memory:         %u\n",  devProp.totalConstMem);
	fprintf(file, "Texture alignment:             %u\n",  devProp.textureAlignment);
	fprintf(file, "Concurrent copy and execution: %s\n",  (devProp.deviceOverlap ? "Yes" : "No"));
	fprintf(file, "Number of multiprocessors:     %d\n",  devProp.multiProcessorCount);
	fprintf(file, "Kernel execution timeout:      %s\n",  (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

void printGPUDevices(FILE* file) {
    int count;
    hipGetDeviceCount(&count);
    hipDeviceProp_t devProp;
    fprintf(file, "Available GPUs:\n");
    fprintf(file, "ID: NAME (RAM)\n");
    fprintf(file, "---------------------------\n");
    for (int deviceId=0; deviceId<count; deviceId++) {
        hipGetDeviceProperties(&devProp, deviceId);
        fprintf(file, "%2d: %s (%d MB) %s\n", deviceId, devProp.name,
                (int)ceil(devProp.totalGlobalMem/1024.0/1024.0),
                devProp.kernelExecTimeoutEnabled ? "(Timeout Enabled)":""
                );
    }
    fprintf(file, "\n");
}

int getGPUProportion(int* proportion, int n) {
    int count;
    hipGetDeviceCount(&count);
    if (count > n-1) {
        count = n-1;
    }
    hipDeviceProp_t devProp;
    int sum = 0;
    proportion[0] = 0;
    for (int deviceId=0; deviceId<count && deviceId < n; deviceId++) {
        hipGetDeviceProperties(&devProp, deviceId);
        int speed = devProp.clockRate*devProp.multiProcessorCount;
        sum += speed;
        proportion[deviceId+1] = sum;
    }
    return count;
}


hipDeviceProp_t selectGPU(int id, FILE* file) {
    int deviceId;
    if (id == -1) {
        deviceId = cutGetMaxGflopsDeviceId();
    } else {
        deviceId = id;
    }

    cutilSafeCall(hipSetDevice( deviceId ));
    cutilCheckMsg("hipSetDevice failed");
    
    hipDeviceProp_t devProp;
    cutilSafeCall(hipGetDeviceProperties(&devProp, deviceId));

    printDevProp(devProp, file);
    return devProp;
}


void printDevMem(FILE* file) {
	size_t freeMem;
	size_t totalMem;
	hipMemGetInfo(&freeMem, &totalMem);
	fprintf(file, " Free Mem: %u\n", freeMem/1024);
	fprintf(file, "Total Mem: %u\n", totalMem/1024);
	fprintf(file, " Used Mem: %u\n", (totalMem-freeMem)/1024);	
	
	
// 	char buf[30];
// 	snprintf(buf, 30, "/proc/%u/statm", (unsigned)getpid());
// 	FILE* pf = fopen(buf, "r");
// 	if (pf) {
// 		unsigned size; //       total program size
// 		unsigned resident;//   resident set size
// 		unsigned share;//      shared pages
// 		unsigned text;//       text (code)
// 		unsigned lib;//        library
// 		unsigned data;//       data/stack
// 		unsigned dt;//         dirty pages (unused in Linux 2.6)
// 		fscanf(pf, "%u" /* %u %u %u %u %u"*/, &size/*, &resident, &share, &text, &lib, &data*/);
// 		fprintf(file, "(?)Used RAM Mem: %u KB\n", size);
// 	}
	//fclose(pf);
	
	
	
	/*struct rusage usage; 
	int ret; 
	
	ret=getrusage(RUSAGE_SELF,&usage);
	fprintf(file, "(?)Used RAM Mem: %u (%d)\n", usage.ru_maxrss, ret);	*/
	
	
}
