#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>

#include <pthread.h>
#include <sys/stat.h>
#include <sys/types.h>

#include "../common/cudalign.hpp"

#include <cutil_inline.h>

#define H_MAX (2*64*1024)

#define NUM_THREADS 2

#define PRINT (0)


typedef struct {
    int partition0;
    int partition1;

    int h0[H_MAX];
    int e0[H_MAX];

    int h1[H_MAX];
    int e1[H_MAX];

    Job* job;
    /* OUT */
    midpoint_t *out_pos;
} split_args_t;

static midpoint_t split(Sequence *seq0, Sequence *seq1, int i0, int j0, int i1, int j1, 
						int type_s, int type_e, int score_s, int score_e,
						int *h0, int *h1, int *e0, int *e1);
static midpoint_t ort_split(Sequence *seq0, Sequence *seq1, int i0, int j0, int i1, int j1, 
						int type_s, int type_e, int score_s, int score_e,
						int *h0, int *h1, int *e0, int *e1);
												
static void *split_thread(void *thread_arg) {
    static int inv_type[] = {0,2,1};

    split_args_t* args = (split_args_t*)thread_arg;

    Job* job = args->job;

    int i0, j0, i1, j1, start_type, prev_type, start_score, prev_score;

    int partition_id = args->partition0;
    i1 = job->midpoints[partition_id].i;
    j1 = job->midpoints[partition_id].j;
    prev_type = job->midpoints[partition_id].type;
	prev_score = job->midpoints[partition_id].score;

    float last_percent = 0;
    for (int k=args->partition0+1; k<args->partition1; k++) {
        float percent = 100.0f*(k-args->partition0)/(args->partition1-args->partition0);
        if (percent > last_percent+1) {
            printf("Split: %5.1f (%6d/%6d)\n", percent , k, job->midpoints.size());
            last_percent = percent;
        }
        partition_id++;
        i0 = job->midpoints[k].i;
        j0 = job->midpoints[k].j;
        start_type = job->midpoints[k].type;
        start_score = job->midpoints[k].score;

        int delta_i = i1-i0;
        int delta_j = j1-j0;

        int inverse = (delta_i < delta_j);
        if (inverse) {
			if (j0 < j1-job->stage4_maximum_partition_size) {
				midpoint_t out_tmp;
                if ( job->stage4_orthogonal_execution ) {
                    out_tmp = ort_split ( &job->seq[1], &job->seq[0], j0, i0, j1, i1,
                                          inv_type[start_type], inv_type[prev_type], start_score, prev_score,
                                          args->h0, args->h1, args->e0, args->e1 );
                } else {
                    out_tmp = split ( &job->seq[1], &job->seq[0], j0, i0, j1, i1,
                                      inv_type[start_type], inv_type[prev_type], start_score, prev_score,
                                      args->h0, args->h1, args->e0, args->e1 );
                }
                /*midpoint_t out_tmp2 = ort_split(&job->seq[1], &job->seq[0], j0, i0, j1, i1, 
										   inv_type[start_type], inv_type[prev_type], start_score, prev_score,
										   args->h0, args->h1, args->e0, args->e1);
				if (out_tmp.i != out_tmp2.i || out_tmp.j != out_tmp2.j  || out_tmp.type != out_tmp2.type || out_tmp.score != out_tmp2.score) {
					fprintf(stderr, "WARN: %d %d %d %d   /    %d %d %d %d\n",
						out_tmp.i, out_tmp.j, out_tmp.type, out_tmp.score,
						out_tmp2.i, out_tmp2.j, out_tmp2.type, out_tmp2.score);
				}*/
				args->out_pos[k].i = out_tmp.j;
                args->out_pos[k].j = out_tmp.i;
                args->out_pos[k].type = inv_type[out_tmp.type];
                args->out_pos[k].score = out_tmp.score;
            } else {
                args->out_pos[k].type = -1;
            }
        } else {
			if (i0 < i1-job->stage4_maximum_partition_size) {
				midpoint_t out_tmp;
                if ( job->stage4_orthogonal_execution ) {
                    out_tmp = ort_split ( &job->seq[0], &job->seq[1], i0, j0, i1, j1,
                                          start_type, prev_type, start_score, prev_score,
                                          args->h0, args->h1, args->e0, args->e1 );
                } else {
                    out_tmp = split ( &job->seq[0], &job->seq[1], i0, j0, i1, j1,
                                          start_type, prev_type, start_score, prev_score,
                                          args->h0, args->h1, args->e0, args->e1 );
                }
                /*midpoint_t out_tmp2 = ort_split(&job->seq[0], &job->seq[1], i0, j0, i1, j1, 
										 start_type, prev_type, start_score, prev_score,
										 args->h0, args->h1, args->e0, args->e1);
				if (out_tmp.i != out_tmp2.i || out_tmp.j != out_tmp2.j  || out_tmp.type != out_tmp2.type || out_tmp.score != out_tmp2.score) {
					fprintf(stderr, "WARN: %d %d %d %d   /    %d %d %d %d\n",
						out_tmp.i, out_tmp.j, out_tmp.type, out_tmp.score,
						out_tmp2.i, out_tmp2.j, out_tmp2.type, out_tmp2.score);
				}*/
										 
				args->out_pos[k] = out_tmp;
            } else {
                args->out_pos[k].type = -1;
            }
        }

        prev_type = start_type;
		prev_score = start_score;
        i1 = i0;
        j1 = j0;
    }

    pthread_exit(NULL);
}

static void create_split_thread(split_args_t* args, pthread_t* pthread) {
    pthread_attr_t attr;
    pthread_attr_init(&attr);
    pthread_attr_setdetachstate(&attr, PTHREAD_CREATE_JOINABLE);
    int rc = pthread_create(pthread, &attr, split_thread, (void *)args);
    if (rc) {
        printf("ERROR; return code from pthread_create() is %d\n", rc);
        exit(-1);
    }
}


static midpoint_t ort_split(Sequence *seq0, Sequence *seq1, int i0, int j0, int i1, int j1, 
						int type_s, int type_e, int score_s, int score_e,
						int *h0, int *h1, int *e0, int *e1) {
	
	if (PRINT) printf("%d %d %d %d %d %d\n", i0, j0, i1, j1, type_s, type_e, score_s, score_e);
	
	int seq0_len = i1-i0;
	int seq1_len = j1-j0;
	
	if (seq1_len >= H_MAX) {
		fprintf(stderr, "Partition size is too large (%d > %d).\n", seq1_len, H_MAX);
		exit(1);
	}
	
	int mid = seq0_len/2;
	int mid0 = mid;
	int mid1 = seq0_len - mid;
	
	/* Forward */
	
	char* s0 = seq0->forward_data+(i0-1);
	char* s1 = seq1->forward_data+(j0-1);
	
	for (int j=1; j<=seq1_len; j++) {
		h0[j] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=2);
		e0[j] = -INF;
	}
	h0[0] = (type_s!=0)?-INF:0;
	e0[0] = (type_s!=1)?-INF:0;
	
	for (int i=1; i<=mid0; i++) {
		int h_tmp = h0[0];
		int h_next;
		h_next = h0[0] = e0[0] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=1);
		int f0 = -INF;
		const char s=s0[i];
		if (PRINT) printf("%2d/%2d ", h0[0], e0[0]);
		for (int j=1; j<=seq1_len; j++) {
			e0[j] = MAX(h0[j]-DNA_GAP_FIRST, e0[j]-DNA_GAP_EXT);
			f0 = MAX(h_next-DNA_GAP_FIRST, f0-DNA_GAP_EXT);
			h_next = MAX3(h_tmp+((s==s1[j])?DNA_MATCH:DNA_MISMATCH), e0[j], f0);
			h_tmp = h0[j];
			h0[j] = h_next;
			if (PRINT) printf("%2d/%2d ", h_next, e0[j]);
		}
		if (PRINT) printf("\n");
		//printf("FW: %d/%d\n", i, mid0);
	}
	
	
	if (PRINT) printf("---------\n");
	/* Reverse */
	
	
	
	//int diff = 	(score_e + (type_e == 1 ? DNA_GAP_OPEN : 0)) - (score_s + (type_s == 1 ? DNA_GAP_OPEN : 0));
	int diff = 	(score_e + (type_e == 0 ? 0 : DNA_GAP_OPEN)) - score_s;
	
	
	s0 = seq0->forward_data+(i1-1);
	s1 = seq1->forward_data+(j1-1);
	
	for (int i=1; i<=mid1; i++) {
		h1[i] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=1);
		e1[i] = -INF;
	}
	h1[0] = (type_e!=0)?-INF:0;
	e1[0] = (type_e!=2)?-INF:0;
	
	{
		int ff = -DNA_GAP_OPEN*(type_e!=1) - DNA_GAP_EXT*mid1;
		int hh = h1[mid1];
		if (PRINT) printf("%2d: %2d/%2d\n", (seq1_len-1) + j0, hh, ff);	
		
		int* _h0 = &h0[seq1_len];
		int* _e0 = &e0[seq1_len];
		
		int sum_match = _h0[0] + hh;
		int sum_gap = _e0[0] + ff + DNA_GAP_OPEN;
		
		if (sum_match == diff) {
			midpoint_t pt;
			pt.i = mid0+i0;
			pt.j = (seq1_len)+j0;
			pt.type = 0;
			pt.score = score_s + _h0[0];
			return pt;
		} else if (sum_gap == diff) {
			midpoint_t pt;
			pt.i = mid0+i0;
			pt.j = (seq1_len)+j0;
			pt.type = 1;
			pt.score = score_s + _e0[0];
			return pt;
		}
	}
	
	int* _h0 = &h0[seq1_len];
	int* _e0 = &e0[seq1_len];
	
	
	for (int j=1; j<=seq1_len; j++) {
		int h_tmp = h1[0];
		int h_next;
		h_next = h1[0] = e1[0] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=2);
		int f1 = -INF;
		const char s=s1[-(j-1)];
		if (PRINT) printf("%2d/%2d ", h1[0], e1[0]);
		for (int i=1; i<=mid1; i++) {
			e1[i] = MAX(h1[i]-DNA_GAP_FIRST, e1[i]-DNA_GAP_EXT);
			f1 = MAX(h_next-DNA_GAP_FIRST, f1-DNA_GAP_EXT);
			h_next = MAX3(h_tmp+((s==s0[-(i-1)])?DNA_MATCH:DNA_MISMATCH), e1[i], f1);
			h_tmp = h1[i];
			h1[i] = h_next;
			
			if (PRINT) printf("%2d/%2d ", h_next, e1[i]);
		}
		if (PRINT) printf("\n");
		
		int sum_match = _h0[-j] + h1[mid1];
		int sum_gap = _e0[-j] + f1 + DNA_GAP_OPEN;
		
		if (sum_match == diff) {
			midpoint_t pt;
			pt.i = mid0+i0;
			pt.j = (seq1_len-j)+j0;
			pt.type = 0;
			pt.score = score_s + _h0[-j];
			return pt;
		} else if (sum_gap == diff) {
			midpoint_t pt;
			pt.i = mid0+i0;
			pt.j = (seq1_len-j)+j0;
			pt.type = 1;
			pt.score = score_s + _e0[-j];
			return pt;
		}
			
			if (PRINT) printf("%2d: %2d/%2d   %2d/%2d    %2d\n", (seq1_len-j-1)+j0, h1[mid1], f1, sum_match, sum_gap, diff);	
	}
	
	if (PRINT) printf("NOT FOUND %d\n", diff);
	fprintf(stderr, "NOT FOUND %d\n", diff);
	/*{
		midpoint_t pt;
		pt.type = -1;
		return pt;
	}*/
	exit(1);
	
	
	
	/*
	s0 = seq0->forward_data+(i1-1);
	s1 = seq1->forward_data+(j1-1);
	
	for (int j=1; j<=seq1_len; j++) {
		h1[j] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=2);
		e1[j] = -INF;
	}
	h1[0] = (type_e!=0)?-INF:0;
	e1[0] = (type_e!=1)?-INF:0;
	
	for (int i=1; i<=mid1; i++) {
		int h_tmp = h1[0];
		int h_next;
		h_next = h1[0] = e1[0] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=1);
		int f1 = -INF;
		const char s=s0[-(i-1)];
		if (PRINT) printf("%2d/%2d ", h1[0], e1[0]);
		for (int j=1; j<=seq1_len; j++) {
			e1[j] = MAX(h1[j]-DNA_GAP_FIRST, e1[j]-DNA_GAP_EXT);
			f1 = MAX(h_next-DNA_GAP_FIRST, f1-DNA_GAP_EXT);
			h_next = MAX3(h_tmp+((s==s1[-(j-1)])?DNA_MATCH:DNA_MISMATCH), e1[j], f1);
			h_tmp = h1[j];
			h1[j] = h_next;
			
			if (PRINT) printf("%2d/%2d ", h_next, e1[j]);
		}
		if (PRINT) printf("\n");
		//printf("RV: %d/%d\n", i, mid0);
	}
	*/
	
	/* Compare */
	
	
	/*
	s0 = seq0->forward_data+(i0-1);
	s1 = seq1->forward_data+(j0-1);
	
	int ii, jj, tt, best, ss;
	best = -INF;
	//tt = -1;
	ii = mid0;
	char s = s0[ii];
	if (PRINT) printf("mid0: %d (%d)\n", mid0 , ii+i0);
	if (PRINT) printf("mid1: %d (%d)\n", mid1 , ii+i0);
	if (PRINT) printf("len: s0 %d   s1 %d\n", seq0_len, seq1_len);
	
	int* _h0 = h0;
	int* _e0 = e0;
	int* _h1 = &h1[seq1_len-(1-1)];
	int* _e1 = &e1[seq1_len-(1-1)];
	
	for (int j=seq1_len; j>=0; j--) { 
		int sum_match = _h0[j] + _h1[-j];
		int sum_gap = _e0[j] + _e1[-j] + DNA_GAP_OPEN;
		
		if (true || PRINT) printf("%4d: %c[%c]%c x %c[%c]%c   [%2d/%2d][%2d/%2d] %2d/%2d\n", 
			(j-1)+j0, s0[ii-1],s0[ii],s0[ii+1], s1[j-1],s1[j],s1[j+1],
						  _h0[j], _e0[j],
						  _h1[-j], _e1[-j], 
						  sum_match, sum_gap);
						  
						  if (sum_match > best) {
							  best = sum_match;
							  jj = j;
							  tt = 0;
							  ss = score_s + _h0[j];
						  } 
						  if (sum_gap > best) {
							  best = sum_gap;
							  jj = j;
							  tt = 1;
							  ss = score_s + _e0[j];// + DNA_GAP_OPEN;
							  						  } 
	}
	if (PRINT) printf("best: %d (score: %d  ss:%d)\n", best, score_s, ss);
	
	printf("DIFF: %d MAX: %d\n", diff, best);
	if (best != diff) fprintf(stderr, "DIFF: %d MAX: %d\n", diff, best);
	
	midpoint_t pt;
	pt.i = ii+i0;
	pt.j = jj+j0;
	pt.type = tt;
	pt.score = ss;
	if (PRINT) printf("pt: %d,%d,%d,%d\n", pt.i, pt.j, pt.type, pt.score);
	return pt;
	*/
}















/*
*/

static midpoint_t split(Sequence *seq0, Sequence *seq1, int i0, int j0, int i1, int j1, 
						int type_s, int type_e, int score_s, int score_e,
						int *h0, int *h1, int *e0, int *e1) {

	if (PRINT) printf("%d %d %d %d %d %d\n", i0, j0, i1, j1, type_s, type_e, score_s, score_e);

    int seq0_len = i1-i0;
    int seq1_len = j1-j0;

    if (seq1_len >= H_MAX) {
        fprintf(stderr, "Partition size is too large (%d > %d).\n", seq1_len, H_MAX);
        exit(1);
    }

    int mid = seq0_len/2;
    int mid0 = mid;
    int mid1 = seq0_len - mid;


    /* Forward */

    char* s0 = seq0->forward_data+(i0-1);
    char* s1 = seq1->forward_data+(j0-1);

    for (int j=1; j<=seq1_len; j++) {
        h0[j] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=2);
        e0[j] = -INF;
    }
    h0[0] = (type_s!=0)?-INF:0;
    e0[0] = (type_s!=1)?-INF:0;

    for (int i=1; i<=mid0; i++) {
        int h_tmp = h0[0];
        int h_next;
        h_next = h0[0] = e0[0] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=1);
        int f0 = -INF;
        const char s=s0[i];
        if (PRINT) printf("%2d/%2d ", h0[0], e0[0]);
        for (int j=1; j<=seq1_len; j++) {
            e0[j] = MAX(h0[j]-DNA_GAP_FIRST, e0[j]-DNA_GAP_EXT);
            f0 = MAX(h_next-DNA_GAP_FIRST, f0-DNA_GAP_EXT);
            h_next = MAX3(h_tmp+((s==s1[j])?DNA_MATCH:DNA_MISMATCH), e0[j], f0);
            h_tmp = h0[j];
            h0[j] = h_next;
            if (PRINT) printf("%2d/%2d ", h_next, e0[j]);
        }
        if (PRINT) printf("\n");
        //printf("FW: %d/%d\n", i, mid0);
    }


    if (PRINT) printf("---------\n");
    /* Reverse */


    s0 = seq0->forward_data+(i1-1);
    s1 = seq1->forward_data+(j1-1);

    for (int j=1; j<=seq1_len; j++) {
        h1[j] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=2);
        e1[j] = -INF;
    }
    h1[0] = (type_e!=0)?-INF:0;
    e1[0] = (type_e!=1)?-INF:0;

    for (int i=1; i<=mid1; i++) {
        int h_tmp = h1[0];
        int h_next;
        h_next = h1[0] = e1[0] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_e!=1);
        int f1 = -INF;
        const char s=s0[-(i-1)];
        if (PRINT) printf("%2d/%2d ", h1[0], e1[0]);
        for (int j=1; j<=seq1_len; j++) {
            e1[j] = MAX(h1[j]-DNA_GAP_FIRST, e1[j]-DNA_GAP_EXT);
            f1 = MAX(h_next-DNA_GAP_FIRST, f1-DNA_GAP_EXT);
            h_next = MAX3(h_tmp+((s==s1[-(j-1)])?DNA_MATCH:DNA_MISMATCH), e1[j], f1);
            h_tmp = h1[j];
            h1[j] = h_next;

            if (PRINT) printf("%2d/%2d ", h_next, e1[j]);
        }
        if (PRINT) printf("\n");
        //printf("RV: %d/%d\n", i, mid0);
    }


    /* Compare */



    s0 = seq0->forward_data+(i0-1);
    s1 = seq1->forward_data+(j0-1);

    int ii, jj, tt, best, ss;
    best = -INF;
    //tt = -1;
    ii = mid0;
    char s = s0[ii];
    if (PRINT) printf("mid0: %d (%d)\n", mid0 , ii+i0);
    if (PRINT) printf("mid1: %d (%d)\n", mid1 , ii+i0);
    if (PRINT) printf("len: s0 %d   s1 %d\n", seq0_len, seq1_len);

    int* _h0 = h0;
    int* _e0 = e0;
    int* _h1 = &h1[seq1_len-(1-1)];
    int* _e1 = &e1[seq1_len-(1-1)];

	for (int j=seq1_len; j>=0; j--) { 
        int sum_match = _h0[j] + _h1[-j];
        int sum_gap = _e0[j] + _e1[-j] + DNA_GAP_OPEN;

        if (PRINT) printf("%4d: %c[%c]%c x %c[%c]%c   [%2d/%2d][%2d/%2d] %2d/%2d\n", 
                (j-1)+j0, s0[ii-1],s0[ii],s0[ii+1], s1[j-1],s1[j],s1[j+1],
                _h0[j], _e0[j],
                _h1[-j], _e1[-j], 
                sum_match, sum_gap);

        if (sum_match > best) {
            best = sum_match;
            jj = j;
            tt = 0;
            ss = score_s + _h0[j];
        } 
        if (sum_gap > best) {
            best = sum_gap;
            jj = j;
            tt = 1;
            ss = score_s + _e0[j];// + DNA_GAP_OPEN;
            /*if (j==0) {
                printf("FULL GAP\n");
            }*/
        } 
    }
    if (PRINT) printf("best: %d (score: %d  ss:%d)\n", best, score_s, ss);
    midpoint_t pt;
    pt.i = ii+i0;
    pt.j = jj+j0;
    pt.type = tt;
    pt.score = ss;
    if (PRINT) printf("pt: %d,%d,%d,%d\n", pt.i, pt.j, pt.type, pt.score);
    return pt;
}

int merge_partitions(Job* job, midpoint_t *new_positions) {
    vector<midpoint_t> merged_partitions;
    merged_partitions.clear();
    merged_partitions.push_back(job->midpoints[0]);
    int has_new_pos = 0;
    for (int i=1; i<job->midpoints.size(); i++) {
        bool diff_pos = (new_positions[i].i!=job->midpoints[i-1].i || new_positions[i].j!=job->midpoints[i-1].j);
        if (new_positions[i].type != -1 && diff_pos) {
            has_new_pos = 1;
            merged_partitions.push_back(new_positions[i]);
        }
        merged_partitions.push_back(job->midpoints[i]);
    }
    if (has_new_pos) {
        job->midpoints = merged_partitions;
    }
    return has_new_pos;
}

int reduce_partitions(Job* job) {
    int i0, j0, i1, j1, start_type, prev_type, prev_score;

    int partition_id = 0;
    i1 = job->midpoints[partition_id].i;
    j1 = job->midpoints[partition_id].j;
    prev_type = job->midpoints[partition_id].type;
    prev_score = job->midpoints[partition_id].score;

    split_args_t args[NUM_THREADS];
    pthread_t thread[NUM_THREADS];

    midpoint_t *new_partitions = (midpoint_t *)malloc(job->midpoints.size()*sizeof(midpoint_t));
    for (int i=0; i<NUM_THREADS; i++) {
        args[i].job = job;
        args[i].out_pos = new_partitions;
        args[i].partition0 = job->midpoints.size()*i/NUM_THREADS;
        args[i].partition1 = job->midpoints.size()*(i+1)/NUM_THREADS;
        if (i>0) {
            args[i].partition0--;
        }
        //printf("%d: %d-%d  (%d)\n", i, args[i].partition0, args[i].partition1, job->partitions_count);
        create_split_thread(&args[i], &thread[i]);
    }
    for (int i=0; i<NUM_THREADS; i++) {
        int rc = pthread_join(thread[i], NULL);
        if (rc) {
            printf("ERROR; return code from pthread_join() is %d\n", rc);
            exit(-1);
        }
    }
    int has_new_partitions = merge_partitions(job, new_partitions);
    return has_new_partitions;
}


void stage4(Job* job) {
	FILE* stats = job->fopenStatistics(STAGE_4);

	fprintf(stats, "SW PARAM: %d/%d/%d/%d\n", DNA_MATCH, DNA_MISMATCH, DNA_GAP_FIRST, DNA_GAP_EXT);
	
	fprintf(stats, "--Alignment sequences:\n", job);
	fprintf(stats, ">%s (%d)\n", job->seq[0].name.c_str(), job->seq[0].getLen());
	fprintf(stats, ">%s (%d)\n", job->seq[1].name.c_str(), job->seq[1].getLen());	
	
	fprintf(stats, "MAXIMUM PARTITION SIZE: %d\n", job->stage4_maximum_partition_size);
	fprintf(stats, "ORTHOGONAL EXECUTION: %s\n", job->stage4_orthogonal_execution?"YES":"NO");
	
	Timer timer2;
	
	hipEvent_t ev_step = timer2.createEvent("STEP");
	hipEvent_t ev_start = timer2.createEvent("START");
	hipEvent_t ev_end = timer2.createEvent("END");
	hipEvent_t ev_crosspoints = timer2.createEvent("CROSSPOINTS");
	hipEvent_t ev_write = timer2.createEvent("WRITE");
	
	timer2.eventRecord(ev_start);
	
	job->loadMidpoints(-1);
	
	timer2.eventRecord(ev_crosspoints);
	
	int must_write_partitions = 0;
	int step = 1;
	int max_i, max_j;
	float step_sum = 0;
	while (job->getLargestMidpointSize(&max_i, &max_j) > job->stage4_maximum_partition_size) {
		int midpoints_count = job->midpoints.size();
		if (step == 1) {
			fprintf(stats, "-step %2d  max size: %5dx%5d crosspoints: %8d   time: %.4f   sum:%.4f\n", 
					0, max_i, max_j, midpoints_count, 0, 0);
			fflush(stats);
		}
		if (!reduce_partitions(job)) {
            // TODO tratar erro? não houve redução!
            break;
        }
        must_write_partitions = 1;
		float step_diff = timer2.eventRecord(ev_step);
		step_sum += step_diff;
		fprintf(stats, " step %2d  max size: %5dx%5d crosspoints: %8d   time: %.4f   sum:%.4f\n", 
				step, max_i, max_j, midpoints_count, step_diff, step_sum);
		fflush(stats);
		//job->writeMidpoints(1000+step);
		//timer2.eventRecord(ev_write);
		step++;
	}
	float step_diff = timer2.eventRecord(ev_step);
	step_sum += step_diff;
	fprintf(stats, "-step %2d  max size: %5dx%5d crosspoints: %8d   time: %.4f   sum:%.4f\n", 
			step, max_i, max_j, job->midpoints.size(), step_diff, step_sum);
	fflush(stats);
	timer2.eventRecord(ev_start);
	
    if (must_write_partitions) {
        job->writeMidpoints();
    }
	timer2.eventRecord(ev_write);
	
    if (PRINT) printf("\n");
	timer2.eventRecord(ev_end);
	
	fprintf(stats, "CUDA times:\n");
	float diff = timer2.printStatistics(stats);
	
	fprintf(stats, "        total: %.4f\n", diff);
	fclose(stats);
}

