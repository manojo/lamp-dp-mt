#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#include <sys/stat.h>
#include <sys/types.h>

#include "../stage2/sw_stage2_common.inc.cu"

int4 match_found22(int seq0_len, int yLen, int4 *d_match_out, int j0r, int i0r, int goal, int baseXr, int step, bus_t* h_busBase, bus_t* d_busBase, bus_t* d_outV, int blocks) {
    static int4 h_match_out[ALPHA];
    dim3 grid(ALPHA, 1, 1);
    dim3 threads(PHASE2_THREADS_COUNT, 1, 1);
    int i_pos = yLen - ALPHA*PHASE2_THREADS_COUNT*(step-blocks+1)-2; // TODO por que -2??
    //printf("seq0_len - i_pos - PHASE2_THREADS_COUNT*ALPHA = %d\n", (seq0_len - i_pos - PHASE2_THREADS_COUNT*ALPHA));
    int adjust = 0;
    if (i_pos < 0) {
        adjust = -i_pos; // Do not access negative offset memory at h_busBase
    }
    //printf("i_pos %d  (adjust: %d)\n", i_pos, adjust);
    // ALPHA*PHASE2_THREADS_COUNT+1 -> +1 because we need extra cell for (m-1)
    for (int i=0; i<(ALPHA*PHASE2_THREADS_COUNT+1); i++) {
        //printf("%d/%d ", h_busBase[i_pos+adjust+i].x, h_busBase[i_pos+adjust+i].y);
    }
    printf("\n");
    cutilSafeCall( hipMemcpy(d_busBase+adjust, &h_busBase[i_pos+adjust], (ALPHA*PHASE2_THREADS_COUNT+1-adjust)*sizeof(bus_t), hipMemcpyHostToDevice));
    kernel_match <<< grid, threads, 0 >>>(i0r+yLen, d_match_out, i0r + (step - blocks) * PHASE2_THREADS_COUNT * ALPHA, goal, baseXr, d_busBase, d_outV + (step - blocks) * PHASE2_THREADS_COUNT * ALPHA);
    cutilSafeCall(hipMemcpy(h_match_out, d_match_out, ALPHA * sizeof (int4), hipMemcpyDeviceToHost));
    for (int k = 0; k < ALPHA; k++) {
        if (h_match_out[k].x != 0) {
			printf("%d: %d %d %d %d   : goal: %d\n", k, h_match_out[k].x, h_match_out[k].y, h_match_out[k].w, h_match_out[k].z, goal);
			return h_match_out[k];
        }
    }
    return make_int4(0,0,0,0);
}

midpoint_t find_next_midpoint22 ( Job* job, Sequence* seq_vertical, Sequence* seq_horizontal, int reverse, midpoint_t midpoint, midpoint_t midpoint1, host_structures_t &host, cuda_structures_t &cuda, SpecialRowReader* specialRow/*, int &line_index*/ ) {
    midpoint_t next_midpoint;
    const int seq0_len = seq_vertical->getLen();
    const int seq1_len = seq_horizontal->getLen();
    const int bus_size = seq1_len*sizeof ( bus_t );
    const int baseX = specialRow->getCol() +1;
	const int start_type = midpoint.type;

    int i0r;
    int j0r;
    int baseXr;

    if ( reverse ) {
        i0r = seq0_len - midpoint.i;
        j0r = seq1_len - midpoint.j;
        baseXr = seq1_len - baseX - 1;
    } else {
        i0r = midpoint.i-1; // TODO VALIDAR
        j0r = midpoint.j-1; // TODO VALIDAR
        baseXr = baseX-1;
    }

	i0r++;
	j0r++;

    /*if (midpoint0.type == 0) {
    if (seq_horizontal->data[midpoint0.j-1] == seq_vertical->data[i0-1]) {
    	printf("DNA_MATCH (%c x %c)\n", seq_horizontal->data[midpoint0.j-1], seq_vertical->data[i0-1]);
    	goal = goal - DNA_MATCH;
    } else {
    printf("DNA_MISMATCH (%c x %c)\n", seq_horizontal->data[midpoint0.j-1], seq_vertical->data[i0-1]);
    goal = goal - DNA_MISMATCH;
    }
    }*/




    printf ( "baseXr = %d\n", baseXr );
    printf ( "j0r = %d\n", j0r );
    printf ( "i0r = %d\n", i0r );
    printf ( "midpoint1.j = %d\n", midpoint1.j );
    printf ( "midpoint1.i = %d\n", midpoint1.i );
    if ( specialRow->getRow() > midpoint1.i ) {
        printf ( "Partition Finished\n" );
		next_midpoint = midpoint1;
		/*next_midpoint.i = job->midpoints[partition_id].i;
		next_midpoint.j = job->midpoints[partition_id].j;
		next_midpoint.type = job->midpoints[partition_id].type;*/
		next_midpoint.score = 0;
        //line_index--;

        return next_midpoint;
    } else {
        printf ( "Partition Continue (%d <= %d)\n", specialRow->getRow(), midpoint1.i );
    }
    //printf("%X %d\n", file, 0);

    //fread(h_busBase, job->seq[1].len, sizeof(bus_t), file);
    //int p = midpoint0.j;


    cutilSafeCall ( hipMemset ( cuda.d_4out, 0, sizeof ( int4 ) ) );

    const int xLen = baseXr-j0r+1; // exclusive baseX
    printf ( "xLen = %d-%d=%d\n", baseXr, j0r, baseXr-j0r );
    {
        dim3 threads ( 512,1,1 );
        dim3 blocks ( xLen/threads.x+1,1,1 );
        kernel_initialize_busH<<<threads, blocks>>> ( cuda.d_busH, j0r, xLen, start_type );
    }

    const int yLen = specialRow->getRow()-midpoint.i;
    printf ( "yLen: %d\n", yLen );

    cutilSafeCall ( hipBindTexture ( 0, t_busH, cuda.d_busH, bus_size ) );

	int blocks = MULTIPROCESSORS*2;
	if ( xLen <= 2*blocks*PHASE2_THREADS_COUNT ) {
		blocks = xLen/2/PHASE2_THREADS_COUNT;
		if (blocks > MULTIPROCESSORS) {
			blocks = (blocks/MULTIPROCESSORS)*MULTIPROCESSORS;
		}
		if (blocks <= 1) {
			blocks = 1;
		}
	}
	
	printf ( "SIZES xLen: %d  B: %d  xLen/B: %d  2*height: %d   %s\n",
			 xLen, blocks, xLen/blocks, 2*PHASE2_THREADS_COUNT, blocks==1?"ERROR":"OK" );
			 
	int h = ( yLen/PHASE2_THREADS_COUNT/ALPHA+blocks+1 ); // TODO validar
    printf ( "READ\n" );
    specialRow->open ( host.h_busBase, yLen );
    int pend1 = specialRow->read ( host.h_busBase, yLen - ALPHA*PHASE2_THREADS_COUNT-3 ); // TODO + em vez de -

	
    int h_split[PHASE2_BLOCKS_COUNT + 1];
	createSplitPositions(j0r, xLen, h_split, blocks);
	
    cutilSafeCall ( hipMemcpyToSymbol(HIP_SYMBOL( d_split), h_split, sizeof ( h_split ) ) );
    //is_too_small = true;


    printf ( "(%d,%d)\n", xLen*DNA_GAP_EXT, host.h_busBase[yLen-1].y );
    printf ( "full_gap: %d\n", ( xLen-1 ) *-DNA_GAP_EXT + host.h_busBase[yLen-1].y + DNA_GAP_OPEN );
    // TODO validar
    int full_gap = 0;
    if ( start_type==1 && ( ( xLen-1 ) *-DNA_GAP_EXT + host.h_busBase[yLen-1].y + DNA_GAP_OPEN == midpoint.score ) ) {
        full_gap = 1;
		next_midpoint.i = midpoint.i;
		next_midpoint.j = baseXr;
		next_midpoint.score = host.h_busBase[yLen-1].y + DNA_GAP_OPEN;
		next_midpoint.type = start_type;
		printf ( "FULL GAP: %d %d %d %d\n", midpoint.i, midpoint.j, start_type, midpoint.score );
    }


    // TODO analisar casos limitrofes. ex.: (i0,j0)=(1,1) - perfect match
    int d;
    dim3  threads ( PHASE2_THREADS_COUNT/*THREADS_COUNT*/, 1, 1 );
    for ( d=0; d<h; d++ ) {
        if ( full_gap ) break;

        if ( blocks == 1 ) {
            dim3  grid ( 1, 1, 1 );
			printf ("KERNEL_NW_SMALL_QUICK(%d,%d,%d,%d,%d,%d,...,%d)\n",i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, d );
			kernel_nw_single_phase<PHASE2_THREADS_COUNT, false><<< grid, threads, 0>>> ( i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, cuda.d_4out, d, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_outV, start_type );
        } else {
			dim3  grid ( blocks, 1, 1 );
            //printf("KERNEL_NW_LARGE_QUICK(%d) xLen: %d  xLen/B: %d  height: %d\n", d, xLen, xLen/N_BLOCKS_COUNT, N_GRID_HEIGHT);
			kernel_nw_short_phase<false><<< grid, threads, 0>>> ( i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, cuda.d_4out, d, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_outV, start_type );
			kernel_nw_long_phase<false><<< grid, threads, 0>>> ( i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, cuda.d_4out, d, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_outV, start_type );
        }

        //printf ( "pos: %d-%d=%d\n", yLen, ALPHA*PHASE2_THREADS_COUNT* ( d+2 ), yLen-ALPHA*PHASE2_THREADS_COUNT* ( d+2 ) );
        pend1 = specialRow->read ( host.h_busBase, yLen-ALPHA*PHASE2_THREADS_COUNT* ( d+2 )-5 );

        if ( d >= blocks ) {

            hipStreamSynchronize ( 0 );
			if ( 1 && PRINT ) {
				printDebugMatch ( &cuda, baseXr, yLen, i0r, reverse, i0r, j0r, seq_vertical, seq_horizontal, d, blocks, seq0_len, &host );
			}

            int4 found = match_found22 ( seq0_len, yLen, cuda.d_match_out, j0r, i0r, midpoint.score, baseXr, d, host.h_busBase, cuda.d_busBase, cuda.d_outV, blocks );
            if ( found.x < 0 ) {
                fprintf ( stderr, "ERROR: Backtrace lost! Can't continue." );
                exit ( 1 );
            } else if ( found.x > 0 ) {
				next_midpoint.i = found.y;
				next_midpoint.j = found.x;
				next_midpoint.score = found.w;// - ((seq_horizontal_data[baseXr] == seq_vertical_data[midpoint0.i-1]) ? 0 : 4);
				next_midpoint.type = found.z;//inv_type[found.z];
                break;
            }
        }

        /*cutilSafeCall(hipMemcpy(h_busH, d_busH, (midpoint0.i-baseY+1)*sizeof(cell2_t), hipMemcpyDeviceToHost));
        for (int i=(midpoint0.i-baseY+1)-10; i<(midpoint0.i-baseY+1); i++) {
         		   printf("%d: %5d %5d\n", i, h_busH[i].x, h_busH[i].y);
        }
        printf("\n");*/

    }


	long long cells_updates = 0;
	for (int i=1; i<=d && i<=blocks; i++) {
		long long delta_h = h_split[i]-h_split[0];
		cells_updates += delta_h * ALPHA * PHASE2_THREADS_COUNT;
	}
	if (d >= blocks) {
		long long delta_h = h_split[blocks]-h_split[0];
		cells_updates += (d-blocks+1)*delta_h * ALPHA * PHASE2_THREADS_COUNT;
	}
	job->cells_updates += cells_updates;
	fprintf ( stdout, "D:%d BLOCKS:%d xLen: %d   cells: %lld    total(%.f mi)\n", d,blocks,xLen, cells_updates, job->cells_updates/1000000.0f);


    if ( d == h ) {
        fprintf ( stderr, "ERROR: Backtrace lost! End of matrix reached." );
        exit ( 1 );
    }

    return next_midpoint;
}


void stage3(Job* job) {
	FILE* stats = job->fopenStatistics(STAGE_3);
	
	job->loadMidpoints(1);
	job->loadSpecialRows(STAGE_2);
	job->cells_updates = 0;
	
	int reverse = 0;

	Sequence* seq_vertical;
	Sequence* seq_horizontal;
	if (reverse) {
		seq_vertical = new Sequence(job->seq[1], reverse);
		seq_horizontal = new Sequence(job->seq[0], reverse);
	} else {
		seq_vertical = new Sequence(job->seq[0], reverse);
		seq_horizontal = new Sequence(job->seq[1], reverse);
	}
	
	int seq0_len = seq_vertical->getLen();
	int seq1_len = seq_horizontal->getLen();
	
    MidpointsFile* midpointsFile = job->fopenMidpointsFile(2);

    /*if (reverse) {
        midpoint0.i = job->phase2_j0;
        midpoint0.j = job->phase2_i0;
    } else {
        midpoint0.i = job->phase2_i0;
        midpoint0.j = job->phase2_j0;
    }
    max = job->phase2_max;*/


	fprintf(stats, "SW PARAM: %d/%d/%d/%d\n", DNA_MATCH, DNA_MISMATCH, DNA_GAP_FIRST, DNA_GAP_EXT);
	
	fprintf(stats, "--Alignment sequences:\n", job);
	fprintf(stats, ">%s (%d)\n", job->seq[0].name.c_str(), seq0_len);
	fprintf(stats, ">%s (%d)\n", job->seq[1].name.c_str(), seq1_len);	
	
    selectGPU(job->gpu, stats);

    Timer timer2;

    hipEvent_t ev_step = timer2.createEvent("STEP");
    hipEvent_t ev_start = timer2.createEvent("START");
    hipEvent_t ev_end = timer2.createEvent("END");
    hipEvent_t ev_copy = timer2.createEvent("COPY");
    hipEvent_t ev_alloc = timer2.createEvent("ALLOC");
    hipEvent_t ev_kernel = timer2.createEvent("KERNEL");
    hipEvent_t ev_writeback = timer2.createEvent("WRITEBACK");

	host_structures_t host;
	cuda_structures_t cuda;
	allocHostStructures(seq_vertical, seq_horizontal, &host);
	allocCudaStructures(seq_vertical, seq_horizontal, &cuda);

	printDevMem(stats);
	
    timer2.eventRecord(ev_start);

    timer2.eventRecord(ev_copy);

    timer2.eventRecord(ev_alloc);



    // setup execution parameters
    dim3  threads_alpha( ALPHA*PHASE2_THREADS_COUNT, 1, 1);

    // execute the kernel
    //int* d_out = (int*)allocCuda0(sizeof(int));
    uint2* d_pos = (uint2*)allocCuda0(sizeof(uint2)*PHASE2_BLOCKS_COUNT);
	
    int line_index=0;//job->special_lines1.size()-1;

	midpoint_t midpoint0;
	midpoint_t midpoint1;
	
	int partition_id = job->midpoints.size()-1;
    //partition_id -= 1;
	midpoint1.i = job->midpoints[partition_id].i;
    midpoint1.j = job->midpoints[partition_id].j;
    midpoint1.type = job->midpoints[partition_id].type;
    midpoint1.score = job->midpoints[partition_id].score;

    while ( partition_id > 0 ) {
        partition_id--;
        midpoint0 = midpoint1;
        midpoint1 = job->midpoints[partition_id];
        midpoint_t midpoint = midpoint0;

		//midpoint.score = midpoint1.score - midpoint0.score;
		midpoint.score = (midpoint1.score + (midpoint1.type == 1 ? DNA_GAP_OPEN : 0)) 
				- (midpoint0.score + (midpoint0.type == 1 ? DNA_GAP_OPEN : 0));
        printf ( "MAX: (%d) %d-%d = %d\n", partition_id, midpoint0.score, midpoint1.score, midpoint.score );
		fprintf(stdout, "Millions Cells Updates: %.3f\n", job->cells_updates/1000000.0f);
		
        /*if ( midpoint0.type == 1 ) {
            if ( midpoint1.type == 0 ) {
                fprintf ( stderr, "PREV_TYPE 1\n" );
                midpoint.score -= DNA_GAP_OPEN;
            }
        }
        if ( midpoint1.type == 1 ) {
            if ( midpoint0.type == 0 ) {
                fprintf ( stderr, "CURR_TYPE 1\n" );
                midpoint.score += DNA_GAP_OPEN;
            }
        }
        if ( midpoint1.type == 1 && midpoint0.type == 1 ) {
            fprintf ( stderr, "CURR_TYPE PREV_TYPE 1\n" );
        }*/
		/*if ( midpoint0.type == 1 ) {
			fprintf ( stderr, "PREV_TYPE 1\n" );
			midpoint.score -= DNA_GAP_OPEN;
		}
		if ( midpoint1.type == 1 ) {
			fprintf ( stderr, "CURR_TYPE 1\n" );
			midpoint.score += DNA_GAP_OPEN;
		}*/
		
/////////


        midpointsFile->write ( midpoint0.i, midpoint0.j, midpoint0.score, midpoint0.type );

        while ( true ) { // todo era pra ser goal != -INF?
            fprintf ( stdout, ">> %d %d %d\n", midpoint.i, midpoint.j, midpoint.score );
            fprintf ( stdout, ">> %08X %08X %d\n", midpoint.i, midpoint.j, midpoint.score );
            SpecialRowReader* specialRow = job->fopenNextSpecialRow ( STAGE_2, midpoint.i, midpoint.j, PHASE2_THREADS_COUNT, &line_index );
            if ( specialRow == NULL ) {
                printf ( "No more special Rows.. (?)\n" );
                midpoint = midpoint1;
                midpoint.score = 0;
                break;
            }
            midpoint = find_next_midpoint22 ( job, seq_vertical, seq_horizontal, reverse, midpoint, midpoint1, host, cuda, specialRow );
            if ( midpoint.score==0 ) break;
            //cutilSafeCall(hipUnbindTexture(t_busH)); // TODO necessario?

            specialRow->close();
#ifdef SW_BUS_FLUSH
            //unflush_bus(baseX);
#endif
            int goal_adj = midpoint1.score - midpoint.score + ( midpoint1.type==0?0:DNA_GAP_OPEN );
            static int inv_type[] = {0,2,1};
            int type_adj = inv_type[midpoint.type];
            midpointsFile->write ( midpoint.i, midpoint.j, goal_adj, type_adj );
        }
    }

//////////

    timer2.eventRecord(ev_kernel);
    //hipEventRecord(ev_kernel,0);
    /*fprintf(stderr, "%d,%d,%d,%d\n", midpoint1.type, midpoint1.i, midpoint1.j, midpoint1.score);
    fprintf(partitions_file, "%d,%d,%d,%d\n", midpoint1.type, midpoint1.i, midpoint1.j, midpoint1.score);*/
    /*fprintf(stderr, "START\n");
    fprintf(partitions_file, "START\n");
    fclose(partitions_file);*/
    midpointsFile->write(midpoint1.i, midpoint1.j, midpoint1.score, midpoint1.type);
    midpointsFile->close();

	fprintf(stats, "CUDA times:\n");
	float diff = timer2.printStatistics(stats);
	
	fprintf(stats, "        total: %.4f\n", diff);
	fprintf(stats, "     Mi.Cells: %.4e\n", (float)job->cells_updates);
	fprintf(stats, "        MCUPS: %.4f\n", job->cells_updates/1000000.0f/(diff/1000.0f));
	
	printDevMem(stats);
	fprintf(stats, " FreeCudaStructures\n");
	freeCudaStructures(&cuda);
	freeHostStructures(&host);
	printDevMem(stats);
	
	fclose(stats);
	hipDeviceReset();
	cutilCheckMsg("hipDeviceReset failed");
}
