#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>

#include <sys/stat.h>
#include <sys/types.h>

#include "../common/cudalign.hpp"

/*#include "../common/sw_configs.hpp"
#include "../common/cuda_configs.hpp"
#include "../common/macros.hpp"

#include "../common/sequence.c"*/ // TODO resolver isso!!

//seq_t seq[2];

#define H_MAX (1*1024)
#define W_MAX (1*1024)

static int h[W_MAX][H_MAX];
static int e[W_MAX][H_MAX];
static int f[W_MAX][H_MAX];



#define PRINT (0)


static void dot(Alignment* alignment, Sequence *seq0, Sequence *seq1, int i, int j, int type) {
    //int pt;
    char* s0 = seq0->forward_data-1;
    char* s1 = seq1->forward_data-1;
    if (PRINT) printf("(%5d,%5d) ", i, j);
    if (type == 0) {
        if (PRINT) printf("%c%s%c [                 ]", s0[i], s0[i]==s1[j]?"-":" ", s1[j]);
    } else if (type == 1) {
        if (PRINT) printf("%c - [ add(1, %7d) ]", s0[i], j);
        alignment->addGapInSeq1(j);
    } else if (type == 2) {
        if (PRINT) printf("- %c [ add(0, %7d) ]", s1[j], i);
        alignment->addGapInSeq0(i);
    }
}


static int sw(Alignment* alignment, Sequence *seq0, Sequence *seq1, int i0, int j0, int i1, int j1, int type_s, int type_e) {
    i0++;
    j0++;

    // TODO conferir
    if (type_e == 0) {
        j1++;
        i1++;
    } else if (type_e == 1) {
        //j1++;
    }

    if (PRINT) printf("%d %d %d %d %d %d\n", i0, j0, i1, j1, type_s, type_e);

    int seq0_len = i1-i0+1;
    int seq1_len = j1-j0+1;


    char* s0 = seq0->forward_data+(i0-1);
    char* s1 = seq1->forward_data+(j0-1);

    for (int j=1; j<=seq1_len; j++) {
        h[0][j] = -j*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=2);
        e[0][j] = -INF;
    }
    h[0][0] = (type_s!=0?-INF:0);
    e[0][0] = (type_s!=1?-INF:0);

    for (int i=1; i<=seq0_len; i++) {
        h[i][0] = -i*DNA_GAP_EXT - DNA_GAP_OPEN*(type_s!=1);
        f[i][0] = -INF;
        const char s=s0[i-1];

        int* h0 = h[i];
        int* h1 = h[i-1];
        int* e0 = e[i];
        int* e1 = e[i-1];
        int* f0 = f[i];
        for (int j=1; j<=seq1_len; j++) {
            e0[j] = MAX(h1[j]-DNA_GAP_FIRST, e1[j]-DNA_GAP_EXT);
            f0[j] = MAX(h0[j-1]-DNA_GAP_FIRST, f0[j-1]-DNA_GAP_EXT);
            h0[j] = MAX3(h1[j-1]+((s==s1[j-1])?DNA_MATCH:DNA_MISMATCH), e0[j], f0[j]);
        }
        //printf("\nj:%4d %c SW: %4d\n", j1-j, s1[j-1], h[j][seq0_len]);
    }

    if (false && PRINT) {
        printf("%8s:\t.\t", "HEADER");
        for (int j=0; j<=seq1_len; j++) {
            printf("%c(%2d)\t", j==0?'*':s1[j-1], (j+j0-1)%100);
        }
        printf("\n");
        for (int i=0; i<=seq0_len; i++) {
            printf("%8d:\t%c\t", i+i0-1, i==0?'*':s0[i-1]);
            for (int j=0; j<=seq1_len; j++) {
                int v=h[i][j];
                if (v <= -INF) {
                    printf("%5s\t", "INF");
                } else {
                    printf("%5d\t", v);
                }
            }
            printf("\n");
        }
    }


    int sum = 0;
    int i=seq0_len;
    int j=seq1_len;
    int score;
    int c;
    
    score = h[seq0_len][seq1_len];
    if (type_e==0) {
        i--;
        j--;
        c=0;
    } else if (type_e==1) {
        score -= DNA_GAP_OPEN;
        int ss = e[seq0_len][seq1_len];
        if (true || ss >= score) { // TODO validar esse if
            score = ss;
            c=1;
        } else {
            c=0;
        }
    } else if (type_e==2) {
        score -= DNA_GAP_OPEN;
        int ss = f[seq0_len][seq1_len];
        if (true || ss > score) { // TODO validar esse if
            score = ss;
            c=2;
        } else {
            c=0;
        }
    }

    if (PRINT) printf ("Score: %5d %d%d\n", score, type_s, type_e);

    while (i>0 && j>0) {

        int dir;

        int _eh = h[i-1][j]-DNA_GAP_FIRST;
        int _fh = h[i][j-1]-DNA_GAP_FIRST;
        int _h11 = h[i-1][j-1]+((s0[i-1]==s1[j-1])?DNA_MATCH:DNA_MISMATCH);
        int _h10 = e[i][j];
        int _h01 = f[i][j];
        int _h00 = h[i][j];

        if (c==0) {
            if (_h00 == _h11) {
                dir = 0;
                c = 0;
            } else if (_h00 == _h10) {
                dir = 1;
                if (_h10==_eh) {
                    c = 0;
                } else {
                    c = 1;
                }
            } else if (_h00 == _h01) {
                dir = 2;
                if (_h01==_fh) {
                    c = 0;
                } else {
                    c = 2;
                }
            }             
        } else if (c==1) {
            dir = 1;
            if (_h10==_eh) {
                c = 0;
            } else {
                c = 1;
            }
        } else if (c==2) {
            dir = 2;
            if (_h01==_fh) {
                c = 0;
            } else {
                c = 2;
            }
        }

        if (i<=0) {
            dir = 2;
            c=2; // necessario somente para debug (pt)
            //printf("??? 2\n");
        }
        if (j<=0) {
            dir = 1;
            c=1; // necessario somente para debug (pt)
            //printf("??? 1\n");
        }

        int pt=0;


        dot(alignment, seq0, seq1, i0+(i-1), j0+(j-1), dir);
        if (dir == 0) {
            pt = ((s0[i-1]==s1[j-1])?DNA_MATCH:DNA_MISMATCH);
            i--;
            j--;
        } else if (dir == 1) {
            pt = c==0?-DNA_GAP_FIRST:-DNA_GAP_EXT;
            i--;
        } else if (dir == 2) {
            pt = c==0?-DNA_GAP_FIRST:-DNA_GAP_EXT;
            j--;
        }
        sum += pt;
        if (PRINT) printf("   %2d %5d {c:%d, dir:%d} (%d,%d)\n", pt, sum, c, dir, i, j);

    }
    while (i>0) {
        dot(alignment, seq0, seq1, i0+(i-1), j0+(j-1), 1);
		int pt = -DNA_GAP_EXT;
		i--;
        c=1;
        sum += pt;
        if (PRINT) printf("   %2d %5d {c:%d, dir:%d} (%d,%d)*\n", pt, sum, c, 2, i, j);
    }
    while (j>0) {
        dot(alignment, seq0, seq1, i0+(i-1), j0+(j-1), 2);
		int pt = -DNA_GAP_EXT;
		j--;
        c=2;
        sum += pt;
        if (PRINT) printf("   %2d %5d {c:%d, dir:%d} (%d,%d)*\n", pt, sum, c, 2, i, j);
    }
	if (type_s==0 && c!=0) {
		sum -= (DNA_GAP_OPEN);
	}
	return sum;
}


void stage5(Job* job) {
	FILE* stats = job->fopenStatistics(STAGE_5);
	
	fprintf(stats, "SW PARAM: %d/%d/%d/%d\n", DNA_MATCH, DNA_MISMATCH, DNA_GAP_FIRST, DNA_GAP_EXT);
	
	fprintf(stats, "--Alignment sequences:\n", job);
	fprintf(stats, ">%s (%d)\n", job->seq[0].name.c_str(), job->seq[0].getLen());
	fprintf(stats, ">%s (%d)\n", job->seq[1].name.c_str(), job->seq[1].getLen());	
	
	job->loadMidpoints(-1);
	
    //Alignment alignment(job);
	Alignment* alignment = job->alignment;
    
    int i0, j0, i1, j1, start_type, prev_type;
    midpoint_t prev = job->midpoints.front();

    int partition_id = 1;
    /*i1 = job->partitions[partition_id].i;
    j1 = job->partitions[partition_id].j;
    prev_type = job->partitions[partition_id].type;*/

    /*int ii = i1;
    int jj = j1;*/

    int max_size = job->getLargestMidpointSize();
    if (max_size > W_MAX || max_size > H_MAX) {
        fprintf(stderr, "ERROR: MAX SIZE: %d\n", max_size);
        exit(1);
    }
    fprintf(stats, "Largest Block: %d\n", max_size);

	Timer timer2;
	
	hipEvent_t ev_step = timer2.createEvent("STEP");
	hipEvent_t ev_start = timer2.createEvent("START");
	hipEvent_t ev_finalize = timer2.createEvent("FINALIZE");
	hipEvent_t ev_write_binary = timer2.createEvent("WRITE_BINARY");
	hipEvent_t ev_write_text = timer2.createEvent("WRITE_TEXT");
	hipEvent_t ev_end = timer2.createEvent("END");
	
	timer2.eventRecord(ev_start);
	
    int score = 0;
    for (; partition_id<job->midpoints.size(); partition_id++) {
        //if (partition_id<job->partitions.size()-1) partition_id++;
        //if (partition_id<job->partitions.size()-1) partition_id++;

    //while (1) {
        midpoint_t curr = job->midpoints[partition_id];
        /*i0 = job->partitions[partition_id].i;
        j0 = job->partitions[partition_id].j;
        start_type = job->partitions[partition_id].type;*/

        if (curr.i == 0 && curr.j == 0) break;

        //printf("ID: %d   %d x %d\n", partition_id, prev.i-curr->i, prev.j-curr->j);
        //dot(&alignment, &job->seq[0], &job->seq[1], prev.i, prev.j, prev.type);
        
        int sum = sw(alignment, &job->seq[0], &job->seq[1], curr.i, curr.j, prev.i, prev.j, curr.type, prev.type);
		
        score += sum;
        if (PRINT) printf("> SW   %5d/%d\n", sum, score);
        if (true || PRINT) {
			int goal_diff = (prev.score)-
					(curr.score);
            if (goal_diff!=sum) {
                fprintf(stderr, "[%s] GOAL DIFF: %8d   SUM: %8d   (%d,%d,%d)-(%d,%d,%d)\n", goal_diff==sum?"OK":"ERROR", goal_diff, sum,
                        curr.i, curr.j, curr.type, prev.i, prev.j, prev.type);
            }
        }

        if (PRINT) printf("\n");

		//timer2.eventRecord(ev_step);
		
        /*prev_type = start_type;
        i1 = i0;
        j1 = j0;*/
        prev = curr; 
    }
	timer2.eventRecord(ev_step);
	
    midpoint_t first = job->midpoints.front();
    midpoint_t last = job->midpoints.back();
    //dot(&alignment, &job->seq[0], &job->seq[1], last.i, last.j, last.type);

    if (PRINT) printf("\n");

    fprintf(stats, "(%d,%d)\n", last.type, first.type);
	fprintf(stats, "(%d,%d)->(%d,%d)\n", last.i, last.j, first.i, first.j);

	alignment->finalize(last.i, last.j, first.i, first.j);
	timer2.eventRecord(ev_finalize);
	
	alignment->printBinary(job->alignment_binary_filename);
	timer2.eventRecord(ev_write_binary);
	
	
	/*alignment->printText(job->alignment_filename);
	timer2.eventRecord(ev_write_text);*/
	
	//alignment.drawAlignment("alignment.svg");
    
    fprintf(stats, "Goal Diff: %d\n", first.score-last.score);
	
	timer2.eventRecord(ev_end);
	
	fprintf(stats, "CUDA times:\n");
	float diff = timer2.printStatistics(stats);
	
	fprintf(stats, "        total: %.4f\n", diff);
	fclose(stats);
	
}
