#include "hip/hip_runtime.h"
#include "../common/cudalign.hpp"

// includes, project
#include <cutil_inline.h>


#define LOG(format, args...)  \
    printf( "%10.4f: ", cutGetTimerValue(timer)); \
    printf (format , ## args); \
    printf ("\n");  \
    cutilCheckError(cutResetTimer( timer )); \


#ifdef SW_NUCLEOTIDE
    typedef unsigned char letter_t;
#else
    typedef char* letter_t;
#endif


#define SW_BUS_FLUSH
#define FLUSH_STEP (2048/PHASE2_THREADS_COUNT)

typedef int2 bus_t;


typedef int cell_t;
typedef int2 cell2_t;
typedef int3 cell3_t;
typedef int4 cell4_t;


#define PRINT (0)

// TODO avaliar THREADS_COUNT
static __shared__   int  s_colx [2][PHASE2_THREADS_COUNT]; // Could be PHASE2_THREADS_COUNT-1
static __shared__   int  s_coly [2][PHASE2_THREADS_COUNT];

static texture<unsigned char, 1, hipReadModeElementType> t_seq0;
static texture<unsigned char, 1, hipReadModeElementType> t_seq1;
static texture<      bus_t, 1, hipReadModeElementType> t_busH;

static __constant__ int d_split[PHASE2_BLOCKS_COUNT+1];








typedef struct {
	cell2_t* h_busOut;
	bus_t* h_busH;
	bus_t* h_busBase;
} host_structures_t;

typedef struct {
    unsigned char* d_seq0;
    unsigned char* d_seq1;
    bus_t* d_busH;
    cell4_t* d_busV1;
    cell4_t* d_busV2;
    cell3_t* d_busV3;
    bus_t *d_outV;
    int* d_out;
    bus_t *d_busBase;
    int4 *d_4out;
    int4 *d_match_out;
} cuda_structures_t;

static void allocHostStructures(Sequence* seq_vertical, Sequence* seq_horizontal, host_structures_t* host_structures) {
    const int seq0_len = seq_vertical->getLen();
    const int seq1_len = seq_horizontal->getLen();
    const int bus_size = seq1_len*sizeof(bus_t);
    const int busbase_size = seq0_len*sizeof(bus_t);

	host_structures->h_busH = (bus_t*)malloc(bus_size);
	host_structures->h_busOut = (cell2_t*)malloc(PHASE2_GRID_HEIGHT*ALPHA*sizeof(cell2_t));
	host_structures->h_busBase = (bus_t*)malloc(busbase_size);

	memset(host_structures->h_busOut, 0, PHASE2_GRID_HEIGHT*ALPHA*sizeof(cell2_t));
}

static void freeHostStructures(host_structures_t* host_structures) {
	free(host_structures->h_busH);
	free(host_structures->h_busOut);
	free(host_structures->h_busBase);
}

static void allocCudaStructures(Sequence* seq_vertical, Sequence* seq_horizontal,
        cuda_structures_t* cuda_structures) {
    const int seq0_len = seq_vertical->getLen();
    const int seq1_len = seq_horizontal->getLen();
    const int bus_size = seq1_len*sizeof(bus_t);
    const int busbase_size = seq0_len*sizeof(bus_t);

	cuda_structures->d_seq0 = allocCudaSeq(seq_vertical);
	cuda_structures->d_seq1 = allocCudaSeq(seq_horizontal);

    cutilSafeCall(hipBindTexture(0, t_seq0, cuda_structures->d_seq0, seq0_len+PHASE2_GRID_HEIGHT));
    cutilSafeCall(hipBindTexture(0, t_seq1, cuda_structures->d_seq1, seq1_len+PHASE2_GRID_HEIGHT));

    cuda_structures->d_busV1    = (cell4_t*)allocCuda0(PHASE2_GRID_HEIGHT*sizeof(cell4_t));
    cuda_structures->d_busV2    = (cell4_t*)allocCuda0(PHASE2_GRID_HEIGHT*sizeof(cell4_t));
    cuda_structures->d_busV3    = (cell3_t*)allocCuda0(PHASE2_GRID_HEIGHT*sizeof(cell3_t));
    cuda_structures->d_outV     =   (bus_t*)allocCuda0(busbase_size);
    cuda_structures->d_out      =     (int*)allocCuda0(sizeof(int));
    cuda_structures->d_busH     =   (bus_t*)allocCuda0(bus_size);
    cuda_structures->d_busBase  =   (bus_t*)allocCuda0((PHASE2_THREADS_COUNT*ALPHA+1)*sizeof(bus_t));
    cuda_structures->d_4out     =    (int4*)allocCuda0(PHASE2_BLOCKS_COUNT*sizeof(int4));
    cuda_structures->d_match_out =   (int4*)allocCuda0(ALPHA*sizeof(int4));
}

static void freeCudaStructures(cuda_structures_t* cuda) {
    cutilSafeCall(hipFree(cuda->d_seq0));
    cutilSafeCall(hipFree(cuda->d_seq1));
    cutilSafeCall(hipUnbindTexture(t_seq1));
    cutilSafeCall(hipUnbindTexture(t_seq0));
    cutilSafeCall(hipFree(cuda->d_busV1));
    cutilSafeCall(hipFree(cuda->d_busV2));
    cutilSafeCall(hipFree(cuda->d_busV3));
    cutilSafeCall(hipFree(cuda->d_outV));
    cutilSafeCall(hipFree(cuda->d_out));
    cutilSafeCall(hipFree(cuda->d_busH));
    cutilSafeCall(hipFree(cuda->d_busBase));
    cutilSafeCall(hipFree(cuda->d_4out));
    cutilSafeCall(hipFree(cuda->d_match_out));

}

static void createSplitPositions(int j0, int xLen, int* h_split, int blocks) {
    for (int i = 0; i < blocks; i++) {
        int xMod = xLen % blocks;
        h_split[i] = j0 + ((xLen / blocks) * i + (xMod > i ? i : xMod));
    }
    h_split[blocks] = (j0+xLen) - 1;
}

static void printDebugMatch ( cuda_structures_t* cuda, int baseXr, int baseLen, int base0, int reverse, int i0r, int j0r, Sequence* seq_vertical, Sequence* seq_horizontal, int d, int blocks, int seq0_len, host_structures_t* host ) {
    cutilSafeCall ( hipMemcpy ( host->h_busOut, cuda->d_outV + ( d - blocks ) * ALPHA*PHASE2_THREADS_COUNT, ALPHA * PHASE2_THREADS_COUNT * sizeof ( cell2_t ), hipMemcpyDeviceToHost ) );

	printf ( "BUSOUT (%d %d %d):\n", j0r, ( d - blocks ) * ALPHA*PHASE2_THREADS_COUNT, PHASE2_THREADS_COUNT*ALPHA );
    for ( int i = 0; i < PHASE2_THREADS_COUNT * ALPHA; i++ ) {
        int my_i = i0r + ( i + ( d - blocks ) * PHASE2_THREADS_COUNT * ALPHA );
		int my_ir = baseLen - (my_i-base0) - 1;
        if ( my_i >= seq0_len ) break;

        int sum_match = host->h_busBase[my_ir-1].x + host->h_busOut[i].x;
        int sum_gap = host->h_busBase[my_ir-1].y + host->h_busOut[i].y + DNA_GAP_OPEN;

        printf ( "i:%8d(%8d) %.5s[%.1s]%.5s %.5s[%.1s]%.5s SW: %4d/%4d  BUS_H: (%4d/%4d)  SUM: %4d/%4d\n",
                 my_i, reverse? (seq0_len-my_i) : (my_i+1),
				 &seq_horizontal->forward_data[baseXr - 5], &seq_horizontal->forward_data[baseXr], &seq_horizontal->forward_data[baseXr + 1],
				 &seq_vertical->forward_data[my_i - 5], &seq_vertical->forward_data[my_i], &seq_vertical->forward_data[my_i +1],
                 host->h_busOut[i].x, host->h_busOut[i].y,
                 host->h_busBase[my_ir-1].x, host->h_busBase[my_ir-1].y,
                 sum_match, sum_gap );
    }

}




/**
 * Returns the maximum of two numbers.
 * @param a first value
 * @param b second value
 */
static __device__ int my_max(int a, int b) {
    return (a>b)?a:b;
}

static __device__ void kernel_load2(const int idx, const int bank, const int i, bus_t* busH, int *h, int *f, unsigned char *s) {
    *s = tex1Dfetch(t_seq1, i);
    if (idx) {
        *h = s_colx[bank][idx];
        *f = s_coly[bank][idx];
    } else {
        int2 temp = tex1Dfetch(t_busH,i);
        *h = temp.x;
        *f = temp.y;
    }
}

static __device__ void kernel_flush2(const int idx, const int bank, const int i, bus_t* busH, const int h, const int f) {
    if (idx == PHASE2_THREADS_COUNT-1) {
        int2 temp = make_int2(h, f);
        busH[i] = temp;
    } else {
        s_colx[bank][idx+1] = h;
        s_coly[bank][idx+1] = f;
    }
}

/**
 * After incrementing the column j, this procedure must be called to check if j overflows the limit of the sequence seq_1.
 * If this happens, we must set j to zero and continue the computation in the proper line and all registers must be
 * reinitialized in order to represent the first column of the matrix (only in this situation the output params will
 * be updated, otherwise the values will be kept).
 *
 * @param seq0_len size of the sequence seq0.
 * @param seq1_len size of the sequence seq1.
 * @param i [in/out] the row of the cell to be processed.
 * @param j [in/out] the column of the cell to be processed.
 * @param ss [in/out] the variable containing seq0[i] information. If DNA/RNA, ss is the nucleotide.
 *           otherwise ss is the substitution vector for aminoaciad seq0[i].
 * @param e00 [in/out] value of E(i,j-1)
 * @param h01 [in/out] value of H(i,j-1)
 * @param h11 [in/out] value of H(i-1,j-1)
 * @param jump [in/out] true if the line i, after update, is out of bound of seq_0.
 * @param H how many lines must be jumped if j overflows.
 */
//__device__ void kernel_check_bound2(int xLen, int j0, int *i, int *j, letter_t* ss, cell_t *e00, cell_t *h01, cell_t *h11, int *jump, int2* busOut, int2* busH, const int type_f, const int H) {
static __device__ void kernel_check_bound2(const int seq0_len, const int xLen, const int j0, const int i0, int *j, int *i,
        uchar4* ss, cell4_t *ee, cell_t *h10,  cell4_t *h00, int *jump, int2* busOut, const int type_f, const int H) {

    if (*j >= j0+xLen) {
        if (*i >= i0) {
            busOut[*i-i0].x = h00->x;
            busOut[*i-i0].y = ee->x;
            busOut[*i-i0+1].x = h00->y;
            busOut[*i-i0+1].y = ee->y;
            busOut[*i-i0+2].x = h00->z;
            busOut[*i-i0+2].y = ee->z;
            busOut[*i-i0+3].x = h00->w;
            busOut[*i-i0+3].y = ee->w;
        }
        *j=j0;
        *i+=H;
        *jump = (*i>=seq0_len); // TODO validar
        ee->x = -INF;
        ee->y = -INF;
        ee->z = -INF;
        ee->w = -INF;
        if (i0-*i == 0) {
            *h10 = (type_f==0?0:-INF);
        } else {
            *h10 = -(*i-i0)*DNA_GAP_EXT - DNA_GAP_OPEN*(type_f!=2);
        }
        h00->x = -(*i-i0+1)*DNA_GAP_EXT - DNA_GAP_OPEN*(type_f!=2);
        h00->y = -(*i-i0+2)*DNA_GAP_EXT - DNA_GAP_OPEN*(type_f!=2);
        h00->z = -(*i-i0+3)*DNA_GAP_EXT - DNA_GAP_OPEN*(type_f!=2);
        h00->w = -(*i-i0+4)*DNA_GAP_EXT - DNA_GAP_OPEN*(type_f!=2);

        // TODO violação i<0 !!!
        ss->x = tex1Dfetch(t_seq0,*i);
        ss->y = tex1Dfetch(t_seq0,*i+1);
        ss->z = tex1Dfetch(t_seq0,*i+2);
        ss->w = tex1Dfetch(t_seq0,*i+3);
    }
}



/**
 * This procedure makes the smith waterman computation for the cell (i,j).
 *
 * @param idx the Thread Id.
 * @param i line of the cell (0..|seq0|-1)
 * @param j column of the cell (0..|seq1|-1)
 * @param busH the last special row saved
 * @param ss the variable containing seq0[i] information. If DNA/RNA, ss is the nucleotide.
 *           otherwise ss is the substitution vector for aminoaciad seq0[i].
 * @param max the maximum score found until now.
 * @param pos the best score found for each block.
 * @param e00 Input: value of E(i,j-1). Output: value of E(i,j)
 * @param h01 Input: value of H(i,j-1). Output: value of H(i,j)
 * @param h11 Input: value of H(i-1,j-1)
 * @param h10 Output: value of H(i-1,j)
 * @param load index of s_col for read (s_col[load] is used for reading; s_col[1-load] is used for writing).
 */
template <bool CHECK_GOAL>
static __device__ void kernel_nw(unsigned char s1, const unsigned char ss, const int i, const int j, cell_t *e00, cell_t *f00, const cell_t h01, const cell_t h11, const cell_t h10, cell_t *h00, const int goal, int4* out) {

    *e00 = my_max(h01-DNA_GAP_FIRST, *e00-DNA_GAP_EXT);
    *f00 = my_max(h10-DNA_GAP_FIRST, *f00-DNA_GAP_EXT);
    int v1 = h11+((ss==s1)?DNA_MATCH:DNA_MISMATCH);
    *h00 = my_max(v1, my_max(*e00, *f00));

	if (CHECK_GOAL) {
    //#ifdef PHASE2_CHECK_GOAL
    if (*h00 == goal) {
        if (out->x == 0) {
            out->w = 0;
            out->z = 0;
            out->x = j;
            out->y = i;
        }
    }
    //#endif
	}
}

template <int THREADS, bool CHECK_GOAL>
static __global__ void kernel_nw_single_phase(const int i0, const int j0, const int baseX, const int seq0_len, const int seq1_len, const int goal, int4 *out, const int step,
        bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3, int2* busOut, int type_f)
{
    int idx = threadIdx.x;

    s_colx[0][idx] = s_colx[1][idx] = busV3[idx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV3[idx].y;

    __syncthreads();

    const int xLen = baseX-j0; // exclusive baseY
    int by = step;
    int j_=xLen-idx;
	int i_=by*THREADS + idx - THREADS;
    i_ = i_*ALPHA;
    int j = j0+j_;
    int i = i0+i_;


    if (i < seq0_len) { // TODO estava i<seq0_len-1. Estava correto? já mudei.
        cell4_t h01=busV1[idx];
        cell4_t ee =busV2[idx];
        cell_t  h11=busV3[idx].z;

        // TODO otimizar retornando uchar4 = int
        uchar4 ss;

        ss.x = tex1Dfetch(t_seq0,i);
        ss.y = tex1Dfetch(t_seq0,i+1);
        ss.z = tex1Dfetch(t_seq0,i+2);
        ss.w = tex1Dfetch(t_seq0,i+3);

        __syncthreads();

        int jump = (i < i0);

        int _k = xLen;

        // TODO teste quando xlen é muito pequeno (i.e. xlen < NUM_THREADS)
        // REMOVER E VER SE GANHAMOS PERFORMANCE. CASO PERCA MUITO DESEMPENHO,
        // MELHOR TENTAR OUTRA ALTERNATIVA
        /*if (j_ <= 0) {
            _k+=1-j_;
        }*/

        if (_k&1) { // if odd
            cell4_t h00;
            cell_t h10;
            cell_t f00;

			kernel_check_bound2(seq0_len, xLen, j0, i0, &j, &i, &ss, &ee, &h11, &h01, &jump, busOut, type_f, THREADS*ALPHA);
            if (!jump) {
                unsigned char s1;

                kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, out);
                kernel_flush2(idx, 0, j, busH, h00.w, f00);

            }
            j++;
            __syncthreads();
            s_colx[1][idx] = s_colx[0][idx];
            s_coly[1][idx] = s_coly[0][idx];
            h11 = h10;
            h01 = h00;
            __syncthreads();
            _k--;
        }

        _k >>= 1;

        for (; _k; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;

			kernel_check_bound2(seq0_len, xLen, j0, i0, &j, &i, &ss, &ee, &h11, &h01, &jump, busOut, type_f, THREADS*ALPHA); // TODO precisa?
            if (!jump) {
                unsigned char s1;
                kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, out);
                kernel_flush2(idx, 0, j, busH, h00.w, f00);
            }

            j++;
            __syncthreads();

			kernel_check_bound2(seq0_len, xLen, j0, i0, &j, &i, &ss, &ee, &h10, &h00, &jump, busOut, type_f, THREADS*ALPHA);// TODO precisa?
            if (!jump) {
                unsigned char s1;
                kernel_load2(idx, 0, j, busH, &h11, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i, j, &ee.x, &f00, h00.x, h10  , h11  , &h01.x, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h00.y, h00.x, h01.x, &h01.y, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h00.z, h00.y, h01.y, &h01.z, goal, out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h00.w, h00.z, h01.z, &h01.w, goal, out);
                kernel_flush2(idx, 1, j, busH, h01.w, f00);
            }

            j++;
            __syncthreads();
        }

        busV1[idx]=h01;
        busV2[idx]=ee;
        busV3[idx].x=s_colx[1][idx];
        busV3[idx].y=s_coly[1][idx];
        busV3[idx].z=h11;
    }
}

template <bool CHECK_GOAL>
static __global__ void kernel_nw_short_phase(const int i0, const int j0, const int baseX, const int seq0_len, const int seq1_len, const int goal, int4 *blockResult, const int step,
        bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3, int2* busOut, const int type_f)
{
    int bx = blockIdx.x;
    int by = step-bx;
    if (by < 0) return;

    int idx = threadIdx.x;

    //int i0_ = d_split[0];
    const int totalLen = baseX-j0; // exclusive baseY

    const volatile int x0 = d_split[bx]-j0;
    const int xLen = d_split[bx+1] - d_split[bx];

	int i_=(by*blockDim.x)+idx;
    int j_=x0-idx;
	int tidx = (i_ % (gridDim.x*blockDim.x));
    i_ = i_*ALPHA;
    if (j_<=0) {
        j_ += totalLen;
		i_ -= (gridDim.x*blockDim.x)*ALPHA;
    }
    int j = j0+j_;
    int i = i0+i_;

    s_colx[0][idx] = s_colx[1][idx] = busV3[tidx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV3[tidx].y;
    __syncthreads();

    int4 out = make_int4(0,0,0,0);

    if (i < seq0_len) { // TODO estava i<seq0_len-1. Estava correto? já mudei.
        cell4_t h01=busV1[tidx];
        cell4_t ee =busV2[tidx];
        cell_t  h11=busV3[tidx].z;

        // TODO otimizar retornando uchar4 = int
        uchar4 ss;

        ss.x = tex1Dfetch(t_seq0,i);
        ss.y = tex1Dfetch(t_seq0,i+1);
        ss.z = tex1Dfetch(t_seq0,i+2);
        ss.w = tex1Dfetch(t_seq0,i+3);

        __syncthreads();

        int jump = (i < i0);
		int _k = blockDim.x >> 1;
        // We need N-1 Steps to complete the pending cells
        for (; _k>1; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;
            kernel_check_bound2(seq0_len, totalLen, j0, i0, &j, &i, &ss, &ee, &h11, &h01, &jump, busOut, type_f, blockDim.x*gridDim.x*ALPHA);
            if (!jump) {
                unsigned char s1;
                kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, &out);
                kernel_flush2(idx, 0, j, busH, h00.w, f00);
            }

            j++;
            __syncthreads();

			kernel_check_bound2(seq0_len, totalLen, j0, i0, &j, &i, &ss, &ee, &h10, &h00, &jump, busOut, type_f, blockDim.x*gridDim.x*ALPHA);
            if (!jump) {
                unsigned char s1;
                kernel_load2(idx, 0, j, busH, &h11, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h00.x, h10  , h11  , &h01.x, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h00.y, h00.x, h01.x, &h01.y, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h00.z, h00.y, h01.y, &h01.z, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h00.w, h00.z, h01.z, &h01.w, goal, &out);
                kernel_flush2(idx, 1, j, busH, h01.w, f00);
            }

            j++;
            __syncthreads();
        }
        {
            cell4_t h00;
            cell_t h10;
            cell_t f00;
			kernel_check_bound2(seq0_len, totalLen, j0, i0, &j, &i, &ss, &ee, &h11, &h01, &jump, busOut, type_f, blockDim.x*gridDim.x*ALPHA);
            if (!jump) {
                unsigned char s1;
                kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
				kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, &out);
				kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, &out);
                kernel_flush2(idx, 0, j, busH, h00.w, f00);
            }

            j++;
            __syncthreads();

			kernel_check_bound2(seq0_len, totalLen, j0, i0, &j, &i, &ss, &ee, &h10, &h00, &jump, busOut, type_f, blockDim.x*gridDim.x*ALPHA);
            if (!jump) { // TODO pode remover esse jump?
                busV1[tidx]=h00;
                busV2[tidx]=ee;
                busV3[tidx].x=s_colx[0][idx];
                busV3[tidx].y=s_coly[0][idx];
                busV3[tidx].z=h10;
            }

        }
    }

    int out_idx = -1;
    if (out.x > 0) {
        out_idx = idx;
    }
    __syncthreads();
    if (out_idx == idx) {
        blockResult[blockIdx.x] = out;
    }
}


template <bool CHECK_GOAL>
static __global__ void kernel_nw_long_phase(const int i0, const int j0, const int baseX, const int seq0_len, const int seq1_len, const int goal, int4 *blockResult, const int step,
        bus_t* busH, cell4_t* busV1, cell4_t* busV2, cell3_t* busV3, int2* busOut, int type_f)
{
    int bx = blockIdx.x;
    int by = step-bx;
    if (by < 0) return;

    int idx = threadIdx.x;

    //const int totalLen = baseX-j0; // exclusive baseY

	const volatile int x0 = (d_split[bx]-j0) + (blockDim.x-1);
	const int xLen = d_split[bx+1] - d_split[bx] - (blockDim.x-1);

	int i_=(by*blockDim.x)+idx;
    int j_=x0-idx;
	int tidx = (i_ % (gridDim.x*blockDim.x));
    i_ = i_*ALPHA;
    /*if (i<=0) {
        i += totalLen;
        j -= N_GRID_HEIGHT*ALPHA;
    }*/
    int j = j0+j_;
    int i = i0+i_;

    s_colx[0][idx] = s_colx[1][idx] = busV3[tidx].x;
    s_coly[0][idx] = s_coly[1][idx] = busV3[tidx].y;
    __syncthreads();

    int4 out = make_int4(0,0,0,0);

    if (i < seq0_len) { // TODO estava i<seq0_len-1. Estava correto? já mudei.
        cell4_t h01=busV1[tidx];
        cell4_t ee =busV2[tidx];
        cell_t  h11=busV3[tidx].z;

        // TODO otimizar retornando uchar4 = int
        uchar4 ss;

        ss.x = tex1Dfetch(t_seq0,i);
        ss.y = tex1Dfetch(t_seq0,i+1);
        ss.z = tex1Dfetch(t_seq0,i+2);
        ss.w = tex1Dfetch(t_seq0,i+3);

        __syncthreads();

        int _k = xLen;

        if (_k&1) { // if odd
            cell4_t h00;
            cell_t h10;
            cell_t f00;

            unsigned char s1;

            kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
			kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, &out);
            kernel_flush2(idx, 0, j, busH, h00.w, f00);

            j++;
            __syncthreads();
            s_colx[1][idx] = s_colx[0][idx];
            s_coly[1][idx] = s_coly[0][idx];
            h11 = h10;
            h01 = h00;
            __syncthreads();
            _k--;
        }

        _k >>= 1;

        for (; _k; _k--) {
            cell4_t h00;
            cell_t h10;
            cell_t f00;

            unsigned char s1;
            kernel_load2(idx, 1, j, busH, &h10, &f00, &s1);
			kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h01.x, h11  , h10  , &h00.x, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h01.y, h01.x, h00.x, &h00.y, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h01.z, h01.y, h00.y, &h00.z, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h01.w, h01.z, h00.z, &h00.w, goal, &out);
            kernel_flush2(idx, 0, j, busH, h00.w, f00);

            j++;
            __syncthreads();

            kernel_load2(idx, 0, j, busH, &h11, &f00, &s1);
			kernel_nw<CHECK_GOAL>(s1, ss.x, i,   j, &ee.x, &f00, h00.x, h10  , h11  , &h01.x, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.y, i+1, j, &ee.y, &f00, h00.y, h00.x, h01.x, &h01.y, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.z, i+2, j, &ee.z, &f00, h00.z, h00.y, h01.y, &h01.z, goal, &out);
			kernel_nw<CHECK_GOAL>(s1, ss.w, i+3, j, &ee.w, &f00, h00.w, h00.z, h01.z, &h01.w, goal, &out);
            kernel_flush2(idx, 1, j, busH, h01.w, f00);

            j++;
            __syncthreads();
        }

        busV1[tidx]=h01;
        busV2[tidx]=ee;
        busV3[tidx].x=s_colx[1][idx];
        busV3[tidx].y=s_coly[1][idx];
        busV3[tidx].z=h11;
    }
    int out_idx = -1;
    if (out.x > 0) {
        out_idx = idx;
    }
    __syncthreads();
    if (out_idx == idx) {
        blockResult[blockIdx.x] = out;
    }
}



static __global__ void kernel_initialize_busH(bus_t* busH, int i0, int len, int type) {
    int tidx = blockIdx.x*blockDim.x + threadIdx.x;
    while (tidx <= len) {
		busH[i0+tidx].x = -(tidx+1)*DNA_GAP_EXT - DNA_GAP_OPEN*(type!=1);
        busH[i0+tidx].y = -INF;

		tidx += blockDim.x*gridDim.x;
    }
}

static __global__ void kernel_match(int seq0_len, int4* out, const int i0, const int goal, const int baseXr, bus_t* busBase, bus_t* busOut) {
    int idx = blockIdx.x*PHASE2_THREADS_COUNT + threadIdx.x;
    int ii=i0+idx+1;
    //int m=seq1_len-j0_-j;
    int m=PHASE2_THREADS_COUNT*ALPHA-idx;

    if (ii>=seq0_len) return; // TODO check this

    int sum_match = busBase[m].x + busOut[idx].x;
    int sum_gap = busBase[m].y + busOut[idx].y + DNA_GAP_OPEN;


    __shared__ int out_idx;
    out_idx = 0x7FFFFFFF;
    int4 local_out = make_int4(0,0,0,0);
    out[blockIdx.x] = local_out;
    __syncthreads();

    // TODO somente funciona se usar atomicMin(&out_idx, idx);
    // isso aparentemente se deve por causa da ocorrencia do type=2.
    // precisa ser provado!!! so tenho uma intuicao que esta certo


    if (sum_match > goal || sum_gap > goal) {
            local_out.w = busOut[idx].x;
            local_out.z = m;
            local_out.x = -1;
            local_out.y = sum_match;
            //atomicMin(&out_idx, idx);
            out_idx = idx;
    }
    else
    if (sum_gap == goal) {
            local_out.w = busBase[m].y + DNA_GAP_OPEN;
            local_out.z = 1;
            local_out.x = baseXr;
            local_out.y = ii;
            //atomicMin(&out_idx, idx);
            out_idx = idx;
    }
    else
    if (sum_match == goal) {
            local_out.w = busBase[m].x;
            local_out.z = 0;
            local_out.x = baseXr;
            local_out.y = ii;
            //atomicMin(&out_idx, idx);
            out_idx = idx;
   }

    __syncthreads();
    if (out_idx == idx) {
        out[blockIdx.x] = local_out;
    }
}

