#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>

#include <sys/stat.h>
#include <sys/types.h>

#include "sw_stage2_common.inc.cu"


int4 goal_found(int4 *d_4out, int blocks) {
    static int4 h_4out[PHASE2_BLOCKS_COUNT];

	cutilSafeCall(hipMemcpy(h_4out, d_4out, blocks*sizeof(int4), hipMemcpyDeviceToHost));
	for (int k=0; k<blocks; k++) {
        //printf("%d: %d %d %d %d   : near goal\n", k, h_4out[k].x, h_4out[k].y, h_4out[k].w, h_4out[k].z);

        if (h_4out[k].x > 0) {
            printf("GOAL END!\n");
            return h_4out[k];
        }
    }
    return make_int4(0,0,0,0);
}

int4 match_found(int seq0_len, int4 *d_match_out, int j0r, int i0r, int goal, int baseXr, int step, bus_t* h_busBase, bus_t* d_busBase, bus_t* d_outV, int blocks) {
    static int4 h_match_out[ALPHA];
    dim3 grid(ALPHA, 1, 1);
    dim3 threads(PHASE2_THREADS_COUNT, 1, 1);
    int i_pos = (seq0_len-i0r) - ALPHA*PHASE2_THREADS_COUNT*(step-blocks+1)-2;  // TODO porque só funciona com -2 ??
    //printf("j_pos %d\n", j_pos);
    //printf("seq0_len{%d} - i0{%d} + (step - blocks){%d} * PHASE2_THREADS_COUNT * ALPHA = %d\n", seq0_len, i0, (step - blocks), seq0_len - i0 + (step - blocks) * PHASE2_THREADS_COUNT * ALPHA+1);
    int adjust = 0;
    if (i_pos < 0) {
        adjust = -i_pos; // Do not access negative offset memory at h_busBase
    }
    if (PRINT) printf("i_pos %d  (adjust: %d)\n", i_pos, adjust);

    // TODO FIXME acho que (ALPHA*PHASE2_THREADS_COUNT-adjust) deve ser (ALPHA*PHASE2_THREADS_COUNT) sem o - adjust
    cutilSafeCall( hipMemcpy(d_busBase, &h_busBase[i_pos+adjust], (ALPHA*PHASE2_THREADS_COUNT+1)*sizeof(bus_t), hipMemcpyHostToDevice));
    kernel_match <<< grid, threads, 0 >>>(seq0_len, d_match_out, i0r + (step - blocks) * PHASE2_THREADS_COUNT * ALPHA, goal, baseXr, d_busBase, d_outV + (step - blocks) * PHASE2_THREADS_COUNT * ALPHA);
    cutilSafeCall(hipMemcpy(h_match_out, d_match_out, ALPHA * sizeof (int4), hipMemcpyDeviceToHost));
    for (int k = 0; k < ALPHA; k++) {
		if (PRINT) printf("%d: %d %d %d %d   : goal: %d\n", k, h_match_out[k].x, h_match_out[k].y, h_match_out[k].w, h_match_out[k].z, goal);
        if (h_match_out[k].x != 0) {
            return h_match_out[k];
        }
    }
    return make_int4(0,0,0,0);
}

/*void init_score(const char* order_blosum, const char score_blosum[24][24], char out_score[128][128]) {
    memset(out_score, 0, sizeof(out_score));
    for (int i=0; order_blosum[i]; i++) {
        for (int j=0; order_blosum[j]; j++) {
            char c0 = order_blosum[i];
            char c1 = order_blosum[j];
            out_score[c0][c1] = score_blosum[i][j];
        }
    }
}*/

static map<int, SpecialRowWriter*> specialRows; // TODO colocar lá em cima

static void unflush_bus(int i0r) {
    // show content:
    for (map<int, SpecialRowWriter*>::iterator it = specialRows.begin(); it != specialRows.end(); it++) {
        int i = (*it).first;
        SpecialRowWriter* row = (*it).second;
        printf("Unflush: %08X,%08X  (%X)\n", i0r, i, row);
        /*if (row == NULL) {
            printf("NULL row\n");
            continue;
        }*/
        if (i >= i0r) {
            row->cancel();
        } else {
            row->close();
        }
        delete row;
    }
    specialRows.clear();
}
static void flush_bus(int reverse, int blocks, int seq0_len, int seq1_len, int i0r, int j0r, Job* job, int step, int baseXr, int *h_split, bus_t* h_busH, bus_t* d_busH) {
#ifdef SW_BUS_FLUSH
    if (job->flush_interval > 0) {
        for (int k=0; k<blocks && k<=step; k++) {
            int bx = k;
            const int x0 = h_split[bx];
            const int xLen = h_split[bx+1] - x0;
            int by = step-bx;
            int i=i0r + by*PHASE2_THREADS_COUNT*ALPHA;

			if (by % job->flush_interval == 0 && by>0 && i<seq0_len) {
                hipStreamSynchronize(0);
                //printf("offset:%X+%d len:%d\n", d_busH, x0, xLen*sizeof(cell2_t));
                cutilSafeCall(hipMemcpy(h_busH+x0, d_busH+x0, xLen*sizeof(cell2_t), hipMemcpyDeviceToHost));
                hipStreamSynchronize(0);

                int rowId;
                int colId;
                if (reverse) {
                    rowId = seq1_len-j0r;
                    colId = seq0_len-i;
                } else {
                    rowId = i;
                    colId = j0r;
                }

                if (bx==0) {
                    SpecialRowWriter* row;
                    row = job->fopenNewSpecialRow(STAGE_2, rowId, colId);
                    specialRows[i] = row;
                    row->open();
                    printf("Flush: %08X,%08X    (aux %08X,%08X)\n", rowId, colId, baseXr, i);
                }
                /*bus_t dummy;
                dummy.x = 0x99999999;
                dummy.y = 0x99999999;*/
                //specialRows[i]->write(&dummy, 1);
                specialRows[i]->write(&h_busH[x0], xLen);
                //specialRows[i]->write(&dummy, 1);
                if (bx==blocks-1) {
                    //printf("Close: %08X,%08X\n", baseY, j0-i);
                    SpecialRowWriter* row = specialRows[i];
                    specialRows.erase(i);
                    row->close();
                    delete row;
                }

            }
        }
    }
#endif
}


int4 findNearGoal(Job* job, cuda_structures_t* cuda, int reverse, int blocks, int i0r, int j0r, int baseXr, int baseX, int* h_split, bus_t* h_busH, int seq0_len, int seq1_len, int goal, int d, int start_type) {
    printf("NEAR GOAL!!!\n");
	dim3 grid(blocks, 1, 1);
    dim3 threads(PHASE2_THREADS_COUNT, 1, 1);
	kernel_nw_short_phase<true><<< grid, threads, 0>>>(i0r, j0r, baseXr, seq0_len, seq1_len, goal, cuda->d_4out, d, cuda->d_busH, cuda->d_busV1, cuda->d_busV2, cuda->d_busV3, cuda->d_outV, start_type);
    flush_bus(reverse, blocks, seq0_len, seq1_len, i0r, j0r, job, d, baseX, h_split, h_busH, cuda->d_busH);
	kernel_nw_long_phase<true><<< grid, threads, 0>>>(i0r, j0r, baseXr, seq0_len, seq1_len, goal, cuda->d_4out, d, cuda->d_busH, cuda->d_busV1, cuda->d_busV2, cuda->d_busV3, cuda->d_outV, start_type);

    return goal_found(cuda->d_4out, blocks);
}

void findFarGoal(Job* job, cuda_structures_t* cuda, int reverse, int blocks, int i0r, int j0r, int baseXr, int baseX, int* h_split, bus_t* h_busH, int seq0_len, int seq1_len, int goal, int d, int start_type) {
	dim3 grid(blocks, 1, 1);
    dim3 threads(PHASE2_THREADS_COUNT, 1, 1);
	//printf("KERNEL_NW_LARGE_QUICK(%d) xLen: %d  xLen/B: %d  height: %d\n", d, xLen, xLen/blocks, PHASE2_GRID_HEIGHT);
	kernel_nw_short_phase<false><<< grid, threads, 0>>>(i0r, j0r, baseXr, seq0_len, seq1_len, goal, cuda->d_4out, d, cuda->d_busH, cuda->d_busV1, cuda->d_busV2, cuda->d_busV3, cuda->d_outV, start_type);
	flush_bus(reverse, blocks, seq0_len, seq1_len, i0r, j0r, job, d, baseXr, h_split, h_busH, cuda->d_busH);
	kernel_nw_long_phase<false><<< grid, threads, 0>>>(i0r, j0r, baseXr, seq0_len, seq1_len, goal, cuda->d_4out, d, cuda->d_busH, cuda->d_busV1, cuda->d_busV2, cuda->d_busV3, cuda->d_outV, start_type);
}

midpoint_t find_next_midpoint(Job* job, Sequence* seq_vertical, Sequence* seq_horizontal, int reverse, midpoint_t midpoint, host_structures_t &host, cuda_structures_t &cuda, SpecialRowReader* specialRow) {
	midpoint_t next_midpoint;
	const int seq0_len = seq_vertical->getLen();
	const int seq1_len = seq_horizontal->getLen();
	const int bus_size = seq1_len*sizeof(bus_t);
	const int baseX = specialRow==NULL?0:specialRow->getRow();

	/*if (specialRow != NULL) {
		baseX = specialRow->getRow();
	} else {
		baseX = 0;
	}*/
	
	int i0r;
	int j0r;
	int baseXr;
	
	if (reverse) {
		i0r = seq0_len - midpoint.i;
		j0r = seq1_len - midpoint.j;
		baseXr = seq1_len - baseX;
		if (baseXr > seq1_len) baseXr = seq1_len;
	} else {
		i0r = midpoint.i-1; // TODO VALIDAR
		j0r = midpoint.j-1; // TODO VALIDAR
		baseXr = baseX;
	}
	
	cutilSafeCall( hipMemset(cuda.d_4out, 0, sizeof(int4)));
	
	const int xLen = baseXr-j0r+1; // inclusive baseX
	{
		dim3 threads(512,1,1);
		dim3 blocks(PHASE2_BLOCKS_COUNT,1,1);
		printf("kernel_initialize_busH<<<%d, %d>>>(..., j0r:%d, xLen:%d   midpoint.type:%d)\n", threads.x, blocks.x, j0r, xLen, midpoint.type);
		kernel_initialize_busH<<<threads, blocks>>>(cuda.d_busH, j0r, xLen, midpoint.type);
		cutilCheckMsg("Kernel execution failed");
		
		/*cutilSafeCall ( hipMemcpy ( host.h_busH, cuda.d_busH+j0r, xLen * sizeof ( cell2_t ), hipMemcpyDeviceToHost ) );
		for (int i=0; i<xLen; i++) {
			printf("%02d ", host.h_busH[i]);
			if (i%10 == 0) printf("\n");
		}
		printf("\n");*/
			
		
	}
	
	cutilSafeCall(hipBindTexture(0, t_busH, cuda.d_busH, bus_size));
	
	int blocks = MULTIPROCESSORS*2;
	if ( xLen <= 2*blocks*PHASE2_THREADS_COUNT ) {
		blocks = xLen/2/PHASE2_THREADS_COUNT;
		if (blocks > MULTIPROCESSORS) {
			blocks = (blocks/MULTIPROCESSORS)*MULTIPROCESSORS;
		}
		if (blocks <= 1) {
			blocks = 1;
		}
	}
	
	printf ( "SIZES xLen: %d  B: %d  xLen/B: %d  2*height: %d   %s\n",
			 xLen, blocks, xLen/blocks, 2*PHASE2_THREADS_COUNT, blocks==1?"ERROR":"OK" );
			 
	
	int h = ( midpoint.i/PHASE2_THREADS_COUNT/ALPHA+blocks+1 ); // TODO validar

	int pend1;
	if (specialRow != NULL) {	
		specialRow->open ( host.h_busBase, midpoint.i );
		pend1 = specialRow->read ( host.h_busBase, midpoint.i - ALPHA*PHASE2_THREADS_COUNT );
	} else {
		// TODO memset?
	}


    int h_split[PHASE2_BLOCKS_COUNT + 1];
    createSplitPositions ( j0r, xLen, h_split, blocks );
    cutilSafeCall ( hipMemcpyToSymbol(HIP_SYMBOL( d_split), h_split, sizeof ( h_split ) ) );




    // TODO analisar casos limitrofes. ex.: (i0,j0)=(1,1) - perfect match
    int d;
	dim3  threads( PHASE2_THREADS_COUNT, 1, 1);
	printf ( "i0r: %d   , j0r: %d,  baseXr: %d\n", i0r, j0r, baseXr );
	printf ( "i0r: %d   , j0r: %d,  baseXr: %d\n", seq0_len-i0r-1, seq1_len-j0r-1, seq1_len-baseXr-1 );
	for ( d=0; d<h; d++ ) {
        if ( blocks == 1 ) {
            dim3  grid ( 1, 1, 1 );
            if ( midpoint.score <= xLen*DNA_MATCH ) {
				kernel_nw_single_phase<PHASE2_THREADS_COUNT, true><<< grid, threads, 0>>> ( i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, cuda.d_4out, d, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_outV, midpoint.type );
                int4 found = goal_found ( cuda.d_4out , blocks);
                if ( found.x > 0 ) {
					next_midpoint.i = seq0_len-found.y-1;
					next_midpoint.j = seq1_len-found.x-1;
					next_midpoint.score = found.w;
					next_midpoint.type = found.z;
					printf ( "GOAL END (%d,%d)!\n", next_midpoint.i, next_midpoint.j );
                    break;
                }
            } else {
				kernel_nw_single_phase<PHASE2_THREADS_COUNT, false><<< grid, threads, 0>>> ( i0r, j0r, baseXr, seq0_len, seq1_len, midpoint.score, cuda.d_4out, d, cuda.d_busH, cuda.d_busV1, cuda.d_busV2, cuda.d_busV3, cuda.d_outV, midpoint.type );
            }
        } else {
            //printf ( "GOAL: %d    MAX: %d\n", midpoint.score, ( xLen+1 ) *DNA_MATCH );
            if ( midpoint.score <= ( xLen+1 ) *DNA_MATCH ) {
				int4 found = findNearGoal ( job, &cuda, reverse, blocks, i0r, j0r, baseXr, baseX, h_split, host.h_busH, seq0_len, seq1_len, midpoint.score, d, midpoint.type );
                if ( found.x > 0 ) {
					next_midpoint.i = seq0_len-found.y-1;
					next_midpoint.j = seq1_len-found.x-1;
					next_midpoint.score = found.w;
					next_midpoint.type = found.z;
					printf ( "GOAL END (%d,%d)!\n", next_midpoint.i, next_midpoint.j );
                    break;
                }

            } else {
				findFarGoal ( job, &cuda, reverse, blocks, i0r, j0r, baseXr, baseX, h_split, host.h_busH, seq0_len, seq1_len, midpoint.score, d, midpoint.type );
            }
        }

		if (specialRow != NULL) {	
			pend1 = specialRow->read ( host.h_busBase, midpoint.i - ALPHA*PHASE2_THREADS_COUNT* ( d+2 ) -1 ); // TODO precisa do -1?
		} else {
			// TODO memset?
		}

        //int blocks = PHASE2_BLOCKS_COUNT;
        if ( d >= blocks ) {

            hipStreamSynchronize ( 0 );
            if ( PRINT ) {
                printDebugMatch ( &cuda, baseXr, seq0_len, 0, reverse, i0r, j0r, seq_vertical, seq_horizontal, d, blocks, seq0_len, &host );
            }
            // TODO não precisa testar se baseX==0, pois ele já deveria encontrar no metodo goal_found
            int4 found = match_found ( seq0_len, cuda.d_match_out, j0r, i0r, midpoint.score, baseXr, d, host.h_busBase, cuda.d_busBase, cuda.d_outV, blocks );
            if ( found.x < 0 ) {
                fprintf ( stderr, "ERROR: Backtrace lost! Can't continue." );
                exit ( 1 );
            } else if ( found.x > 0 ) {
				next_midpoint.i = seq0_len-found.y;
				next_midpoint.j = seq1_len-found.x;
				next_midpoint.score = found.w;
				next_midpoint.type = found.z;
                break;
            }
        }
    }

	long long cells_updates = 0;
	for (int i=1; i<=d && i<=blocks; i++) {
		long long delta_h = h_split[i]-h_split[0];
		cells_updates += delta_h * ALPHA * PHASE2_THREADS_COUNT;
	}
	if (d >= blocks) {
		long long delta_h = h_split[blocks]-h_split[0];
		cells_updates += (d-blocks+1)*delta_h * ALPHA * PHASE2_THREADS_COUNT;
	}
	job->cells_updates += cells_updates;
	fprintf ( stdout, "D:%d BLOCKS:%d xLen: %d   cells: %lld    total(%.f mi)\n", d,blocks,xLen, cells_updates, job->cells_updates/1000000.0f);
	
    if ( d == h ) {
        fprintf ( stderr, "ERROR: Backtrace lost! End of matrix reached." );
        exit ( 1 );
    }
	
	
	//cutilSafeCall(hipUnbindTexture(t_busH)); // TODO necessario?
	
	return next_midpoint;
}


void stage2(Job* job) {
	FILE* stats = job->fopenStatistics(STAGE_2);
	stats = stderr;
	
	job->loadMidpoints(0);
	job->loadSpecialRows(STAGE_1);
	job->cells_updates = 0;
	
	int reverse = 1;

	
    Sequence* seq_vertical;
    Sequence* seq_horizontal;
    if (reverse) {
        seq_vertical = new Sequence(job->seq[1], reverse);
		seq_horizontal = new Sequence(job->seq[0], reverse);
    } else {
		seq_vertical = new Sequence(job->seq[0], reverse);
		seq_horizontal = new Sequence(job->seq[1], reverse);
    }
	/*printf("A: %X\n", seq_vertical);
	printf("B: %d\n", seq_vertical->getLen());
	printf("C: %c %d\n", seq_vertical->forward_data[0], strlen(seq_vertical->forward_data));
	printf("D:  %.5s\n", seq_vertical->forward_data+seq_vertical->getLen()-5);
	printf("seq_vertical: (%d) %.5s ... %.5s\n", seq_vertical->getLen(), seq_vertical->forward_data, seq_vertical->forward_data+seq_vertical->getLen()-6);
	printf("seq_horizontal: (%d) %.5s ... %.5s\n", seq_horizontal->getLen(), seq_horizontal->forward_data, seq_horizontal->forward_data+seq_horizontal->getLen()-5);*/
	

    int seq0_len = seq_vertical->getLen();
    int seq1_len = seq_horizontal->getLen();

    MidpointsFile* midpointsFile = job->fopenMidpointsFile(1);

	midpoint_t midpoint = job->midpoints.back();

    if (reverse) {
		int aux = midpoint.i;
		midpoint.i = midpoint.j;
		midpoint.j = aux;
    } else {
		//midpoint.i = job->phase2_i0;
		//midpoint.j = job->phase2_j0;
    }
    //midpoint.score = job->phase2_max;
	//midpoint.type = 0;

	if (job->flush_limit > 0) {
		int max_i;
		int max_j;
		job->getLargestSpecialRowInterval(STAGE_1, &max_i, &max_j);

		// Necessary if we do not have any special row.
		if (max_i == 0) {
			max_i = midpoint.i;
		}
		
		int max_len = midpoint.j;
		
		int maximum_recomended_flush_interval = max_i/4/(THREADS_COUNT*ALPHA);
		fprintf(stats, "Maximum special row distance: %lld\n", max_i);
		fprintf(stats, "Maximum recomended flush interval: %lld\n", maximum_recomended_flush_interval);
		
		//job->flush_interval = 6;
		job->flush_interval = (max_len*8LL/(job->flush_limit/max_i))/(THREADS_COUNT*ALPHA)+1; // TODO constante no lugar de 8
		if (job->flush_interval > maximum_recomended_flush_interval) {
			fprintf(stats, "Reducing Flush Interval from %lld to %lld\n", job->flush_interval, maximum_recomended_flush_interval);
			
			// TODO comentei para nao influenciar os testes
			//job->flush_interval = maximum_recomended_flush_interval;
			
			// TODO tratar com um warning, com exit? como ajuste? como erro? verificar
			// TODO fazer o mesmo na fase 1?
		}
		
		long long special_lines_count = (max_len/(THREADS_COUNT*ALPHA*job->flush_interval));
		fprintf(stats, "Special columns: %lld\n", special_lines_count);
		fprintf(stats, "Total size: %lld\n", special_lines_count*max_i*8LL);  // TODO 8*/
	} else {
		job->flush_interval = 0;
		job->flush_limit = 0;
	}
	fprintf(stats, "Flush Interval: %d\n", job->flush_interval);
	fprintf(stats, "Flush limit: %lld\n", job->flush_limit);
	
	fprintf(stats, "SW PARAM: %d/%d/%d/%d\n", DNA_MATCH, DNA_MISMATCH, DNA_GAP_FIRST, DNA_GAP_EXT);
	
	fprintf(stats, "--Alignment sequences:\n", job);
	fprintf(stats, ">%s (%d)\n", job->seq[0].name.c_str(), seq0_len);
	fprintf(stats, ">%s (%d)\n", job->seq[1].name.c_str(), seq1_len);	
	fflush(stats);

    selectGPU(job->gpu, stats);

    Timer timer2;

    hipEvent_t ev_step = timer2.createEvent("STEP");
    hipEvent_t ev_start = timer2.createEvent("START");
    hipEvent_t ev_end = timer2.createEvent("END");
    hipEvent_t ev_copy = timer2.createEvent("COPY");
    hipEvent_t ev_alloc = timer2.createEvent("ALLOC");
    hipEvent_t ev_kernel = timer2.createEvent("KERNEL");
    hipEvent_t ev_writeback = timer2.createEvent("WRITEBACK");

	printDevMem(stats);
	
    timer2.eventRecord(ev_start);

	host_structures_t host;
	cuda_structures_t cuda;
	allocHostStructures(seq_vertical, seq_horizontal, &host);
    allocCudaStructures(seq_vertical, seq_horizontal, &cuda);

    timer2.eventRecord(ev_copy);
    timer2.eventRecord(ev_alloc);

    int line_index=0;

	midpointsFile->write(midpoint.j, midpoint.i, midpoint.score, midpoint.type);
	while (midpoint.score > 0) {
		fprintf(stdout, ">> %d %d %d\n", midpoint.j, midpoint.i, midpoint.score);
		fprintf(stdout, "Millions Cells Updates: %.3f\n", job->cells_updates/1000000.0f);
		
		SpecialRowReader* specialRow = job->fopenNextSpecialRow(STAGE_1, midpoint.j, midpoint.i, PHASE2_THREADS_COUNT, &line_index); // TODO inverter j0 com i0?
		midpoint = find_next_midpoint(job, seq_vertical, seq_horizontal, reverse, midpoint, host, cuda, specialRow);
        if (specialRow != NULL) specialRow->close();
#ifdef SW_BUS_FLUSH
        unflush_bus(seq0_len-midpoint.i-1);
#endif

		midpointsFile->write(midpoint.j, midpoint.i, midpoint.score- (midpoint.type==0?0:DNA_GAP_OPEN), midpoint.type);
    }

//////////

    timer2.eventRecord(ev_kernel);
    midpointsFile->close();

	fprintf(stats, "CUDA times:\n");
	float diff = timer2.printStatistics(stats);
	
	fprintf(stats, "        total: %.4f\n", diff);
	fprintf(stats, "     Mi.Cells: %.4e\n", (float)job->cells_updates);
	fprintf(stats, "        MCUPS: %.4f\n", job->cells_updates/1000000.0f/(diff/1000.0f));
			
	printDevMem(stats);
	fprintf(stats, " FreeCudaStructures\n");
	freeCudaStructures(&cuda);
	freeHostStructures(&host);
	printDevMem(stats);
	
	fclose(stats);
	hipDeviceReset();
	cutilCheckMsg("hipDeviceReset failed");
}

//#include "sw_stage3.cu"