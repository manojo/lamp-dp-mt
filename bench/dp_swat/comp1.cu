#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "comp1.h"

#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) {
  if (err==hipSuccess) return;
  fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); cuReset; exit(EXIT_FAILURE);
}
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
#define cuMap(host,dev,size) { cuErr(hipHostAlloc((void**)&host,size,hipHostMallocMapped)); cuErr(hipHostGetDevicePointer((void**)&dev,host,0)); }
#define cuUnmap(host) cuErr(hipHostFree(host))
#define cuStream(stream) hipStream_t stream; cuErr(hipStreamCreate(&stream));
#define cuSync(stream) cuErr(hipStreamSynchronize(stream))
#define hipStreamDestroy(stream) cuErr(hipStreamDestroy(stream))
#define cuAlloc2(cond,host,dev,size) bool cond = hipMalloc((void**)&dev,size)==hipSuccess; if (!cond) { cuMap(host,dev,size); }
#define cuFree2(host,dev) { if (host!=NULL) { cuUnmap(host); host=NULL; } else cuFree(dev); dev=NULL; }
#define _unroll _Pragma("unroll 5")
#define M_W 1025
#define M_H 1025
#define B_H 32
#define MEM_MATRIX (M_W* ((M_H+B_H-1)/B_H)*B_H  +B_H*B_H)
#define idx(i,j) ({ unsigned _i=(i); (B_H*((j)+(_i%B_H)) + (_i%B_H) + (_i/B_H)*M_W*B_H); })
static input_t *g_in1 = NULL, *g_in2 = NULL;
static cost_t *g_cost = NULL;
static back_t *g_back = NULL;
__device__ static __attribute__((unused)) input_t *_in1=NULL, *_in2=NULL;
__global__ void gpu_input(input_t* in1, input_t* in2) { _in1=in1; _in2=in2; }


__device__ static inline int fun0(int p) { return 0; }
__device__ static inline int fun1(int a, char c2) { return a>3 ? a-3 : 0; }
__device__ static inline int fun2(int a, char c2) { return a>1 ? a-1 : 0; }
__device__ static inline int fun3(char c1, int a) { return a>3 ? a-3 : 0; }
__device__ static inline int fun4(char c1, int a) { return a>1 ? a-1 : 0; }
__device__ static inline int fun5() { return 0; }
__device__ static inline int fun6(char c1, int a, char c2) { return a + (c1==c2 ? 10 : -3); }

__global__ void gpu_solve(const input_t* in1, const input_t* in2, cost_t* cost, back_t* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
  const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned tN = blockDim.x * gridDim.x;
  const unsigned tB = blockIdx.x;
  unsigned tP=s_start; // block progress
  for (unsigned jj=s_start; jj<s_stop; ++jj) {
    for (int i=tI; i<M_H; i+=tN) {
      int j = jj-i; if (j>=0)
      if (j<M_W) {
        back_t _back = {{-1},{-1},{-1}};
        cost_t _cost = {}; // init to 0
        #define VALID(I,J,RULE) (back[idx(I,J)].RULE.rule!=-1)
        /* --- g2[i,j] --- */
        if (0==j) {
          { int _c=fun0((i)); if (_c>_cost.g2 || _back.g2.rule==-1) { _cost.g2=_c; _back.g2=(bt0){7}; } }
        }
        if (1<=j) {
          { int _c=fun1(cost[idx(i,j-1)].M,_in2[j-1]); if (_c>_cost.g2 || _back.g2.rule==-1) { _cost.g2=_c; _back.g2=(bt0){8}; } }
        }
        if (1<=j) {
          { int _c=fun2(cost[idx(i,j-1)].g2,_in2[j-1]); if (_c>_cost.g2 || _back.g2.rule==-1) { _cost.g2=_c; _back.g2=(bt0){9}; } }
        }
        cost[idx(i,j)].g2 = _cost.g2;
        back[idx(i,j)].g2 = _back.g2;
        /* --- g1[i,j] --- */
        if (0==i) {
          { int _c=fun0((j)); if (_c>_cost.g1 || _back.g1.rule==-1) { _cost.g1=_c; _back.g1=(bt0){4}; } }
        }
        if (1<=i) {
          { int _c=fun3(_in1[i-1],cost[idx(i-1,j)].M); if (_c>_cost.g1 || _back.g1.rule==-1) { _cost.g1=_c; _back.g1=(bt0){5}; } }
        }
        if (1<=i) {
          { int _c=fun4(_in1[i-1],cost[idx(i-1,j)].g1); if (_c>_cost.g1 || _back.g1.rule==-1) { _cost.g1=_c; _back.g1=(bt0){6}; } }
        }
        cost[idx(i,j)].g1 = _cost.g1;
        back[idx(i,j)].g1 = _back.g1;
        /* --- M[i,j] --- */
        if (i==j && 0==i && 0==j) {
          { int _c=fun5(); if (_c>_cost.M || _back.M.rule==-1) { _cost.M=_c; _back.M=(bt0){0}; } }
        }
        if (1<=i && 1<=j) {
          { int _c=fun6(_in1[i-1],cost[idx(i-1,j-1)].M,_in2[j-1]); if (_c>_cost.M || _back.M.rule==-1) { _cost.M=_c; _back.M=(bt0){1}; } }
        }
        { typeof(cost[idx(0,0)].g1) _c=cost[idx(i,j)].g1; if (_c>_cost.M || _back.M.rule==-1) { _cost.M=_c; _back.M=(bt0){2}; } }
        { typeof(cost[idx(0,0)].g2) _c=cost[idx(i,j)].g2; if (_c>_cost.M || _back.M.rule==-1) { _cost.M=_c; _back.M=(bt0){3}; } }
        cost[idx(i,j)].M = _cost.M;
        back[idx(i,j)].M = _back.M;
      }
    }
    // Sync between blocks, removing __threadfence() here is incorrect but works
    // __threadfence();
    if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
    __syncthreads();
  }
}

__global__ void gpu_backtrack(trace_t* trace, unsigned* size, back_t* back, int i0, int j0) {
  trace_t *rd=trace, *wr=trace; *size=0;
  #define PUSH_BACK(I,J,RULE) { wr->i=I; wr->j=J; wr->rule=RULE; ++wr; ++(*size); }
  PUSH_BACK(i0,j0,0);
  for(;rd<wr;++rd) {
    bt0* bt;
    switch (rd->rule) {
      case 0: bt=(bt0*)&back[idx(rd->i,rd->j)].M; break;
      case 1: bt=(bt0*)&back[idx(rd->i,rd->j)].M; break;
      case 2: bt=(bt0*)&back[idx(rd->i,rd->j)].M; break;
      case 3: bt=(bt0*)&back[idx(rd->i,rd->j)].M; break;
      case 4: bt=(bt0*)&back[idx(rd->i,rd->j)].g1; break;
      case 5: bt=(bt0*)&back[idx(rd->i,rd->j)].g1; break;
      case 6: bt=(bt0*)&back[idx(rd->i,rd->j)].g1; break;
      case 7: bt=(bt0*)&back[idx(rd->i,rd->j)].g2; break;
      case 8: bt=(bt0*)&back[idx(rd->i,rd->j)].g2; break;
      case 9: bt=(bt0*)&back[idx(rd->i,rd->j)].g2; break;
    }
    rd->rule=bt->rule;
    switch (rd->rule) {
      case 0: break;
      case 1: PUSH_BACK(rd->i-1,rd->j-1,0); break;
      case 2: PUSH_BACK(rd->i,rd->j,4); break;
      case 3: PUSH_BACK(rd->i,rd->j,7); break;
      case 4: break;
      case 5: PUSH_BACK(rd->i-1,rd->j,0); break;
      case 6: PUSH_BACK(rd->i-1,rd->j,4); break;
      case 7: break;
      case 8: PUSH_BACK(rd->i,rd->j-1,0); break;
      case 9: PUSH_BACK(rd->i,rd->j-1,7); break;
    }
  }
}

static cost_t* c_cost=NULL;
static back_t* c_back=NULL;

void g_init(input_t* in1, input_t* in2) {
  int dev=-1; cuErr(hipGetDevice(&dev));
  cuMalloc(g_in1,sizeof(input_t)*(M_H-1)); cuPut(in1,g_in1,sizeof(input_t)*(M_H-1),NULL);
  cuMalloc(g_in2,sizeof(input_t)*(M_W-1)); cuPut(in2,g_in2,sizeof(input_t)*(M_W-1),NULL);
  size_t s_cost = sizeof(cost_t)*MEM_MATRIX;
  size_t s_back = sizeof(back_t)*MEM_MATRIX;
  cuAlloc2(costDev,c_cost,g_cost,s_cost); cuAlloc2(backDev,c_back,g_back,s_back);
  gpu_input<<<1,1>>>(g_in1,g_in2);
  hipDeviceProp_t prop; cuErr(hipGetDeviceProperties(&prop, dev));
  size_t mem = (sizeof(input_t)+sizeof(trace_t))*(M_W+M_H) + s_cost + s_back;
  printf("%-20s : %.2fMb / %.2fMb [in=%ld,tr=%ld,cost=%ld,back=%ld] -> cost:%s, backtrack:%s\n","Memory selection",mem/1048576.0,prop.totalGlobalMem/1048576.0, sizeof(input_t),sizeof(trace_t),sizeof(cost_t),sizeof(back_t), costDev?"device":"host", backDev?"device":"host");
}

void g_free() {
  cuFree(g_in1); cuFree(g_in2);
  cuFree2(c_cost,g_cost); cuFree2(c_back,g_back); cuReset;
}

void g_solve() {
  #define WARP_SIZE 32 // constant over CUDA devices
  unsigned blk_size = WARP_SIZE;
  unsigned blk_num = (M_H+blk_size-1)/blk_size;
  unsigned* lock; cuMalloc(lock,sizeof(unsigned)*blk_num);
  cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
  cuStream(stream);
  for (int i=0;i<1;++i) {
    unsigned s0=((M_W+M_H)*i)/1, s1=((M_W+M_H)*(i+1))/1;
    gpu_solve<<<blk_num, blk_size, 0, stream>>>(g_in1, g_in2, g_cost, g_back, lock, s0, s1);
  }
  cuSync(stream); hipStreamDestroy(stream); cuFree(lock);
}

int g_backtrack(trace_t** trace, unsigned* size) {
  int res; unsigned i0=M_H-1, j0=M_W-1;
  cuGet(&res,&g_cost[idx(i0,j0)].M,sizeof(int),NULL);
  if (trace && size) {
    unsigned mem=(M_W+M_H)*sizeof(trace_t);
    trace_t *g_trace=NULL,*c_trace=NULL; cuAlloc2(traceDev,c_trace,g_trace,mem);
    unsigned *g_size=NULL; cuMalloc(g_size,sizeof(unsigned));
    gpu_backtrack<<<1,1,0,NULL>>>(g_trace, g_size, g_back, i0, j0);
    cuGet(size,g_size,sizeof(unsigned),NULL); cuFree(g_size); mem=(*size)*sizeof(trace_t);
    *trace=(trace_t*)malloc(mem); cuGet(*trace,g_trace,mem,NULL); cuFree2(c_trace,g_trace);
  }
  return res;
}
