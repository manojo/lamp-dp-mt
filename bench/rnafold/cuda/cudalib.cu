#include "hip/hip_runtime.h"

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <errno.h>

#include "config.h"
#include "options.h"


#include <cutil.h>

#define is_suboptimal(a, b, c) abs(a - b) <= c

FILE *energyFile;

#define ENERGYFILE "energies.dat"

// #define DIFF3
// #define SHARED_Z // 2000

// #define SHARED_ENERGY // 8000
// #define SHARED_OFFSET  // 8000

// #define SPARSE_CLOSED
// #define MAX_DIFF 200

#define SHMEM 1000

/* Always needed
  ====================================================================== */

#define min(A, B) ((A) < (B) ? (A) : (B))
#define max(A, B) ((A) > (B) ? (A) : (B))

/* Input handling
  ====================================================================== */

void convert_input(int start, char *z, int n);


/* Correct incomplete phases in adpc
  ====================================================================== */

#define decode(X) ((X)-'0')


/* Memory handling
  ====================================================================== */


/* wrappers for standard C functions
   ------------------------------------------- */

void memerr_exit(char *f);
void *mcalloc(size_t nobj, size_t size);
void *mmalloc(size_t size);
void *mrealloc(void *q, size_t size);


/* Memory management
   ------------------------------------------- */

typedef struct {
  char **address;
  int currentBlock;
  int currentPos;
  int blockSize;
  int numberOfBlocks;
} tmemory;

tmemory *adp_statmem;
tmemory *adp_dynmem;

void      set_adplib_debug(int debug);

void     *myalloc(tmemory *mem, int size);

tmemory  *memory_new();
void      memory_clear(tmemory *mem);
void      memory_free(tmemory *mem);

/* Preprocessing tools
  ====================================================================== */

char arr_iupac_base[128][5];
#define iupac_base(A,B) arr_iupac_base[A][B]
char *calc_contains_region(char *z, int n, int *offset, char *pat1);

/* String tools
  ====================================================================== */

char *mkstr(char *s);

#define dots(i,j) libPP_repeat(i,j,'.')

char *libPP_repeat(int i, int j, char c);

/* File input
  ====================================================================== */

/* A single sequence
   ------------------------------------------- */

typedef struct {
  char success;
  char *descr;
  char *seq;
  int  length;
  char *original_seq;  /* backup for window mode */
  int   original_length;
} tsequence;


tsequence *sequence_new();
tsequence *sequence_free(tsequence *ts);

/* A complete file
   ------------------------------------------- */

#define READSEQ_FILE    1
#define READSEQ_STDIN   2
#define READSEQ_STRING  3

typedef struct {
  char *filename;
  char *start;
  int  current;
  char first_input_read;
  char first_descr_read;
  char *temp;
} treadseq;


treadseq    *readseq_open(char mode, char *filename);
treadseq    *readseq_free(treadseq *rs);

/* reader for different input formats
   ------------------------------------------- */

tsequence *readseq_next_line(treadseq *rs);
tsequence *readseq_next_fasta(treadseq *rs);

/* Functions for results output
  ====================================================================== */

void simple_output_optimal     (toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);
void simple_output_subopt_start(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);
void simple_output_subopt      (toptions *opts, tsequence *seq, char *algebra, int score, char *result_prettyprint);
void simple_output_subopt_end  (toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);

void rna_output_optimal     (toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);
void rna_output_subopt_start(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);
void rna_output_subopt      (toptions *opts, tsequence *seq, char *algebra, int score, char *result_prettyprint);
void rna_output_subopt_end  (toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end);

/* Tokenizer for interactive command shell
  ====================================================================== */

typedef struct {
  char **token;
  int count;
} ttokenizer;


ttokenizer *tokenizer_new();
ttokenizer *tokenizer_free(ttokenizer *t);
void tokenizer_exec(ttokenizer *t, char *name, char *s);

/* wrappers for readline
  ====================================================================== */

void rl_init();
char *rl_gets ();

/* colored output
  ====================================================================== */

#define COLOR_DEFAULT     "\x1b[0m"
#define COLOR_BOLD        "\x1b[1m"
#define COLOR_BLACK       "\x1b[0;30m"
#define COLOR_BLUE        "\x1b[0;34m"
#define COLOR_GREEN       "\x1b[0;32m"
#define COLOR_CYAN        "\x1b[0;36m"
#define COLOR_RED         "\x1b[0;31m"
#define COLOR_PURPLE      "\x1b[0;35m"
#define COLOR_BROWN       "\x1b[0;33m"
#define COLOR_GRAY        "\x1b[0;37m"
#define COLOR_DARKGRAY    "\x1b[1;30m"
#define COLOR_LIGHTBLUE   "\x1b[1;34m"
#define COLOR_LIGHTGREEN  "\x1b[1;32m"
#define COLOR_LIGHTCYAN   "\x1b[1;36m"
#define COLOR_LIGHTRED    "\x1b[1;31m"
#define COLOR_LIGHTPURPLE "\x1b[1;35m"
#define COLOR_YELLOW      "\x1b[1;33m"
#define COLOR_WHITE       "\x1b[1;37m"
#define pcolor(colored,col) if (colored) printf(col)


/* Output format string handling
  ====================================================================== */

/* calculate the number of leading spaces for sequence output */
void fs_init_leading_space(char energy, char shrepProb, char dbString, char shapeString, char prob, char rank);

/* free the leading spaces temp mem. */
void fs_free_leading_space();

/* initialize a new format string */
void format_string_init(char *s);

/* set predefined output modes */
void setOutputMode(int outputMode);

/* main entry function for sequence output */
void print_sequence(toptions *opts, tsequence *seq, int pos, int size);

/* used in window-mode to shift the input sequence */
void shift_input(toptions *opts, tsequence *seq, char output);

/* main entry function for rna result output */
void output_result
  (toptions *opts, tsequence *seq,
   int nres, int *energy, double *shrepProb, char *dbString, char *shapeString, double prob, int rank);

typedef struct format_string{
  char type;
  char *string;
  struct format_string *next;
} tformat_string;



/* Initialize all stuff from adplib
  ====================================================================== */

void adplib_init(toptions *opts, tsequence *seq, char **z, int *n);
void adplib_free(toptions *opts, tsequence *seq);



/* Input handling
  ====================================================================== */

/* The alphabet */
#define A 0
#define C 1
#define G 2
#define U 3
#define N 4

void convert_input(int start, char *z, int n){
  int i;
  char c;

  for (i=start; i<=n; i++) {
    c=z[i];
    if      (c=='a') z[i]=A;
    else if (c=='c') z[i]=C;
    else if (c=='g') z[i]=G;
    else if (c=='u') z[i]=U;
    else if (c=='t') z[i]=U;  /* replace DNA with RNA */
    else if (c=='A') z[i]=A;
    else if (c=='C') z[i]=C;
    else if (c=='G') z[i]=G;
    else if (c=='U') z[i]=U;
    else if (c=='T') z[i]=U;
    else             z[i]=N;  /* all other characters are mapped to N and will not be paired */
  }
}


/* Memory handling
  ====================================================================== */

/* wrappers for standard C functions
   ------------------------------------------- */

void memerr_exit(char *f){
    fprintf(stderr, "\n%s: out of memory\n", f);
    fprintf(stderr, "possible reasons:\n");
    fprintf(stderr, "   input sequence too long\n");
    fprintf(stderr, "   energy range too large (decrease with -e or -c)\n");
    fprintf(stderr, "   shape type not abstract enough (increase with -t)\n");
    exit(1);
}

void *mcalloc(size_t nobj, size_t size){
  void *p;
  if ((p = calloc(nobj, size)) != NULL) return p;
  else memerr_exit("calloc");
  return NULL;
}

void *mmalloc(size_t size){
  void *p;
  if ((p = malloc(size)) != NULL) return p;
  else memerr_exit("malloc");
  return NULL;
}

void *mrealloc(void *q, size_t size){
  void *p;
  if ((p = realloc(q, size)) != NULL) return p;
  else memerr_exit("realloc");
  return NULL;
}

/* Memory management
   ------------------------------------------- */

#define ALIGNMENT 8
#define BLOCKSIZE 1000000

static int adplib_debug = 0;

void set_adplib_debug(int debug){
  adplib_debug = debug;
  printf("adplib_debug set to %d.\n", adplib_debug);
}


void *myalloc(tmemory *mem, int size)
{
  //  return mcalloc(size,sizeof(char));
  //}
   if (adplib_debug>1) printf("myalloc(), currentBlock = , currentPos = \n");
   if (size % ALIGNMENT) {
     size = ((size / ALIGNMENT) + 1) * ALIGNMENT;
     if (adplib_debug>1) printf("realigned: %d\n", size);
   }

   if (mem->currentPos + size >= mem->blockSize) {
      mem->currentBlock++;
      if (mem->currentBlock > mem->numberOfBlocks) {
        mem->address = (char **) mrealloc(mem->address, sizeof (char *) * mem->currentBlock);
        mem->numberOfBlocks = mem->currentBlock;
        mem->address[mem->currentBlock - 1] = (char*) mmalloc(mem->blockSize);
	//        if(adplib_debug>1) printf("address of mem->address[mem->currentBlock - 1]: %d\n", mem->address[mem->currentBlock - 1]);
      }
      mem->currentPos = 0;
      //      if (adplib_debug>1) printf("mrealloc: myalloc(%d), currentBlock = %d, currentPos = %d\n", size, mem->currentBlock, mem->currentPos);
   }
   mem->currentPos = mem->currentPos + size;
   //   if (adplib_debug>1) printf("myalloc: address: %d\n", mem->address[mem->currentBlock - 1] + (mem->currentPos - size));
   return(mem->address[mem->currentBlock - 1] + (mem->currentPos - size));
}

tmemory *memory_new()
{
  tmemory *mem        = (tmemory *) mmalloc(sizeof(tmemory));
  mem->address        = (char **) mmalloc(sizeof(char *));
  mem->address[0]     = (char *) mmalloc(BLOCKSIZE);
  mem->blockSize      = BLOCKSIZE;
  mem->currentBlock   = 1;
  mem->numberOfBlocks = 1;
  mem->currentPos     = 0;
  if (adplib_debug>1) printf("adplib.memory.new(): allocated %d bytes\n", BLOCKSIZE);
  return mem;
}

void memory_clear(tmemory *mem)
{
  mem->currentBlock = 0;
  mem->currentPos   = mem->blockSize + 1;
}

void memory_free(tmemory *mem)
{
  int i;
  if (adplib_debug) printf("freeing %d blocks, blockSize = %d => %d bytes\n",
         mem->numberOfBlocks, mem->blockSize, mem->numberOfBlocks * mem->blockSize);
  for (i=0; i<=mem->numberOfBlocks-1; i++) free(mem->address[i]);
  free(mem->address);
  free(mem);
}

/* Preprocessing tools
  ====================================================================== */


/* ---------------------------------------------------------------------------------------------------- */
/* iupac_base                                                                                           */
/* ---------------------------------------------------------------------------------------------------- */

static void init_iupac_base(){
  int i,j;

  for (i=0;i<128;i++)
    for (j=0;j<5;j++)
      arr_iupac_base[i][j]=0;

  arr_iupac_base['a'][A]=1;     arr_iupac_base['A'][A]=1;
  arr_iupac_base['c'][C]=1;	arr_iupac_base['C'][C]=1;
  arr_iupac_base['g'][G]=1;	arr_iupac_base['G'][G]=1;
  arr_iupac_base['t'][U]=1;	arr_iupac_base['T'][U]=1;
  arr_iupac_base['u'][U]=1;	arr_iupac_base['U'][U]=1;
  arr_iupac_base['r'][A]=1;	arr_iupac_base['R'][A]=1;
  arr_iupac_base['r'][G]=1;	arr_iupac_base['R'][G]=1;
  arr_iupac_base['y'][C]=1;	arr_iupac_base['Y'][C]=1;
  arr_iupac_base['y'][U]=1;	arr_iupac_base['Y'][U]=1;
  arr_iupac_base['s'][G]=1;	arr_iupac_base['S'][G]=1;
  arr_iupac_base['s'][C]=1;	arr_iupac_base['S'][C]=1;
  arr_iupac_base['w'][A]=1;	arr_iupac_base['W'][A]=1;
  arr_iupac_base['w'][U]=1;	arr_iupac_base['W'][U]=1;
  arr_iupac_base['k'][G]=1;	arr_iupac_base['K'][G]=1;
  arr_iupac_base['k'][U]=1;	arr_iupac_base['K'][U]=1;
  arr_iupac_base['m'][A]=1;	arr_iupac_base['M'][A]=1;
  arr_iupac_base['m'][C]=1;	arr_iupac_base['M'][C]=1;
  arr_iupac_base['b'][C]=1;	arr_iupac_base['B'][C]=1;
  arr_iupac_base['b'][G]=1;	arr_iupac_base['B'][G]=1;
  arr_iupac_base['b'][U]=1;	arr_iupac_base['B'][U]=1;
  arr_iupac_base['d'][A]=1;	arr_iupac_base['D'][A]=1;
  arr_iupac_base['d'][G]=1;	arr_iupac_base['D'][G]=1;
  arr_iupac_base['d'][U]=1;	arr_iupac_base['D'][U]=1;
  arr_iupac_base['h'][A]=1;	arr_iupac_base['H'][A]=1;
  arr_iupac_base['h'][C]=1;	arr_iupac_base['H'][C]=1;
  arr_iupac_base['h'][U]=1;	arr_iupac_base['H'][U]=1;
  arr_iupac_base['v'][A]=1;	arr_iupac_base['V'][A]=1;
  arr_iupac_base['v'][C]=1;	arr_iupac_base['V'][C]=1;
  arr_iupac_base['v'][G]=1;	arr_iupac_base['V'][G]=1;
  arr_iupac_base['n'][A]=1;	arr_iupac_base['N'][A]=1;
  arr_iupac_base['n'][C]=1;	arr_iupac_base['N'][C]=1;
  arr_iupac_base['n'][G]=1;	arr_iupac_base['N'][G]=1;
  arr_iupac_base['n'][U]=1;	arr_iupac_base['N'][U]=1;
}

/* char arr_iupac_base[ */

/* A 	Adenine
 */
/* C 	Cytosine
 */
/* G 	Guanine
 */
/* T (or U) 	Thymine (or Uracil)
 */
/* R 	A or G
 */
/* Y 	C or T
 */
/* S 	G or C
 */
/* W 	A or T
 */
/* K 	G or T
 */
/* M 	A or C
 */
/* B 	C or G or T
 */
/* D 	A or G or T
 */
/* H 	A or C or T
 */
/* V 	A or C or G
 */
/* N 	any base
 */
/* . or - 	gap */


char *calc_contains_region(char *z, int n, int *offset, char *pat)
{
  int i,j,k,l,ppos,inPattern;
  char *arr;

  if (adplib_debug>1) printf("entering...\n");
  if (adplib_debug>1) for(i=0; i<=n; i++) printf("z[%d] = %d\n", i, z[i]);
  l = strlen(pat);

  if (adplib_debug>1) for(i=0; i<=l; i++) printf("pat[%d] = %c\n", i, pat[i]);
  arr = (char *) malloc((offset[n]+n+1) * sizeof(char));

  if (adplib_debug>1) printf("calculating...\n");

  for (j=0; j<=n; j++) {
    for (i=0; i<=j; i++) {
      arr[offset[j]+i]=0;  // tab(i,j)
      ppos = 0;
      inPattern = 0;
      if (j-i >= l) {
        for (k=i+1;k<=j;k++) {
          if (inPattern){
            if (iupac_base(pat[ppos],z[k])) {
              ppos++;
            }
            else {
              inPattern = 0;
              ppos = 0;
	      break;
	    }
          }
          else {
            if (iupac_base(pat[ppos],z[k])) {
              inPattern = 1;
              ppos++;
	    }
	  }
          if (ppos==l) arr[offset[j]+i]=1;
	}
      }
    }
  }
  if (adplib_debug>1) {
    for (j=0; j<=n; j++) {
      for (i=0; i<=j; i++) {
        printf("%d ", arr[offset[j]+i]);
      }
      printf("\n");
    }
  }
  return arr;
}

/* String tools
  ====================================================================== */

char *mkstr(char *s){
  return strcpy((char *) malloc(strlen(s)+1 * sizeof(char)), s);
}

char *mkstr_stat(char *s){
  return strcpy((char *) myalloc(adp_statmem, (strlen(s)+1) * sizeof(char)), s);
}

/* ---------------------------------------------------------------------------------------------------- */
/* Dotbracket tools                                                                                     */
/* ---------------------------------------------------------------------------------------------------- */

static char *libPP_hlp;
static char *dots_hlp;

char *libPP_repeat(int i, int j, char c){
  int k;
  for (k=0; k<(j-i); k++) libPP_hlp[k]=c;
  libPP_hlp[k]=0;
  return(libPP_hlp);
}

static void libPP_init(tsequence *seq){
   libPP_hlp = (char *) myalloc(adp_statmem, (seq->length+4) * sizeof(char));
   dots_hlp = mkstr_stat(libPP_repeat(0, seq->length, '.'));
}


/* File input
  ====================================================================== */

/* A single sequence
   ------------------------------------------- */

/* typedef struct { */
/*   char success; */
/*   char *descr; */
/*   char *seq; */
/*   int  length; */
/*   char *original_seq;  /\* backup for window mode *\/ */
/*   int   original_length; */
/* } tsequence; */


tsequence *sequence_new()
{
  tsequence *ts;

  ts = (tsequence *) mmalloc(sizeof(tsequence));
  ts->success         = 0;
  ts->descr           = NULL;
  ts->seq             = NULL;
  ts->length          = 0;
  ts->original_seq    = NULL;
  ts->original_length = 0;

  return ts;
}

tsequence *sequence_free(tsequence *ts)
{
  if (ts->descr)        free(ts->descr);
  if (ts->seq)          free(ts->seq);
  if (ts->original_seq) free(ts->original_seq);
  free(ts);

  return NULL;
}

/* A complete file
   ------------------------------------------- */

#define MAXINPUT 1000000

/* typedef struct { */
/*   char *filename; */
/*   char *start; */
/*   int  current; */
/*   char first_input_read; */
/*   char first_descr_read; */
/*   char *temp; */
/* } treadseq; */

static char *readseq_readfile(FILE *inputStream)
{
  int   inpc, inpn;
  int   tinput_alloc;
  char *tinput;

  inpn = 0;
  tinput = (char *) malloc(MAXINPUT * sizeof(char));
  tinput_alloc = MAXINPUT;

  while ((inpc = getc(inputStream)) != EOF) {
    if (inpn == tinput_alloc-2) {
      tinput = (char *) realloc(tinput, sizeof (char) * (tinput_alloc * 2));
      tinput_alloc *= 2;
    }
    tinput[inpn++] = inpc;
  }
  tinput[inpn] = 0;
  return tinput;
}

treadseq *readseq_open(char mode, char *filename){
  treadseq *rs;
  FILE *inputStream;

  rs = (treadseq *) malloc(sizeof(treadseq));
  rs->current          = 0;
  rs->first_input_read = 0;
  rs->first_descr_read = 0;
  rs->temp             = (char *) malloc(MAXINPUT * sizeof(char));

  if (mode == READSEQ_STRING) {
    rs->start    = mkstr(filename);
    rs->filename = mkstr("command line");
    return rs;
  }

  if (mode == READSEQ_STDIN) {
    inputStream   = stdin;
    rs->filename  = mkstr("stdin");
  }
  if (mode == READSEQ_FILE) {
    inputStream   = fopen(filename, "r");
    rs->filename  = mkstr(filename);
  }

  if (!inputStream) {
    fprintf(stderr, "error opening file %s: %s\n", filename, strerror(errno));
    exit(errno);
  }

  rs->start = readseq_readfile(inputStream);

  if (fclose(inputStream)) {
    fprintf(stderr, "error closing file %s: %s\n", filename, strerror(errno));
    exit(errno);
  }

  return rs;
}

treadseq *readseq_free(treadseq *rs){
  if (rs) {
    free(rs->filename);
    free(rs->start);
    free(rs->temp);
    free(rs);
  }
  return NULL;
}


/* Read next line from file
   ------------------------------------------- */

tsequence *readseq_next_line(treadseq *rs){

  tsequence *ts;
  int pos;

  ts = sequence_new();
  ts->success    = 0;
  if (adplib_debug>1) printf("rs: success = 0\n");

  if (adplib_debug>1) printf("rs: %d\n", rs->start[rs->current]);

  rs->temp[0] = 0; pos = 0;
  if (rs->start[rs->current] != 0) {
    while ((rs->start[rs->current] != '\n') && (rs->start[rs->current] != 0))
       rs->temp[pos++] = rs->start[rs->current++];
    if (rs->start[rs->current]=='\n') rs->current++;

    rs->temp[pos] = 0;
    ts->seq          = mkstr(rs->temp);
    ts->original_seq = mkstr(rs->temp);

    // remove additional #13 for DOS input files:
    if ((pos >= 1) && (rs->temp[pos-1] == 13)) rs->temp[pos-1] = 0;

    // remove additional #13 for DOS input files:
    if ((pos >= 1) && (rs->temp[pos-1] == 13)) rs->temp[pos-1] = 0;

    ts->length = ts->original_length = strlen(ts->seq);
    ts->success    = 1;
    if (adplib_debug>1) printf("rs: success = 1\n");
  }

  return ts;
}


/* Read next fasta sequence from file
   ------------------------------------------- */

tsequence *readseq_next_fasta(treadseq *rs){

  tsequence *ts;
  int pos;
  char inpc,fil;

  ts = sequence_new();

  rs->temp[0] = 0; pos = 0;
  if ((rs->start[rs->current] == '>')  && (rs->start[rs->current] != 0)) {
    if (!rs->first_descr_read && rs->first_input_read) {
      fprintf(stderr, "error in input file: missing description for first sequence\n");
      exit(1);
    }
    rs->first_descr_read = 1;
    rs->current++;
    while ((rs->start[rs->current] != '\n') && (rs->start[rs->current] != 0)) rs->temp[pos++] = rs->start[rs->current++];
    if (rs->start[rs->current]) rs->current++;
  }
  rs->temp[pos] = 0;
  ts->descr = mkstr(rs->temp);

  // remove additional #13 for DOS input files:
  if ((pos >= 1) && (rs->temp[pos-1] == 13)) rs->temp[pos-1] = 0;

  rs->temp[0] = 0; pos = 0;
  fil = 1;
  while ((fil || (rs->start[rs->current] != '>')) && (rs->start[rs->current] != 0)) {
    while (((inpc = rs->start[rs->current]) != '\n') && (rs->start[rs->current] != 0))
       if (((inpc >= 65) && (inpc <= 90)) ||
           ((inpc >= 97) && (inpc <= 122))) rs->temp[pos++] = rs->start[rs->current++];
       else                                 rs->current++;
    fil = 0;
    if (rs->start[rs->current]) rs->current++;
    rs->first_input_read = 1;
  }
  rs->temp[pos] = 0;
  ts->seq           = mkstr(rs->temp);
  ts->original_seq  = mkstr(rs->temp);
  ts->length = ts->original_length = strlen(ts->seq);
  if (ts->seq[0]) ts->success    = 1;
  else            ts->success    = 0;

  return ts;
}

/* Functions for results output
  ====================================================================== */

/* Simple standard output
   ------------------------------------------- */

void simple_output_optimal(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
  printf("\nInput: ");
  pcolor(opts->colored_output,COLOR_BOLD);
  printf("%s", seq->seq);
  pcolor(opts->colored_output,COLOR_DEFAULT);
  printf("\nAlgebra: ");
  pcolor(opts->colored_output,COLOR_BOLD);
  printf("%s", algebra);
  pcolor(opts->colored_output,COLOR_DEFAULT);
  printf(", score: ");
  pcolor(opts->colored_output,COLOR_BOLD);
  printf("%d\n", result_score);
  pcolor(opts->colored_output,COLOR_DEFAULT);
}


void simple_output_subopt_start(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
  if (strcmp(algebra, "count")) {
    printf("Suboptimal range: [%d - %d]\n", range_begin, range_end);
    printf("\n");
    printf(" Score | Candidate\n");
    // pcolor(opts->colored_output,COLOR_BOLD);
    printf("-----------------------------------------------------------------\n");
    // pcolor(opts->colored_output,COLOR_DEFAULT);
  }
}

void simple_output_subopt(toptions *opts, tsequence *seq, char *algebra, int score, char *result_prettyprint)
{
  if (strcmp(algebra, "count")) {
    printf("%6d | %s\n", score, result_prettyprint);
  }
}

void simple_output_subopt_end(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
  printf("\n");
  // pcolor(opts->colored_output,COLOR_BOLD);
  printf("=================================================================\n");
  // pcolor(opts->colored_output,COLOR_DEFAULT);

}

/* RNA output
   ------------------------------------------- */

void rna_output_optimal(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
  if (!opts->window_mode) {
    printf("\n");
    pcolor(opts->colored_output,COLOR_BOLD);
    printf("%s: ", algebra);
    printf("%.2f kcal/mol", ((float) result_score) / 100);
    pcolor(opts->colored_output,COLOR_DEFAULT);
    printf("\n");
  }
}

void rna_output_subopt_start(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
  if (!opts->window_mode && (strcmp(algebra, "count"))) {
    printf("Suboptimal range: [%.2f kcal/mol - %.2f kcal/mol]\n", ((float) range_begin)/100, ((float) range_end)/100);
    printf("\n");

    if (seq->descr && seq->descr[0]) {
      pcolor(opts->colored_output,COLOR_BOLD);
      printf(">%s", seq->descr);
      pcolor(opts->colored_output,COLOR_DEFAULT);
      printf("\n");
    }
    if (!opts->split_output_mode) printf("%s\n", seq->original_seq);
  }
  // print subsequence for first window iteration
  // for later iterations, this is done in function shift_input
  if (opts->window_mode && (opts->window_pos==0) && (strcmp(algebra, "count"))) {
    print_sequence(opts, seq, opts->window_pos, opts->window_size);
  }
}

void rna_output_subopt(toptions *opts, tsequence *seq, char *algebra, int score, char *result_prettyprint)
{
  if (strcmp(algebra, "count")) {
    //    printf("%s  (%.2f)\n", result_prettyprint, ((float) score) / 100 );
    output_result(opts, seq, 1, // TODO: number of results
                  &score, NULL, result_prettyprint, NULL, -1, -1);
  }
}

void rna_output_subopt_end(toptions *opts, tsequence *seq, char *algebra, int result_score, int range_begin, int range_end)
{
/*   printf("\n"); */
/*   pcolor(opts->colored_output,COLOR_BOLD); */
/*   printf("=================================================================\n"); */
/*   pcolor(opts->colored_output,COLOR_DEFAULT); */
}

void rna_output_descr(toptions *opts, tsequence *seq)
{
  if (seq->descr && seq->descr[0]) {
    pcolor(opts->colored_output,COLOR_BOLD);
    printf(">%s", seq->descr);
    pcolor(opts->colored_output,COLOR_DEFAULT);
    printf("\n");
  }
}


/* Tokenizer for interactive command shell
  ====================================================================== */

#define MAXTOKEN 500

/* typedef struct { */
/*   char **token; */
/*   int count; */
/* } ttokenizer; */


ttokenizer *tokenizer_new(){
  ttokenizer *t;

  t = (ttokenizer *) malloc(sizeof(ttokenizer));
  t->token = (char **) malloc(MAXTOKEN * sizeof(char *));
  t->count = 0;

  return t;
}

ttokenizer *tokenizer_free(ttokenizer *t){
  int i;
  if (t) {
    for (i=0; i < t->count; i++) free(t->token[i]);
    free(t->token);
    free(t);
  }
  return NULL;
}

void tokenizer_exec(ttokenizer *t, char *name, char *s)
{
  int i;
  char in_quotes;
  char *ttoken;
  char  septoken[]  = {1,0};

  // free old token strings:
  for (i=0; i < t->count; i++) free(t->token[i]);

  // build new token array:
  t->count = 1;

  // replace whitespaces:
  in_quotes = 0;
  for (i=0; i<strlen(s); i++) {
    if (!in_quotes && (s[i] == ' ')) s[i] = 1;
    if (s[i] == 39) {
      in_quotes = 1-in_quotes;
      s[i] = 1;
    }
  }
  // build token list:
  t->token[0]     = mkstr(name);
  while((ttoken = strtok(s,septoken))) {
    t->token[t->count++] = mkstr(ttoken);
    s = NULL;
  }
  // debug output:
  /*   if (debug) { */
  /*        printf("#token: %d\n", t->count-1);  */
  /*        for (i=1;i<t->count;i++) printf("%s\n", t->token[i]);  */
}


/* wrappers for readline
  ====================================================================== */

/* wrappers for readline
  ====================================================================== */

#ifdef HAVE_LIBEDITLINE
extern int rl_insert(int count, int c);
extern int rl_bind_key(int c, int func(int, int));
extern char *readline(const char *prompt);
extern int add_history(const char *line);
#endif

static char *line_read = (char *)NULL;

void rl_init() {
#ifdef HAVE_LIBEDITLINE
  //   printf("readline activated\n");
  //   rl_bind_key('\t', rl_insert);
#else
   line_read = (char *) calloc(MAXINPUT, sizeof(char));
   line_read[0] = 0;
#endif
}

char *rl_gets (){
#ifdef HAVE_LIBEDITLINE
  if (line_read)
    {
      free (line_read);
      line_read = (char *)NULL;
    }

  line_read = readline (NULL);

  if (line_read && *line_read)
    add_history (line_read);
#else
  fgets(line_read, MAXINPUT-2, stdin);
  // remove last newline
  line_read[strlen(line_read)-1] = 0;
#endif

  return (line_read);
}


/* Output format string handling
  ====================================================================== */

/* show shrep probability */
static char shrep_prob_show;

/* PS output */
static char graphics_alloc = 0;
static char *graphics_sequence;
static char *graphics_shapeString;
static char *graphics_preString;
static char *graphics_fileName_temp;
static char *graphics_fileName;

int number_of_graphics;
char colored_db2shape;

/* typedef struct { */
/*   char type; */
/*   char *string; */
/*   struct format_string *next; */
/* } tformat_string; */


static tformat_string *format_string_struct = NULL; // current format string
static char *leading_space;                         // help strings
static char *leading_space_db;

#define FORMAT_LENGTH 1024
#define FORMAT_ENERGY 1
#define FORMAT_SHREPPROB 2
#define FORMAT_DBSTRING 3
#define FORMAT_SHAPESTRING 4
#define FORMAT_PROB 5
#define FORMAT_RANK 6
#define FORMAT_VERB 7

/* create a new format string entry */
static tformat_string *format_string_newentry(char type, char *string){
  tformat_string *f;
  f = (tformat_string *) mcalloc(1, sizeof(tformat_string));
  f->type   = type;
  f->string = mkstr(string);
  f->next   = NULL;
  return f;
}

/* free a complete format string structure */
static void format_string_free(tformat_string *l){
  tformat_string *t, *tn;
  t=l;
  while (t){
    tn = t->next;
    free(t->string);
    free(t);
    t = tn;
  }
}

/* error handling for format strings; print error and use mode -o 2 instead */
static void format_string_error(char *s, char c){
  printf(s, c);
  printf("Using default output mode instead (-o 2).\n");
  setOutputMode(2);
}

/* build a format string structure for the given string */
static tformat_string *format_string_build(char *s){
  char *t;
  char type;
  int pos, tpos;
  tformat_string *fs, *tfs, *ffs;

  if (strlen(s)>FORMAT_LENGTH) {
    format_string_error("Format string too long\n", 0);
    return(format_string_struct);
  }

  t = (char *) mcalloc(FORMAT_LENGTH+1, sizeof(char));
  tfs = NULL; ffs = NULL;

  pos = 0;  tpos = 0;

  while (s[pos]) {
    switch (s[pos++]) {
      case 'E':
        type = FORMAT_ENERGY;
        break;
      case 'R':
        type = FORMAT_SHREPPROB;
        break;
      case 'D':
        type = FORMAT_DBSTRING;
        break;
      case 'S':
        type = FORMAT_SHAPESTRING;
        break;
      case 'P':
        type = FORMAT_PROB;
        break;
      case 'C':
        type = FORMAT_RANK;
        break;
      case 'V':
        type = FORMAT_VERB;
        break;
      default:
        format_string_error("Syntax error in format string: unexpected character '%c'.\n", s[pos-1] == 1 ? ' ' : s[pos-1]);
        return(format_string_struct);
    }
    if (s[pos++] != '{') {
      format_string_error("Syntax error in format string: '{' expected.\n", 0);
      return(format_string_struct);
    }
    else {
      tpos = 0;
      while (s[pos] && (s[pos] != '}')) {
	if (s[pos] == '\\') {
          pos++;
          switch (s[pos++]) {
  	    case 'n':
              t[tpos++] = '\n';
              break;
	    case 't':
              t[tpos++] = '\t';
              break;
	    case 'e':
              t[tpos++] = '\x1b';
              break;
            default:
              format_string_error("Syntax error in format string: unexpected character '\\%c'.\n", s[pos-1]);
              return(format_string_struct);
          }
        }
        else t[tpos++] = s[pos++];
      }
      t[tpos] = 0;
      fs = format_string_newentry(type, t);

      if (!ffs) ffs=fs;
      else      tfs->next = fs;
      tfs = fs;
    }
    pos++;
  }

  free(t);
  return ffs;
}

/* remove all ansi color command from the given string;
   used to calculate the correct number of leading spaces */
static void fs_remove_color_commands(char *s){
  char *t;
  int pos, tpos, l;
  t = s;
  l = strlen(s);
  pos = 0; tpos = 0;
  while (pos < l) {
    if (t[pos] == '\x1b') { while ((pos<l) && (t[pos]!='m')) pos++; }
    else t[tpos++] = t[pos];
    pos++;
  }
  t[tpos] = 0;
}

/* calculate the number of leading spaces for sequence output; example:

   <leading spaces>gucugcaugacugacugacugacuguagcugcaugcaugcaugcacugaugca
   (-20.4)         ....(((.....))).........((((.....))))................

*/
void fs_init_leading_space(char energy, char shrepProb, char dbString, char shapeString, char prob, char rank){
  tformat_string *itr;
  int pos;
  char dbinside;
  char *t, *s;

  s = t            = (char *) myalloc(adp_statmem, (FORMAT_LENGTH+100) * sizeof(char));
  leading_space_db = (char *) myalloc(adp_statmem, 100 * sizeof(char));

  dbinside = 0;
  itr = format_string_struct;
  while (itr) {
    switch(itr->type) {
      case FORMAT_ENERGY:
        if (energy) sprintf(s, itr->string, -10.0);
        break;
      case FORMAT_SHREPPROB:
        if (shrep_prob_show && shrepProb) sprintf(s, itr->string, 0.5);
        break;
      case FORMAT_DBSTRING:
        if (dbString) sprintf(s, "{");
        if (!dbinside) {
          sprintf(leading_space_db, itr->string, "{");
          fs_remove_color_commands(leading_space_db);
          pos = 0;
          while(leading_space_db[pos]) {
            if (leading_space_db[pos]=='{') {
              leading_space_db[pos]=0;
              break;
            }
            if ((leading_space_db[pos]=='\t') || (leading_space_db[pos]=='\n')) pos++;
            else leading_space_db[pos++]=' ';
          }
        }
        dbinside = 1;
        break;
      case FORMAT_SHAPESTRING:
        if (shapeString) sprintf(s, itr->string, "[][]");
        break;
      case FORMAT_PROB:
        if (prob) sprintf(s, itr->string, 0.5);
        break;
      case FORMAT_RANK:
        if (rank) sprintf(s, itr->string, 1);
        break;
      case FORMAT_VERB:
        sprintf(s, "%s", itr->string);
        break;
    }
    s = s + strlen(s);
    itr = itr->next;
  }
  fs_remove_color_commands(t);
  s[0] = 0;
  pos=0;
  while(t[pos]) {
    if (t[pos]=='{') {
      t[pos]=0;
      break;
    }
    if ((t[pos]=='\t') || (t[pos]=='\n')) pos++;
    else t[pos++]=' ';
  }

  if (!dbString || !dbinside) t[0]=0;
  leading_space = t;
}

/* free the leading spaces temp mem. */
void fs_free_leading_space(){
  free(leading_space);
  free(leading_space_db);
}

/* initialize a new format string */
void format_string_init(char *s){
  if (format_string_struct) format_string_free(format_string_struct);
  format_string_struct = format_string_build(s);
}

/* set predefined output modes */
void setOutputMode(int outputMode){
  if        (outputMode == 1) format_string_init("D{%s  }E{(%.2f)  }R{(%.7f)  }P{%.7f  }S{%s}C{  R = %d}V{\n}");
  else   if (outputMode == 2) format_string_init("E{%-8.2f}R{(%.7f)  }D{%s  }P{%.7f  }S{%s}C{  R = %d}V{\n}");
  else   if (outputMode == 3) format_string_init("E{%.2f }R{%.7f }D{%s }P{%.7f }S{%s}C{ %d}V{\n}");
  else                        format_string_init("E{%-8.2f}R{(%.7f)  }D{\x1b[1;31m%s\x1b[0m  }P{\x1b[1;30m%.7f\x1b[0m  }S{%s}C{  R = %d}V{\n}");
}

/* Output handling for sequences
  ====================================================================== */

/* print position numbers for sequence;
   used in window- and split-output modes  */
static void print_subseq_numbers(toptions *opts, int pos, int size){
  int i;
  if (opts->split_output_mode) size = min(opts->window_pos + opts->window_size, pos + opts->split_output_size) - pos;
  pcolor(opts->colored_output, COLOR_BLUE);
  printf("%d", pos +1);
  pcolor(opts->colored_output,COLOR_DEFAULT);

  for (i=1; i<= size - (((int)log10(pos + 1))+1 + ((int)log10(pos + size))+1); i++) printf(" ");
  pcolor(opts->colored_output,COLOR_BLUE);
  printf("%d", pos + size);
  pcolor(opts->colored_output,COLOR_DEFAULT);
  printf("\n");
}

/* print the given subsequence of a sequence */
static void print_subseq(char *fs, char *s, int offset, int pos, int size){
  int i, tpos;
  char *tmp;

  tmp     = (char*) mcalloc(2*strlen(s), sizeof(char));
  tpos = 0;
  for (i=pos+1; i <= pos + size; i++)
    if (i <= strlen(s)) tmp[tpos++] = s[offset+i-1];
    else                tmp[tpos++] = ' ';
  tmp[tpos] = 0;
  printf(fs, tmp);
  free(tmp);
}

/* same as print_subseq; used in color mode */
static void print_subseq_color(char *fs, char *s, int pos, int size){
  int i,c,tpos,ls;
  char e;
  char *lastcol, *tmp, *tmp2;
  int plastcol;

  ls      = strlen(s);
  lastcol = (char*) mcalloc(20,    sizeof(char));
  tmp     = (char*) mcalloc(20*ls, sizeof(char));
  tmp2    = (char*) mcalloc(20*ls, sizeof(char));

  plastcol = -1;

  // go to character pos+1:
  c=0; i=0; e=0;
  while (c<pos+1) {
    if      (s[i] == '\x1b')     { e = 1; plastcol = 0; }
    else if ((s[i] == 'm') && e) e = 0;
    else if (e)                  lastcol[plastcol++] = s[i];
    else                         c++;
    i++;
  }
  if (plastcol != -1) lastcol[plastcol] = 0;
  tpos = 0;
  while (c<=pos+size) {
    if (c <= ls) tmp2[tpos++] = s[i-1];
    else                tmp2[tpos++] = ' ';
    if      (s[i] == '\x1b')       e = 1;
    else if ((s[i] == 'm') && e) e = 0;
    else if (e)                  ;
    else                         c++;
    i++;
  }
  tmp2[tpos] = 0;
  strcat(tmp2, "\x1b[0m");
  tmp[0]=0;
  if (plastcol != -1) sprintf(tmp, "\x1b%sm", lastcol);
  strcat(tmp, tmp2);
  printf(fs, tmp);
  free(lastcol);
  free(tmp);
  free(tmp2);
}

/* main entry function for sequence output */
void print_sequence(toptions *opts, tsequence *seq, int pos, int size){
  if (!opts->split_output_mode) {
    if (opts->window_mode) {
      printf("%s%s", leading_space, leading_space_db);
      print_subseq_numbers(opts, pos, size);
    }
    printf("%s%s", leading_space, leading_space_db);
    print_subseq("%s", seq->original_seq, pos, 0, size);
    printf("\n");
  }
}

/* used in window-mode to shift the input sequence */
void shift_input(toptions *opts, tsequence *seq, char output){
  int i;

/*   printf("window_pos:        %d\n", opts->window_pos); */
/*   printf("window_size:       %d\n", opts->window_size); */
/*   printf("seq->seq:          %s\n", seq->seq); */
/*   printf("seq->original_seq: %s\n", seq->original_seq); */

  for (i=opts->window_pos; i<=opts->window_pos + opts->window_size; i++)
    seq->seq[i-opts->window_pos] = seq->original_seq[i];
  if (number_of_graphics) {
    for (i=opts->window_pos; i<=opts->window_pos + opts->window_size; i++)
       graphics_sequence[i-opts->window_pos] = seq->original_seq[i];
    graphics_sequence[strlen(seq->seq)] = 0;
  }
  convert_input(0, seq->seq, opts->window_size);
  if (output) print_sequence(opts, seq, opts->window_pos, opts->window_size);
}

/* from db2shape-cl */
struct dbcol_result{
  char *dbstr;
  char *shapestr;
};
// TODO  struct dbcol_result *calc_db2shape_cl(char *input, int st, int _maxloop);

/* main entry function for rna result output */
void output_result
  (toptions *opts, tsequence *seq,
   int nres, int *energy, double *shrepProb, char *dbString, char *shapeString, double prob, int rank){

  int pos, size;
  tformat_string *itr;
  char colors;

  itr = format_string_struct;
  colors = colored_db2shape && dbString;
  if (colors) {
    // TODO  dbcol = calc_db2shape_cl(dbString, global_shapetype, maxloop);
    // TODO  dbString    = dbcol->dbstr;
    // TODO  shapeString = dbcol->shapestr;
  }
  while (itr) {
    switch(itr->type) {
      case FORMAT_ENERGY:
        if (energy) printf(itr->string, (float) *energy / 100);
        break;
      case FORMAT_SHREPPROB:
        if (shrep_prob_show && shrepProb) printf(itr->string, *shrepProb);
        break;
      case FORMAT_DBSTRING:
        if (dbString) {
          if (!opts->split_output_mode) printf(itr->string, dbString);
          else {
            for (pos = 0; pos < opts->window_size; pos += opts->split_output_size) {
              size = opts->split_output_size;
              if (pos) printf("%s%s", leading_space, leading_space_db);
              else     printf("%s", leading_space_db);
              print_subseq_numbers(opts, pos + opts->window_pos, size);
              printf("%s%s", leading_space, leading_space_db);
              print_subseq("%s", seq->original_seq, opts->window_pos, pos, size);
              printf("\n");
              printf("%s", leading_space);
              if (colors) print_subseq_color(itr->string, dbString, pos, size);
              else        print_subseq      (itr->string, dbString, 0, pos, size);
              if (pos + opts->split_output_size < opts->window_size) printf("\n");
            }
          }
        }
        break;
      case FORMAT_SHAPESTRING:
        if (shapeString) if (shapeString[0]==0) printf(itr->string, "_");
                  else                   printf(itr->string, shapeString);
        break;
      case FORMAT_PROB:
        if (prob >= 0) printf(itr->string, prob);
        break;
      case FORMAT_RANK:
        if (rank >= 0) printf(itr->string, rank);
        break;
      case FORMAT_VERB:
        printf("%s",itr->string);
        break;
    }
    itr = itr->next;
    // free colored strings:
    if (colors) {
      // TODO  free(dbcol->dbstr);
      // TODO  free(dbcol->shapestr);
      // TODO  free(dbcol);
    }
    // TODO if (dbString) rna_plot(nres, dbString_org, energy, prob, shapeString_org);
  }
}


/* Initialize all stuff from adplib
  ====================================================================== */

void adplib_init(toptions *opts, tsequence *seq, char **z, int *n){
  *z = (char *) seq->seq - 1;
  if (opts->window_mode) {
    *n = min(opts->window_size, seq->length);
  }
  else {
    *n = seq->length;
    opts->window_size = seq->length;
  }
  opts->window_size = min(opts->window_size, seq->length);
  adp_dynmem  = memory_new();
  adp_statmem = memory_new();
  libPP_init(seq);

  setOutputMode(1);
  init_iupac_base();
  fs_init_leading_space(1,1,1,1,1,1);
}

void adplib_free(toptions *opts, tsequence *seq){
  memory_free(adp_dynmem);
  memory_free(adp_statmem);
}

/* ---------------------------------------------------------------------------
rnalib.h
RNA energy library, based on Haskell implementation by Jens Reeder
Author: Peter Steffen
$Date: 2006/04/18 08:40:55 $
--------------------------------------------------------------------------- */


// alphabet size (A,C,G,U,N)
#define ASIZE 5
#define inp(I) z[I]
#define d_inp(I) d_z[I]

/* basepair and stackpair predicates */

extern __shared__ char memory[] ;

// --------

#ifdef SHARED_OFFSET
#define d_offset ((int *) (memory))
#define dd_offset(I) d_offset[I]
#define memory_o (memory + 8010)
#else
#define dd_offset(I) (((I)*((I)+1))/2)
#define memory_o memory
#endif


//__device__ char *g_z;
//__device__ int *d_columns ;


// #ifdef SHARED_Z
// #define d_z (memory)
// #define memory_e (memory_o + 2010)
// #else
// #define d_z g_z
// #define memory_e memory_o
// #endif


// ------------

#ifdef SHARED_ENERGY

#define d_canPair (memory_e + 150)
#define d_stack_dg ((int *) (memory_e + 200))
#define d_tstacki_dg ((int *) (memory_e + 2800))
#define d_termaupenalty_ar ((int *) (memory_e + 5400))
#define d_il_ent_ar ((int *) (memory_e + 5600))
#define d_bl_ent_ar ((int *) (memory_e + 5800))
#define d_hl_ent_ar ((int *) (memory_e + 6000))
#define memory_s (memory_e + 6040)

#define d_basepairing(I,J)  ((I+1 < J) && d_canPair[index2(d_inp((I)+1),d_inp(J))])
#define d_stackpairing(I,J) ((I+3 < J) && d_canPair[index2(d_inp((I)+1),d_inp(J))] && d_canPair[index2(d_inp((I)+2),d_inp((J)-1))])

#define d_stack_dg_ac(I,J,K,L) d_stack_dg [index4(d_inp(I),d_inp(J),d_inp(K),d_inp(L))]
#define d_sr_energy(I,J) d_stack_dg [index4(d_inp((I)),d_inp((I)+1),d_inp((J)-1),d_inp((J)))]
#define d_il_stack(I,J,K,L) (d_tstacki_dg[index4(d_inp((I)),d_inp((I)+1),d_inp((L)),d_inp((L)+1))] + \
			     d_tstacki_dg[index4(d_inp((J)+1),d_inp((J)),d_inp((K)+1),d_inp((K)))])
#define d_termaupenalty(I,J) d_termaupenalty_ar[index2(d_inp(I),d_inp(J))]

#else

#define memory_s memory_e

#define d_canPair g_canPair
#define d_stack_dg g_stack_dg
#define d_tstacki_dg g_tstacki_dg
#define d_termaupenalty_ar g_termaupenalty_ar
#define d_il_ent_ar g_il_ent_ar
#define d_bl_ent_ar g_bl_ent_ar
#define d_hl_ent_ar g_hl_ent_ar

#define d_basepairing(I,J)  ((I+1 < J) && d_canPair[d_inp((I)+1)][d_inp(J)])
#define d_stackpairing(I,J) ((I+3 < J) && d_canPair[d_inp((I)+1)][d_inp(J)] && d_canPair[d_inp((I)+2)][d_inp((J)-1)])
#define d_stack_dg_ac(I,J,K,L) d_stack_dg[d_inp(I)][d_inp(J)][d_inp(K)][d_inp(L)]
#define d_sr_energy(I,J) d_stack_dg[d_inp((I))][d_inp((I)+1)][d_inp((J)-1)][d_inp((J))]
#define d_il_stack(I,J,K,L) (d_tstacki_dg[d_inp((I))][d_inp((I)+1)][d_inp((L))][d_inp((L)+1)] + \
                             d_tstacki_dg[d_inp((J)+1)][d_inp((J))][d_inp((K)+1)][d_inp((K))])

#define d_termaupenalty(I,J) d_termaupenalty_ar[d_inp(I)][d_inp(J)]


#endif





/////

#define basepairing(I,J)  ((I+1 < J) && canPair[inp((I)+1)][inp(J)])
char canStackPair[ASIZE][ASIZE][ASIZE][ASIZE];

#define stackpairing(I,J) ((I+3 < J) && canPair[inp((I)+1)][inp(J)] && canPair[inp((I)+2)][inp((J)-1)])

/* alternative definition of basepair, working on characters */
char basepair(int i, int j);
__device__ char d_basepair(int i, int j);

/* Constants     */
/* ------------- */

#define const_e    (2.718281828459)
#define mean_scale (1.34855)

/* Energy tables          */
/* ---------------------- */

/* The Jacobson-Stockmayer term for loop interpolation. */
#define jacobson_stockmayer(size) (107.856*log((size)/30.0))

#define UNDEF 1000000

char canPair[ASIZE][ASIZE];
__device__ __constant__ char g_canPair[ASIZE][ASIZE];

int stack_dg          [ASIZE][ASIZE][ASIZE][ASIZE];
__device__ __constant__ int g_stack_dg          [ASIZE][ASIZE][ASIZE][ASIZE];
int hl_ent_ar             [31];
__device__ __constant__ int g_hl_ent_ar[31];
int tstackh_dg        [ASIZE][ASIZE][ASIZE][ASIZE];
__device__ __constant__ int d_tstackh_dg        [ASIZE][ASIZE][ASIZE][ASIZE];
int hl_tetra          [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
__device__  int d_hl_tetra          [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
int bl_ent_ar             [31];
__device__ __constant__ int g_bl_ent_ar[31];
int il_ent_ar             [31];
__device__ __constant__ int g_il_ent_ar[31];
int tstacki_dg        [ASIZE][ASIZE][ASIZE][ASIZE];
__device__ __constant__ int g_tstacki_dg        [ASIZE][ASIZE][ASIZE][ASIZE];
int dr_dangle_dg      [ASIZE][ASIZE][ASIZE];
__device__ __constant__ int d_dr_dangle_dg      [ASIZE][ASIZE][ASIZE];
int dl_dangle_dg      [ASIZE][ASIZE][ASIZE];
__device__ __constant__ int d_dl_dangle_dg      [ASIZE][ASIZE][ASIZE];
int termaupenalty_ar  [ASIZE][ASIZE];
__device__ __constant__ int g_termaupenalty_ar  [ASIZE][ASIZE];

int intloop11      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
__device__ int d_intloop11      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
int intloop21      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
__device__ int d_intloop21      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
int intloop22      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
__device__ int d_intloop22      [ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE][ASIZE];
double *scale_ar;
__device__ double *d_scale_ar;

//#include "d_energy.cu"

/* Energy Functions                          */
/* ----------------------------------------- */

#define index2(I,J) (I*ASIZE + J)
#define index3(I,J,K) (index2(index2(I, J), K))
#define index4(I,J,K,L) (index3(index2(I, J), K, L))
#define index5(I,J,K,L,M) (index4(index2(I, J), K, L, M))
#define index6(I,J,K,L,M,N) (index5(index2(I, J), K, L, M, N))
#define index7(I,J,K,L,M,N,O) (index6(index2(I, J), K, L, M, N, O))
#define index8(I,J,K,L,M,N,O,P) (index7(index2(I, J), K, L, M, N, O, P))



#define stack_dg_ac(I,J,K,L) stack_dg[inp(I)][inp(J)][inp(K)][inp(L)]
#define sr_energy(I,J) stack_dg[inp((I))][inp((I)+1)][inp((J)-1)][inp((J))]

#define hl_ent(size) ((size) <= 30 ? hl_ent_ar[size] : 769 + jacobson_stockmayer(i))
#define d_hl_ent(size) ((size) <= 30 ? d_hl_ent_ar[size] : 769 + jacobson_stockmayer(i))

#define hl_stack(I,J) tstackh_dg[inp((I))][inp((I)+1)][inp((J)-1)][inp((J))]
#define d_hl_stack(I,J) d_tstackh_dg[d_inp((I))][d_inp((I)+1)][d_inp((J)-1)][d_inp((J))]


int hl_energy(int i, int j);
//__device__ int d_hl_energy(int i, int j);

#define bl_ent(size) ((size) <= 30 ? bl_ent_ar[size] : 609 + jacobson_stockmayer(i))
#define d_bl_ent(size) ((size) <= 30 ? d_bl_ent_ar[size] : 609 + jacobson_stockmayer(i))

int bl_energy(int bl, int i, int j, int br);
//__device__ int d_bl_energy(int bl, int i, int j, int br);
int br_energy(int bl, int i, int j, int br);
//__device__ int d_br_energy(int bl, int i, int j, int br);

#define il_ent(size) ((size) <= 30 ? il_ent_ar[size] : 369 + jacobson_stockmayer(i))
#define d_il_ent(size) ((size) <= 30 ? d_il_ent_ar[size] : 369 + jacobson_stockmayer(i))

#define il_stack(I,J,K,L) (tstacki_dg[inp((I))][inp((I)+1)][inp((L))][inp((L)+1)] + \
                           tstacki_dg[inp((J)+1)][inp((J))][inp((K)+1)][inp((K))])


int il_energy(int i, int j, int k, int l);
//__device__ int d_il_energy(int i, int j, int k, int l);

#define dr_energy(I,J)  dr_dangle_dg[inp((I))][inp((J))][inp((J)+1)]
#define d_dr_energy(I,J)  d_dr_dangle_dg[d_inp((I))][d_inp((J))][d_inp((J)+1)]
#define dli_energy(I,J) dr_dangle_dg[inp((J))][inp((I))][inp((I)+1)]
#define d_dli_energy(I,J) d_dr_dangle_dg[d_inp((J))][d_inp((I))][d_inp((I)+1)]

#define dl_energy(I,J)  dl_dangle_dg[inp((I)-1)][inp((I))][inp((J))]
#define d_dl_energy(I,J)  d_dl_dangle_dg[d_inp((I)-1)][d_inp((I))][d_inp((J))]
#define dri_energy(I,J) dl_dangle_dg[inp((J)-1)][inp((J))][inp((I))]
#define d_dri_energy(I,J) d_dl_dangle_dg[d_inp((J)-1)][d_inp((J))][d_inp((I))]

#define ss_energy(I,J) 0
#define d_ss_energy(I,J) 0

#define dangles(i,j,i2,j2,k,l,k2,l2) ((dli_energy(j,k+1) + dri_energy(j2,k2+1)))
#define d_dangles(i,j,i2,j2,k,l,k2,l2) ((d_dli_energy(j,k+1) + d_dri_energy(j2,k2+1)))
#define sspenalty(a) (npp * (a))
#define d_sspenalty(a) (d_npp * (a))

#define termaupenalty(I,J) termaupenalty_ar[inp(I)][inp(J)]

#define mk_pf(X) (exp ((X)/ (-61.6321)))
#define scale(size) scale_ar[size]
#define d_scale(size) d_scale_ar[size]

/* initializations */
void rnalib_init(toptions *opts, tsequence *seq);
void rnalib_free();

/* ---------------------------------------------------------------------------
rnalib.c
RNA energy library, based on Haskell implementation by Jens Reeder
Author: Peter Steffen
$Date: 2006/04/18 08:40:51 $
--------------------------------------------------------------------------- */


/* ---------------------------------------------------------------------------------------------------- */
/* input handling                                                                                       */
/* ---------------------------------------------------------------------------------------------------- */

/* The alphabet */
#define A 0
#define C 1
#define G 2
#define U 3
#define N 4

static int   n;
static int   d_n;
static char *z;
static char *d_z;

#ifdef DIFF3
#define ROUND_THREADS
#endif



/* initialize basepair predicate */
static void init_canPair(){
  if(fread(canPair, sizeof(char), ASIZE*ASIZE, energyFile) != ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_canPair"), (char *) canPair,
				    ASIZE*ASIZE*sizeof(char), 0,
				    hipMemcpyHostToDevice));
}


/* initialize stackpair predicate */
static void init_canStackPair(){
  if(fread(canStackPair, sizeof(char), ASIZE*ASIZE*ASIZE*ASIZE, energyFile) != ASIZE*ASIZE*ASIZE*ASIZE)
    printf("File read error.");

/* // no, it's recomputed from d_canPair
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_canStackPair"), (char *) canStackPair,
				    ASIZE*ASIZE*ASIZE*ASIZE*sizeof(char), 0,
				    hipMemcpyHostToDevice));
*/
}

/* alternative definition of basepair, working on characters */
char basepair(int i, int j){
  return(((z[i] == 'a') && (z[j] == 'u')) ||
         ((z[i] == 'u') && (z[j] == 'a')) ||
         ((z[i] == 'c') && (z[j] == 'g')) ||
         ((z[i] == 'g') && (z[j] == 'c')) ||
         ((z[i] == 'g') && (z[j] == 'u')) ||
         ((z[i] == 'u') && (z[j] == 'g')));
}

//__device__ char d_basepair(int i, int j){
//  return(((d_z[i] == 'a') && (d_z[j] == 'u')) ||
//         ((d_z[i] == 'u') && (d_z[j] == 'a')) ||
//         ((d_z[i] == 'c') && (d_z[j] == 'g')) ||
//         ((d_z[i] == 'g') && (d_z[j] == 'c')) ||
//         ((d_z[i] == 'g') && (d_z[j] == 'u')) ||
//         ((d_z[i] == 'u') && (d_z[j] == 'g')));
//}


/* ---------------------------------------------------------------------------------------------------- */
/* Energy stuff                                                                                         */
/* ---------------------------------------------------------------------------------------------------- */

/* subword length */
#define size_of(I,J) ((J)-(I))

/* Some constants and utilities */
/* ---------------------------- */


/* ---------------------------------------------------------------------------------------------------- */
/* Stacking Region Energies                                                                             */
/* ---------------------------------------------------------------------------------------------------- */
/*
   Stabilizing energies for canonical basepairs: AU, CG, GU
   Basepairing: Parameters are in 5' 3' order.
   stack_dg a b c d
            ^ ^ ^ ^
            | |_| |
            |_____|
*/

static void init_stack_dg()
{
  if(fread(stack_dg, sizeof(int), ASIZE*ASIZE*ASIZE*ASIZE, energyFile) != ASIZE*ASIZE*ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_stack_dg"), (int *) stack_dg,
				    ASIZE*ASIZE*ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* ---------------------------------------------------------------------------------------------------- */
/* Hairpin Loop Energies                                                                                */
/* ---------------------------------------------------------------------------------------------------- */

static void init_hl_ent_ar()
{
  if(fread(hl_ent_ar, sizeof(int), 31, energyFile) != 31)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_hl_ent_ar"), (int *) hl_ent_ar,
				    31*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* Stacking Interaction           */
/* ------------------------------ */

static void init_tstackh_dg()
{
  if(fread(tstackh_dg, sizeof(int), ASIZE*ASIZE*ASIZE*ASIZE, energyFile) != ASIZE*ASIZE*ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_tstackh_dg"), (int *) tstackh_dg,
				    ASIZE*ASIZE*ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

#define hl_stack(I,J) tstackh_dg[inp((I))][inp((I)+1)][inp((J)-1)][inp((J))]
#define d_hl_stack(I,J) d_tstackh_dg[d_inp((I))][d_inp((I)+1)][d_inp((J)-1)][d_inp((J))]

/* Tetraloop Bonus Energies       */
/* ------------------------------ */
/*  Ultrastable tetra-loops & energy bonus at 37 �C: */

static void init_hl_tetra()
{
  if(fread(hl_tetra, sizeof(int), ASIZE*ASIZE*ASIZE*ASIZE*ASIZE*ASIZE, energyFile) != ASIZE*ASIZE*ASIZE*ASIZE*ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_hl_tetra"), (int *) hl_tetra,
				    ASIZE*ASIZE*ASIZE*ASIZE*ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* Terminal AU penalty is included in hl_stack,  */
/* therefore it must be added explicitely only for (size == 3) */

int hl_energy(int i, int j){
  int size;
  int entropy;
  int tetra_bonus, stack_mismatch;
  int termaupen;

  size           = j-i-1;
  entropy        = hl_ent(size);
  stack_mismatch = hl_stack(i,j);
  tetra_bonus    = hl_tetra[inp(i)][inp(i+1)][inp(i+2)][inp(i+3)][inp(i+4)][inp(i+5)];
  termaupen      = termaupenalty_ar[inp(i)][inp(j)];

  if (size==3) return(entropy + termaupen);
  if (size==4) return(entropy + stack_mismatch + tetra_bonus);
  if (size>4)  return(entropy + stack_mismatch);
  printf("hairpin loop < 3 found. Please use production\n");
  printf("  hl <<< lbase -~~ (region `with` minsize 3)  ~~- lbase\n");
  printf("in your grammar.\n");
  exit(1);
}

//__device__ int d_hl_energy(int i, int j){
//  int size;
//  int entropy;
//  int tetra_bonus, stack_mismatch;
//  int termaupen;
//
//  size           = j-i-1;
//  entropy        = d_hl_ent(size);
//  stack_mismatch = d_hl_stack(i,j);
//  tetra_bonus    = d_hl_tetra[d_inp(i)][d_inp(i+1)][d_inp(i+2)][d_inp(i+3)][d_inp(i+4)][d_inp(i+5)];
//  termaupen      = d_termaupenalty_ar[d_inp(i)][d_inp(j)];
//
//  if (size==3) return(entropy + termaupen);
//  if (size==4) return(entropy + stack_mismatch + tetra_bonus);
//  if (size>4)  return(entropy + stack_mismatch);
//  printf("hairpin loop < 3 found. Please use production\n");
//  printf("  hl <<< lbase -~~ (region `with` minsize 3)  ~~- lbase\n");
//  printf("in your grammar.\n");
//  exit(1);
//}
//
/* ---------------------------------------------------------------------------------------------------- */
/* Bulge Loop Energies                                                                                  */
/* ---------------------------------------------------------------------------------------------------- */

static void init_bl_ent_ar()
{
  if(fread(bl_ent_ar, sizeof(int), 31, energyFile) != 31)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_bl_ent_ar"), (int *) bl_ent_ar,
				    31*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* Bulge Loop Left                */
/* ------------------------------ */
/*
      											  .        .
      											  .        .
										       (bl+3) - (br-2)
 If size == 1 the terminal aupenalty for the stem starting after the bulge (that is    (bl+2) - (br-1))
										    bl+1
										          bl  -   br

 is added possibly. This is unwanted. Since we do not have a chance to check the size of the bulge when parsing the stem
 we substract the possible penalty here!
*/

int bl_energy(int bl, int i, int j, int br){
  int stacking, size, entropy;

  stacking    = stack_dg[inp(bl)][inp(j+1)][inp(br-1)][inp(br)];
  size        = size_of(i,j);
  entropy     = bl_ent(size);

  if     (size==1)  return(stacking + entropy - termaupenalty_ar[inp(bl+2)][inp(br-1)]);
  else if (size>1)  return(entropy + termaupenalty_ar[inp(bl)][inp(br)]);
  else {printf("bl_energy size < 1\n"); exit(-1);}
}

//__device__ int d_bl_energy(int bl, int i, int j, int br){
//  int stacking, size, entropy;
//
//  stacking    = d_stack_dg[d_inp(bl)][d_inp(j+1)][d_inp(br-1)][d_inp(br)];
//  size        = size_of(i,j);
//  entropy     = d_bl_ent(size);
//
//  if     (size==1)  return(stacking + entropy - d_termaupenalty_ar[d_inp(bl+2)][d_inp(br-1)]);
//  else if (size>1)  return(entropy + d_termaupenalty_ar[d_inp(bl)][d_inp(br)]);
//  else {printf("bl_energy size < 1\n"); exit(-1);}
//}


/* Bulge Loop Right               */
/* ------------------------------ */

int br_energy(int bl, int i, int j, int br){
  int stacking, size, entropy;

  stacking    = stack_dg[inp(bl)][inp(bl+1)][inp(i)][inp(br)];
  size        = size_of(i,j);
  entropy     = bl_ent(size);

  if (size==1) return(stacking + entropy - termaupenalty_ar[inp(bl+1)][inp(br-2)]);
  else return(entropy + termaupenalty_ar[inp(bl)][inp(br)]);
}

//__device__ int d_br_energy(int bl, int i, int j, int br){
//  int stacking, size, entropy;
//
//  stacking    = d_stack_dg[d_inp(bl)][d_inp(bl+1)][d_inp(i)][d_inp(br)];
//  size        = size_of(i,j);
//  entropy     = d_bl_ent(size);
//
//  if (size==1) return(stacking + entropy - d_termaupenalty_ar[d_inp(bl+1)][d_inp(br-2)]);
//  if (size>1)  return(entropy + d_termaupenalty_ar[d_inp(bl)][d_inp(br)]);
//}


/* ---------------------------------------------------------------------------------------------------- */
/* Interior Loop Energies                                                                               */
/* ---------------------------------------------------------------------------------------------------- */

/* Entropic Term                  */
/* ------------------------------ */
/*
  DESTABILIZING ENERGIES BY SIZE OF LOOP

  il_ent 1 and 2 undefined in the tables of Mathews et al. since
  special energy values exist
*/


static void init_il_ent_ar()
{
  if(fread(il_ent_ar, sizeof(int), 31, energyFile) != 31)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_il_ent_ar"), (int *) il_ent_ar,
				    31*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* Stacking Interaction           */
/* ------------------------------ */
/*

STACKING ENERGIES : TERMINAL MISMATCHES AND BASE-PAIRS.

Stabilizing energies for canonical basepairs: AU, CG, GU
Basepairing: Paramers are in 5' 3' order.
tstacki_dg a b c d
           ^ ^ ^ ^
           | |_| |
           |_____|

*/

static void init_tstacki_dg()
{
  if(fread(tstacki_dg, sizeof(int), ASIZE*ASIZE*ASIZE*ASIZE, energyFile) != ASIZE*ASIZE*ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_tstacki_dg"), (int *) tstacki_dg,
				    ASIZE*ASIZE*ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/*
the time intensive n^4 version of internal loops
(used in reduced form O(n^2*c^2) where c is the maximal internal loop size)

(i,j) = left region, (k,l) = right region

        i --- l+1
 5'    /        \    3'
 |   i+1         l  / \
 |    |          |   |
\ /   |          |   |
 3'   |          |   5'
      j         k+1
       \        /
        j+1 --- k

*/

/* Ninio's equation */
#define il_asym(SL,SR) min(300,((abs((SL)-(SR)))*50))
#define d_il_asym(SL,SR) min(300,((abs((SL)-(SR)))*50))

/* include internal loop energies */
//#include "intloop11.c"
//#include "intloop21.c"
//#include "intloop22.c"

#define il11_energy(lb,rb) intloop11[inp((lb))][inp((lb)+1)][inp((lb)+2)][inp((rb)-2)][inp((rb)-1)][inp((rb))]
#define d_il11_energy(lb,rb) d_intloop11[d_inp((lb))][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((rb)-2)][d_inp((rb)-1)][d_inp((rb))]

#define il12_energy(lb,rb) intloop21[inp(lb)][inp((lb)+1)][inp((lb)+2)][inp((rb)-3)][inp((rb)-2)][inp((rb)-1)][inp(rb)]
#define d_il12_energy(lb,rb) d_intloop21[d_inp(lb)][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((rb)-3)][d_inp((rb)-2)][d_inp((rb)-1)][d_inp(rb)]
#define il21_energy(lb,rb) intloop21[inp((rb)-2)][inp((rb)-1)][inp(rb)][inp(lb)][inp((lb)+1)][inp((lb)+2)][inp((lb)+3)]
#define d_il21_energy(lb,rb) d_intloop21[d_inp((rb)-2)][d_inp((rb)-1)][d_inp(rb)][d_inp(lb)][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((lb)+3)]

#define int22_energy(a,b,c,d,e,f,g,h) intloop22[inp(a)][inp(b)][inp(c)][inp(d)][inp(e)][inp(f)][inp(g)][inp(h)]
#define d_int22_energy(a,b,c,d,e,f,g,h) d_intloop22[d_inp(a)][d_inp(b)][d_inp(c)][d_inp(d)][d_inp(e)][d_inp(f)][d_inp(g)][d_inp(h)]
#define il22_energy(lb,rb) int22_energy(lb,((lb)+1),((lb)+2),((lb)+3),((rb)-3),((rb)-2),((rb)-1),rb)
#define d_il22_energy(lb,rb) d_int22_energy(lb,((lb)+1),((lb)+2),((lb)+3),((rb)-3),((rb)-2),((rb)-1),rb)


int il_energy(int i, int j, int k, int l)
{
  int sl, sr;
  sl = size_of(i,j);
  sr = size_of(k,l);
  if ((sl > 2) || (sr > 2))
    return((il_ent (sl + sr))
         + (il_stack (i,j,k,l))
         + (il_asym(sl,sr))); else
  if ((sl == 1) && (sr == 1)) return(il11_energy(i,l+1)); else
  if ((sl == 1) && (sr == 2)) return(il12_energy(i,l+1)); else
  if ((sl == 2) && (sr == 1)) return(il21_energy(i,l+1)); else
  if ((sl == 2) && (sr == 2)) return(il22_energy(i,l+1)); else
  return 65000;
}

//__device__ int d_il_energy(int i, int j, int k, int l)
//{
//  int sl, sr;
//  sl = size_of(i,j);
//  sr = size_of(k,l);
//  if ((sl > 2) || (sr > 2))
//    return((d_il_ent (sl + sr))
//         + (d_il_stack (i,j,k,l))
//         + (d_il_asym(sl,sr))); else
//  if ((sl == 1) && (sr == 1)) return(d_il11_energy(i,l+1)); else
//  if ((sl == 1) && (sr == 2)) return(d_il12_energy(i,l+1)); else
//  if ((sl == 2) && (sr == 1)) return(d_il21_energy(i,l+1)); else
//  if ((sl == 2) && (sr == 2)) return(d_il22_energy(i,l+1)); else
//  return 65000;
//}


/* ---------------------------------------------------------------------------------------------------- */
/* Dangling ends                                                                                        */
/* ---------------------------------------------------------------------------------------------------- */

/* dangle right                   */
/* ------------------------------ */

static void init_dr_dangle_dg()
{
  if(fread(dr_dangle_dg, sizeof(int), ASIZE*ASIZE*(ASIZE), energyFile) != ASIZE*ASIZE*(ASIZE))
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_dr_dangle_dg"), (int *) dr_dangle_dg,
				    ASIZE*ASIZE*(ASIZE)*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* dangle left                    */
/* ------------------------------ */

static void init_dl_dangle_dg()
{
  if(fread(dl_dangle_dg, sizeof(int), (ASIZE)*ASIZE*ASIZE, energyFile) != (ASIZE)*ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_dl_dangle_dg"), (int *) dl_dangle_dg,
				    (ASIZE)*ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

#define ss_energy(I,J) 0
#define d_ss_energy(I,J) 0


/* ---------------------------------------------------------------------------------------------------- */
/* special pseudoknot energies                                                                          */
/* ---------------------------------------------------------------------------------------------------- */

/* This are the dangling energies for the bases bridging the stacks */

#define dangles(i,j,i2,j2,k,l,k2,l2) ((dli_energy(j,k+1) + dri_energy(j2,k2+1)))
#define d_dangles(i,j,i2,j2,k,l,k2,l2) ((d_dli_energy(j,k+1) + d_dri_energy(j2,k2+1)))
#define sspenalty(a) (npp * (a))
#define d_sspenalty(a) (d_npp * (a))

/* ---------------------------------------------------------------------------------------------------- */
/* Terminal AU penalty                                                                                  */
/* ---------------------------------------------------------------------------------------------------- */

static void init_termaupenalty_ar()
{
  if(fread(termaupenalty_ar, sizeof(int), ASIZE*ASIZE, energyFile) != ASIZE*ASIZE)
    printf("File read error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("g_termaupenalty_ar"), (int *) termaupenalty_ar,
				    ASIZE*ASIZE*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

/* internal loop energies */
static void  init_intloop11(){
 if(fread(intloop11, sizeof(int), (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE), energyFile) != (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE))
    printf("File write error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_intloop11"), (int *) intloop11,
				    (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

static void  init_intloop21(){
 if(fread(intloop21, sizeof(int), (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE), energyFile) !=
     (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE))
       printf("File write error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_intloop21"), (int *) intloop21,
				    (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}

static void  init_intloop22(){
  if(fread(intloop22, sizeof(int), (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE), energyFile) !=
     (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE))
       printf("File write error.");

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL("d_intloop22"), (int *) intloop22,
				    (ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*(ASIZE)*sizeof(int), 0,
				    hipMemcpyHostToDevice));

}


/* ---------------------------------------------------------------------------------------------------- */
/* Scale                                                                                                */
/* ---------------------------------------------------------------------------------------------------- */

static void init_scale_ar()
{
  int i;
  scale_ar=(double *) mcalloc(n+2, sizeof(double));
  scale_ar[0] = 1.0;
  for (i = 1; i<= n; i++) {
    scale_ar[i] = scale_ar[i-1] / mean_scale;
  }
}

/* ---------------------------------------------------------------------------------------------------- */
/* Initialize rna input and energy tables                                                                         */
/* ---------------------------------------------------------------------------------------------------- */

void rnalib_init(toptions *opts, tsequence *seq)
{
  /* initialize input and pairing tables */
  z    = seq->seq - 1;
  n    = seq->length;

  convert_input(1, z, n);

  CUDA_SAFE_CALL(hipMalloc((void **) &d_z, (n+2)*sizeof(char)));
  hipMemcpy(d_z, z, (n+2)*sizeof(char), hipMemcpyHostToDevice);
  d_n=n;

  if((energyFile=fopen(ENERGYFILE, "rb"))==NULL) {
    printf("Cannot open file %s.\n",ENERGYFILE);
    exit(1);
  }

  init_canPair();
  init_canStackPair();

  /* initialize energies */
  init_stack_dg();
  init_hl_ent_ar();
  init_tstackh_dg();
  init_hl_tetra();
  init_bl_ent_ar();
  init_il_ent_ar();
  init_tstacki_dg();
  init_dr_dangle_dg();
  init_dl_dangle_dg();
  init_termaupenalty_ar();

  init_intloop11();
  init_intloop21();
  init_intloop22();

  init_scale_ar();

  fclose(energyFile);

}

void rnalib_free()
{
  free(scale_ar);
}


/* data structures                                                                  */
/* -------------------------------------------------------------------------------- */
#define size_of(I,J) ((J)-(I))
#define d_il11_energy(lb,rb) d_intloop11[d_inp((lb))][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((rb)-2)][d_inp((rb)-1)][d_inp((rb))]
#define d_il12_energy(lb,rb) d_intloop21[d_inp(lb)][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((rb)-3)][d_inp((rb)-2)][d_inp((rb)-1)][d_inp(rb)]
#define d_il21_energy(lb,rb) d_intloop21[d_inp((rb)-2)][d_inp((rb)-1)][d_inp(rb)][d_inp(lb)][d_inp((lb)+1)][d_inp((lb)+2)][d_inp((lb)+3)]
#define d_int22_energy(a,b,c,d,e,f,g,h) d_intloop22[d_inp(a)][d_inp(b)][d_inp(c)][d_inp(d)][d_inp(e)][d_inp(f)][d_inp(g)][d_inp(h)]
#define d_il22_energy(lb,rb) d_int22_energy(lb,((lb)+1),((lb)+2),((lb)+3),((rb)-3),((rb)-2),((rb)-1),rb)
#define d_il_asym(SL,SR) min(300,((abs((SL)-(SR)))*50))

__device__ int d_hl_energy(int i, int j, char *d_z){
  int size;
  int entropy;
  int tetra_bonus, stack_mismatch;
  int termaupen;

  size           = j-i-1;
  entropy        = d_hl_ent(size);
  stack_mismatch = d_hl_stack(i,j);
  tetra_bonus    = d_hl_tetra[d_inp(i)][d_inp(i+1)][d_inp(i+2)][d_inp(i+3)][d_inp(i+4)][d_inp(i+5)];
  termaupen      = d_termaupenalty(i,j);

  if (size==3) return(entropy + termaupen);
  if (size==4) return(entropy + stack_mismatch + tetra_bonus);
  return(entropy + stack_mismatch);
  //printf("hairpin loop < 3 found. Please use production\n");
  //printf("  hl <<< lbase -~~ (region `with` minsize 3)  ~~- lbase\n");
  //printf("in your grammar.\n");
  //exit(1);
}

__device__ int d_bl_energy(int bl, int i, int j, int br, char *d_z){
  int stacking, size, entropy;

  // stacking    = d_stack_dg[d_inp(bl)][d_inp(j+1)][d_inp(br-1)][d_inp(br)];
  stacking    = d_stack_dg_ac(bl,j+1,br-1,br);
  size        = size_of(i,j);
  entropy     = d_bl_ent(size);

  if     (size==1)  return(stacking + entropy - d_termaupenalty(bl+2,br-1));
  return(entropy + d_termaupenalty(bl,br));
}

__device__ int d_br_energy(int bl, int i, int j, int br, char *d_z){
  int stacking, size, entropy;

  // stacking    = d_stack_dg[d_inp(bl)][d_inp(bl+1)][d_inp(i)][d_inp(br)];
	stacking    = d_stack_dg_ac(bl, bl+1, i, br);
  size        = size_of(i,j);
  entropy     = d_bl_ent(size);

  if (size==1) return(stacking + entropy - d_termaupenalty(bl+1,br-2));
  return(entropy + d_termaupenalty(bl,br));
}

__device__ int d_il_energy(int i, int j, int k, int l, char *d_z)
{
  int sl, sr;
  sl = size_of(i,j);
  sr = size_of(k,l);
  if ((sl > 2) || (sr > 2))
    return((d_il_ent (sl + sr))
         + (d_il_stack (i,j,k,l))
         + (d_il_asym(sl,sr))); else
  if ((sl == 1) && (sr == 1)) return(d_il11_energy(i,l+1)); else
  if ((sl == 1) && (sr == 2)) return(d_il12_energy(i,l+1)); else
  if ((sl == 2) && (sr == 1)) return(d_il21_energy(i,l+1)); else
  if ((sl == 2) && (sr == 2)) return(d_il22_energy(i,l+1)); else
  return 65000;
}
