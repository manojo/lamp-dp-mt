#include "hip/hip_runtime.h"
/* =========================================================================

   RNAfold 1.0




========================================================================= */

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include "config.h"
#include "options.h"
#include "cudalib.cu"
#include "RNAfold_mfe.cu"

#include "RNAfold.h"

/* ======================================================================
   Init default values
   ====================================================================== */

static void init_defaults(toptions *opt) {
  opt->number_of_blocks  = -1;   // Set number of blocks (-b)
  opt->threads_per_block = 128;  // Set number of threads per blocks (-T)
  opt->traceback_diff    = 0;    // Set energy range (kcal/mol) (-e)
  opt->traceback_percent = 10;   // Set energy range (%%) (-c)
  opt->maxloop           = 30;   // Set maximal loop length (-M)
  opt->window_mode       = 0;    // Set window size (-w)
  opt->window_size       = 0;    // Set window size (-w)
  opt->window_step       = 1;    // Set window position increment (-W)
  opt->window_best       = 0;    // Show best results in window mode (-B)
  opt->split_output_mode = 0;    // Specify output width for structures (-S)
  opt->split_output_size = 0;    // Specify output width for structures (-S)
  if (opt->inputfile) free(opt->inputfile);
  opt->inputfile         = NULL; // Read input from file (-f)
  opt->colored_output    = 0;    // Colored output (-z)
  opt->terminate            = 0;
  opt->interactive          = 0;
  /* PS output */
  opt->number_of_graphics   = 0;
  opt->graphics_includeinfo = 0;
  opt->graphics_highlight   = 0;
  /* output mode */
  opt->output_mode          = 0;
  opt->format_string        = NULL;
}



/* ======================================================================
   Process arguments
   ====================================================================== */

static void process_args(toptions *opt, char interactive, int argc, char **argv) {
  int   c;
  char  manopt, manoptmode;

  opterr = 0; optind = 1;

  pcolor(opt->colored_output,COLOR_BLUE);
  while ((c = getopt (argc, argv, "hH:vb:T:e:c:M:w:W:BS:f:z")) != -1) {
    switch (c) {

      // Display this information (-h)
      case 'h':
        if (interactive) {
          printf("Interactive mode:\nEnter sequence directly or use the following commands to change settings:\n");
        }
        else {

        printf("Options:\n");
        }
        printf("  -h           	Display this information\n");
        printf("  -H <option>  	Display detailed information on <option>\n");
        printf("  -v           	Show version\n");
        printf("  -b <value>   	Set number of blocks\n");
        printf("  -T <value>   	Set number of threads per blocks\n");
        printf("  -e <value>   	Set energy range (kcal/mol)\n");
        printf("  -c <value>   	Set energy range (%%)\n");
        printf("  -M <value>   	Set maximal loop length\n");
        printf("  -w <value>   	Set window size\n");
        printf("  -W <value>   	Set window position increment\n");
        printf("  -B           	Show best results in window mode\n");
        printf("  -S <value>   	Specify output width for structures\n");
        printf("  -f <filename>	Read input from file\n");
        printf("  -z           	Colored output\n");
        if (opt->interactive) {
        printf("Additional interactive mode commands:\n");
        printf("  :s           	Show current configuration\n");
        printf("  :d           	Reset configuration\n");
        printf("  :e <string>  	Execute system command\n");
        printf("  :q           	Quit\n");
        }
        opt->terminate = 1;
        break;
      // Display detailed information on <option> (-H)
      case 'H':
        if      (optarg[0]=='-') { manoptmode = '-'; manopt = optarg[1]; }
        else if (optarg[0]==':') { manoptmode = ':'; manopt = optarg[1]; }
        else                     { manoptmode = '-'; manopt = optarg[0]; }
        if (!interactive) printf("\n");
        #include "RNAfold-man.c"
        if (!interactive) printf("\n");
        opt->terminate = 1;
        break;
      // Show version (-v)
      case 'v':
        printf("%s (%s)\n",PACKAGE_STRING,RELEASE_DATE);
        printf("\n");
        opt->terminate = 1;
        break;
      // Set number of blocks (-b)
      case 'b':
        sscanf(optarg,"%d",&(opt->number_of_blocks));
        opt->number_of_blocks = max(1, opt->number_of_blocks);
        if (interactive) printf("Number of blocks set to %d\n", opt->number_of_blocks);
        break;
      // Set number of threads per blocks (-T)
      case 'T':
        sscanf(optarg,"%d",&(opt->threads_per_block));
        opt->threads_per_block = max(1, opt->threads_per_block);
        if (interactive) printf("Number of threads per blocks set to %d\n", opt->threads_per_block);
        break;
      // Set energy range (kcal/mol) (-e)
      case 'e':
        sscanf(optarg,"%f",&(opt->traceback_diff));
        opt->traceback_diff = max(0, opt->traceback_diff);
        if (interactive) printf("Energy range set to %.2f kcal/mol.\n", opt->traceback_diff);

        opt->traceback_percent = 0;
        break;
      // Set energy range (%%) (-c)
      case 'c':
        sscanf(optarg,"%d",&(opt->traceback_percent));
        opt->traceback_percent = max(0, opt->traceback_percent);
        if (interactive) printf("Energy range set to %d%% of mfe.\n", opt->traceback_percent);

        opt->traceback_diff = 0;
        break;
      // Set maximal loop length (-M)
      case 'M':
        sscanf(optarg,"%d",&(opt->maxloop));
        opt->maxloop = max(0, opt->maxloop);
        if (interactive) printf("Set maximal loop length to %d.\n", opt->maxloop);
        break;
      // Set window size (-w)
      case 'w':
        if ((interactive) && (optarg[0] == '-')) {
          printf("Window mode disabled.\n");
          opt->window_mode = 0;
        }
        if (optarg[0] != '-') {
          opt->window_mode = 1;
          sscanf(optarg,"%d",&(opt->window_size));
          opt->window_size = max(1, opt->window_size);
          if (interactive) printf("Set window size to %d. Type -w - to disable.\n", opt->window_size);
        }
        break;
      // Set window position increment (-W)
      case 'W':
        sscanf(optarg,"%d",&(opt->window_step));
        opt->window_step = max(1, opt->window_step);
        if (interactive) printf("Set window position increment to %d.\n", opt->window_step);
        break;
      // Show best results in window mode (-B)
      case 'B':
        if (interactive) {
          printf("Show best results in window mode");
          opt->window_best = 1 - opt->window_best;
          if (opt->window_best) printf (" = ON. Type -B again to switch off.\n");
          else printf (" = OFF. Type -B again to switch on.\n");
        }
        else opt->window_best = 1;
        break;
      // Specify output width for structures (-S)
      case 'S':
        if ((interactive) && (optarg[0] == '-')) {
          printf("Output splitting disabled.\n");
          opt->split_output_mode = 0;
        }
        if (optarg[0] != '-') {
          opt->split_output_mode = 1;
          sscanf(optarg,"%d",&(opt->split_output_size));
          opt->split_output_size = max(1, opt->split_output_size);
          if (interactive) printf("Set output size to %d. Type -S - to disable.\n", opt->split_output_size);
        }
        break;
      // Read input from file (-f)
      case 'f':
        if (opt->inputfile) free(opt->inputfile);
        opt->inputfile = mkstr(optarg);
        break;
      // Colored output (-z)
      case 'z':
        if (interactive) {
          printf("Colored output");
          opt->colored_output = 1 - opt->colored_output;
          if (opt->colored_output) printf (" = ON. Type -z again to switch off.\n");
          else printf (" = OFF. Type -z again to switch on.\n");
        }
        else opt->colored_output = 1;

             if (opt->interactive && !opt->colored_output) printf(COLOR_DEFAULT);

        break;


      case '?':
        if (interactive) printf("Unknown option '-%c'. Type -h for more information.\n", optopt);
        else {
          fprintf (stderr, "%s: unknown option `-%c'.\nTry '%s -h' for more information.\n", argv[0], optopt, argv[0]);
          exit(1);
        }
        break;
      default:
        abort ();
      }
  }
  pcolor(opt->colored_output,COLOR_DEFAULT);
}

/* ======================================================================
   Print settings
   ====================================================================== */

static void print_settings(toptions *opt) {
  printf("Current settings\n-----------------\n");
  printf("Number of blocks  (-b) %d\n", opt->number_of_blocks);
  printf("Number of threads per blocks  (-T) %d\n", opt->threads_per_block);
  printf("Energy range                           %.2f kcal/mol  (-e)\n", opt->traceback_diff);
  printf("Energy range                           %d%% of mfe  (-c)\n", opt->traceback_percent);
  printf("Maximal loop length:                   %d  (-M)\n", opt->maxloop);
  if (opt->window_mode)
  printf("Window size:                           %d  (-w)\n", opt->window_size);
  else
  printf("Window size:                           %s-  (-w)\n","");
  printf("Window position increment:             %d  (-W)\n", opt->window_step);
  printf("Show best result in window mode        %s  (-B)\n", opt->window_best ? "ON" : "OFF" );
  if (opt->split_output_mode)
  printf("Output width for structures:           %d  (-S)\n", opt->split_output_size);
  else
  printf("Output width for structures:           %s-  (-S)\n","");
  printf("Colored output                         %s  (-z)\n", opt->colored_output ? "ON" : "OFF" );
}


/* ======================================================================
   Main interface
   ====================================================================== */

int main (int argc, char **argv) {
  toptions   *opt;
  tsequence  *seq;
  treadseq   *rs        = NULL;
  ttokenizer *tokenizer = NULL;
  char       *command;

  CUT_DEVICE_INIT();

  opt = (toptions *) calloc(1,sizeof(toptions));

  init_defaults(opt);
  process_args(opt, 0, argc, argv);

  if (!opt->terminate) {
    if (optind < argc)               rs = readseq_open(READSEQ_STRING, argv[optind]);
    else if (opt->inputfile)         rs = readseq_open(READSEQ_FILE,   opt->inputfile);
    else if (!isatty(fileno(stdin))) rs = readseq_open(READSEQ_STDIN,  NULL);
    else {
      printf("Interactive mode. Try `./RNAfold -h` for more information.\n");
      rl_init();
      opt->interactive = 1;
      opt->colored_output = 1 - opt->colored_output;
      tokenizer = tokenizer_new();
      rs = readseq_open(READSEQ_STRING, "");
    }

    while (1) {
      if (opt->interactive) {
        if (opt->colored_output)
          printf("%s\nInput sequence (upper or lower case); :q to quit, -h for help.\n....,....1....,....2....,....3....,....4....,....5....,....6....,....7....,....8\n%s",COLOR_RED,COLOR_DEFAULT);
        else
	  printf("\nInput sequence (upper or lower case); :q to quit, -h for help.\n....,....1....,....2....,....3....,....4....,....5....,....6....,....7....,....8\n");
        command = rl_gets();

        if (!command || (command[0] == '@') || ((command[0] == ':') && (command[1] == 'q'))) {
          pcolor(opt->colored_output,COLOR_BLUE);
          printf("Leaving RNAfold.");
          pcolor(opt->colored_output,COLOR_DEFAULT);
          printf("\n");
          exit(0);
        }
        else if (command[0] == ':') {
          pcolor(opt->colored_output,COLOR_BLUE);
          if (command[1] == 's') print_settings(opt);
          if (command[1] == 'd') {
            init_defaults(opt);
            opt->colored_output = 1;
            opt->interactive = 1;
            printf("Activated default configuration.\n");
            pcolor(opt->colored_output,COLOR_DEFAULT);
          }
          if (command[1] == 'e') {
            system(command + 2);
          }
          if (command[1] == 'r') {
            system("make update");
            system("./RNAfold");
            exit(0);
          }
        }
        else if (command[0] == '-') {
          tokenizer_exec(tokenizer, argv[0], command);
          process_args(opt, 1, tokenizer->count, tokenizer->token);
          if (opt->inputfile) {
            rs = readseq_free(rs);
            rs = readseq_open(READSEQ_FILE, opt->inputfile);
          }
          free(opt->inputfile);
          opt->inputfile = NULL;
        }
        else {
          rs = readseq_free(rs);
          rs = readseq_open(READSEQ_STRING, command);
        }
      }

      while (1) {
        seq = readseq_next_fasta(rs);
        if (!(seq->success)) break;
        if (1) {
          main_rnafold_mfe(opt, seq);
        }

        sequence_free(seq);
      }

      if (!opt->interactive) break;
    }
  }
  exit(0);
}
