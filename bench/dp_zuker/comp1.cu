#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include "comp1.h"

#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) {
  if (err==hipSuccess) return;
  fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); cuReset; exit(EXIT_FAILURE);
}
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
#define cuMap(host,dev,size) { cuErr(hipHostAlloc((void**)&host,size,hipHostMallocMapped)); cuErr(hipHostGetDevicePointer((void**)&dev,host,0)); }
#define cuUnmap(host) cuErr(hipHostFree(host))
#define cuStream(stream) hipStream_t stream; cuErr(hipStreamCreate(&stream));
#define cuSync(stream) cuErr(hipStreamSynchronize(stream))
#define hipStreamDestroy(stream) cuErr(hipStreamDestroy(stream))
#define cuAlloc2(cond,host,dev,size) bool cond = hipMalloc((void**)&dev,size)==hipSuccess; if (!cond) { cuMap(host,dev,size); }
#define cuFree2(host,dev) { if (host!=NULL) { cuUnmap(host); host=NULL; } else cuFree(dev); dev=NULL; }
#define _unroll _Pragma("unroll 1")
//#define M_W 1024
//#define M_H 1024
#define MEM_MATRIX ((M_H*(M_H+1))/2)
#define idx(i,j) ({ unsigned _i=(i),_d=M_H+1+_i-(j); MEM_MATRIX - (_d*(_d-1))/2 +_i; })
static input_t *g_in1 = NULL, *g_in2 = NULL;
static cost_t *g_cost = NULL;
static back_t *g_back = NULL;
__device__ static __attribute__((unused)) input_t *_in1=NULL, *_in2=NULL;
__global__ void gpu_input(input_t* in1, input_t* in2) { _in1=in1; _in2=in2; }

// --------------------------------
#include "../../src/librna/vienna/vienna.h"

__constant__ paramT0 param0;

#define my_len (M_H-1)
#define my_seq _in1
#define my_P g_P
#define my_P0 param0

#define my_dev __device__
#include "../../src/librna/librna_impl.h"
#include "../../src/librna/vienna/vienna.c"
#include "../../src/librna/vienna/energy_par.c"

static paramT *cg_P=NULL;
__global__ static void _initP(paramT* params) { g_P=params; }
static inline void rna_init() {
  read_parameter_file("../../src/librna/vienna/rna_turner2004.par");
  paramT* P = get_scaled_parameters();
  hipMemcpyToSymbol(HIP_SYMBOL(param0),&(P->p0),sizeof(paramT0));

  cuMalloc(cg_P,sizeof(paramT));
  cuPut(P,cg_P,sizeof(paramT),NULL);
  _initP<<<1,1>>>(cg_P); free(P);
}
static inline void rna_free() { cuFree(cg_P); }
// --------------------------------

__device__ static inline int fun0(int lb, int e, int rb) { return e + sr_energy(lb,rb); }
__device__ static inline int fun1(int lb, int f1, T2ii x, int f2, int rb) { return hl_energy(f1,f2) + sr_energy(lb,rb); }
__device__ static inline int fun10(int c1, int c) { return c1+c; }
__device__ static inline int fun11(int c1, T2ii e) { return c1; }
__device__ static inline int fun12(int x, int e) { return x+e; }
__device__ static inline int fun13() { return 0; }
__device__ static inline int fun2(int lb, int f1, T2ii b, int x, int f2, int rb) { return x + bl_energy(f1,b._1,b._2-1,f2,f2-1) + sr_energy(lb,rb); }
__device__ static inline int fun3(int lb, int f1, int x, T2ii b, int f2, int rb) { return x + br_energy(f1,b._1,b._2-1,f2,f1+1) + sr_energy(lb,rb); }
__device__ static inline int fun4(int f1, int f2, T2ii r1, int x, T2ii r2, int f3, int f4) { return x + il_energy(f2,r1._2,r2._1-1,f3) + sr_energy(f1,f4); }
__device__ static inline int fun5(int lb, int f1, int x, int f2, int rb) { return ml_energy() + ul_energy() + x + termau_energy(f1,f2) + sr_energy(lb,rb) + ml_mismatch_energy(f1,f2); }
__device__ static inline bool fun6(int i, int j) { return i+4<=j && bp_index(_in1[i],_in1[j-1])!=NO_BP && bp_index(_in1[i+1],_in1[j-2])!=NO_BP; }
__device__ static inline int fun7(int lb, int e) { return e; }
__device__ static inline int fun8(int lb, int e, int rb) { return e + ext_mismatch_energy(lb,rb-1) + termau_energy(lb,rb-1); }
__device__ static inline int fun9(int c1) { return ul_energy()+c1; }

__global__ void gpu_solve(const input_t* in1, const input_t* in2, cost_t* cost, back_t* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
  const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned tN = blockDim.x * gridDim.x;
  const unsigned tB = blockIdx.x;
  unsigned tP=s_start; // block progress
  for (unsigned jj=s_start; jj<s_stop; ++jj) {
    for (int ii=tI; ii<M_H; ii+=tN) {
      int i = M_H-1-ii, j = i+jj;
      if (j<M_W) {
        back_t _back = {{-1,{0,0}},{-1,{0}},{-1,{0}},{-1,{0}}};
        cost_t _cost = {}; // init to 0
        #define VALID(I,J,RULE) (back[idx(I,J)].RULE.rule!=-1)
        /* --- cl[i,j] --- */
        if (fun6(i,j)) {
          if (i+9<=j && VALID(i+1,j-1,cl)) {
            { int _c=fun0((i),cost[idx(i+1,j-1)].cl,(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){0}; } }
          }
          if (i+7<=j) {
            { int _c=fun1((i),(i+1),(T2ii){i+2,j-2},(j-2),(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){1}; } }
          }
          if (i+12<=j) _unroll for(int k=i+3,ku=j-9; k<=ku; ++k) {
            if (k-32<=i && VALID(k,j-2,cl)) {
              { int _c=fun2((i),(i+1),(T2ii){i+2,k},cost[idx(k,j-2)].cl,(j-2),(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){2,{k}}; } }
            }
          }
          if (i+12<=j) _unroll for(int k=i+9,ku=j-3; k<=ku; ++k) {
            if (j-32<=k && VALID(i+2,k,cl)) {
              { int _c=fun3((i),(i+1),cost[idx(i+2,k)].cl,(T2ii){k,j-2},(j-2),(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){3,{k}}; } }
            }
          }
          if (i+13<=j) for(int k=MAX(i+10,j-32),ku=j-3; k<=ku; ++k) {
            _unroll for(int l=i+3,lu=MIN(k-7,i+32); l<=lu; ++l) {
              if (VALID(l,k,cl)) {
                { int _c=fun4((i),(i+1),(T2ii){i+2,l},cost[idx(l,k)].cl,(T2ii){k,j-2},(j-2),(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){4,{l,k}}; } }
              }
            }
          }
          if (i+18<=j && VALID(i+2,j-2,ml)) {
            { int _c=fun5((i),(i+1),cost[idx(i+2,j-2)].ml,(j-2),(j-1)); if (_c<_cost.cl || _back.cl.rule==-1) { _cost.cl=_c; _back.cl=(bt2){5}; } }
          }
        }
        cost[idx(i,j)].cl = _cost.cl;
        back[idx(i,j)].cl = _back.cl;
        /* --- ml[i,j] --- */
        if (i+15<=j && VALID(i+1,j,ml)) {
          { int _c=fun7((i),cost[idx(i+1,j)].ml); if (_c<_cost.ml || _back.ml.rule==-1) { _cost.ml=_c; _back.ml=(bt1){6}; } }
        }
        _unroll for(int k=i+7,ku=j-7; k<=ku; ++k) {
          if (VALID(i,k,cl) && VALID(k,j,ml1)) {
            { int _c=fun10(fun9(fun8((i),cost[idx(i,k)].cl,(k))),cost[idx(k,j)].ml1); if (_c<_cost.ml || _back.ml.rule==-1) { _cost.ml=_c; _back.ml=(bt1){7,{k}}; } }
          }
        }
        cost[idx(i,j)].ml = _cost.ml;
        back[idx(i,j)].ml = _back.ml;
        /* --- ml1[i,j] --- */
        if (i+8<=j && VALID(i+1,j,ml1)) {
          { int _c=fun7((i),cost[idx(i+1,j)].ml1); if (_c<_cost.ml1 || _back.ml1.rule==-1) { _cost.ml1=_c; _back.ml1=(bt1){8}; } }
        }
        _unroll for(int k=i+7,ku=j-7; k<=ku; ++k) {
          if (VALID(i,k,cl) && VALID(k,j,ml1)) {
            { int _c=fun10(fun9(fun8((i),cost[idx(i,k)].cl,(k))),cost[idx(k,j)].ml1); if (_c<_cost.ml1 || _back.ml1.rule==-1) { _cost.ml1=_c; _back.ml1=(bt1){9,{k}}; } }
          }
        }
        if (i+7<=j && VALID(i,j,cl)) {
          { int _c=fun9(fun8((i),cost[idx(i,j)].cl,(j))); if (_c<_cost.ml1 || _back.ml1.rule==-1) { _cost.ml1=_c; _back.ml1=(bt1){10}; } }
        }
        _unroll for(int k=i+7; k<j; ++k) {
          if (VALID(i,k,cl)) {
            { int _c=fun11(fun9(fun8((i),cost[idx(i,k)].cl,(k))),(T2ii){k,j}); if (_c<_cost.ml1 || _back.ml1.rule==-1) { _cost.ml1=_c; _back.ml1=(bt1){11,{k}}; } }
          }
        }
        cost[idx(i,j)].ml1 = _cost.ml1;
        back[idx(i,j)].ml1 = _back.ml1;
        /* --- st[i,j] --- */
        if (i+1<=j) {
          { int _c=fun7((i),cost[idx(i+1,j)].st); if (_c<_cost.st || _back.st.rule==-1) { _cost.st=_c; _back.st=(bt1){12}; } }
        }
        _unroll for(int k=i+7; k<=j; ++k) {
          if (VALID(i,k,cl)) {
            { int _c=fun12(fun8((i),cost[idx(i,k)].cl,(k)),cost[idx(k,j)].st); if (_c<_cost.st || _back.st.rule==-1) { _cost.st=_c; _back.st=(bt1){13,{k}}; } }
          }
        }
        if (i==j) {
          { int _c=fun13(); if (_c<_cost.st || _back.st.rule==-1) { _cost.st=_c; _back.st=(bt1){14}; } }
        }
        cost[idx(i,j)].st = _cost.st;
        back[idx(i,j)].st = _back.st;
      }
    }
    // Sync between blocks, removing __threadfence() here is incorrect but works
    // __threadfence();
    if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
    __syncthreads();
  }
}

__global__ void gpu_backtrack(trace_t* trace, unsigned* size, back_t* back, int i0, int j0) {
  const unsigned trace_len[15] = {2,2,2,2,2,2,1,1,1,1,1,1,1,1,1};
  trace_t *rd=trace, *wr=trace; *size=0;
  #define PUSH_BACK(I,J,RULE) { wr->i=I; wr->j=J; wr->rule=RULE; ++wr; ++(*size); }
  PUSH_BACK(i0,j0,12);
  for(;rd<wr;++rd) {
    bt2* bt;
    switch (rd->rule) {
      case 0: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 1: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 2: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 3: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 4: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 5: bt=(bt2*)&back[idx(rd->i,rd->j)].cl; break;
      case 6: bt=(bt2*)&back[idx(rd->i,rd->j)].ml; break;
      case 7: bt=(bt2*)&back[idx(rd->i,rd->j)].ml; break;
      case 8: bt=(bt2*)&back[idx(rd->i,rd->j)].ml1; break;
      case 9: bt=(bt2*)&back[idx(rd->i,rd->j)].ml1; break;
      case 10: bt=(bt2*)&back[idx(rd->i,rd->j)].ml1; break;
      case 11: bt=(bt2*)&back[idx(rd->i,rd->j)].ml1; break;
      case 12: bt=(bt2*)&back[idx(rd->i,rd->j)].st; break;
      case 13: bt=(bt2*)&back[idx(rd->i,rd->j)].st; break;
      case 14: bt=(bt2*)&back[idx(rd->i,rd->j)].st; break;
    }
    rd->rule=bt->rule;
    for (int i=0,l=trace_len[rd->rule]; i<l; ++i) rd->pos[i]=bt->pos[i];
    switch (rd->rule) {
      case 0: PUSH_BACK(rd->i+1,MAX(rd->i+8,rd->j-1),0); break;
      case 1: break;
      case 2: PUSH_BACK(rd->pos[0],MAX(rd->i+10,MAX(rd->i+11,rd->j-1)-1),0); break;
      case 3: PUSH_BACK(rd->i+2,rd->pos[0],0); break;
      case 4: PUSH_BACK(rd->pos[0],rd->pos[1],0); break;
      case 5: PUSH_BACK(rd->i+2,MAX(rd->i+16,MAX(rd->i+17,rd->j-1)-1),6); break;
      case 6: PUSH_BACK(rd->i+1,rd->j,6); break;
      case 7: PUSH_BACK(rd->i+0,MAX(rd->i+7,rd->pos[0]-0),0); PUSH_BACK(rd->pos[0],rd->j,8); break;
      case 8: PUSH_BACK(rd->i+1,rd->j,8); break;
      case 9: PUSH_BACK(rd->i+0,MAX(rd->i+7,rd->pos[0]-0),0); PUSH_BACK(rd->pos[0],rd->j,8); break;
      case 10: PUSH_BACK(rd->i+0,MAX(rd->i+7,rd->j-0),0); break;
      case 11: PUSH_BACK(rd->i+0,MAX(rd->i+7,rd->pos[0]-0),0); break;
      case 12: PUSH_BACK(rd->i+1,rd->j,12); break;
      case 13: PUSH_BACK(rd->i+0,MAX(rd->i+7,rd->pos[0]-0),0); PUSH_BACK(rd->pos[0],rd->j,12); break;
      case 14: break;
    }
  }
}

static cost_t* c_cost=NULL;
static back_t* c_back=NULL;

void g_init(input_t* in1, input_t* in2) {
  int dev=-1; cuErr(hipGetDevice(&dev));
  cuMalloc(g_in1,sizeof(input_t)*(M_H-1)); cuPut(in1,g_in1,sizeof(input_t)*(M_H-1),NULL);
  g_in2=NULL;
  rna_init();
  size_t s_cost = sizeof(cost_t)*MEM_MATRIX;
  size_t s_back = sizeof(back_t)*MEM_MATRIX;
  cuAlloc2(costDev,c_cost,g_cost,s_cost); cuAlloc2(backDev,c_back,g_back,s_back);
  gpu_input<<<1,1>>>(g_in1,g_in2);
  hipDeviceProp_t prop; cuErr(hipGetDeviceProperties(&prop, dev));
  size_t mem = (sizeof(input_t)+sizeof(trace_t))*(M_W+M_H) + s_cost + s_back;
  printf("%-20s : %.2fMb / %.2fMb [in=%ld,tr=%ld,cost=%ld,back=%ld] -> cost:%s, backtrack:%s\n","Memory selection",mem/1048576.0,prop.totalGlobalMem/1048576.0, sizeof(input_t),sizeof(trace_t),sizeof(cost_t),sizeof(back_t), costDev?"device":"host", backDev?"device":"host");
}

void g_free() {
  cuFree(g_in1); rna_free();
  cuFree2(c_cost,g_cost); cuFree2(c_back,g_back); cuReset;
}

void g_solve() {
  #define WARP_SIZE 32 // constant over CUDA devices
  unsigned blk_size = WARP_SIZE;
  unsigned blk_num = (M_H+blk_size-1)/blk_size;
  unsigned* lock; cuMalloc(lock,sizeof(unsigned)*blk_num);
  cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
  cuStream(stream);

  int steps = (M_W+M_H)/1536;
  steps = steps*steps*steps;
  if (steps<1) steps=1;

  for (int i=0;i<steps;++i) {
    unsigned s0=((M_W)*i)/steps, s1=((M_W)*(i+1))/steps;
    gpu_solve<<<blk_num, blk_size, 0, stream>>>(g_in1, g_in2, g_cost, g_back, lock, s0, s1);
  }
  cuSync(stream); hipStreamDestroy(stream); cuFree(lock);
}

int g_backtrack(trace_t** trace, unsigned* size) {
  int res; unsigned i0=0, j0=M_W-1;
  cuGet(&res,&g_cost[idx(i0,j0)].st,sizeof(int),NULL);
  if (trace && size) {
    unsigned mem=(M_W+M_H)*sizeof(trace_t);
    trace_t *g_trace=NULL,*c_trace=NULL; cuAlloc2(traceDev,c_trace,g_trace,mem);
    unsigned *g_size=NULL; cuMalloc(g_size,sizeof(unsigned));
    gpu_backtrack<<<1,1,0,NULL>>>(g_trace, g_size, g_back, i0, j0);
    cuGet(size,g_size,sizeof(unsigned),NULL); cuFree(g_size); mem=(*size)*sizeof(trace_t);
    *trace=(trace_t*)malloc(mem); cuGet(*trace,g_trace,mem,NULL); cuFree2(c_trace,g_trace);
  }
  return res;
}

// FLAGS := -arch=sm_30

int main() {
  struct timeval ts,te; double delta;
  gettimeofday(&ts,NULL);
  char in1[8193]="aauaaaccacucuuuaucaccucauuucagugauaagugagaaugauacgacgaguucaucgguaauuaagguucugucaccacgaacuuaagcuaugugacaugggacaaagcggucacgucuauauucgauuaauguuaugcggacgccaacauccauggcccccuguccgaugguuuggcucucuguaaagggucuaaagacaaacuuccugacgaaggaccucuaaggauacaucgaaaggguguggcccaguuuggguggcucaugacacggagcaguuaaaagcacgugaugacugguggcgcgugacuugccgaaaauuacgugacgugacugggccaugacggaugagcgcuuccuucuggugcggagcauacccccggacacuuuccaucgcuucccaucaugccaugaacgcucaagcuacccucagugcuugugcccgacuagcgcacaccgaccuggugacgagcuuuggucaccucacgaggggcggcccugucacgaacguaacguaagggacagugguguucaaaagguuaccuuccuaacaaagcccuuaaaguucacaccaucguucuacgcaccggcagucaagauacgugugcucgaaaagcguagauuuuacauauuacuaugacagugauaucaauauauugugagacaaggcgaaucucaacaagaaccggcucguaucgggagaacguagaaagcaugccgaacaugauaauccagucguauaggauggagcgauuugacuacugguugucuugcacugauggagcauagucaaaaguacaucccuuagugccaucucgugaggccuuacaggcauucgcacccaugcccaagaagcuaccuucuaaguggcaauacguacacauggaggacuuagcucgagugcgacggggggaugcuuucauggccccuuaucgcccgguaauccaugccuugaugaucgugugaacagggcuaugcucuuacucaaaugguuggguucaacgguagcgcauagguuuacacauguaaggaaacugguuagagcggcaggauuggauucauaagauuuucauugacauuuacacaccagccgaggagugcguguugcagugguaguuucguuuuucugagcgcucgaaguaauguucgcgccuuugggggagaaucauacgagaucauccaucgcugauaagcuugggaucucguucacuccgggucgucaccgcaguaaccgauucagggugaucagacucguaaccgauuguuucacacugugauugcagugccggugcucgcuaacaauccaaccggcaucgaacgugaagcgcuucgugcucggcuaaacugccguuaaccggaaugacaccgcgcggagaagagauagaaccggguuuacucuacgacuggccgagguuguaccuaauaagcuguacauacgggacgagcgagcaauaaccagucuauauggcaucaaguaaguucgcagaaccagacaaccgcuauggcaacgacacuuggcugacgaggaaaucaguauuugcauuuuucuaccccuauuaguguugguacccgcgcauagucaggauaacaaugucggacagcaaaacuaccaacgcaaucauuuccgguaaacuuuggucgcucguguauaagguuuggcccguugaugacuacacucacgauugugguacacucgcuacagaauuacaggguauucacuauaccgauuacuucgccaaaaauccucggaaccaccggauugcuauuccaacacacuggccuccguguuuucuuucucgaacagaucagccgggugcauaaucguuuguaucguugaagugaugggauaucugaguuggacggcuggcccacaucaauuuaagcacgacagcuuguaggcgacucgaucuggauacgagucuuagucaacccaguacacauuucuagccccucggcuggaaucguuccuaauaaucuacacacggcuugccgucgaugucaaucaaaugccgcuauuugcuuggccacgcuuauaacauguggcgauccugccgauggcgacgaucgucagccccggcacgcaguuuucucccagaaggaaacuuuaagaugaucagacuuagucugaagagaacauuccgucuaucacuugguaggaaugcagcaguucccuuguucggggauccgggaggacgacaacuagcaggaagacucccccccgcuaaccuaaaccaguggguucugccugauacuuuggcuugauauugaccucaaaacauucguuaggucaguguguuagccucuugcaucgccaggugucucccucucuuuuaaaaccauguuuacgugugugaugugguuaucgaacccuauguguugaauuuacguaguagcaauguuggucggaccacagcccaaaugcgacauuaggcuaguucauggcauucaguacgccgcaggaaaagggcuucuggguuuucacggaagugucgaacacgaaauaggggggcugaggcacugagugaguaauucgcaauuccguucugagugaagucccguucauucauaagaccuccaacgagcauuaauaugucacgaucccaauuugagaauugccgcggauauguacagagggugccucauucacucacaagggcuaucacaccuggguacaaaucucgacugugcuacaucagggugcguuccuguccauaggauuuugaagucucgggaucuaucgcaccgcccggauggagauugauaggugugggaguuuccauuaggagguuccacagcaacugcgccgguucuuacuccuggugcuguucgagucgaauaaacuggcuuuauauggcaccucacugucagucccguguuagcgaaugguggaguuacaugcguucuauguuuagcacgcccgugcagggggcauuacgguguuaggccaucauuugauguacauggagaugagcuucgugugaacccguuuggaaccgcaacccgaauugccaauuuuuuguucgauucaauucgaguguguucuacgucgggagggcuaaaacuguacuuacuccccucugccgcgucggaaugccuccaccuguauaauaggaugguuuucucagcuuaccacaugcugcuuuagcgauaauccuaucugcggcuuuggcguggauuggggccaugagguccugacacgggcaguuuaacccacuuggacuccuaaccuaagggcagacgaguagaauaggguuguugucuuguaucggugucaaaucgauauugguguccgguuccaacaggggucaaucuacgcuauugguugccuuauuaaaaauuaccaggguccggucugucgggucaaucgauaacggcauuggacauugauuaacauauaugccggcgagugaauucagguccaaucagcacauaccaagaagaguuagaaagggcggggucacguaacgcugggugcugcgagucauugcagucuuugcgaucgggcgugguuccgcgcuuucauccccuucgaaagucuccagcaagaauuccguucauuugacagaaauuagucgaauaucgaucggagcgcgaaauggguagaaacgcuagagcuuuuacugggguacgacuauaucucuaagcaucuaucccuccaguaaaagugucggccaagcaaugccaccguuaccguaagcuacuacacagucgggggcucgccucaucgaagcauaugaccccccauguucuugggauauacugagguuuauguucaggcucacuugcauagugagagugucccuuagaagguacaauguaaaggacagcggaccaaccugaucuuagaguguaaucuacguucaguucagucuguuuaagucacaagagugagcuccagcuaagaggccgcucuugaccggguuccaauauuggcguuaacguucguuuguuguaucgacgcccgaaggggggcagggaagaaucugaaauauguucgauagguuguaguccaucgaccugccgcaccauuucccgccuaaaggguacgggagccucgagcaaagccacgauaggaagcuagcaagcggugauaaagccuggcccaaguaaacauucggcuccucauaggcccagguagaccgaugacauggcgcgaaacucuugcuguaggucuuucacguggacggcacuaccugcaaaagacgggaaucgaccucaugcaggguucuugguacagcaucgguuuuuaccgggguagggaggcuugugccuucggcucccuacaugaaggcgcuaggacugccauuuuucauuaccccacaucgcgcgcauaaaacugcuucuacccuggauggcaauuguuucauauccacaguuccggaauucuuuagcgagggcaaaggggauuacguaaagauggauaaaaguacuacuuaucaagcgcuguaggggucuugcgcgucgcuucucuaguuggugggcaaaguccaguucauaccuugggguaggcauugaaggauauggcucgcuugucuccagggcuccggcgguaucgcguucuauucucgaguggccuaggauuugucuugccaaaagaauacuucgcagcauagucgucaccccaguagucuagccauacucucgccgucgccucucguccagcuugcgaauuugaggccgaugagccgcgaucugccacagggaagucguugacucgaaugguauagcguggaggggaagaccaacuucacgucagacuaauacgccgugggucgccuucagucgcagaugugaccacagguuauggauauagcaauugucagcccgacaccauuuaguucgcuauuauccuaugccgccgagauucaaggcacugggaagcacacaaggcuccggaucgcccgacuaacccauaccaaucaagcagagaggucgaagguuucuccguguauacaacuauucaaaaaagauggaauacccacggugagccgcgaucaacuuuacgcgaucgcucuuguaccgcuagacucacgucacaccgacgagucguacgucucuugagcuuaccucauuggacagccacccggaaaagacaucugcgaaaugcucgagaugcuuagucgcgcaaggggguuccguuuaaugaguucgacugggauaucgauauggucucgacaguucacuuuugacagcaguuggaauuagaggacuucucuuaaacgcuccgaguccucuuugccaggggagcgugaacaccccgagugauuugagcacaacaaugggucugaucaggggggaggcgcuguggacgaaacacguucccuucaccggauuggaaauauggccgaucuauaguaccagacaccgcgcgaaggauccguuugcacguaguaagggcuuggaaagucuugcggugguaugauucaguuaucguuauaagagcguuuucugaggagagcgggaguagcgaugacucgcggugaggggugauauucaauaguugcgcguauaggggauaccccuaucuaguacuugacgacaaaaacauuuacgaugugguuucuccuguaagggacggcgacugacccuugacugaucuaugcugagcucccgggaccccgauauggguuagggucugccagaucaagcggauugggagggucggggugauauuugcacacgccgggcccgucacgcauugggcuggcaggcacgugcgugugucaagggggaauuucgaggucccuuaacuuaugcgguggugagcccgugcuaucagaacagaucccuccugccgauaaauaacgagcagcgaucgaaccaugcaacgacagucaauugaccauguggcgggcuguaggacgcgugaaucagcgcucccagacagcaagauucagccuaucgcauuaaugggacacaagagcucuuccguaagcuaauacgucgcggagcguauuaaugagccgauuaaugccccgcgaaggucugcgaucaugguucagugggaguauggcguggaaaucacgaaugcgaacccgucgccgaacuggcgaucacaccucacuuggggucagcuuuuacuagaguggccccgcuauaggggucuacgaacgauaucagacagguucuacuuugggucggcaugcuauaguucgcagggguugcuacgagugaauuggcucccguaaucggugacgcuucacugauauuguaguugcgagcugcugcguuaacagguuccaacgccguacagaauggcucuacgccgcagcguauuauaacccgugacaucgcguagcgagggugcacuuugagggccgcccggggcaccgucgcagcccggguauacauaccagaggaaauuccacuugccgauauagaggcuucaagauugcauggcauccuuacaucuuuauuuaaguucggugcacguagacacuugccgcgucuucuuucagaagaauaaaugagggggcacuucuuagacacaccuuuuuuuaauccucgcguccucgaaacagcggaugugcugcugucuguaagggagguccguguuauagcgugucccuagacgcucuagucauccacacugugccugggcauugggacgcuggggucguauugugcgaguaagcggaccgguuccgacaugauaugaggcggagaaacuucuuccuagagacuuucaaguaagauaauucugaacgcuuacgccugaugugucugaacaggcuucgauggguugccuaggcagcacgaaggacguggaagugcuuugaguggagagccgggccuuuguuacaguaaacgucauccggcuagcuaaccaauacguuuuccuacagcacguccggcgcccuguguuuucauguggaauacaacuaccgcauugauaccucgcaccgcaaaaguaguuccccugcagaguacuccuuagguuucguaacaaaaacgagaguaguccgcucuucguagcguuaaacgucgaccgggcgccauuuggaggaagcgugucgaccugggauaggugcgacgguugagagaccgcgaagcuauuuucguuccggacguaaguuggcucacgugugcggcuuuucauuacuauaguuuauagugcugggaucguucuuugcgguaaguuucuaagcggguacucgccgacagucaugucaaaugaagaguuacugaacacgauguggguguccuaguaguugcuaaacauccugguuauaucuauggacgguacaaggauuccgaucauccacacgacgugaucguuggcgcacgcaucuaagagauucacucacaaaaugccaccgcuggccaaacgauaucgucacgaccagccacacgaaggugcaaguauauggccuuggaugaauugaagcgcgcacuggaagcugcaccccguagggucaaagggcaugagccagucucguauggguaacgguagguagaucuuaaaguaggugggucgggcgauacaaauggagaauaagggagugagguucguaaaauucucacugcaauacguaaaggagagauauccugaguaccucacagggucaacaaagaacgguucgcuggaccggguuaccgaacauacuuguccucuugcgauuuuacccaucugcgagauacucccucccgaauccuuggugcauuuuccgaacgagucacggccagaccuaccugugacugccgguauacuugguccaagcauccguuaagucaguagaucuucaaaaugcuaucacaagucuaucucgucucaacgguccaaugcgggacguugucguuuauugggaguaacgccacccuugguucucgcguggggucgagagucagcucuauauauccauuucucaccaaccugguguuauaaccaagcuaauuggagagcguuucccgacaccagugguagaugcacuauaggaacggaggacccacuccguaccgggugcgcaccacgggaugcgcauuaugccugccguagcuggcugaaacuagucuacaugugcgcuucucuagcaugaauuuaguagugagcgguccgguuuacgaggcuggaguuaccauuggcugugucggggaaggggacaggauguccggacguccaccgacucuuaguaccgacacucacacccuuauuuggaaaaagaggaggacucucccgcggcaacguacg";
  for (char* p=in1;*p;++p) switch(*p) {
    case 'a': *p=1; break;
    case 'c': *p=2; break;
    case 'g': *p=3; break;
    case 'u': *p=4; break;
  }
  g_init(in1,NULL);
  // free(in1);

  gettimeofday(&ts,NULL); g_solve(); gettimeofday(&te,NULL);
  delta=(te.tv_sec-ts.tv_sec)*1000.0+(te.tv_usec-ts.tv_usec)/1000.0;
  printf("%-20s : %.3f sec\n","- CUDA compute",delta/1000.0);

  trace_t *trace=NULL; unsigned size=0;

  gettimeofday(&ts,NULL); g_backtrack(&trace,&size); gettimeofday(&te,NULL);
  delta=(te.tv_sec-ts.tv_sec)*1000.0+(te.tv_usec-ts.tv_usec)/1000.0;
  printf("%-20s : %.3f sec\n","- CUDA backtrack",delta/1000.0);

  free(trace);
  g_free();
  return 0;
}
