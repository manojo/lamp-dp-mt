#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "nu.h"

#define cuReset hipDeviceReset()
#define cuDevSync hipDeviceSynchronize()
#define cuErr(err) cuErr_(err,__FILE__,__LINE__)
__attribute__((unused)) static inline void cuErr_(hipError_t err, const char *file, int line) {
  if (err==hipSuccess) return;
  fprintf(stderr,"%s:%i CUDA error %d:%s\n", file, line, err, hipGetErrorString(err)); cuReset; exit(EXIT_FAILURE);
}
#define cuMalloc(ptr,size) cuErr(hipMalloc((void**)&ptr,size))
#define cuFree(ptr) cuErr(hipFree(ptr))
#define cuPut(host,dev,size,stream) cuErr(hipMemcpyAsync(dev,host,size,hipMemcpyHostToDevice,stream))
#define cuGet(host,dev,size,stream) cuErr(hipMemcpyAsync(host,dev,size,hipMemcpyDeviceToHost,stream))
#define cuMap(host,dev,size) { cuErr(hipHostAlloc((void**)&host,size,hipHostMallocMapped)); cuErr(hipHostGetDevicePointer((void**)&dev,host,0)); }
#define cuUnmap(host) cuErr(hipHostFree(host))
#define cuStream(stream) hipStream_t stream; cuErr(hipStreamCreate(&stream));
#define cuSync(stream) cuErr(hipStreamSynchronize(stream))
#define hipStreamDestroy(stream) cuErr(hipStreamDestroy(stream))
#define cuAlloc2(cond,host,dev,size) bool cond = hipMalloc((void**)&dev,size)==hipSuccess; if (!cond) { cuMap(host,dev,size); }
#define cuFree2(host,dev) { if (host!=NULL) { cuUnmap(host); host=NULL; } else cuFree(dev); dev=NULL; }
#define _unroll _Pragma("unroll 5")
#define M_W (SIZE+1)
#define M_H (SIZE+1)
#define MEM_MATRIX ((M_H*(M_H+1))/2)
#define idx(i,j) ({ unsigned _i=(i),_d=M_H+1+_i-(j); MEM_MATRIX - ((_d*(_d-1))>>1) +_i; })
static input_t *g_in1 = NULL, *g_in2 = NULL;
static cost_t *g_cost = NULL;
static back_t *g_back = NULL;
__device__ static __attribute__((unused)) input_t *_in1=NULL, *_in2=NULL;
__global__ void gpu_input(input_t* in1, input_t* in2) { _in1=in1; _in2=in2; }

__device__ static inline int fun0() { return 0; }
__device__ static inline int fun1(char c, int a) { return a; }
__device__ static inline int fun2(int a, char c) { return a; }
__device__ static inline int fun3(char l, int a, char r) { return a+1; }
__device__ static inline bool fun4(int i, int j) { if (i+2>j) return false; char a=_in1[i],b=_in1[j-1]; return (a=='a'&&b=='u') || (a=='u'&&b=='a') || (a=='g'&&b=='u') || (a=='u'&&b=='g') || (a=='c'&&b=='g') || (a=='g'&&b=='c'); }
__device__ static inline int fun5(int l, int r) { return l+r; }

__global__ void gpu_solve(const input_t* in1, const input_t* in2, cost_t* cost, back_t* back, volatile unsigned* lock, unsigned s_start, unsigned s_stop) {
  const unsigned tI = threadIdx.x + blockIdx.x * blockDim.x;
  const unsigned tN = blockDim.x * gridDim.x;
  const unsigned tB = blockIdx.x;
  unsigned tP=s_start; // block progress
  for (unsigned jj=s_start; jj<s_stop; ++jj) {
    for (int ii=tI; ii<M_H; ii+=tN) {
      int i=M_H-1-ii, j=i+jj;
      if (j<M_W) {
        #define VALID(I,J,RULE) (cost[idx(I,J)].RULE!=-1000)
        #include "nu_kern.h"
      }
    }
    // Sync between blocks, removing __threadfence() here is incorrect but works
    // __threadfence();
    if (threadIdx.x==0) { lock[tB]=++tP; if (tB) while(lock[tB-1]<tP) {} }
    __syncthreads();
  }
}

__global__ void gpu_backtrack(trace_t* trace, unsigned* size, back_t* back, int i0, int j0) {
  const unsigned trace_len[5] = {1,1,1,1,1};
  trace_t *rd=trace, *wr=trace; *size=0;
  #define PUSH_BACK(I,J,RULE) { wr->i=I; wr->j=J; wr->rule=RULE; ++wr; ++(*size); }
  PUSH_BACK(i0,j0,0);
  for(;rd<wr;++rd) {
    bt1* bt;
    switch (rd->rule) {
      case 0: bt=(bt1*)&back[idx(rd->i,rd->j)].s; break;
      case 1: bt=(bt1*)&back[idx(rd->i,rd->j)].s; break;
      case 2: bt=(bt1*)&back[idx(rd->i,rd->j)].s; break;
      case 3: bt=(bt1*)&back[idx(rd->i,rd->j)].s; break;
      case 4: bt=(bt1*)&back[idx(rd->i,rd->j)].s; break;
    }
    rd->rule=bt->rule;
    for (int i=0,l=trace_len[rd->rule]; i<l; ++i) rd->pos[i]=bt->pos[i];
    switch (rd->rule) {
      case 0: break;
      case 1: PUSH_BACK(rd->i+1,rd->j,0); break;
      case 2: PUSH_BACK(rd->i,MAX(rd->i+0,rd->j-1),0); break;
      case 3: PUSH_BACK(rd->i+1,MAX(rd->i+1,rd->j-1),0); break;
      case 4: PUSH_BACK(rd->i,rd->pos[0],0); PUSH_BACK(rd->pos[0],rd->j,0); break;
    }
  }
}

static cost_t* c_cost=NULL;
static back_t* c_back=NULL;

void my_init(input_t* in1, input_t* in2) {
  int dev=-1; cuErr(hipGetDevice(&dev));
  cuMalloc(g_in1,sizeof(input_t)*(M_H-1)); cuPut(in1,g_in1,sizeof(input_t)*(M_H-1),NULL);
  g_in2=NULL;
  size_t s_cost = sizeof(cost_t)*MEM_MATRIX;
  size_t s_back = sizeof(back_t)*MEM_MATRIX;
  cuAlloc2(costDev,c_cost,g_cost,s_cost); cuAlloc2(backDev,c_back,g_back,s_back);
  gpu_input<<<1,1>>>(g_in1,g_in2);
  hipDeviceProp_t prop; cuErr(hipGetDeviceProperties(&prop, dev));
}

void my_free() {
  cuFree(g_in1);
  cuFree2(c_cost,g_cost); cuFree2(c_back,g_back); cuReset;
}

void my_solve() {
  #define WARP_SIZE 32 // constant over CUDA devices
  unsigned blk_size = WARP_SIZE;
  unsigned blk_num = (M_H+blk_size-1)/blk_size;
  unsigned* lock; cuMalloc(lock,sizeof(unsigned)*blk_num);
  cuErr(hipMemset(lock,0,sizeof(unsigned)*blk_num));
  cuStream(stream);
  for (int i=0;i<1;++i) {
    unsigned s0=((M_W)*i)/1, s1=((M_W)*(i+1))/1;
    gpu_solve<<<blk_num, blk_size, 0, stream>>>(g_in1, g_in2, g_cost, g_back, lock, s0, s1);
  }
  cuSync(stream); hipStreamDestroy(stream); cuFree(lock);
}

int my_backtrack(trace_t** trace, unsigned* size) {
  int res; unsigned i0=0, j0=M_W-1;
  cuGet(&res,&g_cost[idx(i0,j0)].s,sizeof(int),NULL);
  if (trace && size) {
    unsigned mem=(M_W+M_H)*sizeof(trace_t);
    trace_t *g_trace=NULL,*c_trace=NULL; cuAlloc2(traceDev,c_trace,g_trace,mem);
    unsigned *g_size=NULL; cuMalloc(g_size,sizeof(unsigned));
    gpu_backtrack<<<1,1,0,NULL>>>(g_trace, g_size, g_back, i0, j0);
    cuGet(size,g_size,sizeof(unsigned),NULL); cuFree(g_size); mem=(*size)*sizeof(trace_t);
    *trace=(trace_t*)malloc(mem); cuGet(*trace,g_trace,mem,NULL); cuFree2(c_trace,g_trace);
  }
  return res;
}
